#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: dlnetwork.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "dlnetwork.h"
#include "Resize2DLayer.h"
#include "deployedPointPillarModel.h"
#include "deployedPointPillarModel_data.h"
#include "deployedPointPillarModel_internal_types.h"
#include "ipermute.h"
#include "permute.h"
#include "rt_nonfinite.h"
#include "MWBatchNormalizationLayer.hpp"
#include "MWCNNLayer.hpp"
#include "MWConcatenationLayer.hpp"
#include "MWConvLayer.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudnnCustomLayerBase.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWFusedConvActivationLayer.hpp"
#include "MWInputLayer.hpp"
#include "MWMaxPoolingLayer.hpp"
#include "MWOutputLayer.hpp"
#include "MWSigmoidLayer.hpp"
#include "MWSplittingLayer.hpp"
#include "MWTensor.hpp"
#include "MWTensorBase.hpp"
#include "coder_array.h"
#include "stdio.h"
#include <cstdlib>
#include <cstring>

// Named Constants
const char *errorString{
    "Abnormal termination due to: %s.\nError in %s (line %d)."};

const char *errStringBase{
    "Error during execution of the generated code. %s at line: %d, file: "
    "%s\nExiting program execution ...\n"};

// Variable Definitions
static bool gpuConstsCopied_dlnetwork_layerPredictWithColMajority;

static bool b_gpuConstsCopied_dlnetwork_layerPredictWithColMajority;

// Function Declarations
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int b_line);

static void checkRunTimeError(const char *errMsg, const char *file,
                              unsigned int b_line);

static __global__ void
dlnetwork_layerPredictWithColMajority_kernel21(double dv[124]);

static __global__ void
dlnetwork_layerPredictWithColMajority_kernel26(double dv[62]);

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void detector0_0::allocate()
{
  targetImpl->allocatePermuteBuffers(13713408, 3);
  targetImpl->allocate(76800000, 4);
  for (int idx{0}; idx < 57; idx++) {
    layers[idx]->allocate();
  }
  (static_cast<MWTensor<float> *>(inputTensors[0]))
      ->setData(layers[3]->getLayerOutput(0));
  (static_cast<MWTensor<float> *>(inputTensors[1]))
      ->setData(layers[0]->getLayerOutput(0));
}

//
// Arguments    : void
// Return Type  : void
//
void detector0_0::cleanup()
{
  deallocate();
  for (int idx{0}; idx < 57; idx++) {
    layers[idx]->cleanup();
  }
  if (targetImpl) {
    targetImpl->cleanup();
  }
  isInitialized = false;
  checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
}

//
// Arguments    : void
// Return Type  : void
//
void detector0_0::deallocate()
{
  targetImpl->deallocate();
  for (int idx{0}; idx < 57; idx++) {
    layers[idx]->deallocate();
  }
}

//
// Arguments    : void
// Return Type  : void
//
void detector0_0::postsetup()
{
  targetImpl->postSetup(layers, numLayers);
}

//
// Arguments    : void
// Return Type  : void
//
void detector0_0::resetState()
{
}

//
// Arguments    : void
// Return Type  : void
//
void detector0_0::setSize()
{
  for (int idx{0}; idx < 57; idx++) {
    layers[idx]->propagateSize();
  }
  allocate();
  postsetup();
}

//
// Arguments    : void
// Return Type  : void
//
void detector0_0::setup()
{
  if (isInitialized) {
    resetState();
  } else {
    targetImpl->preSetup();
    targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(layers[0]))
        ->createInputLayer(targetImpl, inputTensors[1], "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[1]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[0]->getOutputTensor(0), 1, 1, 9, 64, 1, 1, 0,
            0, 0, 0, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_pillars_conv2d_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_pillars_conv2d_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWMaxPoolingLayer *>(layers[2]))
        ->createMaxPoolingLayer<float, float>(
            targetImpl, layers[1]->getOutputTensor(0), 1, 100, 1, 100, 0, 0, 0,
            0, 0, 0, "FLOAT", 1, "SSCB", 0);
    (static_cast<MWInputLayer *>(layers[3]))
        ->createInputLayer(targetImpl, inputTensors[0], "SSCB", 2);
    (static_cast<MWScatterLayer_detector0_01 *>(layers[4]))
        ->createCustomLayer(targetImpl, layers[2]->getOutputTensor(0),
                            layers[3]->getOutputTensor(0), "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[5]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[4]->getOutputTensor(0), 3, 3, 64, 64, 2, 2, 0,
            1, 0, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_conv2d0_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_conv2d0_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[6]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[5]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn0_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn0_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn0_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn0_trainedVariance.bin",
            64, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[7]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[6]->getOutputTensor(0), 3, 3, 64, 64, 1, 1, 1,
            1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_conv2d1_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_conv2d1_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[8]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[7]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn1_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn1_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn1_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn1_trainedVariance.bin",
            64, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[9]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[8]->getOutputTensor(0), 3, 3, 64, 64, 1, 1, 1,
            1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_conv2d2_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_conv2d2_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[10]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[9]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn2_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn2_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn2_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn2_trainedVariance.bin",
            64, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[11]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[10]->getOutputTensor(0), 3, 3, 64, 64, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_conv2d3_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_conv2d3_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[12]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[11]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn3_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn3_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn3_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block1_bn3_trainedVariance.bin",
            64, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[13]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[12]->getOutputTensor(0), 3, 3, 64, 128, 2, 2,
            0, 1, 0, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d0_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d0_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[14]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[13]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn0_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn0_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn0_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn0_trainedVariance.bin",
            128, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[15]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[14]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d1_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d1_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[16]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[15]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn1_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn1_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn1_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn1_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[17]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[16]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d2_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d2_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[18]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[17]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn2_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn2_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn2_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn2_trainedVariance.bin",
            128, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[19]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[18]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d3_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d3_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[20]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[19]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn3_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn3_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn3_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn3_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[21]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[20]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d4_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d4_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[22]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[21]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn4_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn4_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn4_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn4_trainedVariance.bin",
            128, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[23]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[22]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d5_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_conv2d5_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[24]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[23]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn5_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn5_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn5_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block2_bn5_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[25]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[12]->getOutputTensor(0), 3, 3, 64, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up1_conv2dt_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up1_conv2dt_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[26]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[25]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up1_bn_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up1_bn_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up1_bn_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up1_bn_trainedVariance.bin",
            128, "SSCB", 0);
    (static_cast<MWResize2DLayer_detector0_01 *>(layers[27]))
        ->createCustomLayer(targetImpl, layers[24]->getOutputTensor(0), "SSCB",
                            1);
    (static_cast<MWFusedConvActivationLayer *>(layers[28]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[27]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up2_conv2dt_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up2_conv2dt_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            3);
    (static_cast<MWBatchNormalizationLayer *>(layers[29]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[28]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up2_bn_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up2_bn_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up2_bn_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up2_bn_trainedVariance.bin",
            128, "SSCB", 3);
    (static_cast<MWFusedConvActivationLayer *>(layers[30]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[24]->getOutputTensor(0), 3, 3, 128, 128, 2, 2,
            0, 1, 0, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d0_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d0_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[31]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[30]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn0_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn0_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn0_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn0_trainedVariance.bin",
            128, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[32]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[31]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d1_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d1_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[33]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[32]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn1_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn1_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn1_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn1_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[34]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[33]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d2_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d2_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[35]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[34]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn2_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn2_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn2_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn2_trainedVariance.bin",
            128, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[36]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[35]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d3_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d3_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[37]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[36]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn3_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn3_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn3_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn3_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[38]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[37]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d4_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d4_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[39]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[38]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn4_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn4_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn4_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn4_trainedVariance.bin",
            128, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[40]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[39]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d5_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_conv2d5_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[41]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[40]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn5_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn5_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn5_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_block3_bn5_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWResize2DLayer_detector0_02 *>(layers[42]))
        ->createCustomLayer(targetImpl, layers[41]->getOutputTensor(0), "SSCB",
                            1);
    (static_cast<MWFusedConvActivationLayer *>(layers[43]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[42]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up3_conv2dt_w.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up3_conv2dt_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[44]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[43]->getOutputTensor(0), 0.001F,
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up3_bn_offset.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up3_bn_scale.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up3_bn_trainedMean.bin",
            "./codegen/exe/deployedPointPillarModel/"
            "cnn_detector0_0_cnn_up3_bn_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWConcatenationLayer *>(layers[45]))
        ->createConcatenationLayer(
            targetImpl, 3, layers[26]->getOutputTensor(0),
            layers[29]->getOutputTensor(0), layers[44]->getOutputTensor(0), 3,
            "SSCB", 1);
    (static_cast<MWConvLayer *>(layers[46]))
        ->createConvLayer(targetImpl, layers[45]->getOutputTensor(0), 1, 1, 384,
                          20, 1, 1, 0, 0, 0, 0, 1, 1, 1,
                          "./codegen/exe/deployedPointPillarModel/"
                          "cnn_detector0_0_angle_conv2d_w_fused.bin",
                          "./codegen/exe/deployedPointPillarModel/"
                          "cnn_detector0_0_angle_conv2d_b_fused.bin",
                          "SSCB", 0);
    (static_cast<MWSplittingLayer *>(layers[47]))
        ->createSplittingLayer(targetImpl, layers[46]->getOutputTensor(0), 6, 2,
                               2, 2, 2, 6, 6, 6, "SSCB", "SSCB", "SSCB", "SSCB",
                               "SSCB", "SSCB", 0, 0, 0, 0, 0, 0);
    (static_cast<MWSigmoidLayer *>(layers[48]))
        ->createSigmoidLayer(targetImpl, layers[47]->getOutputTensor(2), "SSCB",
                             0);
    (static_cast<MWOutputLayer *>(layers[49]))
        ->createOutputLayer(targetImpl, layers[48]->getOutputTensor(0), "SSCB",
                            0);
    (static_cast<MWOutputLayer *>(layers[50]))
        ->createOutputLayer(targetImpl, layers[47]->getOutputTensor(0), "SSCB",
                            0);
    (static_cast<MWSigmoidLayer *>(layers[51]))
        ->createSigmoidLayer(targetImpl, layers[47]->getOutputTensor(1), "SSCB",
                             0);
    (static_cast<MWOutputLayer *>(layers[52]))
        ->createOutputLayer(targetImpl, layers[51]->getOutputTensor(0), "SSCB",
                            0);
    (static_cast<MWOutputLayer *>(layers[53]))
        ->createOutputLayer(targetImpl, layers[47]->getOutputTensor(4), "SSCB",
                            0);
    (static_cast<MWSigmoidLayer *>(layers[54]))
        ->createSigmoidLayer(targetImpl, layers[47]->getOutputTensor(3), "SSCB",
                             0);
    (static_cast<MWOutputLayer *>(layers[55]))
        ->createOutputLayer(targetImpl, layers[54]->getOutputTensor(0), "SSCB",
                            0);
    (static_cast<MWOutputLayer *>(layers[56]))
        ->createOutputLayer(targetImpl, layers[47]->getOutputTensor(5), "SSCB",
                            0);
    outputTensors[0] = layers[49]->getOutputTensor(0);
    outputTensors[1] = layers[50]->getOutputTensor(0);
    outputTensors[2] = layers[52]->getOutputTensor(0);
    outputTensors[3] = layers[53]->getOutputTensor(0);
    outputTensors[4] = layers[55]->getOutputTensor(0);
    outputTensors[5] = layers[56]->getOutputTensor(0);
    setSize();
  }
  isInitialized = true;
}

//
// Arguments    : hipError_t errCode
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int b_line)
{
  if ((errCode != hipSuccess) && (errCode != hipErrorDeinitialized)) {
    printf(errorString, hipGetErrorString(errCode), file, b_line);
  }
}

//
// Arguments    : const char *errMsg
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
static void checkRunTimeError(const char *errMsg, const char *file,
                              unsigned int b_line)
{
  printf(errStringBase, errMsg, b_line, file);
  exit(EXIT_FAILURE);
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double dv[124]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void dlnetwork_layerPredictWithColMajority_kernel21(double dv[124])
{
  unsigned long long threadId;
  int i;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 124) {
    dv[i] = static_cast<double>(i) + 1.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double dv[62]
// Return Type  : void
//
static __global__ __launch_bounds__(
    64, 1) void dlnetwork_layerPredictWithColMajority_kernel26(double dv[62])
{
  unsigned long long threadId;
  int i;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 62) {
    dv[i] = static_cast<double>(i) + 1.0;
  }
}

//
// Arguments    : int layerIdx
// Return Type  : void
//
void detector0_0::activations(int layerIdx)
{
  for (int idx{0}; idx <= layerIdx; idx++) {
    layers[idx]->predict();
  }
}

//
// Arguments    : void
// Return Type  : ::detector0_0
//
detector0_0::detector0_0()
{
  numLayers = 57;
  isInitialized = false;
  targetImpl = 0;
  layers[0] = new MWInputLayer;
  layers[0]->setName("pillars|input");
  layers[1] = new MWFusedConvActivationLayer;
  layers[1]->setName("pillars|conv2d_pillars|relu");
  layers[2] = new MWMaxPoolingLayer;
  layers[2]->setName("pillars|reshape");
  layers[3] = new MWInputLayer;
  layers[3]->setName("pillars|indices|reshape");
  layers[4] = new MWScatterLayer_detector0_01;
  layers[4]->setName("pillars|scatter_nd");
  layers[5] = new MWFusedConvActivationLayer;
  layers[5]->setName("cnn|block1|conv2d0_cnn|block1|conv2d0_relu");
  layers[6] = new MWBatchNormalizationLayer;
  layers[6]->setName("cnn|block1|bn0");
  layers[6]->setInPlaceIndex(0, 0);
  layers[7] = new MWFusedConvActivationLayer;
  layers[7]->setName("cnn|block1|conv2d1_cnn|block1|conv2d1_relu");
  layers[8] = new MWBatchNormalizationLayer;
  layers[8]->setName("cnn|block1|bn1");
  layers[8]->setInPlaceIndex(0, 0);
  layers[9] = new MWFusedConvActivationLayer;
  layers[9]->setName("cnn|block1|conv2d2_cnn|block1|conv2d2_relu");
  layers[10] = new MWBatchNormalizationLayer;
  layers[10]->setName("cnn|block1|bn2");
  layers[10]->setInPlaceIndex(0, 0);
  layers[11] = new MWFusedConvActivationLayer;
  layers[11]->setName("cnn|block1|conv2d3_cnn|block1|conv2d3_relu");
  layers[12] = new MWBatchNormalizationLayer;
  layers[12]->setName("cnn|block1|bn3");
  layers[12]->setInPlaceIndex(0, 0);
  layers[13] = new MWFusedConvActivationLayer;
  layers[13]->setName("cnn|block2|conv2d0_cnn|block2|conv2d0_relu");
  layers[14] = new MWBatchNormalizationLayer;
  layers[14]->setName("cnn|block2|bn0");
  layers[14]->setInPlaceIndex(0, 0);
  layers[15] = new MWFusedConvActivationLayer;
  layers[15]->setName("cnn|block2|conv2d1_cnn|block2|conv2d1_relu");
  layers[16] = new MWBatchNormalizationLayer;
  layers[16]->setName("cnn|block2|bn1");
  layers[16]->setInPlaceIndex(0, 0);
  layers[17] = new MWFusedConvActivationLayer;
  layers[17]->setName("cnn|block2|conv2d2_cnn|block2|conv2d2_relu");
  layers[18] = new MWBatchNormalizationLayer;
  layers[18]->setName("cnn|block2|bn2");
  layers[18]->setInPlaceIndex(0, 0);
  layers[19] = new MWFusedConvActivationLayer;
  layers[19]->setName("cnn|block2|conv2d3_cnn|block2|conv2d3_relu");
  layers[20] = new MWBatchNormalizationLayer;
  layers[20]->setName("cnn|block2|bn3");
  layers[20]->setInPlaceIndex(0, 0);
  layers[21] = new MWFusedConvActivationLayer;
  layers[21]->setName("cnn|block2|conv2d4_cnn|block2|conv2d4_relu");
  layers[22] = new MWBatchNormalizationLayer;
  layers[22]->setName("cnn|block2|bn4");
  layers[22]->setInPlaceIndex(0, 0);
  layers[23] = new MWFusedConvActivationLayer;
  layers[23]->setName("cnn|block2|conv2d5_cnn|block2|conv2d5_relu");
  layers[24] = new MWBatchNormalizationLayer;
  layers[24]->setName("cnn|block2|bn5");
  layers[24]->setInPlaceIndex(0, 0);
  layers[25] = new MWFusedConvActivationLayer;
  layers[25]->setName("cnn|up1|conv2dt_cnn|up1|conv2dt_relu");
  layers[26] = new MWBatchNormalizationLayer;
  layers[26]->setName("cnn|up1|bn");
  layers[26]->setInPlaceIndex(0, 0);
  layers[27] = new MWResize2DLayer_detector0_01;
  layers[27]->setName("cnn|add|resize|2");
  layers[28] = new MWFusedConvActivationLayer;
  layers[28]->setName("cnn|up2|conv2dt_cnn|up2|conv2dt_relu");
  layers[29] = new MWBatchNormalizationLayer;
  layers[29]->setName("cnn|up2|bn");
  layers[29]->setInPlaceIndex(0, 0);
  layers[30] = new MWFusedConvActivationLayer;
  layers[30]->setName("cnn|block3|conv2d0_cnn|block3|conv2d0_relu");
  layers[31] = new MWBatchNormalizationLayer;
  layers[31]->setName("cnn|block3|bn0");
  layers[31]->setInPlaceIndex(0, 0);
  layers[32] = new MWFusedConvActivationLayer;
  layers[32]->setName("cnn|block3|conv2d1_cnn|block3|conv2d1_relu");
  layers[33] = new MWBatchNormalizationLayer;
  layers[33]->setName("cnn|block3|bn1");
  layers[33]->setInPlaceIndex(0, 0);
  layers[34] = new MWFusedConvActivationLayer;
  layers[34]->setName("cnn|block3|conv2d2_cnn|block3|conv2d2_relu");
  layers[35] = new MWBatchNormalizationLayer;
  layers[35]->setName("cnn|block3|bn2");
  layers[35]->setInPlaceIndex(0, 0);
  layers[36] = new MWFusedConvActivationLayer;
  layers[36]->setName("cnn|block3|conv2d3_cnn|block3|conv2d3_relu");
  layers[37] = new MWBatchNormalizationLayer;
  layers[37]->setName("cnn|block3|bn3");
  layers[37]->setInPlaceIndex(0, 0);
  layers[38] = new MWFusedConvActivationLayer;
  layers[38]->setName("cnn|block3|conv2d4_cnn|block3|conv2d4_relu");
  layers[39] = new MWBatchNormalizationLayer;
  layers[39]->setName("cnn|block3|bn4");
  layers[39]->setInPlaceIndex(0, 0);
  layers[40] = new MWFusedConvActivationLayer;
  layers[40]->setName("cnn|block3|conv2d5_cnn|block3|conv2d5_relu");
  layers[41] = new MWBatchNormalizationLayer;
  layers[41]->setName("cnn|block3|bn5");
  layers[41]->setInPlaceIndex(0, 0);
  layers[42] = new MWResize2DLayer_detector0_02;
  layers[42]->setName("cnn|add|resize|3");
  layers[43] = new MWFusedConvActivationLayer;
  layers[43]->setName("cnn|up3|conv2dt_cnn|up3|conv2dt_relu");
  layers[44] = new MWBatchNormalizationLayer;
  layers[44]->setName("cnn|up3|bn");
  layers[44]->setInPlaceIndex(0, 0);
  layers[45] = new MWConcatenationLayer;
  layers[45]->setName("cnn|concatenate");
  layers[46] = new MWConvLayer;
  layers[46]->setName("angle|conv2d_fused");
  layers[47] = new MWSplittingLayer;
  layers[47]->setName("angle|conv2d_fused_channelSplit");
  layers[47]->setInPlaceIndex(0, 0);
  layers[47]->setInPlaceIndex(1, 0);
  layers[47]->setInPlaceIndex(2, 0);
  layers[47]->setInPlaceIndex(3, 0);
  layers[47]->setInPlaceIndex(4, 0);
  layers[47]->setInPlaceIndex(5, 0);
  layers[48] = new MWSigmoidLayer;
  layers[48]->setName("activation");
  layers[48]->setInPlaceIndex(0, 0);
  layers[49] = new MWOutputLayer;
  layers[49]->setName("output_activation");
  layers[49]->setInPlaceIndex(0, 0);
  layers[50] = new MWOutputLayer;
  layers[50]->setName("output_angle|conv2d");
  layers[50]->setInPlaceIndex(0, 0);
  layers[51] = new MWSigmoidLayer;
  layers[51]->setName("heading|conv2dSigmoid");
  layers[51]->setInPlaceIndex(0, 0);
  layers[52] = new MWOutputLayer;
  layers[52]->setName("output_heading|conv2dSigmoid");
  layers[52]->setInPlaceIndex(0, 0);
  layers[53] = new MWOutputLayer;
  layers[53]->setName("output_loc|conv2d");
  layers[53]->setInPlaceIndex(0, 0);
  layers[54] = new MWSigmoidLayer;
  layers[54]->setName("occupancy|conv2dSigmoid");
  layers[54]->setInPlaceIndex(0, 0);
  layers[55] = new MWOutputLayer;
  layers[55]->setName("output_occupancy|conv2dSigmoid");
  layers[55]->setInPlaceIndex(0, 0);
  layers[56] = new MWOutputLayer;
  layers[56]->setName("output_size|conv2d");
  layers[56]->setInPlaceIndex(0, 0);
  targetImpl = new MWCudnnTarget::MWTargetNetworkImpl;
  inputTensors[0] = new MWTensor<float>;
  inputTensors[0]->setHeight(12000);
  inputTensors[0]->setWidth(2);
  inputTensors[0]->setChannels(1);
  inputTensors[0]->setBatchSize(1);
  inputTensors[0]->setSequenceLength(1);
  inputTensors[1] = new MWTensor<float>;
  inputTensors[1]->setHeight(12000);
  inputTensors[1]->setWidth(100);
  inputTensors[1]->setChannels(9);
  inputTensors[1]->setBatchSize(1);
  inputTensors[1]->setSequenceLength(1);
}

//
// Arguments    : void
// Return Type  : void
//
detector0_0::~detector0_0()
{
  try {
    if (isInitialized) {
      cleanup();
    }
    for (int idx{0}; idx < 57; idx++) {
      delete layers[idx];
    }
    if (targetImpl) {
      delete targetImpl;
    }
    delete inputTensors[0];
    delete inputTensors[1];
  } catch (...) {
  }
}

//
// Arguments    : void
// Return Type  : int
//
int detector0_0::getBatchSize()
{
  return inputTensors[0]->getBatchSize();
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *detector0_0::getInputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(inputTensors[b_index]))->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *detector0_0::getInputDataPointer()
{
  return (static_cast<MWTensor<float> *>(inputTensors[0]))->getData();
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : float *
//
float *detector0_0::getLayerOutput(int layerIndex, int portIndex)
{
  return layers[layerIndex]->getLayerOutput(portIndex);
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : int
//
int detector0_0::getLayerOutputSize(int layerIndex, int portIndex)
{
  return static_cast<unsigned int>(
             layers[layerIndex]->getOutputTensor(portIndex)->getNumElements()) *
         sizeof(float);
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *detector0_0::getOutputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(outputTensors[b_index]))->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *detector0_0::getOutputDataPointer()
{
  return (static_cast<MWTensor<float> *>(outputTensors[0]))->getData();
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : int
//
int detector0_0::getOutputSequenceLength(int layerIndex, int portIndex)
{
  return layers[layerIndex]->getOutputTensor(portIndex)->getSequenceLength();
}

//
// Arguments    : const float varargin_1[428544]
//                float varargout_1[6856704]
// Return Type  : void
//
namespace coder {
namespace internal {
void b_dlnetwork_layerPredictWithColMajority(const float varargin_1[428544],
                                             float varargout_1[6856704])
{
  static const double cpu_dv[248]{
      1.0,    1.0,    1.125,  1.375,  1.625,  1.875,  2.125,  2.375,  2.625,
      2.875,  3.125,  3.375,  3.625,  3.875,  4.125,  4.375,  4.625,  4.875,
      5.125,  5.375,  5.625,  5.875,  6.125,  6.375,  6.625,  6.875,  7.125,
      7.375,  7.625,  7.875,  8.125,  8.375,  8.625,  8.875,  9.125,  9.375,
      9.625,  9.875,  10.125, 10.375, 10.625, 10.875, 11.125, 11.375, 11.625,
      11.875, 12.125, 12.375, 12.625, 12.875, 13.125, 13.375, 13.625, 13.875,
      14.125, 14.375, 14.625, 14.875, 15.125, 15.375, 15.625, 15.875, 16.125,
      16.375, 16.625, 16.875, 17.125, 17.375, 17.625, 17.875, 18.125, 18.375,
      18.625, 18.875, 19.125, 19.375, 19.625, 19.875, 20.125, 20.375, 20.625,
      20.875, 21.125, 21.375, 21.625, 21.875, 22.125, 22.375, 22.625, 22.875,
      23.125, 23.375, 23.625, 23.875, 24.125, 24.375, 24.625, 24.875, 25.125,
      25.375, 25.625, 25.875, 26.125, 26.375, 26.625, 26.875, 27.125, 27.375,
      27.625, 27.875, 28.125, 28.375, 28.625, 28.875, 29.125, 29.375, 29.625,
      29.875, 30.125, 30.375, 30.625, 30.875, 31.125, 31.375, 31.625, 31.875,
      32.125, 32.375, 32.625, 32.875, 33.125, 33.375, 33.625, 33.875, 34.125,
      34.375, 34.625, 34.875, 35.125, 35.375, 35.625, 35.875, 36.125, 36.375,
      36.625, 36.875, 37.125, 37.375, 37.625, 37.875, 38.125, 38.375, 38.625,
      38.875, 39.125, 39.375, 39.625, 39.875, 40.125, 40.375, 40.625, 40.875,
      41.125, 41.375, 41.625, 41.875, 42.125, 42.375, 42.625, 42.875, 43.125,
      43.375, 43.625, 43.875, 44.125, 44.375, 44.625, 44.875, 45.125, 45.375,
      45.625, 45.875, 46.125, 46.375, 46.625, 46.875, 47.125, 47.375, 47.625,
      47.875, 48.125, 48.375, 48.625, 48.875, 49.125, 49.375, 49.625, 49.875,
      50.125, 50.375, 50.625, 50.875, 51.125, 51.375, 51.625, 51.875, 52.125,
      52.375, 52.625, 52.875, 53.125, 53.375, 53.625, 53.875, 54.125, 54.375,
      54.625, 54.875, 55.125, 55.375, 55.625, 55.875, 56.125, 56.375, 56.625,
      56.875, 57.125, 57.375, 57.625, 57.875, 58.125, 58.375, 58.625, 58.875,
      59.125, 59.375, 59.625, 59.875, 60.125, 60.375, 60.625, 60.875, 61.125,
      61.375, 61.625, 61.875, 62.0,   62.0};
  double(*gpu_dv)[62];
  float(*c_gpu_tmp)[6856704];
  float(*b_gpu_tmp)[1714176];
  float(*gpu_tmp)[1714176];
  if (!b_gpuConstsCopied_dlnetwork_layerPredictWithColMajority) {
    b_gpuConstsCopied_dlnetwork_layerPredictWithColMajority = true;
    hipMemcpy(*b_global_gpu_dv, cpu_dv, sizeof(double[248]),
               hipMemcpyHostToDevice);
  }
  hipMalloc(&gpu_tmp, 6856704ULL);
  hipMalloc(&b_gpu_tmp, 6856704ULL);
  hipMalloc(&c_gpu_tmp, 27426816ULL);
  hipMalloc(&gpu_dv, 496ULL);
  dlnetwork_layerPredictWithColMajority_kernel26<<<dim3(1U, 1U, 1U),
                                                   dim3(64U, 1U, 1U)>>>(
      *gpu_dv);
  nnet::internal::cnn::coder::b_interpAlongSpatialDim(varargin_1, *gpu_tmp);
  b_permute(*gpu_tmp, *b_gpu_tmp);
  nnet::internal::cnn::coder::b_interp1CustomImpl(*gpu_dv, *b_gpu_tmp,
                                                  *b_global_gpu_dv, *c_gpu_tmp);
  ipermute(*c_gpu_tmp, varargout_1);
  hipFree(*gpu_dv);
  hipFree(*c_gpu_tmp);
  hipFree(*b_gpu_tmp);
  hipFree(*gpu_tmp);
}

//
// Arguments    : const float gpu_varargin_1[768000]
//                const float gpu_varargin_2[24000]
//                float gpu_varargout_1[13713408]
// Return Type  : void
//
void dlnetwork_layerPredictWithColMajority(const float gpu_varargin_1[768000],
                                           const float gpu_varargin_2[24000],
                                           float gpu_varargout_1[13713408])
{
  array<float, 2U> maps;
  array<int, 1U> iv1;
  array<int, 1U> r;
  array<int, 1U> r1;
  float(*cpu_varargout_1)[13713408];
  float(*cpu_varargin_1)[768000];
  float(*cpu_varargin_2)[24000];
  int iv1_idx_0;
  int trueCount;
  bool indices[12000];
  bool varargin_1_outdatedOnCpu;
  bool varargin_2_outdatedOnCpu;
  cpu_varargout_1 = (float(*)[13713408])malloc(54853632U);
  cpu_varargin_2 = (float(*)[24000])malloc(96000U);
  cpu_varargin_1 = (float(*)[768000])malloc(3072000U);
  varargin_2_outdatedOnCpu = true;
  varargin_1_outdatedOnCpu = true;
  hipMemcpy(*cpu_varargout_1, gpu_varargout_1, 54853632ULL,
             hipMemcpyDeviceToHost);
  std::memset(&(*cpu_varargout_1)[0], 0, 13713408U * sizeof(float));
  trueCount = 0;
  for (int i{0}; i < 12000; i++) {
    bool b;
    if (varargin_2_outdatedOnCpu) {
      hipMemcpy(*cpu_varargin_2, gpu_varargin_2, 96000ULL,
                 hipMemcpyDeviceToHost);
    }
    varargin_2_outdatedOnCpu = false;
    b = ((*cpu_varargin_2)[i] >= 1.0F);
    indices[i] = b;
    if (b) {
      trueCount++;
    }
  }
  r.set_size(trueCount);
  trueCount = 0;
  for (int i{0}; i < 12000; i++) {
    if (indices[i]) {
      r[trueCount] = i;
      trueCount++;
    }
  }
  maps.set_size(r.size(0), 2);
  for (trueCount = 0; trueCount < 2; trueCount++) {
    for (int i{0}; i < r.size(0); i++) {
      if (varargin_2_outdatedOnCpu) {
        hipMemcpy(*cpu_varargin_2, gpu_varargin_2, 96000ULL,
                   hipMemcpyDeviceToHost);
      }
      varargin_2_outdatedOnCpu = false;
      maps[i + maps.size(0) * trueCount] =
          (*cpu_varargin_2)[r[i] + 12000 * trueCount] - 1.0F;
    }
  }
  iv1.set_size(maps.size(0));
  for (trueCount = 0; trueCount < maps.size(0); trueCount++) {
    iv1[trueCount] = static_cast<int>((maps[trueCount + maps.size(0)] * 432.0F +
                                       maps[trueCount]) +
                                      1.0F) -
                     1;
  }
  trueCount = 0;
  for (int i{0}; i < 12000; i++) {
    if (indices[i]) {
      trueCount++;
    }
  }
  r1.set_size(trueCount);
  trueCount = 0;
  for (int i{0}; i < 12000; i++) {
    if (indices[i]) {
      r1[trueCount] = i;
      trueCount++;
    }
  }
  iv1_idx_0 = iv1.size(0);
  for (trueCount = 0; trueCount < 64; trueCount++) {
    for (int i{0}; i < iv1_idx_0; i++) {
      if (varargin_1_outdatedOnCpu) {
        hipMemcpy(*cpu_varargin_1, gpu_varargin_1, 3072000ULL,
                   hipMemcpyDeviceToHost);
      }
      varargin_1_outdatedOnCpu = false;
      (*cpu_varargout_1)[iv1[i] + 214272 * trueCount] =
          (*cpu_varargin_1)[r1[i] + 12000 * trueCount];
    }
  }
  hipMemcpy(gpu_varargout_1, *cpu_varargout_1, 54853632ULL,
             hipMemcpyHostToDevice);
  free(*cpu_varargin_1);
  free(*cpu_varargin_2);
  free(*cpu_varargout_1);
}

//
// Arguments    : const float varargin_1[1714176]
//                float varargout_1[6856704]
// Return Type  : void
//
void dlnetwork_layerPredictWithColMajority(const float varargin_1[1714176],
                                           float varargout_1[6856704])
{
  static const double cpu_dv[248]{
      1.0,    1.25,   1.75,   2.25,   2.75,   3.25,   3.75,   4.25,   4.75,
      5.25,   5.75,   6.25,   6.75,   7.25,   7.75,   8.25,   8.75,   9.25,
      9.75,   10.25,  10.75,  11.25,  11.75,  12.25,  12.75,  13.25,  13.75,
      14.25,  14.75,  15.25,  15.75,  16.25,  16.75,  17.25,  17.75,  18.25,
      18.75,  19.25,  19.75,  20.25,  20.75,  21.25,  21.75,  22.25,  22.75,
      23.25,  23.75,  24.25,  24.75,  25.25,  25.75,  26.25,  26.75,  27.25,
      27.75,  28.25,  28.75,  29.25,  29.75,  30.25,  30.75,  31.25,  31.75,
      32.25,  32.75,  33.25,  33.75,  34.25,  34.75,  35.25,  35.75,  36.25,
      36.75,  37.25,  37.75,  38.25,  38.75,  39.25,  39.75,  40.25,  40.75,
      41.25,  41.75,  42.25,  42.75,  43.25,  43.75,  44.25,  44.75,  45.25,
      45.75,  46.25,  46.75,  47.25,  47.75,  48.25,  48.75,  49.25,  49.75,
      50.25,  50.75,  51.25,  51.75,  52.25,  52.75,  53.25,  53.75,  54.25,
      54.75,  55.25,  55.75,  56.25,  56.75,  57.25,  57.75,  58.25,  58.75,
      59.25,  59.75,  60.25,  60.75,  61.25,  61.75,  62.25,  62.75,  63.25,
      63.75,  64.25,  64.75,  65.25,  65.75,  66.25,  66.75,  67.25,  67.75,
      68.25,  68.75,  69.25,  69.75,  70.25,  70.75,  71.25,  71.75,  72.25,
      72.75,  73.25,  73.75,  74.25,  74.75,  75.25,  75.75,  76.25,  76.75,
      77.25,  77.75,  78.25,  78.75,  79.25,  79.75,  80.25,  80.75,  81.25,
      81.75,  82.25,  82.75,  83.25,  83.75,  84.25,  84.75,  85.25,  85.75,
      86.25,  86.75,  87.25,  87.75,  88.25,  88.75,  89.25,  89.75,  90.25,
      90.75,  91.25,  91.75,  92.25,  92.75,  93.25,  93.75,  94.25,  94.75,
      95.25,  95.75,  96.25,  96.75,  97.25,  97.75,  98.25,  98.75,  99.25,
      99.75,  100.25, 100.75, 101.25, 101.75, 102.25, 102.75, 103.25, 103.75,
      104.25, 104.75, 105.25, 105.75, 106.25, 106.75, 107.25, 107.75, 108.25,
      108.75, 109.25, 109.75, 110.25, 110.75, 111.25, 111.75, 112.25, 112.75,
      113.25, 113.75, 114.25, 114.75, 115.25, 115.75, 116.25, 116.75, 117.25,
      117.75, 118.25, 118.75, 119.25, 119.75, 120.25, 120.75, 121.25, 121.75,
      122.25, 122.75, 123.25, 123.75, 124.0};
  double(*gpu_dv)[124];
  float(*c_gpu_tmp)[6856704];
  float(*b_gpu_tmp)[3428352];
  float(*gpu_tmp)[3428352];
  if (!gpuConstsCopied_dlnetwork_layerPredictWithColMajority) {
    gpuConstsCopied_dlnetwork_layerPredictWithColMajority = true;
    hipMemcpy(*global_gpu_dv, cpu_dv, sizeof(double[248]),
               hipMemcpyHostToDevice);
  }
  hipMalloc(&gpu_tmp, 13713408ULL);
  hipMalloc(&b_gpu_tmp, 13713408ULL);
  hipMalloc(&c_gpu_tmp, 27426816ULL);
  hipMalloc(&gpu_dv, 992ULL);
  dlnetwork_layerPredictWithColMajority_kernel21<<<dim3(1U, 1U, 1U),
                                                   dim3(128U, 1U, 1U)>>>(
      *gpu_dv);
  nnet::internal::cnn::coder::interpAlongSpatialDim(varargin_1, *gpu_tmp);
  permute(*gpu_tmp, *b_gpu_tmp);
  nnet::internal::cnn::coder::interp1CustomImpl(*gpu_dv, *b_gpu_tmp,
                                                *global_gpu_dv, *c_gpu_tmp);
  ipermute(*c_gpu_tmp, varargout_1);
  hipFree(*gpu_dv);
  hipFree(*c_gpu_tmp);
  hipFree(*b_gpu_tmp);
  hipFree(*gpu_tmp);
}

//
// Arguments    : detector0_0 *obj
// Return Type  : void
//
void dlnetwork_setup(detector0_0 *obj)
{
  try {
    obj->setup();
  } catch (std::runtime_error const &err) {
    obj->cleanup();
    checkRunTimeError(err.what(), __FILE__, __LINE__);
  } catch (...) {
    obj->cleanup();
    checkRunTimeError("", __FILE__, __LINE__);
  }
}

} // namespace internal
} // namespace coder

//
// File trailer for dlnetwork.cu
//
// [EOF]
//
