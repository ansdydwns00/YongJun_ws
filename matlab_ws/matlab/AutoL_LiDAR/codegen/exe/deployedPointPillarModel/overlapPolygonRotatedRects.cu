#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: overlapPolygonRotatedRects.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "overlapPolygonRotatedRects.h"
#include "div.h"
#include "inpolygon.h"
#include "mtimes.h"
#include "rt_nonfinite.h"
#include "coder_array.h"
#include "coder_gpu_array.h"
#include <cmath>

// Function Declarations
static void binary_expand_op_12(coder::array<bool, 2U> &in1,
                                const coder::array<float, 2U> &in2, int in3,
                                int in4, int in5);

// Function Definitions
//
// Arguments    : coder::array<bool, 2U> &in1
//                const coder::array<float, 2U> &in2
//                int in3
//                int in4
//                int in5
// Return Type  : void
//
static void binary_expand_op_12(coder::array<bool, 2U> &in1,
                                const coder::array<float, 2U> &in2, int in3,
                                int in4, int in5)
{
  int b;
  int stride_0_1;
  int stride_1_1;
  if ((in5 - in4) + 1 == 1) {
    b = in3 + 1;
  } else {
    b = (in5 - in4) + 1;
  }
  in1.set_size(1, b);
  stride_0_1 = (in3 + 1 != 1);
  stride_1_1 = ((in5 - in4) + 1 != 1);
  if ((in5 - in4) + 1 == 1) {
    b = in3 + 1;
  } else {
    b = (in5 - in4) + 1;
  }
  for (int i{0}; i < b; i++) {
    in1[i] = (in2[i * stride_0_1] * in2[in4 + i * stride_1_1] <= 0.0F);
  }
}

//
// Arguments    : const array<float, 1U> &xIndices1
//                const array<float, 1U> &yIndices1
//                const array<float, 1U> &xIndices2
//                const array<float, 1U> &yIndices2
//                array<float, 2U> &overlapedPolyPts
// Return Type  : void
//
namespace coder {
namespace vision {
namespace internal {
namespace detector {
void overlapPolygonRotatedRects_findOverlapPolyPoints(
    const array<float, 1U> &xIndices1, const array<float, 1U> &yIndices1,
    const array<float, 1U> &xIndices2, const array<float, 1U> &yIndices2,
    array<float, 2U> &overlapedPolyPts)
{
  array<float, 2U> b_cpu_b;
  array<float, 2U> b_polyEdges1;
  array<float, 2U> b_polyEdges2;
  array<float, 2U> cn;
  array<float, 2U> cpu_b;
  array<float, 2U> cpu_crossProd;
  array<float, 2U> cpu_transPolyEdge2;
  array<float, 2U> cpu_y1;
  array<float, 2U> cpu_y2;
  array<float, 2U> crossProdSign;
  array<float, 2U> d_y;
  array<float, 2U> e_y;
  array<float, 2U> intersectPoints;
  array<float, 2U> padEdges2;
  array<float, 2U> polyEdges1;
  array<float, 2U> polyEdges2;
  array<float, 2U> x1;
  array<float, 2U> x2;
  array<int, 2U> b_y;
  array<int, 2U> c_y;
  array<int, 2U> ii;
  array<int, 2U> r2;
  array<int, 1U> r;
  array<int, 1U> r1;
  array<bool, 2U> b_x;
  array<bool, 2U> c_x;
  array<bool, 2U> indices;
  array<bool, 2U> insideInd1;
  array<bool, 2U> insideInd2;
  gpu_array<float, 2U> b_gpu_b;
  gpu_array<float, 2U> gpu_b;
  gpu_array<float, 2U> gpu_crossProd;
  gpu_array<float, 2U> gpu_transPolyEdge2;
  gpu_array<float, 2U> gpu_y1;
  gpu_array<float, 2U> gpu_y2;
  float cpu_dist[2];
  float(*gpu_dist)[2];
  int acoef;
  int csz_idx_1;
  int i;
  int idx;
  int k;
  int nx;
  unsigned int sz_idx_1;
  bool b_b_outdatedOnCpu;
  bool b_b_outdatedOnGpu;
  bool b_outdatedOnCpu;
  bool b_outdatedOnGpu;
  bool crossProd_outdatedOnCpu;
  bool crossProd_outdatedOnGpu;
  bool exitg1;
  bool p;
  bool transPolyEdge2_outdatedOnCpu;
  bool transPolyEdge2_outdatedOnGpu;
  bool y1_outdatedOnCpu;
  bool y1_outdatedOnGpu;
  bool y2_outdatedOnCpu;
  bool y2_outdatedOnGpu;
  hipMalloc(&gpu_dist, 8ULL);
  y2_outdatedOnCpu = false;
  y2_outdatedOnGpu = false;
  b_b_outdatedOnCpu = false;
  b_b_outdatedOnGpu = false;
  y1_outdatedOnCpu = false;
  y1_outdatedOnGpu = false;
  b_outdatedOnCpu = false;
  b_outdatedOnGpu = false;
  crossProd_outdatedOnCpu = false;
  crossProd_outdatedOnGpu = false;
  transPolyEdge2_outdatedOnCpu = false;
  transPolyEdge2_outdatedOnGpu = false;
  polyEdges1.set_size(2, xIndices1.size(0));
  nx = xIndices1.size(0);
  for (k = 0; k < nx; k++) {
    polyEdges1[2 * k] = xIndices1[k];
  }
  nx = yIndices1.size(0);
  for (k = 0; k < nx; k++) {
    polyEdges1[2 * k + 1] = yIndices1[k];
  }
  polyEdges2.set_size(2, xIndices2.size(0));
  nx = xIndices2.size(0);
  for (k = 0; k < nx; k++) {
    polyEdges2[2 * k] = xIndices2[k];
  }
  nx = yIndices2.size(0);
  for (k = 0; k < nx; k++) {
    polyEdges2[2 * k + 1] = yIndices2[k];
  }
  p = true;
  k = 0;
  exitg1 = false;
  while ((!exitg1) && (k < 2)) {
    if (!(polyEdges1[k] == polyEdges1[k + 2 * (polyEdges1.size(1) - 1)])) {
      p = false;
      exitg1 = true;
    } else {
      k++;
    }
  }
  if (!p) {
    b_polyEdges1.set_size(2, polyEdges1.size(1) + 1);
    nx = polyEdges1.size(1);
    for (k = 0; k < nx; k++) {
      b_polyEdges1[2 * k] = polyEdges1[2 * k];
      b_polyEdges1[2 * k + 1] = polyEdges1[2 * k + 1];
    }
    b_polyEdges1[2 * polyEdges1.size(1)] = polyEdges1[0];
    b_polyEdges1[2 * polyEdges1.size(1) + 1] = polyEdges1[1];
    polyEdges1.set_size(2, b_polyEdges1.size(1));
    for (k = 0; k < (b_polyEdges1.size(1) << 1); k++) {
      polyEdges1[k] = b_polyEdges1[k];
    }
  }
  p = true;
  k = 0;
  exitg1 = false;
  while ((!exitg1) && (k < 2)) {
    if (!(polyEdges2[k] == polyEdges2[k + 2 * (polyEdges2.size(1) - 1)])) {
      p = false;
      exitg1 = true;
    } else {
      k++;
    }
  }
  if (!p) {
    b_polyEdges2.set_size(2, polyEdges2.size(1) + 1);
    nx = polyEdges2.size(1);
    for (k = 0; k < nx; k++) {
      b_polyEdges2[2 * k] = polyEdges2[2 * k];
      b_polyEdges2[2 * k + 1] = polyEdges2[2 * k + 1];
    }
    b_polyEdges2[2 * polyEdges2.size(1)] = polyEdges2[0];
    b_polyEdges2[2 * polyEdges2.size(1) + 1] = polyEdges2[1];
    polyEdges2.set_size(2, b_polyEdges2.size(1));
    for (k = 0; k < (b_polyEdges2.size(1) << 1); k++) {
      polyEdges2[k] = b_polyEdges2[k];
    }
  }
  if (polyEdges1.size(1) > polyEdges2.size(1)) {
    padEdges2.set_size(2, polyEdges1.size(1));
    for (k = 0; k < (polyEdges1.size(1) << 1); k++) {
      padEdges2[k] = polyEdges1[k];
    }
    polyEdges1.set_size(2, polyEdges2.size(1));
    for (k = 0; k < (polyEdges2.size(1) << 1); k++) {
      polyEdges1[k] = polyEdges2[k];
    }
    polyEdges2.set_size(2, padEdges2.size(1));
    for (k = 0; k < (padEdges2.size(1) << 1); k++) {
      polyEdges2[k] = padEdges2[k];
    }
  }
  intersectPoints.set_size(2, 0);
  i = polyEdges1.size(1);
  if (polyEdges1.size(1) - 2 >= 0) {
    csz_idx_1 = polyEdges2.size(1);
  }
  for (int n{0}; n <= i - 2; n++) {
    float point2_idx_0;
    float point2_idx_1;
    float u;
    cpu_transPolyEdge2.set_size(2, csz_idx_1);
    if (!transPolyEdge2_outdatedOnGpu) {
      gpu_transPolyEdge2.set_size(&cpu_transPolyEdge2);
    }
    if (csz_idx_1 != 0) {
      acoef = (polyEdges2.size(1) != 1);
      nx = csz_idx_1 - 1;
      for (k = 0; k <= nx; k++) {
        idx = acoef * k;
        if (transPolyEdge2_outdatedOnCpu) {
          arrayCopyGpuToCpu(cpu_transPolyEdge2, gpu_transPolyEdge2);
        }
        cpu_transPolyEdge2[2 * k] = polyEdges2[2 * idx] - polyEdges1[2 * n];
        cpu_transPolyEdge2[2 * k + 1] =
            polyEdges2[2 * idx + 1] - polyEdges1[2 * n + 1];
        transPolyEdge2_outdatedOnCpu = false;
        transPolyEdge2_outdatedOnGpu = true;
      }
    }
    point2_idx_0 = polyEdges1[2 * (n + 1)] - polyEdges1[2 * n];
    point2_idx_1 = polyEdges1[2 * (n + 1) + 1] - polyEdges1[2 * n + 1];
    cpu_dist[0] = point2_idx_1;
    cpu_dist[1] = -point2_idx_0;
    hipMemcpy(*gpu_dist, cpu_dist, 8ULL, hipMemcpyHostToDevice);
    ::coder::internal::blas::mtimes(
        *gpu_dist, cpu_transPolyEdge2, &transPolyEdge2_outdatedOnCpu,
        gpu_transPolyEdge2, &transPolyEdge2_outdatedOnGpu, cpu_crossProd,
        &crossProd_outdatedOnCpu, gpu_crossProd, &crossProd_outdatedOnGpu);
    crossProdSign.set_size(1, cpu_crossProd.size(1));
    for (k = 0; k < cpu_crossProd.size(1); k++) {
      if (crossProd_outdatedOnCpu) {
        arrayCopyGpuToCpu(cpu_crossProd, gpu_crossProd);
      }
      crossProd_outdatedOnCpu = false;
      u = cpu_crossProd[k];
      if (std::isnan(u)) {
        u = rtNaNF;
      } else if (u < 0.0F) {
        u = -1.0F;
      } else {
        u = (u > 0.0F);
      }
      crossProdSign[k] = u;
    }
    if (crossProdSign.size(1) - 1 < 1) {
      idx = 0;
    } else {
      idx = crossProdSign.size(1) - 1;
    }
    if (crossProdSign.size(1) < 2) {
      acoef = 0;
      nx = 0;
    } else {
      acoef = 1;
      nx = crossProdSign.size(1);
    }
    if (idx == nx - acoef) {
      indices.set_size(1, idx);
      for (k = 0; k < idx; k++) {
        indices[k] = (crossProdSign[k] * crossProdSign[acoef + k] <= 0.0F);
      }
    } else {
      binary_expand_op_12(indices, crossProdSign, idx - 1, acoef, nx - 1);
    }
    p = false;
    nx = 1;
    exitg1 = false;
    while ((!exitg1) && (nx <= indices.size(1))) {
      if (indices[nx - 1]) {
        p = true;
        exitg1 = true;
      } else {
        nx++;
      }
    }
    if (p) {
      nx = indices.size(1);
      idx = 0;
      ii.set_size(1, indices.size(1));
      acoef = 1;
      exitg1 = false;
      while ((!exitg1) && (acoef <= nx)) {
        if (indices[acoef - 1]) {
          idx++;
          ii[idx - 1] = acoef;
          if (idx >= nx) {
            exitg1 = true;
          } else {
            acoef++;
          }
        } else {
          acoef++;
        }
      }
      if (indices.size(1) == 1) {
        if (idx == 0) {
          ii.set_size(1, 0);
        }
      } else {
        if (idx < 1) {
          idx = 0;
        }
        ii.set_size(ii.size(0), idx);
      }
      x1.set_size(1, ii.size(1));
      x2.set_size(1, ii.size(1));
      for (k = 0; k < ii.size(1); k++) {
        if (crossProd_outdatedOnCpu) {
          arrayCopyGpuToCpu(cpu_crossProd, gpu_crossProd);
        }
        x1[k] = cpu_crossProd[ii[k] - 1];
        crossProd_outdatedOnCpu = false;
        x2[k] = cpu_crossProd[ii[k]];
      }
      u = point2_idx_0 * point2_idx_0 + point2_idx_1 * point2_idx_1;
      cpu_dist[0] = point2_idx_0 / u;
      cpu_dist[1] = point2_idx_1 / u;
      cpu_b.set_size(2, ii.size(1));
      if (!b_outdatedOnGpu) {
        gpu_b.set_size(&cpu_b);
      }
      for (k = 0; k < ii.size(1); k++) {
        if (b_outdatedOnCpu) {
          arrayCopyGpuToCpu(cpu_b, gpu_b);
        }
        if (transPolyEdge2_outdatedOnCpu) {
          arrayCopyGpuToCpu(cpu_transPolyEdge2, gpu_transPolyEdge2);
        }
        cpu_b[2 * k] = cpu_transPolyEdge2[2 * (ii[k] - 1)];
        transPolyEdge2_outdatedOnCpu = false;
        cpu_b[2 * k + 1] = cpu_transPolyEdge2[2 * (ii[k] - 1) + 1];
        b_outdatedOnCpu = false;
        b_outdatedOnGpu = true;
      }
      hipMemcpy(*gpu_dist, cpu_dist, 8ULL, hipMemcpyHostToDevice);
      ::coder::internal::blas::mtimes(
          *gpu_dist, cpu_b, &b_outdatedOnCpu, gpu_b, &b_outdatedOnGpu, cpu_y1,
          &y1_outdatedOnCpu, gpu_y1, &y1_outdatedOnGpu);
      b_cpu_b.set_size(2, ii.size(1));
      if (!b_b_outdatedOnGpu) {
        b_gpu_b.set_size(&b_cpu_b);
      }
      for (k = 0; k < ii.size(1); k++) {
        if (b_b_outdatedOnCpu) {
          arrayCopyGpuToCpu(b_cpu_b, b_gpu_b);
        }
        if (transPolyEdge2_outdatedOnCpu) {
          arrayCopyGpuToCpu(cpu_transPolyEdge2, gpu_transPolyEdge2);
        }
        b_cpu_b[2 * k] = cpu_transPolyEdge2[2 * ii[k]];
        transPolyEdge2_outdatedOnCpu = false;
        b_cpu_b[2 * k + 1] = cpu_transPolyEdge2[2 * ii[k] + 1];
        b_b_outdatedOnCpu = false;
        b_b_outdatedOnGpu = true;
      }
      ::coder::internal::blas::mtimes(
          *gpu_dist, b_cpu_b, &b_b_outdatedOnCpu, b_gpu_b, &b_b_outdatedOnGpu,
          cpu_y2, &y2_outdatedOnCpu, gpu_y2, &y2_outdatedOnGpu);
      if (cpu_y1.size(1) == 1) {
        nx = x2.size(1);
      } else {
        nx = cpu_y1.size(1);
      }
      if (cpu_y2.size(1) == 1) {
        acoef = x1.size(1);
      } else {
        acoef = cpu_y2.size(1);
      }
      if (cpu_y1.size(1) == 1) {
        k = x2.size(1);
      } else {
        k = cpu_y1.size(1);
      }
      if (k == 1) {
        if (cpu_y2.size(1) == 1) {
          k = x1.size(1);
        } else {
          k = cpu_y2.size(1);
        }
      } else if (cpu_y1.size(1) == 1) {
        k = x2.size(1);
      } else {
        k = cpu_y1.size(1);
      }
      if (x2.size(1) == 1) {
        idx = x1.size(1);
      } else {
        idx = x2.size(1);
      }
      if ((cpu_y1.size(1) == x2.size(1)) && (cpu_y2.size(1) == x1.size(1)) &&
          (nx == acoef) && (x2.size(1) == x1.size(1)) && (k == idx)) {
        d_y.set_size(1, cpu_y1.size(1));
        for (k = 0; k < cpu_y1.size(1); k++) {
          float f;
          u = x2[k];
          f = x1[k];
          if (y1_outdatedOnCpu) {
            arrayCopyGpuToCpu(cpu_y1, gpu_y1);
          }
          y1_outdatedOnCpu = false;
          if (y2_outdatedOnCpu) {
            arrayCopyGpuToCpu(cpu_y2, gpu_y2);
          }
          y2_outdatedOnCpu = false;
          d_y[k] = (cpu_y1[k] * u - cpu_y2[k] * f) / (u - f);
        }
      } else {
        if (y1_outdatedOnCpu) {
          arrayCopyGpuToCpu(cpu_y1, gpu_y1);
        }
        y1_outdatedOnCpu = false;
        if (y2_outdatedOnCpu) {
          arrayCopyGpuToCpu(cpu_y2, gpu_y2);
        }
        y2_outdatedOnCpu = false;
        binary_expand_op_11(d_y, cpu_y1, x2, cpu_y2, x1);
      }
      indices.set_size(1, d_y.size(1));
      for (k = 0; k < d_y.size(1); k++) {
        indices[k] = ((d_y[k] >= 0.0F) && (d_y[k] < 1.0F));
      }
      p = false;
      nx = 1;
      exitg1 = false;
      while ((!exitg1) && (nx <= indices.size(1))) {
        if (indices[nx - 1]) {
          p = true;
          exitg1 = true;
        } else {
          nx++;
        }
      }
      if (p) {
        nx = indices.size(1) - 1;
        acoef = 0;
        for (idx = 0; idx <= nx; idx++) {
          if (indices[idx]) {
            acoef++;
          }
        }
        r2.set_size(1, acoef);
        acoef = 0;
        for (idx = 0; idx <= nx; idx++) {
          if (indices[idx]) {
            r2[acoef] = idx;
            acoef++;
          }
        }
        nx = r2.size(1);
        e_y.set_size(2, r2.size(1));
        for (k = 0; k < nx; k++) {
          e_y[2 * k] = point2_idx_0 * d_y[r2[k]];
          e_y[2 * k + 1] = point2_idx_1 * d_y[r2[k]];
        }
        cn.set_size(2, e_y.size(1));
        if (e_y.size(1) != 0) {
          nx = (e_y.size(1) != 1);
          idx = e_y.size(1) - 1;
          for (k = 0; k <= idx; k++) {
            acoef = nx * k;
            cn[2 * k] = polyEdges1[2 * n] + e_y[2 * acoef];
            cn[2 * k + 1] = polyEdges1[2 * n + 1] + e_y[2 * acoef + 1];
          }
        }
      } else {
        cn.set_size(2, 0);
      }
    } else {
      cn.set_size(2, 0);
    }
    if (cn.size(1) != 0) {
      k = intersectPoints.size(1);
      intersectPoints.set_size(2, intersectPoints.size(1) + cn.size(1));
      nx = cn.size(1);
      for (idx = 0; idx < nx; idx++) {
        intersectPoints[2 * (idx + k)] = cn[2 * idx];
        intersectPoints[2 * (idx + k) + 1] = cn[2 * idx + 1];
      }
    }
  }
  images::internal::coder::inpolygon(xIndices1, yIndices1, xIndices2, yIndices2,
                                     insideInd1);
  images::internal::coder::inpolygon(xIndices2, yIndices2, xIndices1, yIndices1,
                                     insideInd2);
  overlapedPolyPts.set_size(2, 0);
  if (intersectPoints.size(1) != 0) {
    overlapedPolyPts.set_size(2, intersectPoints.size(1));
    nx = intersectPoints.size(1);
    for (k = 0; k < nx; k++) {
      overlapedPolyPts[2 * k] = intersectPoints[2 * k];
      overlapedPolyPts[2 * k + 1] = intersectPoints[2 * k + 1];
    }
  }
  nx = insideInd1.size(0);
  if ((insideInd1.size(0) == 0) || (insideInd1.size(1) == 0)) {
    sz_idx_1 = static_cast<unsigned int>(insideInd1.size(1));
    b_y.set_size(1, insideInd1.size(1));
    for (k = 0; k < static_cast<int>(sz_idx_1); k++) {
      b_y[k] = 0;
    }
  } else {
    idx = insideInd1.size(1);
    b_y.set_size(1, insideInd1.size(1));
    for (i = 0; i < idx; i++) {
      acoef = i * insideInd1.size(0);
      b_y[i] = insideInd1[acoef];
      for (k = 0; k <= nx - 2; k++) {
        b_y[i] = b_y[i] + insideInd1[(acoef + k) + 1];
      }
    }
  }
  b_x.set_size(1, b_y.size(1));
  for (k = 0; k < b_y.size(1); k++) {
    b_x[k] = (b_y[k] > 0);
  }
  p = (b_x.size(1) != 0);
  if (p) {
    k = 1;
    exitg1 = false;
    while ((!exitg1) && (k <= b_x.size(1))) {
      if (!b_x[k - 1]) {
        p = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
  }
  if (p) {
    nx = insideInd1.size(0) * insideInd1.size(1) - 1;
    acoef = 0;
    for (i = 0; i <= nx; i++) {
      if (insideInd1[i]) {
        acoef++;
      }
    }
    r.set_size(acoef);
    acoef = 0;
    for (i = 0; i <= nx; i++) {
      if (insideInd1[i]) {
        r[acoef] = i;
        acoef++;
      }
    }
    k = overlapedPolyPts.size(1);
    overlapedPolyPts.set_size(2, r.size(0) + overlapedPolyPts.size(1));
    nx = r.size(0);
    for (idx = 0; idx < nx; idx++) {
      overlapedPolyPts[2 * (idx + k)] = xIndices1[r[idx]];
      overlapedPolyPts[2 * (idx + k) + 1] = yIndices1[r[idx]];
    }
  }
  nx = insideInd2.size(0);
  if ((insideInd2.size(0) == 0) || (insideInd2.size(1) == 0)) {
    sz_idx_1 = static_cast<unsigned int>(insideInd2.size(1));
    c_y.set_size(1, insideInd2.size(1));
    for (k = 0; k < static_cast<int>(sz_idx_1); k++) {
      c_y[k] = 0;
    }
  } else {
    idx = insideInd2.size(1);
    c_y.set_size(1, insideInd2.size(1));
    for (i = 0; i < idx; i++) {
      acoef = i * insideInd2.size(0);
      c_y[i] = insideInd2[acoef];
      for (k = 0; k <= nx - 2; k++) {
        c_y[i] = c_y[i] + insideInd2[(acoef + k) + 1];
      }
    }
  }
  c_x.set_size(1, c_y.size(1));
  for (k = 0; k < c_y.size(1); k++) {
    c_x[k] = (c_y[k] > 0);
  }
  p = (c_x.size(1) != 0);
  if (p) {
    k = 1;
    exitg1 = false;
    while ((!exitg1) && (k <= c_x.size(1))) {
      if (!c_x[k - 1]) {
        p = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
  }
  if (p) {
    nx = insideInd2.size(0) * insideInd2.size(1) - 1;
    acoef = 0;
    for (i = 0; i <= nx; i++) {
      if (insideInd2[i]) {
        acoef++;
      }
    }
    r1.set_size(acoef);
    acoef = 0;
    for (i = 0; i <= nx; i++) {
      if (insideInd2[i]) {
        r1[acoef] = i;
        acoef++;
      }
    }
    k = overlapedPolyPts.size(1);
    overlapedPolyPts.set_size(2, r1.size(0) + overlapedPolyPts.size(1));
    nx = r1.size(0);
    for (idx = 0; idx < nx; idx++) {
      overlapedPolyPts[2 * (idx + k)] = xIndices2[r1[idx]];
      overlapedPolyPts[2 * (idx + k) + 1] = yIndices2[r1[idx]];
    }
  }
  hipFree(*gpu_dist);
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<float, 2U> &in3
// Return Type  : void
//
} // namespace detector
} // namespace internal
} // namespace vision
} // namespace coder
void minus(coder::array<float, 2U> &in1, const coder::array<float, 2U> &in2,
           const coder::array<float, 2U> &in3)
{
  int aux_0_1;
  int aux_1_1;
  int b;
  int stride_0_1;
  int stride_1_1;
  if (in3.size(1) == 1) {
    b = in2.size(1);
  } else {
    b = in3.size(1);
  }
  in1.set_size(2, b);
  stride_0_1 = (in2.size(1) != 1);
  stride_1_1 = (in3.size(1) != 1);
  aux_0_1 = 0;
  aux_1_1 = 0;
  if (in3.size(1) == 1) {
    b = in2.size(1);
  } else {
    b = in3.size(1);
  }
  for (int i{0}; i < b; i++) {
    in1[2 * i] = in2[2 * aux_0_1] - in3[2 * aux_1_1];
    in1[2 * i + 1] = in2[2 * aux_0_1 + 1] - in3[2 * aux_1_1 + 1];
    aux_1_1 += stride_1_1;
    aux_0_1 += stride_0_1;
  }
}

//
// File trailer for overlapPolygonRotatedRects.cu
//
// [EOF]
//
