//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: sensor_msgs_PointCloud2Struct.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "sensor_msgs_PointCloud2Struct.h"
#include "deployedPointPillarModel_types.h"
#include "rt_nonfinite.h"
#include "sensor_msgs_PointFieldStruct.h"
#include "std_msgs_HeaderStruct.h"
#include "coder_array.h"

// Function Definitions
//
// Message struct definition for sensor_msgs/PointCloud2
//
// Arguments    : sensor_msgs_PointCloud2Struct_T *msg
// Return Type  : void
//
void sensor_msgs_PointCloud2Struct(sensor_msgs_PointCloud2Struct_T *msg)
{
  static const char b_cv[23]{'s', 'e', 'n', 's', 'o', 'r', '_', 'm',
                             's', 'g', 's', '/', 'P', 'o', 'i', 'n',
                             't', 'C', 'l', 'o', 'u', 'd', '2'};
  sensor_msgs_PointFieldStruct_T expl_temp;
  for (int i{0}; i < 23; i++) {
    msg->MessageType[i] = b_cv[i];
  }
  std_msgs_HeaderStruct(&msg->header);
  sensor_msgs_PointFieldStruct(&expl_temp);
  msg->height = 0U;
  msg->width = 0U;
  msg->fields.set_size(1);
  for (int i{0}; i < 22; i++) {
    msg->fields[0].MessageType[i] = expl_temp.MessageType[i];
  }
  msg->fields[0].INT8 = expl_temp.INT8;
  msg->fields[0].UINT8 = expl_temp.UINT8;
  msg->fields[0].INT16 = expl_temp.INT16;
  msg->fields[0].UINT16 = expl_temp.UINT16;
  msg->fields[0].INT32 = expl_temp.INT32;
  msg->fields[0].UINT32 = expl_temp.UINT32;
  msg->fields[0].FLOAT32 = expl_temp.FLOAT32;
  msg->fields[0].FLOAT64 = expl_temp.FLOAT64;
  msg->fields[0].name.set_size(1, expl_temp.name.size(1));
  for (int i{0}; i < expl_temp.name.size(1); i++) {
    msg->fields[0].name[i] = expl_temp.name[i];
  }
  msg->fields[0].offset = expl_temp.offset;
  msg->fields[0].datatype = expl_temp.datatype;
  msg->fields[0].count = expl_temp.count;
  msg->is_bigendian = false;
  msg->point_step = 0U;
  msg->row_step = 0U;
  msg->data.set_size(1);
  msg->data[0] = 0U;
  msg->is_dense = false;
  msg->fields.set_size(0);
  //(msg);
}

//
// File trailer for sensor_msgs_PointCloud2Struct.cu
//
// [EOF]
//
