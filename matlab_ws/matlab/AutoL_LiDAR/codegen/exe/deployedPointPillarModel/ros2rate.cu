#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: ros2rate.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "ros2rate.h"
#include "builtin_interfaces_TimeStruct.h"
#include "deployedPointPillarModel_data.h"
#include "deployedPointPillarModel_internal_types.h"
#include "deployedPointPillarModel_types.h"
#include "rt_nonfinite.h"
#include "coder_posix_time.h"
#include "mlros2_time.h"
#include "rclcpp/rclcpp.hpp"
#include <cmath>

// Function Definitions
//
// Arguments    : void
// Return Type  : double
//
namespace coder {
double ros2rate_getCurrentTime()
{
  builtin_interfaces_TimeStruct_T timeStruct;
  bool isSimTime;
  builtin_interfaces_TimeStruct();
  isSimTime = getSimTime();
  time2struct(&timeStruct, !isSimTime);
  return static_cast<double>(timeStruct.sec) +
         static_cast<double>(timeStruct.nanosec) * 1.0E-9;
}

//
// Arguments    : const ros2rate *obj
//                double sleepTime
// Return Type  : void
//
void ros2rate_internalSleep(const ros2rate *obj, double sleepTime)
{
  coderTimespec b_timespec;
  double elapsedTime;
  double endTime;
  double lastWakeTime;
  bool exitg1;
  lastWakeTime = ros2rate_getCurrentTime() - obj->StartTime;
  endTime = lastWakeTime + sleepTime;
  elapsedTime = ros2rate_getCurrentTime() - obj->StartTime;
  exitg1 = false;
  while ((!exitg1) && (elapsedTime < endTime)) {
    elapsedTime = std::fmin(sleepTime, obj->CheckPeriod);
    if (pauseState == 0) {
      double delayInt;
      if ((elapsedTime < 0.0) || std::isnan(elapsedTime)) {
        elapsedTime = 0.0;
      }
      delayInt = std::floor(elapsedTime);
      if (delayInt > 4.294967295E+9) {
        delayInt = 4.294967295E+9;
      }
      elapsedTime -= delayInt;
      if (elapsedTime > 0.0) {
        b_timespec.tv_nsec = std::floor(elapsedTime * 1.0E+9);
      } else {
        b_timespec.tv_nsec = 0.0;
      }
      b_timespec.tv_sec = delayInt;
      coderTimeSleep(&b_timespec);
    }
    elapsedTime = ros2rate_getCurrentTime() - obj->StartTime;
    if (elapsedTime < lastWakeTime) {
      exitg1 = true;
    } else {
      lastWakeTime = elapsedTime;
      sleepTime = endTime - elapsedTime;
    }
  }
}

//
// Arguments    : ros2rate *obj
// Return Type  : ros2rate *
//
ros2rate *ros2rate_ros2rate(ros2rate *obj)
{
  ros2rate *b_obj;
  bool isTimeValid;
  b_obj = obj;
  b_obj->CheckPeriod = 0.01;
  isTimeValid = false;
  while (!isTimeValid) {
    isTimeValid = (ros2rate_getCurrentTime() > 0.0);
  }
  b_obj->DesiredRate = 20.0;
  b_obj->DesiredPeriod = 1.0 / b_obj->DesiredRate;
  b_obj->PreviousPeriod = ros2rate_getCurrentTime();
  b_obj->OverrunAction[0] = 's';
  b_obj->OverrunAction[1] = 'l';
  b_obj->OverrunAction[2] = 'i';
  b_obj->OverrunAction[3] = 'p';
  b_obj->LastWakeTime = rtNaN;
  b_obj->LastWakeTime = 0.0;
  b_obj->StartTime = ros2rate_getCurrentTime();
  b_obj->NextExecutionIndex = 0.0;
  b_obj->ExecutionStartTime = 0.0;
  b_obj->ExecutionStartTime = ros2rate_getCurrentTime();
  return b_obj;
}

} // namespace coder

//
// File trailer for ros2rate.cu
//
// [EOF]
//
