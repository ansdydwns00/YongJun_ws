//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: rtGetNaN.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Abstract:
//       MATLAB for code generation function to initialize non-finite, NaN
// Include Files
#include "rtGetNaN.h"
#include "rt_nonfinite.h"

// Function: rtGetNaN
// ======================================================================
//  Abstract:
// Initialize rtNaN needed by the generated code.
//  NaN is initialized as non-signaling. Assumes IEEE.
real_T rtGetNaN(void)
{
  return rtNaN;
}

// Function: rtGetNaNF
// =====================================================================
//  Abstract:
//  Initialize rtNaNF needed by the generated code.
//  NaN is initialized as non-signaling. Assumes IEEE
real32_T rtGetNaNF(void)
{
  return rtNaNF;
}

//
// File trailer for rtGetNaN.cu
//
// [EOF]
//
