#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: rosReadField.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "rosReadField.h"
#include "PointCloud2Types.h"
#include "deployedPointPillarModel_internal_types.h"
#include "deployedPointPillarModel_types.h"
#include "rt_nonfinite.h"
#include "coder_array.h"
#include <cmath>
#include <cstddef>
#include <cstring>

// Function Definitions
//
// Arguments    : unsigned int msg_height
//                unsigned int msg_width
//                const array<sensor_msgs_PointFieldStruct_T, 1U> &msg_fields
//                unsigned int msg_point_step
//                const array<unsigned char, 1U> &msg_data
//                array<float, 2U> &fieldData
// Return Type  : void
//
namespace coder {
void rosReadField(unsigned int msg_height, unsigned int msg_width,
                  const array<sensor_msgs_PointFieldStruct_T, 1U> &msg_fields,
                  unsigned int msg_point_step,
                  const array<unsigned char, 1U> &msg_data,
                  array<float, 2U> &fieldData)
{
  static const char b_cv[9]{'i', 'n', 't', 'e', 'n', 's', 'i', 't', 'y'};
  array<cell_wrap_6, 2U> allFieldNames;
  array<double, 2U> byteIdx;
  array<double, 2U> c;
  array<double, 2U> c_y;
  array<float, 2U> validPoints;
  array<float, 1U> fv;
  array<unsigned int, 2U> dv;
  array<unsigned int, 2U> pointIndices;
  array<int, 2U> r;
  array<int, 2U> r1;
  array<int, 2U> r2;
  array<unsigned int, 1U> a;
  array<char, 2U> a__4;
  array<unsigned char, 1U> rawData;
  array<bool, 1U> b_x;
  double b_y;
  unsigned long long u;
  int acoef;
  int bcoef;
  int fieldIdx;
  int n;
  unsigned int numPointsActual;
  unsigned int qY;
  int varargin_3;
  bool b_bool;
  bool exitg2;
  if (msg_fields.size(0) == 0) {
    allFieldNames.set_size(0, 0);
  } else {
    bcoef = msg_fields.size(0);
    allFieldNames.set_size(1, msg_fields.size(0));
    for (acoef = 0; acoef < bcoef; acoef++) {
      allFieldNames[acoef].f1.set_size(1, msg_fields[acoef].name.size(1));
      for (varargin_3 = 0; varargin_3 < msg_fields[acoef].name.size(1);
           varargin_3++) {
        allFieldNames[acoef].f1[varargin_3] =
            msg_fields[acoef].name[varargin_3];
      }
    }
  }
  fieldIdx = -1;
  if ((allFieldNames.size(0) == 0) || (allFieldNames.size(1) == 0)) {
    n = 0;
  } else {
    n = allFieldNames.size(1);
  }
  for (acoef = 0; acoef < n; acoef++) {
    b_bool = false;
    if (allFieldNames[acoef].f1.size(1) == 9) {
      bcoef = 0;
      int exitg1;
      do {
        exitg1 = 0;
        if (bcoef + 1 < 10) {
          if (allFieldNames[acoef].f1[bcoef] != b_cv[bcoef]) {
            exitg1 = 1;
          } else {
            bcoef++;
          }
        } else {
          b_bool = true;
          exitg1 = 1;
        }
      } while (exitg1 == 0);
    }
    if (b_bool) {
      fieldIdx = acoef;
    }
  }
  u = static_cast<unsigned long long>(msg_width) * msg_height;
  if (u > 4294967295ULL) {
    u = 4294967295ULL;
  }
  if (static_cast<unsigned int>(u) < 1U) {
    n = 0;
  } else {
    n = static_cast<int>(u);
  }
  pointIndices.set_size(1, n);
  for (int k{0}; k < n; k++) {
    pointIndices[k] = static_cast<unsigned int>(k + 1);
  }
  u = static_cast<unsigned long long>(msg_height) * msg_width;
  if (u > 4294967295ULL) {
    u = 4294967295ULL;
  }
  b_y = std::trunc(static_cast<double>(msg_data.size(0)) /
                   static_cast<double>(msg_point_step));
  if (b_y < 4.294967296E+9) {
    qY = static_cast<unsigned int>(b_y);
  } else if (b_y >= 4.294967296E+9) {
    qY = MAX_uint32_T;
  } else {
    qY = 0U;
  }
  if ((!std::isnan(b_y)) && (static_cast<unsigned int>(u) > b_y)) {
    numPointsActual = qY;
  } else {
    numPointsActual = static_cast<unsigned int>(u);
  }
  ros::msg::sensor_msgs::internal::PointCloud2Types_rosToMATLABType(
      msg_fields[fieldIdx].datatype, a__4, &b_y);
  b_y *= static_cast<double>(msg_fields[fieldIdx].count);
  byteIdx.set_size(pointIndices.size(1), static_cast<int>(b_y));
  for (varargin_3 = 0;
       varargin_3 < pointIndices.size(1) * static_cast<int>(b_y);
       varargin_3++) {
    byteIdx[varargin_3] = 0.0;
  }
  if (b_y < 1.0) {
    c_y.set_size(1, 0);
  } else {
    c_y.set_size(1, static_cast<int>(b_y - 1.0) + 1);
    for (varargin_3 = 0; varargin_3 <= static_cast<int>(b_y - 1.0);
         varargin_3++) {
      c_y[varargin_3] = static_cast<double>(varargin_3) + 1.0;
    }
  }
  n = pointIndices.size(1) - 1;
  bcoef = 0;
  for (acoef = 0; acoef <= n; acoef++) {
    qY = pointIndices[acoef];
    if ((static_cast<int>(qY) > 0) && (qY <= numPointsActual)) {
      bcoef++;
    }
  }
  a.set_size(bcoef);
  bcoef = 0;
  for (acoef = 0; acoef <= n; acoef++) {
    qY = pointIndices[acoef];
    if ((static_cast<int>(qY) > 0) && (qY <= numPointsActual)) {
      unsigned int q0;
      q0 = msg_fields[fieldIdx].offset;
      u = static_cast<unsigned long long>(msg_point_step) * (qY - 1U);
      if (u > 4294967295ULL) {
        u = 4294967295ULL;
      }
      qY = q0 + static_cast<unsigned int>(u);
      if (qY < q0) {
        qY = MAX_uint32_T;
      }
      a[bcoef] = qY;
      bcoef++;
    }
  }
  c.set_size(a.size(0), c_y.size(1));
  if ((a.size(0) != 0) && (c_y.size(1) != 0)) {
    bcoef = (c_y.size(1) != 1);
    n = c_y.size(1) - 1;
    acoef = (a.size(0) != 1);
    for (int k{0}; k <= n; k++) {
      int i3;
      varargin_3 = bcoef * k;
      i3 = c.size(0) - 1;
      for (int b_k{0}; b_k <= i3; b_k++) {
        c[b_k + c.size(0) * k] =
            static_cast<double>(a[acoef * b_k]) + c_y[varargin_3];
      }
    }
  }
  n = pointIndices.size(1) - 1;
  bcoef = 0;
  for (acoef = 0; acoef <= n; acoef++) {
    qY = pointIndices[acoef];
    if ((static_cast<int>(qY) > 0) && (qY <= numPointsActual)) {
      bcoef++;
    }
  }
  r.set_size(1, bcoef);
  bcoef = 0;
  for (acoef = 0; acoef <= n; acoef++) {
    qY = pointIndices[acoef];
    if ((static_cast<int>(qY) > 0) && (qY <= numPointsActual)) {
      r[bcoef] = acoef;
      bcoef++;
    }
  }
  n = r.size(1);
  bcoef = static_cast<int>(b_y);
  for (varargin_3 = 0; varargin_3 < bcoef; varargin_3++) {
    for (acoef = 0; acoef < n; acoef++) {
      byteIdx[r[acoef] + byteIdx.size(0) * varargin_3] =
          c[acoef + n * varargin_3];
    }
  }
  n = pointIndices.size(1) - 1;
  bcoef = 0;
  for (acoef = 0; acoef <= n; acoef++) {
    qY = pointIndices[acoef];
    if ((static_cast<int>(qY) > 0) && (qY <= numPointsActual)) {
      bcoef++;
    }
  }
  r1.set_size(1, bcoef);
  bcoef = 0;
  for (acoef = 0; acoef <= n; acoef++) {
    qY = pointIndices[acoef];
    if ((static_cast<int>(qY) > 0) && (qY <= numPointsActual)) {
      r1[bcoef] = acoef;
      bcoef++;
    }
  }
  dv.set_size(byteIdx.size(1), r1.size(1));
  for (varargin_3 = 0; varargin_3 < r1.size(1); varargin_3++) {
    for (acoef = 0; acoef < byteIdx.size(1); acoef++) {
      dv[acoef + dv.size(0) * varargin_3] = static_cast<unsigned int>(
          byteIdx[r1[varargin_3] + byteIdx.size(0) * acoef]);
    }
  }
  rawData.set_size(dv.size(0) * dv.size(1));
  for (varargin_3 = 0; varargin_3 < dv.size(0) * dv.size(1); varargin_3++) {
    rawData[varargin_3] = msg_data[static_cast<int>(dv[varargin_3]) - 1];
  }
  if (dv.size(0) * dv.size(1) == 0) {
    bcoef = 0;
  } else {
    bcoef = (dv.size(0) * dv.size(1)) >> 2;
  }
  fv.set_size(bcoef);
  std::memcpy((void *)&(fv.data())[0], (void *)&(rawData.data())[0],
              (unsigned int)((size_t)bcoef * sizeof(float)));
  if (static_cast<int>(msg_fields[fieldIdx].count) > 0) {
    qY = msg_fields[fieldIdx].count;
    if (qY == 0U) {
      bcoef = MAX_int32_T;
    } else {
      bcoef = static_cast<int>(static_cast<unsigned int>(fv.size(0)) / qY);
    }
  } else {
    bcoef = 0;
  }
  n = static_cast<int>(msg_fields[fieldIdx].count);
  validPoints.set_size(bcoef, static_cast<int>(msg_fields[fieldIdx].count));
  for (varargin_3 = 0; varargin_3 < n; varargin_3++) {
    for (acoef = 0; acoef < bcoef; acoef++) {
      validPoints[acoef + validPoints.size(0) * varargin_3] =
          fv[varargin_3 + n * acoef];
    }
  }
  bcoef = pointIndices.size(1);
  b_x.set_size(pointIndices.size(1));
  for (varargin_3 = 0; varargin_3 < bcoef; varargin_3++) {
    qY = pointIndices[varargin_3];
    b_x[varargin_3] = ((static_cast<int>(qY) <= 0) || (qY > numPointsActual));
  }
  b_bool = false;
  bcoef = 1;
  exitg2 = false;
  while ((!exitg2) && (bcoef <= b_x.size(0))) {
    if (b_x[bcoef - 1]) {
      b_bool = true;
      exitg2 = true;
    } else {
      bcoef++;
    }
  }
  if (b_bool) {
    fieldData.set_size(pointIndices.size(1),
                       static_cast<int>(msg_fields[fieldIdx].count));
    bcoef = pointIndices.size(1);
    for (varargin_3 = 0;
         varargin_3 < bcoef * static_cast<int>(msg_fields[fieldIdx].count);
         varargin_3++) {
      fieldData[varargin_3] = rtNaNF;
    }
    n = pointIndices.size(1) - 1;
    bcoef = 0;
    for (acoef = 0; acoef <= n; acoef++) {
      qY = pointIndices[acoef];
      if ((static_cast<int>(qY) > 0) && (qY <= numPointsActual)) {
        bcoef++;
      }
    }
    r2.set_size(1, bcoef);
    bcoef = 0;
    for (acoef = 0; acoef <= n; acoef++) {
      qY = pointIndices[acoef];
      if ((static_cast<int>(qY) > 0) && (qY <= numPointsActual)) {
        r2[bcoef] = acoef;
        bcoef++;
      }
    }
    n = r2.size(1);
    bcoef = static_cast<int>(msg_fields[fieldIdx].count);
    for (varargin_3 = 0; varargin_3 < bcoef; varargin_3++) {
      for (acoef = 0; acoef < n; acoef++) {
        fieldData[r2[acoef] + fieldData.size(0) * varargin_3] =
            validPoints[acoef + n * varargin_3];
      }
    }
  } else {
    fieldData.set_size(validPoints.size(0), validPoints.size(1));
    for (varargin_3 = 0; varargin_3 < validPoints.size(0) * validPoints.size(1);
         varargin_3++) {
      fieldData[varargin_3] = validPoints[varargin_3];
    }
  }
}

} // namespace coder

//
// File trailer for rosReadField.cu
//
// [EOF]
//
