//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: geometry_msgs_PoseStruct.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "geometry_msgs_PoseStruct.h"
#include "deployedPointPillarModel_types.h"
#include "geometry_msgs_PointStruct.h"
#include "geometry_msgs_QuaternionStruct.h"
#include "rt_nonfinite.h"

// Function Definitions
//
// Message struct definition for geometry_msgs/Pose
//
// Arguments    : geometry_msgs_PoseStruct_T *msg
// Return Type  : void
//
void geometry_msgs_PoseStruct(geometry_msgs_PoseStruct_T *msg)
{
  static const char b_cv[18]{'g', 'e', 'o', 'm', 'e', 't', 'r', 'y', '_',
                             'm', 's', 'g', 's', '/', 'P', 'o', 's', 'e'};
  for (int i{0}; i < 18; i++) {
    msg->MessageType[i] = b_cv[i];
  }
  msg->position = geometry_msgs_PointStruct();
  msg->orientation = geometry_msgs_QuaternionStruct();
  //(msg);
}

//
// File trailer for geometry_msgs_PoseStruct.cu
//
// [EOF]
//
