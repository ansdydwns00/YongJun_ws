#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnSigmoidLayerImpl.hpp"
#include <cstdarg>
#include <cassert>
 namespace MWCudnnTarget { MWSigmoidLayerImpl::MWSigmoidLayerImpl(MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) { 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&rISNTTiSXOTdHqHTtNiB)); 
createAndAddDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); } 
MWSigmoidLayerImpl::~MWSigmoidLayerImpl() { } void 
MWSigmoidLayerImpl::propagateSize() { MWTensorBase* opTensor = 
getLayer()->getOutputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor)); 
CUDNN_CALL(hipdnnSetActivationDescriptor( rISNTTiSXOTdHqHTtNiB, 
HIPDNN_ACTIVATION_SIGMOID,  HIPDNN_NOT_PROPAGATE_NAN, 0)); } void 
MWSigmoidLayerImpl::predict() { MWCNNLayer* SigmoidLayer = getLayer(); 
MWTensorBase* ipTensorBase = SigmoidLayer->getInputTensor(0); MWTensorBase* 
opTensorBase = SigmoidLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t ipDesc = 
MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
CUDNN_CALL(hipdnnActivationForward( *cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
rISNTTiSXOTdHqHTtNiB, getOnePtr(), ipDesc, ipTensor->getData(), getZeroPtr(), 
*desc, opTensor->getData())); } void MWSigmoidLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(rISNTTiSXOTdHqHTtNiB)); } } 