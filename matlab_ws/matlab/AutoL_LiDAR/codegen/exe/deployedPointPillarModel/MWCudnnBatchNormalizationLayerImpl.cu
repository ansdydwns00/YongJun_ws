#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnBatchNormalizationLayerImpl.hpp"
#include <cstdio>
#include <cassert>
#include <vector>
 namespace MWCudnnTarget { 
MWBatchNormalizationLayerImpl::MWBatchNormalizationLayerImpl( MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl, float GbdgxISzcqHOpzQEBrvP, const 
char* HhKGcPZwrclEFnIdWerH, const char* HoqiuUUuJnXGkfDodicJ, const char* 
IAlDgIFcchbwRGBSfVfA, const char* IIiwAtyrOtLzLWAUlTey, int 
eybNKlJCSDUvsznWynwK) : MWCNNLayerImpl(layer, ntwk_impl) , 
olKGEIcsxmLSoMhRhEtP(NULL) , fYaOQTeunPwVjnhhTECh(NULL) , tGsvtyAVkrDznETdweDC(NULL) 
, tiuPuNpHUkyfgdeGvHSN(NULL) , PQjbchiGbyJfmpiqPpOC(GbdgxISzcqHOpzQEBrvP) 
, cwCXkgHfZmFQRzNVUlCO(eybNKlJCSDUvsznWynwK) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&KHClOltUSuqFVVErSxVb)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
const size_t numBytes = sizeof(float) * cwCXkgHfZmFQRzNVUlCO; 
CUDA_CALL(hipMalloc((void**)&olKGEIcsxmLSoMhRhEtP, numBytes)); 
CUDA_CALL(hipMalloc((void**)&fYaOQTeunPwVjnhhTECh, numBytes)); 
CUDA_CALL(hipMalloc((void**)&tGsvtyAVkrDznETdweDC, numBytes)); 
CUDA_CALL(hipMalloc((void**)&tiuPuNpHUkyfgdeGvHSN, numBytes)); 
loadScale(HoqiuUUuJnXGkfDodicJ); loadOffset(HhKGcPZwrclEFnIdWerH); 
loadTrainedMean(IAlDgIFcchbwRGBSfVfA); 
loadTrainedVariance(IIiwAtyrOtLzLWAUlTey); } 
MWBatchNormalizationLayerImpl::~MWBatchNormalizationLayerImpl() { } void 
MWBatchNormalizationLayerImpl::propagateSize() { MWCNNLayer* BNLayer = 
getLayer(); MWTensorBase* ipTensor = BNLayer->getInputTensor(); MWTensorBase* 
opTensor = BNLayer->getOutputTensor(); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor)); 
bQjijJlpNAVdwDDQgpaX = HIPDNN_BATCHNORM_SPATIAL; 
CUDNN_CALL(hipdnnDeriveBNTensorDescriptor(KHClOltUSuqFVVErSxVb, *desc, 
bQjijJlpNAVdwDDQgpaX)); } void MWBatchNormalizationLayerImpl::iLoadParamOntoGPU(char 
const* const PtRNGuserCxHAQfyEjFc, int const dMxIKDGTITyhdLqIHBLA, float* 
QwUuNuQNtlPXrIwRNiSZ) { FILE* QMgBqCuvjnbWHWiVPEwn = 
MWCNNLayer::openBinaryFile(PtRNGuserCxHAQfyEjFc); assert(QMgBqCuvjnbWHWiVPEwn); int 
const LHIWBuIwgwCuuNBzenxH = sizeof(float) * dMxIKDGTITyhdLqIHBLA; float* 
KHjdvykTFbUxdfZTFbqy = (float*)malloc(LHIWBuIwgwCuuNBzenxH); 
fread(KHjdvykTFbUxdfZTFbqy, sizeof(float), dMxIKDGTITyhdLqIHBLA, QMgBqCuvjnbWHWiVPEwn); 
fclose(QMgBqCuvjnbWHWiVPEwn); CUDA_CALL(hipMemcpy(QwUuNuQNtlPXrIwRNiSZ, 
KHjdvykTFbUxdfZTFbqy, LHIWBuIwgwCuuNBzenxH, hipMemcpyHostToDevice)); 
free(KHjdvykTFbUxdfZTFbqy); } void MWBatchNormalizationLayerImpl::loadScale(const 
char* PtRNGuserCxHAQfyEjFc) { iLoadParamOntoGPU(PtRNGuserCxHAQfyEjFc, 
cwCXkgHfZmFQRzNVUlCO, olKGEIcsxmLSoMhRhEtP); } void 
MWBatchNormalizationLayerImpl::loadOffset(const char* PtRNGuserCxHAQfyEjFc) { 
iLoadParamOntoGPU(PtRNGuserCxHAQfyEjFc, cwCXkgHfZmFQRzNVUlCO, 
fYaOQTeunPwVjnhhTECh); } void MWBatchNormalizationLayerImpl::loadTrainedMean(const 
char* PtRNGuserCxHAQfyEjFc) { iLoadParamOntoGPU(PtRNGuserCxHAQfyEjFc, 
cwCXkgHfZmFQRzNVUlCO, tGsvtyAVkrDznETdweDC); } void 
MWBatchNormalizationLayerImpl::loadTrainedVariance(const char* 
PtRNGuserCxHAQfyEjFc) { iLoadParamOntoGPU(PtRNGuserCxHAQfyEjFc, 
cwCXkgHfZmFQRzNVUlCO, tiuPuNpHUkyfgdeGvHSN); } void 
MWBatchNormalizationLayerImpl::setLearnables(std::vector<float*> learnables) { 
assert(learnables.size() == 2); float* gCYwEfkibolsgZAumsuW = learnables[0]; 
CUDA_CALL(hipMemcpy(fYaOQTeunPwVjnhhTECh, gCYwEfkibolsgZAumsuW, sizeof(float) * 
cwCXkgHfZmFQRzNVUlCO, hipMemcpyHostToDevice)); float* omxlPZbBePZdWaJOBUUG = 
learnables[1]; CUDA_CALL(hipMemcpy(olKGEIcsxmLSoMhRhEtP, omxlPZbBePZdWaJOBUUG, 
sizeof(float) * cwCXkgHfZmFQRzNVUlCO, hipMemcpyHostToDevice)); } void 
MWBatchNormalizationLayerImpl::predict() { MWCNNLayer* BNLayer = getLayer(); 
MWTensorBase* ipTensorBase = BNLayer->getInputTensor(); MWTensorBase* 
opTensorBase = BNLayer->getOutputTensor(); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); const hipdnnTensorDescriptor_t 
TaAJDyqFVJXfAfCJhOuU = MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); float* 
TbrNrGxaFFHrzKUcfHNZ = ipTensor->getData(); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t jLmklYtHcmTxayQTpmRw = *desc; float* jLyhrFjMmVnNjoeDJCwH = 
opTensor->getData(); CUDNN_CALL(hipdnnBatchNormalizationForwardInference( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), bQjijJlpNAVdwDDQgpaX, getOnePtr(), 
getZeroPtr(),  TaAJDyqFVJXfAfCJhOuU, TbrNrGxaFFHrzKUcfHNZ, jLmklYtHcmTxayQTpmRw, 
jLyhrFjMmVnNjoeDJCwH, KHClOltUSuqFVVErSxVb, olKGEIcsxmLSoMhRhEtP, 
fYaOQTeunPwVjnhhTECh, tGsvtyAVkrDznETdweDC, tiuPuNpHUkyfgdeGvHSN, 
PQjbchiGbyJfmpiqPpOC)); } void MWBatchNormalizationLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(KHClOltUSuqFVVErSxVb)); if 
(olKGEIcsxmLSoMhRhEtP) { CUDA_FREE_CALL(olKGEIcsxmLSoMhRhEtP); olKGEIcsxmLSoMhRhEtP = NULL; 
} if (fYaOQTeunPwVjnhhTECh) { CUDA_FREE_CALL(fYaOQTeunPwVjnhhTECh); fYaOQTeunPwVjnhhTECh 
= NULL; } if (tGsvtyAVkrDznETdweDC) { CUDA_FREE_CALL(tGsvtyAVkrDznETdweDC); 
tGsvtyAVkrDznETdweDC = NULL; } if (tiuPuNpHUkyfgdeGvHSN) { 
CUDA_FREE_CALL(tiuPuNpHUkyfgdeGvHSN); tiuPuNpHUkyfgdeGvHSN = NULL; 
} } } 