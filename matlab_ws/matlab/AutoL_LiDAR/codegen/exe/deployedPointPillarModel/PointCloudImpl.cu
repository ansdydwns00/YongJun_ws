#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: PointCloudImpl.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "PointCloudImpl.h"
#include "getSubsetPoints.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include "coder_array.h"
#include "coder_gpu_array.h"

// Function Declarations
static __global__ void PointCloudImpl_extractValidPoints_kernel4(
    const double numPoints, const coder::gpu_array<float, 2U> ptCloudCoords,
    const int i, coder::gpu_array<bool, 1U> validCoords);

static __global__ void PointCloudImpl_subsetImpl_kernel10(
    const double numElements, const coder::gpu_array<unsigned char, 2U> color,
    const double outSize, const coder::gpu_array<double, 1U> outIndex,
    const coder::gpu_array<bool, 1U> indices, const int b_indices,
    coder::gpu_array<unsigned char, 2U> outCol);

static __global__ void
PointCloudImpl_subsetImpl_kernel11(const coder::gpu_array<bool, 1U> indices,
                                   const int b_indices,
                                   coder::gpu_array<double, 1U> outIndex);

static __global__ void
PointCloudImpl_subsetImpl_kernel12(const coder::gpu_array<double, 1U> outIndex,
                                   double *outSize);

static __global__ void PointCloudImpl_subsetImpl_kernel13(
    const coder::gpu_array<float, 2U> intensity, const double outSize,
    const coder::gpu_array<double, 1U> outIndex,
    const coder::gpu_array<bool, 1U> indices, const int i,
    coder::gpu_array<float, 1U> outIntensity);

static __global__ void PointCloudImpl_subsetImpl_kernel14(
    const coder::gpu_array<float, 1U> outIntensity, const int b_outIntensity,
    coder::gpu_array<float, 2U> c_outIntensity);

static __global__ void
PointCloudImpl_subsetImpl_kernel5(const coder::gpu_array<bool, 1U> indices,
                                  const int b_indices,
                                  coder::gpu_array<double, 1U> outIndex);

static __global__ void
PointCloudImpl_subsetImpl_kernel6(const coder::gpu_array<double, 1U> outIndex,
                                  double *outSize);

static __global__ void PointCloudImpl_subsetImpl_kernel7(
    const double numElements, const coder::gpu_array<float, 2U> location,
    const double outSize, const coder::gpu_array<double, 1U> outIndex,
    const coder::gpu_array<bool, 1U> indices, const int b_indices,
    coder::gpu_array<float, 2U> outLoc);

static __global__ void
PointCloudImpl_subsetImpl_kernel8(const coder::gpu_array<bool, 1U> indices,
                                  const int b_indices,
                                  coder::gpu_array<double, 1U> outIndex);

static __global__ void
PointCloudImpl_subsetImpl_kernel9(const coder::gpu_array<double, 1U> outIndex,
                                  double *outSize);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double numPoints
//                const coder::gpu_array<float, 2U> ptCloudCoords
//                const int i
//                coder::gpu_array<bool, 1U> validCoords
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void PointCloudImpl_extractValidPoints_kernel4(
        const double numPoints, const coder::gpu_array<float, 2U> ptCloudCoords,
        const int i, coder::gpu_array<bool, 1U> validCoords)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long long>(i) - 1ULL;
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int b_i;
    b_i = static_cast<int>(idx);
    if ((static_cast<bool>(!static_cast<int>(isinf(ptCloudCoords[b_i])))) &&
        (static_cast<bool>(!static_cast<int>(isnan(ptCloudCoords[b_i]))))) {
      float b_x;
      b_x = ptCloudCoords[b_i + static_cast<int>(numPoints)];
      if ((static_cast<bool>(!static_cast<int>(isinf(b_x)))) &&
          (static_cast<bool>(!static_cast<int>(isnan(b_x))))) {
        b_x = ptCloudCoords[b_i + (static_cast<int>(numPoints) << 1)];
        if ((static_cast<bool>(!static_cast<int>(isinf(b_x)))) &&
            (static_cast<bool>(!static_cast<int>(isnan(b_x))))) {
          validCoords[b_i] = true;
        } else {
          validCoords[b_i] = false;
        }
      } else {
        validCoords[b_i] = false;
      }
    } else {
      validCoords[b_i] = false;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double numElements
//                const coder::gpu_array<unsigned char, 2U> color
//                const double outSize
//                const coder::gpu_array<double, 1U> outIndex
//                const coder::gpu_array<bool, 1U> indices
//                const int b_indices
//                coder::gpu_array<unsigned char, 2U> outCol
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void PointCloudImpl_subsetImpl_kernel10(
        const double numElements,
        const coder::gpu_array<unsigned char, 2U> color, const double outSize,
        const coder::gpu_array<double, 1U> outIndex,
        const coder::gpu_array<bool, 1U> indices, const int b_indices,
        coder::gpu_array<unsigned char, 2U> outCol)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<unsigned long long>(b_indices) + 1ULL) * 3ULL - 1ULL;
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int col;
    int itr;
    itr = static_cast<int>(idx %
                           (static_cast<unsigned long long>(b_indices) + 1ULL));
    col = static_cast<int>((idx - static_cast<unsigned long long>(itr)) /
                           (static_cast<unsigned long long>(b_indices) + 1ULL));
    if (indices[itr]) {
      outCol[static_cast<int>(outIndex[itr] +
                              static_cast<double>(col) * outSize) -
             1] =
          color[static_cast<int>((static_cast<double>(itr) + 1.0) +
                                 static_cast<double>(col) * numElements) -
                1];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<bool, 1U> indices
//                const int b_indices
//                coder::gpu_array<double, 1U> outIndex
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void PointCloudImpl_subsetImpl_kernel11(
        const coder::gpu_array<bool, 1U> indices, const int b_indices,
        coder::gpu_array<double, 1U> outIndex)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long long>(b_indices);
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int col;
    col = static_cast<int>(idx);
    outIndex[col] = static_cast<double>(indices[col]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<double, 1U> outIndex
//                double *outSize
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void PointCloudImpl_subsetImpl_kernel12(
        const coder::gpu_array<double, 1U> outIndex, double *outSize)
{
  unsigned long long threadId;
  int i;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 2) {
    *outSize = outIndex[outIndex.size(0) - 1];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<float, 2U> intensity
//                const double outSize
//                const coder::gpu_array<double, 1U> outIndex
//                const coder::gpu_array<bool, 1U> indices
//                const int i
//                coder::gpu_array<float, 1U> outIntensity
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void PointCloudImpl_subsetImpl_kernel13(
        const coder::gpu_array<float, 2U> intensity, const double outSize,
        const coder::gpu_array<double, 1U> outIndex,
        const coder::gpu_array<bool, 1U> indices, const int i,
        coder::gpu_array<float, 1U> outIntensity)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long long>(i) - 1ULL;
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int itr;
    itr = static_cast<int>(idx);
    if (indices[itr]) {
      outIntensity[static_cast<int>(outIndex[itr] + 0.0 * outSize) - 1] =
          intensity[itr];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<float, 1U> outIntensity
//                const int b_outIntensity
//                coder::gpu_array<float, 2U> c_outIntensity
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void PointCloudImpl_subsetImpl_kernel14(
        const coder::gpu_array<float, 1U> outIntensity,
        const int b_outIntensity, coder::gpu_array<float, 2U> c_outIntensity)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long long>(b_outIntensity);
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int col;
    col = static_cast<int>(idx);
    c_outIntensity[col] = outIntensity[col];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<bool, 1U> indices
//                const int b_indices
//                coder::gpu_array<double, 1U> outIndex
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void PointCloudImpl_subsetImpl_kernel5(
        const coder::gpu_array<bool, 1U> indices, const int b_indices,
        coder::gpu_array<double, 1U> outIndex)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long long>(b_indices);
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int col;
    col = static_cast<int>(idx);
    outIndex[col] = static_cast<double>(indices[col]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<double, 1U> outIndex
//                double *outSize
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void PointCloudImpl_subsetImpl_kernel6(
        const coder::gpu_array<double, 1U> outIndex, double *outSize)
{
  unsigned long long threadId;
  int i;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 2) {
    *outSize = outIndex[outIndex.size(0) - 1];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double numElements
//                const coder::gpu_array<float, 2U> location
//                const double outSize
//                const coder::gpu_array<double, 1U> outIndex
//                const coder::gpu_array<bool, 1U> indices
//                const int b_indices
//                coder::gpu_array<float, 2U> outLoc
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void PointCloudImpl_subsetImpl_kernel7(
        const double numElements, const coder::gpu_array<float, 2U> location,
        const double outSize, const coder::gpu_array<double, 1U> outIndex,
        const coder::gpu_array<bool, 1U> indices, const int b_indices,
        coder::gpu_array<float, 2U> outLoc)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<unsigned long long>(b_indices) + 1ULL) * 3ULL - 1ULL;
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int col;
    int itr;
    itr = static_cast<int>(idx %
                           (static_cast<unsigned long long>(b_indices) + 1ULL));
    col = static_cast<int>((idx - static_cast<unsigned long long>(itr)) /
                           (static_cast<unsigned long long>(b_indices) + 1ULL));
    if (indices[itr]) {
      outLoc[static_cast<int>(outIndex[itr] +
                              static_cast<double>(col) * outSize) -
             1] =
          location[static_cast<int>((static_cast<double>(itr) + 1.0) +
                                    static_cast<double>(col) * numElements) -
                   1];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<bool, 1U> indices
//                const int b_indices
//                coder::gpu_array<double, 1U> outIndex
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void PointCloudImpl_subsetImpl_kernel8(
        const coder::gpu_array<bool, 1U> indices, const int b_indices,
        coder::gpu_array<double, 1U> outIndex)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long long>(b_indices);
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int col;
    col = static_cast<int>(idx);
    outIndex[col] = static_cast<double>(indices[col]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<double, 1U> outIndex
//                double *outSize
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void PointCloudImpl_subsetImpl_kernel9(
        const coder::gpu_array<double, 1U> outIndex, double *outSize)
{
  unsigned long long threadId;
  int i;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 2) {
    *outSize = outIndex[outIndex.size(0) - 1];
  }
}

//
// Arguments    : const array<float, 2U> &cpu_ptCloudCoords
//                array<bool, 1U> &cpu_validCoords
//                bool *validCoords_outdatedOnCpu
//                gpu_array<bool, 1U> &gpu_validCoords
//                bool *validCoords_outdatedOnGpu
// Return Type  : void
//
namespace coder {
namespace vision {
namespace internal {
namespace codegen {
namespace gpu {
void PointCloudImpl_extractValidPoints(
    const array<float, 2U> &cpu_ptCloudCoords, array<bool, 1U> &cpu_validCoords,
    bool *validCoords_outdatedOnCpu, gpu_array<bool, 1U> &gpu_validCoords,
    bool *validCoords_outdatedOnGpu)
{
  gpu_array<float, 2U> gpu_ptCloudCoords;
  dim3 block;
  dim3 grid;
  double numPoints;
  bool validLaunchParams;
  numPoints = static_cast<double>(cpu_ptCloudCoords.size(0) * 3) / 3.0;
  cpu_validCoords.set_size(static_cast<int>(numPoints));
  if (!*validCoords_outdatedOnGpu) {
    gpu_validCoords.set_size(&cpu_validCoords);
  }
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>(static_cast<int>(numPoints)),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    arrayCopyCpuToGpu(gpu_ptCloudCoords, cpu_ptCloudCoords);
    if (*validCoords_outdatedOnGpu) {
      arrayCopyCpuToGpu(gpu_validCoords, cpu_validCoords);
    }
    PointCloudImpl_extractValidPoints_kernel4<<<grid, block>>>(
        numPoints, gpu_ptCloudCoords, static_cast<int>(numPoints),
        gpu_validCoords);
    *validCoords_outdatedOnGpu = false;
    *validCoords_outdatedOnCpu = true;
  }
}

//
// Arguments    : const array<float, 2U> &cpu_location
//                const array<unsigned char, 2U> &cpu_color
//                const array<float, 2U> &cpu_normal
//                const array<float, 2U> &cpu_intensity
//                const array<float, 2U> &cpu_rangeData
//                array<bool, 1U> &cpu_indices
//                bool *indices_outdatedOnCpu
//                gpu_array<bool, 1U> &gpu_indices
//                bool *indices_outdatedOnGpu
//                array<float, 2U> &cpu_outLoc
//                array<unsigned char, 2U> &cpu_outCol
//                bool *outCol_outdatedOnCpu
//                gpu_array<unsigned char, 2U> &gpu_outCol
//                bool *outCol_outdatedOnGpu
//                array<float, 2U> &cpu_outNorm
//                bool *outNorm_outdatedOnCpu
//                gpu_array<float, 2U> &gpu_outNorm
//                bool *outNorm_outdatedOnGpu
//                array<float, 2U> &cpu_outIntensity
//                bool *outIntensity_outdatedOnCpu
//                gpu_array<float, 2U> &gpu_outIntensity
//                bool *outIntensity_outdatedOnGpu
//                array<float, 2U> &cpu_outRangeData
//                bool *outRangeData_outdatedOnCpu
//                gpu_array<float, 2U> &gpu_outRangeData
//                bool *outRangeData_outdatedOnGpu
// Return Type  : void
//
void PointCloudImpl_subsetImpl(
    const array<float, 2U> &cpu_location,
    const array<unsigned char, 2U> &cpu_color,
    const array<float, 2U> &cpu_normal, const array<float, 2U> &cpu_intensity,
    const array<float, 2U> &cpu_rangeData, array<bool, 1U> &cpu_indices,
    bool *indices_outdatedOnCpu, gpu_array<bool, 1U> &gpu_indices,
    bool *indices_outdatedOnGpu, array<float, 2U> &cpu_outLoc,
    array<unsigned char, 2U> &cpu_outCol, bool *outCol_outdatedOnCpu,
    gpu_array<unsigned char, 2U> &gpu_outCol, bool *outCol_outdatedOnGpu,
    array<float, 2U> &cpu_outNorm, bool *outNorm_outdatedOnCpu,
    gpu_array<float, 2U> &gpu_outNorm, bool *outNorm_outdatedOnGpu,
    array<float, 2U> &cpu_outIntensity, bool *outIntensity_outdatedOnCpu,
    gpu_array<float, 2U> &gpu_outIntensity, bool *outIntensity_outdatedOnGpu,
    array<float, 2U> &cpu_outRangeData, bool *outRangeData_outdatedOnCpu,
    gpu_array<float, 2U> &gpu_outRangeData, bool *outRangeData_outdatedOnGpu)
{
  array<double, 1U> b_cpu_outIndex;
  array<double, 1U> c_cpu_outIndex;
  array<double, 1U> cpu_outIndex;
  array<float, 1U> b_cpu_outIntensity;
  gpu_array<double, 1U> b_gpu_outIndex;
  gpu_array<double, 1U> c_gpu_outIndex;
  gpu_array<double, 1U> gpu_outIndex;
  gpu_array<float, 2U> gpu_intensity;
  gpu_array<float, 2U> gpu_location;
  gpu_array<float, 2U> gpu_normal;
  gpu_array<float, 2U> gpu_outLoc;
  gpu_array<float, 2U> gpu_rangeData;
  gpu_array<float, 1U> b_gpu_outIntensity;
  gpu_array<unsigned char, 2U> gpu_color;
  dim3 block;
  dim3 grid;
  double b_cpu_outSize;
  double c_cpu_outSize;
  double cpu_outSize;
  double numElements;
  double *b_gpu_outSize;
  double *c_gpu_outSize;
  double *gpu_outSize;
  int indices;
  bool normal_outdatedOnGpu;
  bool outLoc_outdatedOnCpu;
  bool rangeData_outdatedOnGpu;
  bool validLaunchParams;
  hipMalloc(&gpu_outSize, 8ULL);
  hipMalloc(&b_gpu_outSize, 8ULL);
  hipMalloc(&c_gpu_outSize, 8ULL);
  outLoc_outdatedOnCpu = false;
  rangeData_outdatedOnGpu = true;
  normal_outdatedOnGpu = true;
  if ((cpu_location.size(0) == 0) || (cpu_indices.size(0) == 0)) {
    cpu_outLoc.set_size(0, 3);
  } else {
    numElements = static_cast<double>(cpu_location.size(0) * 3) / 3.0;
    cpu_outIndex.set_size(cpu_indices.size(0));
    c_gpu_outIndex.set_size(&cpu_outIndex);
    indices = cpu_indices.size(0) - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(indices + 1LL), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (*indices_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_indices, cpu_indices);
      }
      *indices_outdatedOnGpu = false;
      PointCloudImpl_subsetImpl_kernel5<<<grid, block>>>(gpu_indices, indices,
                                                         c_gpu_outIndex);
    }
    if (cpu_outIndex.size(0) != 1) {
      mwCallThrustScan1D(&(c_gpu_outIndex.data())[0], false,
                         cpu_outIndex.size(0));
    }
    PointCloudImpl_subsetImpl_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        c_gpu_outIndex, c_gpu_outSize);
    hipMemcpy(&cpu_outSize, c_gpu_outSize, 8ULL, hipMemcpyDeviceToHost);
    cpu_outLoc.set_size(static_cast<int>(cpu_outSize), 3);
    indices = cpu_indices.size(0) - 1;
    validLaunchParams =
        mwGetLaunchParameters1D(static_cast<double>((indices + 1LL) * 3LL),
                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      arrayCopyCpuToGpu(gpu_location, cpu_location);
      if (*indices_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_indices, cpu_indices);
      }
      *indices_outdatedOnGpu = false;
      arrayCopyCpuToGpu(gpu_outLoc, cpu_outLoc);
      PointCloudImpl_subsetImpl_kernel7<<<grid, block>>>(
          numElements, gpu_location, cpu_outSize, c_gpu_outIndex, gpu_indices,
          indices, gpu_outLoc);
      outLoc_outdatedOnCpu = true;
    }
  }
  if ((cpu_color.size(0) == 0) || (cpu_color.size(1) == 0) ||
      (cpu_indices.size(0) == 0)) {
    cpu_outCol.set_size(0, 3);
    if (!*outCol_outdatedOnGpu) {
      gpu_outCol.set_size(&cpu_outCol);
    }
  } else {
    numElements =
        static_cast<double>(cpu_color.size(0) * cpu_color.size(1)) / 3.0;
    b_cpu_outIndex.set_size(cpu_indices.size(0));
    b_gpu_outIndex.set_size(&b_cpu_outIndex);
    indices = cpu_indices.size(0) - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(indices + 1LL), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (*indices_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_indices, cpu_indices);
      }
      *indices_outdatedOnGpu = false;
      PointCloudImpl_subsetImpl_kernel8<<<grid, block>>>(gpu_indices, indices,
                                                         b_gpu_outIndex);
    }
    if (b_cpu_outIndex.size(0) != 1) {
      mwCallThrustScan1D(&(b_gpu_outIndex.data())[0], false,
                         b_cpu_outIndex.size(0));
    }
    PointCloudImpl_subsetImpl_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        b_gpu_outIndex, b_gpu_outSize);
    hipMemcpy(&b_cpu_outSize, b_gpu_outSize, 8ULL, hipMemcpyDeviceToHost);
    cpu_outCol.set_size(static_cast<int>(b_cpu_outSize), 3);
    if (!*outCol_outdatedOnGpu) {
      gpu_outCol.set_size(&cpu_outCol);
    }
    indices = cpu_indices.size(0) - 1;
    validLaunchParams =
        mwGetLaunchParameters1D(static_cast<double>((indices + 1LL) * 3LL),
                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      arrayCopyCpuToGpu(gpu_color, cpu_color);
      if (*indices_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_indices, cpu_indices);
      }
      *indices_outdatedOnGpu = false;
      if (*outCol_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_outCol, cpu_outCol);
      }
      PointCloudImpl_subsetImpl_kernel10<<<grid, block>>>(
          numElements, gpu_color, b_cpu_outSize, b_gpu_outIndex, gpu_indices,
          indices, gpu_outCol);
      *outCol_outdatedOnGpu = false;
      *outCol_outdatedOnCpu = true;
    }
  }
  getSubsetPoints_anonFcn2(
      *(array<float, 2U> *)&cpu_normal, gpu_normal, &normal_outdatedOnGpu,
      cpu_indices, indices_outdatedOnCpu, gpu_indices, indices_outdatedOnGpu,
      cpu_outNorm, outNorm_outdatedOnCpu, gpu_outNorm, outNorm_outdatedOnGpu);
  if ((cpu_intensity.size(0) == 0) || (cpu_intensity.size(1) == 0) ||
      (cpu_indices.size(0) == 0)) {
    b_cpu_outIntensity.set_size(0);
    b_gpu_outIntensity.set_size(&b_cpu_outIntensity);
  } else {
    c_cpu_outIndex.set_size(cpu_indices.size(0));
    gpu_outIndex.set_size(&c_cpu_outIndex);
    indices = cpu_indices.size(0) - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(indices + 1LL), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (*indices_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_indices, cpu_indices);
      }
      *indices_outdatedOnGpu = false;
      PointCloudImpl_subsetImpl_kernel11<<<grid, block>>>(gpu_indices, indices,
                                                          gpu_outIndex);
    }
    if (c_cpu_outIndex.size(0) != 1) {
      mwCallThrustScan1D(&(gpu_outIndex.data())[0], false,
                         c_cpu_outIndex.size(0));
    }
    PointCloudImpl_subsetImpl_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        gpu_outIndex, gpu_outSize);
    hipMemcpy(&c_cpu_outSize, gpu_outSize, 8ULL, hipMemcpyDeviceToHost);
    b_cpu_outIntensity.set_size(static_cast<int>(c_cpu_outSize));
    b_gpu_outIntensity.set_size(&b_cpu_outIntensity);
    indices = cpu_indices.size(0);
    validLaunchParams = mwGetLaunchParameters1D(static_cast<double>(indices),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      arrayCopyCpuToGpu(gpu_intensity, cpu_intensity);
      if (*indices_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_indices, cpu_indices);
      }
      *indices_outdatedOnGpu = false;
      PointCloudImpl_subsetImpl_kernel13<<<grid, block>>>(
          gpu_intensity, c_cpu_outSize, gpu_outIndex, gpu_indices, indices,
          b_gpu_outIntensity);
    }
  }
  cpu_outIntensity.set_size(b_cpu_outIntensity.size(0), 1);
  if (!*outIntensity_outdatedOnGpu) {
    gpu_outIntensity.set_size(&cpu_outIntensity);
  }
  indices = b_cpu_outIntensity.size(0) - 1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<double>(indices + 1LL), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    if (*outIntensity_outdatedOnGpu) {
      arrayCopyCpuToGpu(gpu_outIntensity, cpu_outIntensity);
    }
    PointCloudImpl_subsetImpl_kernel14<<<grid, block>>>(
        b_gpu_outIntensity, indices, gpu_outIntensity);
    *outIntensity_outdatedOnGpu = false;
    *outIntensity_outdatedOnCpu = true;
  }
  getSubsetPoints_anonFcn2(
      *(array<float, 2U> *)&cpu_rangeData, gpu_rangeData,
      &rangeData_outdatedOnGpu, cpu_indices, indices_outdatedOnCpu, gpu_indices,
      indices_outdatedOnGpu, cpu_outRangeData, outRangeData_outdatedOnCpu,
      gpu_outRangeData, outRangeData_outdatedOnGpu);
  if (outLoc_outdatedOnCpu) {
    arrayCopyGpuToCpu(cpu_outLoc, gpu_outLoc);
  }
  hipFree(c_gpu_outSize);
  hipFree(b_gpu_outSize);
  hipFree(gpu_outSize);
}

} // namespace gpu
} // namespace codegen
} // namespace internal
} // namespace vision
} // namespace coder

//
// File trailer for PointCloudImpl.cu
//
// [EOF]
//
