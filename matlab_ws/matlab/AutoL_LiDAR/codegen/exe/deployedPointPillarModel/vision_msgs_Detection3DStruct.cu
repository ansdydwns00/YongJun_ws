//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: vision_msgs_Detection3DStruct.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "vision_msgs_Detection3DStruct.h"
#include "deployedPointPillarModel_types.h"
#include "rt_nonfinite.h"
#include "std_msgs_HeaderStruct.h"
#include "vision_msgs_BoundingBox3DStruct.h"
#include "vision_msgs_ObjectHypothesisWithPoseStruct.h"
#include "coder_array.h"

// Function Definitions
//
// Message struct definition for vision_msgs/Detection3D
//
// Arguments    : vision_msgs_Detection3DStruct_T *msg
// Return Type  : void
//
void vision_msgs_Detection3DStruct(vision_msgs_Detection3DStruct_T *msg)
{
  static const char b_cv[23]{'v', 'i', 's', 'i', 'o', 'n', '_', 'm',
                             's', 'g', 's', '/', 'D', 'e', 't', 'e',
                             'c', 't', 'i', 'o', 'n', '3', 'D'};
  vision_msgs_ObjectHypothesisWithPoseStruct_T expl_temp;
  for (int i{0}; i < 23; i++) {
    msg->MessageType[i] = b_cv[i];
  }
  std_msgs_HeaderStruct(&msg->header);
  vision_msgs_ObjectHypothesisWithPoseStruct(&expl_temp);
  vision_msgs_BoundingBox3DStruct(&msg->bbox);
  msg->results.set_size(1);
  for (int i{0}; i < 36; i++) {
    msg->results[0].MessageType[i] = expl_temp.MessageType[i];
  }
  for (int i{0}; i < 28; i++) {
    msg->results[0].hypothesis.MessageType[i] =
        expl_temp.hypothesis.MessageType[i];
  }
  msg->results[0].hypothesis.class_id.set_size(
      1, expl_temp.hypothesis.class_id.size(1));
  for (int i{0}; i < expl_temp.hypothesis.class_id.size(1); i++) {
    msg->results[0].hypothesis.class_id[i] = expl_temp.hypothesis.class_id[i];
  }
  msg->results[0].hypothesis.score = expl_temp.hypothesis.score;
  for (int i{0}; i < 32; i++) {
    msg->results[0].pose.MessageType[i] = expl_temp.pose.MessageType[i];
  }
  for (int i{0}; i < 18; i++) {
    msg->results[0].pose.pose.MessageType[i] =
        expl_temp.pose.pose.MessageType[i];
  }
  for (int i{0}; i < 19; i++) {
    msg->results[0].pose.pose.position.MessageType[i] =
        expl_temp.pose.pose.position.MessageType[i];
  }
  msg->results[0].pose.pose.position.x = expl_temp.pose.pose.position.x;
  msg->results[0].pose.pose.position.y = expl_temp.pose.pose.position.y;
  msg->results[0].pose.pose.position.z = expl_temp.pose.pose.position.z;
  for (int i{0}; i < 24; i++) {
    msg->results[0].pose.pose.orientation.MessageType[i] =
        expl_temp.pose.pose.orientation.MessageType[i];
  }
  msg->results[0].pose.pose.orientation.x = expl_temp.pose.pose.orientation.x;
  msg->results[0].pose.pose.orientation.y = expl_temp.pose.pose.orientation.y;
  msg->results[0].pose.pose.orientation.z = expl_temp.pose.pose.orientation.z;
  msg->results[0].pose.pose.orientation.w = expl_temp.pose.pose.orientation.w;
  for (int i{0}; i < 36; i++) {
    msg->results[0].pose.covariance[i] = expl_temp.pose.covariance[i];
  }
  msg->id.set_size(1, 0);
  msg->results.set_size(0);
  //(msg);
}

//
// File trailer for vision_msgs_Detection3DStruct.cu
//
// [EOF]
//
