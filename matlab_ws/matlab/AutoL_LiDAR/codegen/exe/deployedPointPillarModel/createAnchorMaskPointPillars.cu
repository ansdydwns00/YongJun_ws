#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: createAnchorMaskPointPillars.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "createAnchorMaskPointPillars.h"
#include "rt_nonfinite.h"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include "coder_array.h"
#include <cmath>
#include <cstring>

// Function Definitions
//
// Arguments    : const float pillarIndices[24000]
//                const double pcRange[6]
//                const array<double, 2U> &anchorsBEV
//                array<double, 1U> &anchorMask
// Return Type  : void
//
namespace coder {
namespace lidar {
namespace internal {
namespace cnn {
void createAnchorMaskPointPillars(const float pillarIndices[24000],
                                  const double pcRange[6],
                                  const array<double, 2U> &anchorsBEV,
                                  array<double, 1U> &anchorMask)
{
  static double cpu_sparseVoxelMap[214272];
  array<int, 1U> iv;
  double(*gpu_sparseVoxelMap)[214272];
  double numPillars;
  double xMin;
  double yMin;
  int inDims[2];
  int n;
  bool sparseVoxelMap_outdatedOnCpu;
  hipMalloc(&gpu_sparseVoxelMap, 1714176ULL);
  xMin = pcRange[0];
  yMin = pcRange[2];
  std::memset(&cpu_sparseVoxelMap[0], 0, 214272U * sizeof(double));
  n = 0;
  for (int k{0}; k < 24000; k++) {
    if (pillarIndices[k] != 0.0F) {
      n++;
    }
  }
  numPillars = static_cast<double>(n) / 2.0;
  if (numPillars < 1.0) {
    n = 0;
  } else {
    n = static_cast<int>(numPillars);
  }
  iv.set_size(n);
  for (int k{0}; k < n; k++) {
    iv[k] = static_cast<int>(pillarIndices[k]) +
            432 * (static_cast<int>(pillarIndices[k + 12000]) - 1);
  }
  for (int k{0}; k < iv.size(0); k++) {
    cpu_sparseVoxelMap[iv[k] - 1] = 1.0;
  }
  hipMemcpy(*gpu_sparseVoxelMap, cpu_sparseVoxelMap, 1714176ULL,
             hipMemcpyHostToDevice);
  mwCallThrustScanNDEdge(&(*gpu_sparseVoxelMap)[0], 432, false, 214272);
  inDims[0] = 432;
  inDims[1] = 496;
  mwCallThrustScanNDOther(&(*gpu_sparseVoxelMap)[0], 2, &inDims[0], 1, false,
                          214272, false);
  sparseVoxelMap_outdatedOnCpu = true;
  anchorMask.set_size(anchorsBEV.size(0));
  for (int k{0}; k < anchorsBEV.size(0); k++) {
    double xmaxGrid;
    double ymaxGrid;
    double yminGrid;
    numPillars = std::fmax(std::floor((anchorsBEV[k] - xMin) / 0.16), 1.0);
    yminGrid = std::fmax(
        std::floor((anchorsBEV[k + anchorsBEV.size(0)] - yMin) / 0.16), 1.0);
    xmaxGrid = std::fmin(
        std::floor((anchorsBEV[k + anchorsBEV.size(0) * 2] - xMin) / 0.16),
        432.0);
    ymaxGrid = std::fmin(
        std::floor((anchorsBEV[k + anchorsBEV.size(0) * 3] - yMin) / 0.16),
        496.0);
    if (sparseVoxelMap_outdatedOnCpu) {
      hipMemcpy(cpu_sparseVoxelMap, *gpu_sparseVoxelMap, 1714176ULL,
                 hipMemcpyDeviceToHost);
    }
    sparseVoxelMap_outdatedOnCpu = false;
    anchorMask[k] =
        ((cpu_sparseVoxelMap[(static_cast<int>(xmaxGrid) +
                              432 * (static_cast<int>(ymaxGrid) - 1)) -
                             1] -
          cpu_sparseVoxelMap[(static_cast<int>(xmaxGrid) +
                              432 * (static_cast<int>(yminGrid) - 1)) -
                             1]) -
         cpu_sparseVoxelMap[(static_cast<int>(numPillars) +
                             432 * (static_cast<int>(ymaxGrid) - 1)) -
                            1]) +
        cpu_sparseVoxelMap[(static_cast<int>(numPillars) +
                            432 * (static_cast<int>(yminGrid) - 1)) -
                           1];
  }
  hipFree(*gpu_sparseVoxelMap);
}

} // namespace cnn
} // namespace internal
} // namespace lidar
} // namespace coder

//
// File trailer for createAnchorMaskPointPillars.cu
//
// [EOF]
//
