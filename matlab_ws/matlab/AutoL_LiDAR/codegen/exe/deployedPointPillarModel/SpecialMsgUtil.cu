//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: SpecialMsgUtil.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "SpecialMsgUtil.h"
#include "rt_nonfinite.h"
#include "coder_array.h"
#include <cstddef>
#include <cstring>

// Function Definitions
//
// Arguments    : const array<unsigned char, 1U> &b_data
//                const array<double, 2U> &byteIdx
//                const array<bool, 2U> &pointIdxIsValid
//                array<float, 1U> &fieldPoints
// Return Type  : void
//
namespace coder {
namespace ros {
namespace internal {
namespace ros2 {
void SpecialMsgUtil_readFieldFromData(const array<unsigned char, 1U> &b_data,
                                      const array<double, 2U> &byteIdx,
                                      const array<bool, 2U> &pointIdxIsValid,
                                      array<float, 1U> &fieldPoints)
{
  array<float, 1U> fv;
  array<float, 1U> validPoints;
  array<unsigned int, 2U> dv;
  array<int, 2U> r;
  array<int, 2U> r1;
  array<unsigned char, 1U> rawData;
  array<bool, 1U> b_x;
  int end;
  int ny;
  bool b_y;
  bool exitg1;
  end = pointIdxIsValid.size(1) - 1;
  ny = 0;
  for (int i{0}; i <= end; i++) {
    if (pointIdxIsValid[i]) {
      ny++;
    }
  }
  r.set_size(1, ny);
  ny = 0;
  for (int i{0}; i <= end; i++) {
    if (pointIdxIsValid[i]) {
      r[ny] = i;
      ny++;
    }
  }
  dv.set_size(byteIdx.size(1), r.size(1));
  for (end = 0; end < r.size(1); end++) {
    for (ny = 0; ny < byteIdx.size(1); ny++) {
      dv[ny + dv.size(0) * end] =
          static_cast<unsigned int>(byteIdx[r[end] + byteIdx.size(0) * ny]);
    }
  }
  rawData.set_size(dv.size(0) * dv.size(1));
  for (end = 0; end < dv.size(0) * dv.size(1); end++) {
    rawData[end] = b_data[static_cast<int>(dv[end]) - 1];
  }
  if (dv.size(0) * dv.size(1) == 0) {
    ny = 0;
  } else {
    ny = (dv.size(0) * dv.size(1)) >> 2;
  }
  fv.set_size(ny);
  std::memcpy((void *)&(fv.data())[0], (void *)&(rawData.data())[0],
              (unsigned int)((size_t)ny * sizeof(float)));
  ny = fv.size(0);
  validPoints.set_size(fv.size(0));
  for (end = 0; end < ny; end++) {
    validPoints[end] = fv[end];
  }
  b_x.set_size(pointIdxIsValid.size(1));
  for (end = 0; end < pointIdxIsValid.size(1); end++) {
    b_x[end] = !pointIdxIsValid[end];
  }
  b_y = false;
  ny = 1;
  exitg1 = false;
  while ((!exitg1) && (ny <= b_x.size(0))) {
    if (b_x[ny - 1]) {
      b_y = true;
      exitg1 = true;
    } else {
      ny++;
    }
  }
  if (b_y) {
    fieldPoints.set_size(pointIdxIsValid.size(1));
    for (end = 0; end < pointIdxIsValid.size(1); end++) {
      fieldPoints[end] = rtNaNF;
    }
    end = pointIdxIsValid.size(1) - 1;
    ny = 0;
    for (int i{0}; i <= end; i++) {
      if (pointIdxIsValid[i]) {
        ny++;
      }
    }
    r1.set_size(1, ny);
    ny = 0;
    for (int i{0}; i <= end; i++) {
      if (pointIdxIsValid[i]) {
        r1[ny] = i;
        ny++;
      }
    }
    ny = r1.size(1);
    for (end = 0; end < ny; end++) {
      fieldPoints[r1[end]] = validPoints[end];
    }
  } else {
    fieldPoints.set_size(validPoints.size(0));
    for (end = 0; end < validPoints.size(0); end++) {
      fieldPoints[end] = validPoints[end];
    }
  }
}

} // namespace ros2
} // namespace internal
} // namespace ros
} // namespace coder

//
// File trailer for SpecialMsgUtil.cu
//
// [EOF]
//
