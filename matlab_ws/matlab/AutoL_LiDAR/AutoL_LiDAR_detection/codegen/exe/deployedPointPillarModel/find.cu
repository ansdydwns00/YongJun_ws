//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: find.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "find.h"
#include "rt_nonfinite.h"
#include "coder_array.h"

// Function Definitions
//
// Arguments    : const array<bool, 1U> &b_x
//                array<int, 1U> &i
// Return Type  : void
//
namespace coder {
void eml_find(const array<bool, 1U> &b_x, array<int, 1U> &i)
{
  int idx;
  int ii;
  int nx;
  bool exitg1;
  nx = b_x.size(0);
  idx = 0;
  i.set_size(b_x.size(0));
  ii = 1;
  exitg1 = false;
  while ((!exitg1) && (ii <= nx)) {
    if (b_x[ii - 1]) {
      idx++;
      i[idx - 1] = ii;
      if (idx >= nx) {
        exitg1 = true;
      } else {
        ii++;
      }
    } else {
      ii++;
    }
  }
  if (b_x.size(0) == 1) {
    if (idx == 0) {
      i.set_size(0);
    }
  } else {
    if (idx < 1) {
      idx = 0;
    }
    i.set_size(idx);
  }
}

} // namespace coder

//
// File trailer for find.cu
//
// [EOF]
//
