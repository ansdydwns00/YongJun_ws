#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: pointPillarsObjectDetector.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "pointPillarsObjectDetector.h"
#include "PointCloudImpl.h"
#include "arrayfun.h"
#include "bbox2poly.h"
#include "cellstr_sort.h"
#include "cellstr_strtrim.h"
#include "cellstr_unique.h"
#include "cosd.h"
#include "createAnchorMaskPointPillars.h"
#include "createAnchorsPointPillars.h"
#include "deployedPointPillarModel_data.h"
#include "deployedPointPillarModel_internal_types.h"
#include "find.h"
#include "getCategoryNames.h"
#include "overlapPolygonRotatedRects.h"
#include "pointCloudBase.h"
#include "predict.h"
#include "rt_nonfinite.h"
#include "sind.h"
#include "sort.h"
#include "strcmp.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include "MWSortFunctors.h"
#include "MWSortWithIndexUtility.h"
#include "coder_array.h"
#include "coder_gpu_array.h"
#include "createPillars.hpp"
#include "rt_defines.h"
#include <cmath>

// Variable Definitions
static const char cv[10]{'P', 'e', 'd', 'e', 's', 't', 'r', 'a', 'i', 'n'};

// Function Declarations
namespace coder {
namespace internal {
static void iGenerateDetections(const float YPredictions_f1_Data[321408],
                                const float YPredictions_f2_Data[321408],
                                const float YPredictions_f3_Data[107136],
                                const float YPredictions_f4_Data[107136],
                                const float YPredictions_f5_Data[107136],
                                const float YPredictions_f6_Data[107136],
                                const float pillarIndices[24000],
                                double params_Threshold,
                                array<float, 2U> &boxPreds);

static void pointPillarsObjectDetector_iPostprocessSingleDetection(
    const float YPredData_f1_Data[321408],
    const float YPredData_f2_Data[321408],
    const float YPredData_f3_Data[107136],
    const float YPredData_f4_Data[107136],
    const float YPredData_f5_Data[107136],
    const float YPredData_f6_Data[107136], const float pillarIndices[24000],
    array<double, 2U> &boxPred, array<float, 1U> &scores,
    array<unsigned int, 2U> &labelNames_codes,
    array<cell_wrap_6, 1U> &labelNames_categoryNames);

static void
returnCategoricalLabels(array<unsigned int, 2U> &labelNamesMod_codes,
                        array<cell_wrap_6, 1U> &labelNamesMod_categoryNames);

static void
returnCategoricalLabels(double numBBoxes,
                        array<unsigned int, 2U> &labelNamesMod_codes,
                        array<cell_wrap_6, 1U> &labelNamesMod_categoryNames);

} // namespace internal
} // namespace coder
static __global__ void
iGenerateDetections_kernel30(const coder::gpu_array<bool, 3U> a,
                             coder::gpu_array<bool, 3U> anchorMask);

static __global__ void pointPillarsObjectDetector_detect_kernel1(
    const int result_dim0, const coder::gpu_array<float, 2U> intensity,
    const int result_dim1, const int result, const int sizes_idx_1,
    coder::gpu_array<float, 2U> dataIn);

static __global__ void
pointPillarsObjectDetector_detect_kernel2(float pillarIndices[24000]);

static __global__ void pointPillarsObjectDetector_detect_kernel3(
    float dlPillarFeatures_Data[10800000]);

static float rt_atan2f_snf(float u0, float u1);

// Function Definitions
//
// Arguments    : const float YPredictions_f1_Data[321408]
//                const float YPredictions_f2_Data[321408]
//                const float YPredictions_f3_Data[107136]
//                const float YPredictions_f4_Data[107136]
//                const float YPredictions_f5_Data[107136]
//                const float YPredictions_f6_Data[107136]
//                const float pillarIndices[24000]
//                double params_Threshold
//                array<float, 2U> &boxPreds
// Return Type  : void
//
namespace coder {
namespace internal {
static void iGenerateDetections(const float YPredictions_f1_Data[321408],
                                const float YPredictions_f2_Data[321408],
                                const float YPredictions_f3_Data[107136],
                                const float YPredictions_f4_Data[107136],
                                const float YPredictions_f5_Data[107136],
                                const float YPredictions_f6_Data[107136],
                                const float pillarIndices[24000],
                                double params_Threshold,
                                array<float, 2U> &boxPreds)
{
  static const double dv[10]{0.824900632523975,
                             0.824900632523975,
                             0.93545500918179725,
                             0.93545500918179725,
                             1.7943301367067914,
                             1.7943301367067914,
                             -1.78,
                             -1.78,
                             0.0,
                             1.5707963267948966};
  static const double dv1[6]{0.0, 69.12, -39.68, 39.68, -5.0, 5.0};
  static float predAngle[107136];
  static const signed char dv2[3]{2, 3, 1};
  static bool b_x[107136];
  array<cell_wrap_60, 2U> newAnchors;
  array<double, 2U> a__6;
  array<double, 2U> anchorsBEV;
  array<double, 1U> anchorMask;
  array<double, 1U> anchorNum;
  array<double, 1U> c_r1;
  array<double, 1U> c_x;
  array<double, 1U> cls;
  array<double, 1U> col;
  array<double, 1U> r;
  array<double, 1U> row;
  array<float, 1U> angGt;
  array<float, 1U> hdGt;
  array<float, 1U> varargin_1;
  array<float, 1U> varargin_2;
  array<float, 1U> varargin_4;
  array<float, 1U> varargin_5;
  array<float, 1U> varargin_6;
  array<float, 1U> varargin_8;
  array<float, 1U> varargout_1;
  array<float, 1U> zGt;
  array<int, 1U> b_ii;
  array<int, 1U> v1;
  array<short, 1U> vk;
  array<signed char, 1U> varargout_6;
  array<bool, 3U> cpu_a;
  array<bool, 3U> cpu_anchorMask;
  gpu_array<bool, 3U> gpu_a;
  gpu_array<bool, 3U> gpu_anchorMask;
  cell_wrap_5 b_r1[1];
  cell_wrap_5 r1;
  int k;
  int plast;
  bool a_outdatedOnGpu;
  bool anchorMask_outdatedOnCpu;
  bool b;
  bool exitg1;
  anchorMask_outdatedOnCpu = false;
  a_outdatedOnGpu = false;
  for (k = 0; k < 10; k++) {
    r1.f1[k] = dv[k];
  }
  b_r1[0] = r1;
  lidar::internal::cnn::createAnchorsPointPillars(b_r1, a__6, anchorsBEV);
  lidar::internal::cnn::createAnchorMaskPointPillars(pillarIndices, dv1,
                                                     anchorsBEV, anchorMask);
  newAnchors.set_size(1, 2);
  for (plast = 0; plast < 2; plast++) {
    int ii;
    ii = plast + 1;
    for (k = 0; k < 5; k++) {
      newAnchors[ii - 1].f1[k] = dv[plast + (k << 1)];
    }
  }
  cpu_a.set_size(2, 216, 248);
  gpu_a.set_size(&cpu_a);
  for (k = 0; k < 107136; k++) {
    cpu_a[k] = (anchorMask[k] > 1.0);
    a_outdatedOnGpu = true;
  }
  b = true;
  plast = 0;
  k = 0;
  exitg1 = false;
  while ((!exitg1) && (k + 1 < 4)) {
    if (plast > dv2[k]) {
      b = false;
      exitg1 = true;
    } else {
      plast = dv2[k];
      k++;
    }
  }
  if (b) {
    cpu_anchorMask.set_size(216, 248, 2);
    gpu_anchorMask.set_size(&cpu_anchorMask);
    for (k = 0; k < 107136; k++) {
      cpu_anchorMask[k] = cpu_a[k];
    }
  } else {
    cpu_anchorMask.set_size(216, 248, 2);
    gpu_anchorMask.set_size(&cpu_anchorMask);
    if (a_outdatedOnGpu) {
      arrayCopyCpuToGpu(gpu_a, cpu_a);
    }
    iGenerateDetections_kernel30<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        gpu_a, gpu_anchorMask);
    anchorMask_outdatedOnCpu = true;
  }
  for (k = 0; k < 107136; k++) {
    predAngle[k] = YPredictions_f6_Data[k];
  }
  for (k = 0; k < 107136; k++) {
    if (anchorMask_outdatedOnCpu) {
      arrayCopyGpuToCpu(cpu_anchorMask, gpu_anchorMask);
    }
    anchorMask_outdatedOnCpu = false;
    b_x[k] =
        ((YPredictions_f3_Data[k] > params_Threshold) && cpu_anchorMask[k]);
  }
  plast = 0;
  b_ii.set_size(107136);
  k = 1;
  exitg1 = false;
  while ((!exitg1) && (k < 107137)) {
    if (b_x[k - 1]) {
      plast++;
      b_ii[plast - 1] = k;
      if (plast >= 107136) {
        exitg1 = true;
      } else {
        k++;
      }
    } else {
      k++;
    }
  }
  if (plast < 1) {
    plast = 0;
  }
  b_ii.set_size(plast);
  v1.set_size(b_ii.size(0));
  for (k = 0; k < b_ii.size(0); k++) {
    v1[k] = b_ii[k] - 1;
  }
  vk.set_size(v1.size(0));
  for (k = 0; k < v1.size(0); k++) {
    vk[k] = static_cast<short>(v1[k] / 53568);
  }
  varargout_6.set_size(vk.size(0));
  for (k = 0; k < vk.size(0); k++) {
    varargout_6[k] = static_cast<signed char>(vk[k] + 1);
    v1[k] = v1[k] - vk[k] * 53568;
  }
  vk.set_size(v1.size(0));
  for (k = 0; k < v1.size(0); k++) {
    vk[k] = static_cast<short>(v1[k] / 216);
  }
  for (k = 0; k < v1.size(0); k++) {
    v1[k] = v1[k] - vk[k] * 216;
  }
  row.set_size(v1.size(0));
  col.set_size(vk.size(0));
  anchorNum.set_size(varargout_6.size(0));
  for (k = 0; k < v1.size(0); k++) {
    row[k] = static_cast<double>(v1[k]) + 1.0;
    col[k] = static_cast<double>(vk[k]) + 1.0;
    anchorNum[k] = varargout_6[k];
  }
  varargin_8.set_size(row.size(0));
  plast = row.size(0);
  r.set_size(row.size(0));
  c_r1.set_size(col.size(0));
  arrayfun(YPredictions_f2_Data, newAnchors, row, col, anchorNum, zGt);
  varargin_4.set_size(row.size(0));
  varargin_5.set_size(row.size(0));
  varargin_6.set_size(row.size(0));
  hdGt.set_size(row.size(0));
  for (k = 0; k < plast; k++) {
    varargin_8[k] =
        YPredictions_f3_Data[((static_cast<int>(row[k]) +
                               216 * (static_cast<int>(col[k]) - 1)) +
                              53568 * (static_cast<int>(anchorNum[k]) - 1)) -
                             1];
    r[k] = 0.32 * (row[k] - 1.0) + 0.16;
    c_r1[k] = (0.32 * (col[k] - 1.0) - 39.68) + 0.16;
    varargin_4[k] =
        std::exp(YPredictions_f1_Data[((static_cast<int>(row[k]) +
                                        216 * (static_cast<int>(col[k]) - 1)) +
                                       53568 * (static_cast<int>(anchorNum[k]) -
                                                1)) -
                                      1]) *
        static_cast<float>(
            newAnchors[static_cast<int>(anchorNum[k]) - 1].f1[0]);
    varargin_5[k] =
        std::exp(YPredictions_f1_Data[((static_cast<int>(row[k]) +
                                        216 * (static_cast<int>(col[k]) - 1)) +
                                       53568 * (static_cast<int>(anchorNum[k]) -
                                                1)) +
                                      107135]) *
        static_cast<float>(
            newAnchors[static_cast<int>(anchorNum[k]) - 1].f1[1]);
    varargin_6[k] =
        std::exp(YPredictions_f1_Data[((static_cast<int>(row[k]) +
                                        216 * (static_cast<int>(col[k]) - 1)) +
                                       53568 * (static_cast<int>(anchorNum[k]) -
                                                1)) +
                                      214271]) *
        static_cast<float>(
            newAnchors[static_cast<int>(anchorNum[k]) - 1].f1[2]);
    hdGt[k] =
        YPredictions_f5_Data[((static_cast<int>(row[k]) +
                               216 * (static_cast<int>(col[k]) - 1)) +
                              53568 * (static_cast<int>(anchorNum[k]) - 1)) -
                             1];
  }
  applyArrayFunction(YPredictions_f2_Data, newAnchors, row, col, anchorNum, r,
                     varargin_1);
  b_applyArrayFunction(YPredictions_f2_Data, newAnchors, row, col, anchorNum,
                       c_r1, varargin_2);
  plast = hdGt.size(0) - 1;
  for (k = 0; k <= plast; k++) {
    if (hdGt[k] >= 0.5F) {
      hdGt[k] = 1.0F;
    }
  }
  plast = hdGt.size(0) - 1;
  for (k = 0; k <= plast; k++) {
    if (hdGt[k] < 0.5F) {
      hdGt[k] = -1.0F;
    }
  }
  for (k = 0; k < 107136; k++) {
    float f;
    f = YPredictions_f6_Data[k];
    if ((f > 1.0F) || (f < -1.0F)) {
      predAngle[k] = 0.0F;
    }
  }
  arrayfun(predAngle, newAnchors, row, col, anchorNum, hdGt, angGt);
  plast = angGt.size(0) - 1;
  for (k = 0; k <= plast; k++) {
    if (angGt[k] > 3.1415926535897931) {
      angGt[k] = angGt[k] - 6.28318548F;
    }
  }
  plast = angGt.size(0) - 1;
  for (k = 0; k <= plast; k++) {
    if (angGt[k] < -3.1415926535897931) {
      angGt[k] = angGt[k] + 6.28318548F;
    }
  }
  varargout_1.set_size(row.size(0));
  plast = row.size(0);
  for (k = 0; k < plast; k++) {
    varargout_1[k] =
        YPredictions_f4_Data[((static_cast<int>(row[k]) +
                               216 * (static_cast<int>(col[k]) - 1)) +
                              53568 * (static_cast<int>(anchorNum[k]) - 1)) -
                             1];
  }
  c_x.set_size(varargout_1.size(0));
  for (k = 0; k < varargout_1.size(0); k++) {
    c_x[k] = -static_cast<double>(varargout_1[k]);
  }
  plast = c_x.size(0);
  for (k = 0; k < plast; k++) {
    c_x[k] = std::exp(c_x[k]);
  }
  cls.set_size(c_x.size(0));
  for (k = 0; k < c_x.size(0); k++) {
    cls[k] = 1.0 / (c_x[k] + 1.0);
  }
  plast = cls.size(0) - 1;
  for (k = 0; k <= plast; k++) {
    if (cls[k] >= params_Threshold) {
      cls[k] = 1.0;
    }
  }
  plast = cls.size(0) - 1;
  for (k = 0; k <= plast; k++) {
    if (cls[k] < params_Threshold) {
      cls[k] = 0.0;
    }
  }
  boxPreds.set_size(varargin_1.size(0), 9);
  plast = varargin_1.size(0);
  for (k = 0; k < plast; k++) {
    boxPreds[k] = varargin_1[k];
  }
  plast = varargin_2.size(0);
  for (k = 0; k < plast; k++) {
    boxPreds[k + boxPreds.size(0)] = varargin_2[k];
  }
  plast = zGt.size(0);
  for (k = 0; k < plast; k++) {
    boxPreds[k + boxPreds.size(0) * 2] = 0.5F * zGt[k];
  }
  plast = varargin_4.size(0);
  for (k = 0; k < plast; k++) {
    boxPreds[k + boxPreds.size(0) * 3] = varargin_4[k];
    boxPreds[k + boxPreds.size(0) * 4] = varargin_5[k];
    boxPreds[k + boxPreds.size(0) * 5] = varargin_6[k];
  }
  plast = angGt.size(0);
  for (k = 0; k < plast; k++) {
    boxPreds[k + boxPreds.size(0) * 6] = angGt[k];
  }
  plast = varargin_8.size(0);
  for (k = 0; k < plast; k++) {
    boxPreds[k + boxPreds.size(0) * 7] = varargin_8[k];
    boxPreds[k + boxPreds.size(0) * 8] = static_cast<float>(cls[k]);
  }
}

//
// Arguments    : const float YPredData_f1_Data[321408]
//                const float YPredData_f2_Data[321408]
//                const float YPredData_f3_Data[107136]
//                const float YPredData_f4_Data[107136]
//                const float YPredData_f5_Data[107136]
//                const float YPredData_f6_Data[107136]
//                const float pillarIndices[24000]
//                array<double, 2U> &boxPred
//                array<float, 1U> &scores
//                array<unsigned int, 2U> &labelNames_codes
//                array<cell_wrap_6, 1U> &labelNames_categoryNames
// Return Type  : void
//
static void pointPillarsObjectDetector_iPostprocessSingleDetection(
    const float YPredData_f1_Data[321408],
    const float YPredData_f2_Data[321408],
    const float YPredData_f3_Data[107136],
    const float YPredData_f4_Data[107136],
    const float YPredData_f5_Data[107136],
    const float YPredData_f6_Data[107136], const float pillarIndices[24000],
    array<double, 2U> &boxPred, array<float, 1U> &scores,
    array<unsigned int, 2U> &labelNames_codes,
    array<cell_wrap_6, 1U> &labelNames_categoryNames)
{
  static const signed char iv4[7]{0, 1, 2, 3, 4, 5, 8};
  static const signed char iv[5]{0, 1, 3, 4, 6};
  array<double, 1U> cpu_idx;
  array<float, 2U> X;
  array<float, 2U> Y;
  array<float, 2U> area;
  array<float, 2U> b_bbox;
  array<float, 2U> b_overlapedPolyPts;
  array<float, 2U> boxPreds;
  array<float, 2U> c_bbox;
  array<float, 2U> d_bbox;
  array<float, 2U> distVec;
  array<float, 2U> fv1;
  array<float, 2U> overlapPoly;
  array<float, 2U> overlapedPolyPts;
  array<float, 2U> r3;
  array<float, 2U> r4;
  array<float, 2U> theta;
  array<float, 2U> u;
  array<float, 2U> v;
  array<float, 1U> b_X;
  array<float, 1U> b_Y;
  array<float, 1U> b_r;
  array<float, 1U> cpu_out;
  array<float, 1U> r;
  array<float, 1U> r1;
  array<float, 1U> r2;
  array<float, 1U> xIndices1;
  array<float, 1U> yIndices1;
  array<int, 2U> iidx;
  array<int, 1U> b_index;
  array<int, 1U> posIdx;
  array<bool, 1U> b_boxPreds;
  array<bool, 1U> b_selectedIndex;
  array<bool, 1U> selectedIndex;
  gpu_array<double, 1U> gpu_idx;
  gpu_array<float, 1U> gpu_out;
  int inDims[2];
  bool idx_outdatedOnGpu;
  bool out_outdatedOnGpu;
  idx_outdatedOnGpu = false;
  out_outdatedOnGpu = false;
  iGenerateDetections(YPredData_f1_Data, YPredData_f2_Data, YPredData_f3_Data,
                      YPredData_f4_Data, YPredData_f5_Data, YPredData_f6_Data,
                      pillarIndices, 0.3, boxPreds);
  if (boxPreds.size(0) != 0) {
    int i;
    int sortDim;
    b_boxPreds.set_size(boxPreds.size(0));
    for (i = 0; i < boxPreds.size(0); i++) {
      b_boxPreds[i] = (boxPreds[i + boxPreds.size(0) * 8] > 0.15);
    }
    eml_find(b_boxPreds, posIdx);
    sortDim = boxPreds.size(0);
    for (i = 0; i < sortDim; i++) {
      boxPreds[i + boxPreds.size(0) * 6] =
          57.2957802F * boxPreds[i + boxPreds.size(0) * 6];
    }
    if (posIdx.size(0) != 0) {
      float center_idx_0;
      int currentBox;
      int inidx;
      int numOfBbox;
      int vidx;
      scores.set_size(posIdx.size(0));
      for (i = 0; i < posIdx.size(0); i++) {
        scores[i] = boxPreds[(posIdx[i] + boxPreds.size(0) * 7) - 1];
      }
      sortDim = 2;
      if (posIdx.size(0) != 1) {
        sortDim = 1;
      }
      inDims[0] = posIdx.size(0);
      inDims[1] = 1;
      cpu_out.set_size(scores.size(0));
      gpu_out.set_size(&cpu_out);
      cpu_idx.set_size(posIdx.size(0));
      gpu_idx.set_size(&cpu_idx);
      for (i = 0; i < scores.size(0); i++) {
        cpu_out[i] = scores[i];
        out_outdatedOnGpu = true;
        cpu_idx[i] = 0.0;
        idx_outdatedOnGpu = true;
      }
      if (out_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_out, cpu_out);
      }
      if (idx_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_idx, cpu_idx);
      }
      gpu_thrust_sort_with_index(&(gpu_out.data())[0], &(gpu_idx.data())[0], 2,
                                 &inDims[0], sortDim, 'd', false);
      idx_outdatedOnGpu = true;
      b_bbox.set_size(cpu_idx.size(0), 5);
      for (i = 0; i < 5; i++) {
        for (inidx = 0; inidx < cpu_idx.size(0); inidx++) {
          if (idx_outdatedOnGpu) {
            arrayCopyGpuToCpu(cpu_idx, gpu_idx);
          }
          idx_outdatedOnGpu = false;
          b_bbox[inidx + b_bbox.size(0) * i] =
              boxPreds[(posIdx[static_cast<int>(cpu_idx[inidx]) - 1] +
                        boxPreds.size(0) * iv[i]) -
                       1];
        }
      }
      selectedIndex.set_size(cpu_idx.size(0));
      for (i = 0; i < cpu_idx.size(0); i++) {
        selectedIndex[i] = true;
      }
      X.set_size(4, cpu_idx.size(0));
      Y.set_size(4, cpu_idx.size(0));
      for (i = 0; i < (cpu_idx.size(0) << 2); i++) {
        X[i] = 0.0F;
        Y[i] = 0.0F;
      }
      r.set_size(cpu_idx.size(0));
      for (i = 0; i < cpu_idx.size(0); i++) {
        r[i] = b_bbox[i + b_bbox.size(0) * 4];
      }
      b_r.set_size(r.size(0));
      for (i = 0; i < r.size(0); i++) {
        b_r[i] = r[i];
      }
      b_cosd(b_r);
      r1.set_size(r.size(0));
      for (i = 0; i < r.size(0); i++) {
        r1[i] = r[i];
      }
      b_sind(r1);
      if (cpu_idx.size(0) == b_r.size(0)) {
        c_bbox.set_size(cpu_idx.size(0), 2);
        sortDim = cpu_idx.size(0);
        for (i = 0; i < sortDim; i++) {
          center_idx_0 = b_bbox[i + b_bbox.size(0) * 2];
          c_bbox[i] = center_idx_0 / 2.0F;
          c_bbox[i + c_bbox.size(0)] = center_idx_0 / 2.0F;
        }
        r3.set_size(b_r.size(0), 2);
        sortDim = b_r.size(0);
        for (i = 0; i < sortDim; i++) {
          r3[i] = b_r[i];
        }
        sortDim = r1.size(0);
        for (i = 0; i < sortDim; i++) {
          r3[i + r3.size(0)] = r1[i];
        }
        u.set_size(c_bbox.size(0), 2);
        for (i = 0; i < (c_bbox.size(0) << 1); i++) {
          u[i] = c_bbox[i] * r3[i];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op_9(u, b_bbox, cpu_idx, b_r, r1);
      }
      r2.set_size(r.size(0));
      for (i = 0; i < r.size(0); i++) {
        r2[i] = r[i];
      }
      b_sind(r2);
      b_cosd(r);
      if (cpu_idx.size(0) == r2.size(0)) {
        d_bbox.set_size(cpu_idx.size(0), 2);
        sortDim = cpu_idx.size(0);
        for (i = 0; i < sortDim; i++) {
          center_idx_0 = b_bbox[i + b_bbox.size(0) * 3];
          d_bbox[i] = center_idx_0 / 2.0F;
          d_bbox[i + d_bbox.size(0)] = center_idx_0 / 2.0F;
        }
        r4.set_size(r2.size(0), 2);
        sortDim = r2.size(0);
        for (i = 0; i < sortDim; i++) {
          r4[i] = -r2[i];
        }
        sortDim = r.size(0);
        for (i = 0; i < sortDim; i++) {
          r4[i + r4.size(0)] = r[i];
        }
        v.set_size(d_bbox.size(0), 2);
        for (i = 0; i < (d_bbox.size(0) << 1); i++) {
          v[i] = d_bbox[i] * r4[i];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op_8(v, b_bbox, cpu_idx, r2, r);
      }
      if (cpu_idx.size(0) == 1) {
        sortDim = u.size(0);
      } else {
        sortDim = cpu_idx.size(0);
      }
      if ((cpu_idx.size(0) == u.size(0)) && (sortDim == v.size(0))) {
        sortDim = cpu_idx.size(0);
        for (i = 0; i < sortDim; i++) {
          X[4 * i] = (b_bbox[i] - u[i]) - v[i];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op_7(X, cpu_idx, b_bbox, u, v);
      }
      if (cpu_idx.size(0) == 1) {
        sortDim = u.size(0);
      } else {
        sortDim = cpu_idx.size(0);
      }
      if ((cpu_idx.size(0) == u.size(0)) && (sortDim == v.size(0))) {
        sortDim = cpu_idx.size(0);
        for (i = 0; i < sortDim; i++) {
          Y[4 * i] = (b_bbox[i + b_bbox.size(0)] - u[i + u.size(0)]) -
                     v[i + v.size(0)];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op_6(Y, cpu_idx, b_bbox, u, v);
      }
      if (cpu_idx.size(0) == 1) {
        sortDim = u.size(0);
      } else {
        sortDim = cpu_idx.size(0);
      }
      if ((cpu_idx.size(0) == u.size(0)) && (sortDim == v.size(0))) {
        sortDim = X.size(1);
        for (i = 0; i < sortDim; i++) {
          X[4 * i + 1] = (b_bbox[i] + u[i]) - v[i];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op_5(X, b_bbox, cpu_idx, u, v);
      }
      if (cpu_idx.size(0) == 1) {
        sortDim = u.size(0);
      } else {
        sortDim = cpu_idx.size(0);
      }
      if ((cpu_idx.size(0) == u.size(0)) && (sortDim == v.size(0))) {
        sortDim = Y.size(1);
        for (i = 0; i < sortDim; i++) {
          Y[4 * i + 1] = (b_bbox[i + b_bbox.size(0)] + u[i + u.size(0)]) -
                         v[i + v.size(0)];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op_4(Y, b_bbox, cpu_idx, u, v);
      }
      if (cpu_idx.size(0) == 1) {
        sortDim = u.size(0);
      } else {
        sortDim = cpu_idx.size(0);
      }
      if ((cpu_idx.size(0) == u.size(0)) && (sortDim == v.size(0))) {
        sortDim = X.size(1);
        for (i = 0; i < sortDim; i++) {
          X[4 * i + 2] = (b_bbox[i] + u[i]) + v[i];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op_3(X, b_bbox, cpu_idx, u, v);
      }
      if (cpu_idx.size(0) == 1) {
        sortDim = u.size(0);
      } else {
        sortDim = cpu_idx.size(0);
      }
      if ((cpu_idx.size(0) == u.size(0)) && (sortDim == v.size(0))) {
        sortDim = Y.size(1);
        for (i = 0; i < sortDim; i++) {
          Y[4 * i + 2] = (b_bbox[i + b_bbox.size(0)] + u[i + u.size(0)]) +
                         v[i + v.size(0)];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op_2(Y, b_bbox, cpu_idx, u, v);
      }
      if (cpu_idx.size(0) == 1) {
        sortDim = u.size(0);
      } else {
        sortDim = cpu_idx.size(0);
      }
      if ((cpu_idx.size(0) == u.size(0)) && (sortDim == v.size(0))) {
        sortDim = X.size(1);
        for (i = 0; i < sortDim; i++) {
          X[4 * i + 3] = (b_bbox[i] - u[i]) + v[i];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op_1(X, b_bbox, cpu_idx, u, v);
      }
      if (cpu_idx.size(0) == 1) {
        sortDim = u.size(0);
      } else {
        sortDim = cpu_idx.size(0);
      }
      if ((cpu_idx.size(0) == u.size(0)) && (sortDim == v.size(0))) {
        sortDim = Y.size(1);
        for (i = 0; i < sortDim; i++) {
          Y[4 * i + 3] = (b_bbox[i + b_bbox.size(0)] - u[i + u.size(0)]) +
                         v[i + v.size(0)];
        }
      } else {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        binary_expand_op(Y, b_bbox, cpu_idx, u, v);
      }
      area.set_size(1, X.size(1));
      sortDim = X.size(1);
      for (int b_i{0}; b_i < sortDim; b_i++) {
        inidx = b_i << 2;
        vidx = b_i << 2;
        area[b_i] = std::abs(
            (((X[vidx] - X[vidx + 3]) * (Y[vidx] + Y[vidx + 3]) / 2.0F +
              (X[inidx + 1] - X[inidx]) * (Y[inidx + 1] + Y[inidx]) / 2.0F) +
             (X[inidx + 2] - X[inidx + 1]) * (Y[inidx + 2] + Y[inidx + 1]) /
                 2.0F) +
            (X[inidx + 3] - X[inidx + 2]) * (Y[inidx + 3] + Y[inidx + 2]) /
                2.0F);
      }
      numOfBbox = cpu_idx.size(0) - 2;
      currentBox = 0;
      for (int b_i{0}; b_i <= numOfBbox + 1; b_i++) {
        currentBox = b_i + 1;
        xIndices1.set_size(5);
        xIndices1[4] = X[4 * b_i];
        yIndices1.set_size(5);
        xIndices1[0] = X[4 * b_i];
        yIndices1[0] = Y[4 * b_i];
        xIndices1[1] = X[4 * b_i + 1];
        yIndices1[1] = Y[4 * b_i + 1];
        xIndices1[2] = X[4 * b_i + 2];
        yIndices1[2] = Y[4 * b_i + 2];
        xIndices1[3] = X[4 * b_i + 3];
        yIndices1[3] = Y[4 * b_i + 3];
        yIndices1[4] = Y[4 * b_i];
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        if (std::isnan(boxPreds[(posIdx[static_cast<int>(cpu_idx[b_i]) - 1] +
                                 boxPreds.size(0) * 8) -
                                1])) {
          selectedIndex[b_i] = false;
        } else {
          if (selectedIndex[b_i]) {
            sortDim = 2;
          } else {
            sortDim = 1;
          }
          if (sortDim != 1) {
            int i3;
            i3 = numOfBbox - b_i;
            for (int j{0}; j <= i3; j++) {
              unsigned int b_j;
              b_j = (static_cast<unsigned int>(b_i) +
                     static_cast<unsigned int>(j)) +
                    2U;
              if (selectedIndex[static_cast<int>(b_j) - 1] &&
                  (!(boxPreds[(posIdx[static_cast<int>(
                                          cpu_idx[static_cast<int>(b_j) - 1]) -
                                      1] +
                               boxPreds.size(0) * 8) -
                              1] !=
                     boxPreds[(posIdx[static_cast<int>(cpu_idx[b_i]) - 1] +
                               boxPreds.size(0) * 8) -
                              1]))) {
                b_X.set_size(5);
                b_X[4] = X[4 * (static_cast<int>(b_j) - 1)];
                b_Y.set_size(5);
                b_X[0] = X[4 * (static_cast<int>(b_j) - 1)];
                b_Y[0] = Y[4 * (static_cast<int>(b_j) - 1)];
                b_X[1] = X[4 * (static_cast<int>(b_j) - 1) + 1];
                b_Y[1] = Y[4 * (static_cast<int>(b_j) - 1) + 1];
                b_X[2] = X[4 * (static_cast<int>(b_j) - 1) + 2];
                b_Y[2] = Y[4 * (static_cast<int>(b_j) - 1) + 2];
                b_X[3] = X[4 * (static_cast<int>(b_j) - 1) + 3];
                b_Y[3] = Y[4 * (static_cast<int>(b_j) - 1) + 3];
                b_Y[4] = Y[4 * (static_cast<int>(b_j) - 1)];
                vision::internal::detector::
                    overlapPolygonRotatedRects_findOverlapPolyPoints(
                        xIndices1, yIndices1, b_X, b_Y, overlapedPolyPts);
                if (overlapedPolyPts.size(1) != 0) {
                  float center_idx_1;
                  sortDim = overlapedPolyPts.size(1);
                  center_idx_0 = overlapedPolyPts[0];
                  center_idx_1 = overlapedPolyPts[1];
                  for (vidx = 0; vidx <= sortDim - 2; vidx++) {
                    inidx = (vidx + 1) << 1;
                    center_idx_0 += overlapedPolyPts[inidx];
                    center_idx_1 += overlapedPolyPts[inidx + 1];
                  }
                  center_idx_0 /= static_cast<float>(overlapedPolyPts.size(1));
                  center_idx_1 /= static_cast<float>(overlapedPolyPts.size(1));
                  fv1.set_size(2, overlapedPolyPts.size(1));
                  sortDim = overlapedPolyPts.size(1);
                  for (vidx = 0; vidx < sortDim; vidx++) {
                    inidx = vidx << 1;
                    fv1[inidx] = center_idx_0;
                    fv1[inidx + 1] = center_idx_1;
                  }
                  if (overlapedPolyPts.size(1) == fv1.size(1)) {
                    distVec.set_size(2, overlapedPolyPts.size(1));
                    for (i = 0; i < (overlapedPolyPts.size(1) << 1); i++) {
                      distVec[i] = overlapedPolyPts[i] - fv1[i];
                    }
                  } else {
                    minus(distVec, overlapedPolyPts, fv1);
                  }
                  i = distVec.size(1);
                  theta.set_size(1, distVec.size(1));
                  for (vidx = 0; vidx < i; vidx++) {
                    theta[vidx] =
                        rt_atan2f_snf(distVec[2 * vidx + 1], distVec[2 * vidx]);
                  }
                  sort(theta, iidx);
                  b_overlapedPolyPts.set_size(2, iidx.size(1));
                  for (i = 0; i < iidx.size(1); i++) {
                    b_overlapedPolyPts[2 * i] =
                        overlapedPolyPts[2 * (iidx[i] - 1)];
                    b_overlapedPolyPts[2 * i + 1] =
                        overlapedPolyPts[2 * (iidx[i] - 1) + 1];
                  }
                  overlapedPolyPts.set_size(2, b_overlapedPolyPts.size(1));
                  for (i = 0; i < (b_overlapedPolyPts.size(1) << 1); i++) {
                    overlapedPolyPts[i] = b_overlapedPolyPts[i];
                  }
                  overlapPoly.set_size(2, overlapedPolyPts.size(1) + 1);
                  sortDim = overlapedPolyPts.size(1);
                  for (i = 0; i < sortDim; i++) {
                    overlapPoly[2 * i] = overlapedPolyPts[2 * i];
                    overlapPoly[2 * i + 1] = overlapedPolyPts[2 * i + 1];
                  }
                  overlapPoly[2 * overlapedPolyPts.size(1)] =
                      overlapedPolyPts[0];
                  overlapPoly[2 * overlapedPolyPts.size(1) + 1] =
                      overlapedPolyPts[1];
                  center_idx_0 =
                      (overlapPoly[0] -
                       overlapPoly[2 * (overlapPoly.size(1) - 1)]) *
                      (overlapPoly[1] +
                       overlapPoly[2 * (overlapPoly.size(1) - 1) + 1]) /
                      2.0F;
                  sortDim = overlapPoly.size(1);
                  for (vidx = 0; vidx <= sortDim - 2; vidx++) {
                    center_idx_0 +=
                        (overlapPoly[2 * (vidx + 1)] - overlapPoly[2 * vidx]) *
                        (overlapPoly[2 * (vidx + 1) + 1] +
                         overlapPoly[2 * vidx + 1]) /
                        2.0F;
                  }
                  if (std::abs(center_idx_0) /
                          std::fmin(area[b_i],
                                    area[static_cast<int>(b_j) - 1]) >
                      0.1) {
                    selectedIndex[static_cast<int>(b_j) - 1] = false;
                  }
                }
              }
            }
          }
        }
      }
      if (static_cast<unsigned int>(currentBox) + 1U >
          static_cast<unsigned int>(selectedIndex.size(0))) {
        currentBox = 0;
        sortDim = 0;
      } else {
        sortDim = selectedIndex.size(0);
      }
      sortDim -= currentBox;
      for (i = 0; i < sortDim; i++) {
        selectedIndex[currentBox + i] = false;
      }
      b_selectedIndex.set_size(selectedIndex.size(0));
      for (i = 0; i < selectedIndex.size(0); i++) {
        b_selectedIndex[i] = selectedIndex[i];
      }
      for (i = 0; i < b_selectedIndex.size(0); i++) {
        if (idx_outdatedOnGpu) {
          arrayCopyGpuToCpu(cpu_idx, gpu_idx);
        }
        idx_outdatedOnGpu = false;
        selectedIndex[static_cast<int>(cpu_idx[i]) - 1] = b_selectedIndex[i];
      }
      sortDim = selectedIndex.size(0) - 1;
      inidx = 0;
      vidx = 0;
      for (int b_i{0}; b_i <= sortDim; b_i++) {
        if (selectedIndex[b_i]) {
          inidx++;
          scores[vidx] = scores[b_i];
          vidx++;
        }
      }
      scores.set_size(inidx);
      eml_find(selectedIndex, b_index);
      boxPred.set_size(b_index.size(0), 9);
      for (i = 0; i < b_index.size(0) * 9; i++) {
        boxPred[i] = 0.0;
      }
      sortDim = b_index.size(0);
      for (i = 0; i < 7; i++) {
        for (inidx = 0; inidx < sortDim; inidx++) {
          boxPred[inidx + boxPred.size(0) * iv4[i]] =
              boxPreds[(b_index[inidx] + boxPreds.size(0) * i) - 1];
        }
      }
      returnCategoricalLabels(static_cast<double>(boxPred.size(0)),
                              labelNames_codes, labelNames_categoryNames);
    } else {
      boxPred.set_size(0, 9);
      scores.set_size(0);
      returnCategoricalLabels(labelNames_codes, labelNames_categoryNames);
    }
  } else {
    boxPred.set_size(0, 9);
    scores.set_size(0);
    returnCategoricalLabels(labelNames_codes, labelNames_categoryNames);
  }
}

//
// Arguments    : array<unsigned int, 2U> &labelNamesMod_codes
//                array<cell_wrap_6, 1U> &labelNamesMod_categoryNames
// Return Type  : void
//
static void
returnCategoricalLabels(array<unsigned int, 2U> &labelNamesMod_codes,
                        array<cell_wrap_6, 1U> &labelNamesMod_categoryNames)
{
  array<cell_wrap_6, 2U> rv;
  array<cell_wrap_6, 1U> b_rv;
  array<cell_wrap_6, 1U> c_rv;
  array<cell_wrap_6, 1U> uB;
  array<cell_wrap_9, 2U> valueset;
  array<double, 1U> ib;
  int rv_idx_0;
  valueset.set_size(1, 1);
  for (rv_idx_0 = 0; rv_idx_0 < 10; rv_idx_0++) {
    valueset[0].f1[rv_idx_0] = cv[rv_idx_0];
  }
  matlab::internal::coder::datatypes::cellstr_strtrim(valueset, rv);
  rv_idx_0 = rv.size(1);
  b_rv = rv.reshape(rv_idx_0);
  matlab::internal::coder::datatypes::cellstr_unique(b_rv, c_rv);
  rv_idx_0 = rv.size(1);
  b_rv = rv.reshape(rv_idx_0);
  categorical_getCategoryNames(b_rv, labelNamesMod_categoryNames);
  rv_idx_0 = rv.size(1);
  b_rv = rv.reshape(rv_idx_0);
  matlab::internal::coder::datatypes::cellstr_unique(b_rv, uB, ib);
  labelNamesMod_codes.set_size(1, 0);
}

//
// Arguments    : double numBBoxes
//                array<unsigned int, 2U> &labelNamesMod_codes
//                array<cell_wrap_6, 1U> &labelNamesMod_categoryNames
// Return Type  : void
//
static void
returnCategoricalLabels(double numBBoxes,
                        array<unsigned int, 2U> &labelNamesMod_codes,
                        array<cell_wrap_6, 1U> &labelNamesMod_categoryNames)
{
  array<char, 2U> *b_x;
  array<char, 2U> *b_y;
  array<cell_wrap_6, 2U> rv;
  array<cell_wrap_6, 1U> b_rv;
  array<cell_wrap_6, 1U> c;
  array<cell_wrap_6, 1U> inData;
  array<cell_wrap_6, 1U> labelCells;
  array<cell_wrap_6, 1U> uA;
  array<cell_wrap_6, 1U> uB;
  array<cell_wrap_9, 2U> valueset;
  array<double, 1U> cpu_icA;
  array<double, 1U> ib;
  array<double, 1U> icA;
  array<double, 1U> locbOfUA;
  array<int, 2U> c_y;
  array<int, 2U> d_y;
  array<int, 1U> a__2;
  array<int, 1U> dIdx;
  array<int, 1U> idx;
  array<char, 1U> b_v;
  array<bool, 1U> d;
  array<bool, 1U> r;
  gpu_array<double, 1U> gpu_icA;
  double b_d;
  int b_i;
  int b_j1;
  int c_i;
  int i;
  int j;
  int j2;
  char v[10];
  bool icA_outdatedOnCpu;
  bool icA_outdatedOnGpu;
  icA_outdatedOnCpu = false;
  icA_outdatedOnGpu = false;
  labelCells.set_size(static_cast<int>(numBBoxes));
  i = static_cast<int>(numBBoxes);
  for (b_i = 0; b_i < i; b_i++) {
    b_j1 = 0;
    for (j = 0; j < 10; j++) {
      b_j1 = j + 1;
    }
    b_v.set_size(b_j1);
    for (j = 0; j < 10; j++) {
      b_v[j] = cv[j];
    }
    labelCells[b_i].f1.set_size(1, b_v.size(0));
    for (c_i = 0; c_i < b_v.size(0); c_i++) {
      labelCells[b_i].f1[c_i] = b_v[c_i];
    }
  }
  for (j = 0; j < 10; j++) {
    v[j] = cv[j];
  }
  valueset.set_size(1, 1);
  for (c_i = 0; c_i < 10; c_i++) {
    valueset[0].f1[c_i] = v[c_i];
  }
  inData.set_size(labelCells.size(0));
  j = labelCells.size(0);
  for (i = 0; i < j; i++) {
    b_x = &labelCells[i].f1;
    b_y = &inData[i].f1;
    b_j1 = 1;
    while ((b_j1 <= b_x->size(1)) &&
           bv[static_cast<unsigned char>((*b_x)[b_j1 - 1]) & 127] &&
           ((*b_x)[b_j1 - 1] != '\x00')) {
      b_j1++;
    }
    j2 = b_x->size(1);
    while ((j2 > 0) && bv[static_cast<unsigned char>((*b_x)[j2 - 1]) & 127] &&
           ((*b_x)[j2 - 1] != '\x00')) {
      j2--;
    }
    if (b_j1 > j2) {
      b_i = 0;
      j2 = 0;
    } else {
      b_i = b_j1 - 1;
    }
    b_y->set_size(1, j2 - b_i);
    for (c_i = 0; c_i < j2 - b_i; c_i++) {
      (*b_y)[c_i] = (*b_x)[b_i + c_i];
    }
  }
  matlab::internal::coder::datatypes::cellstr_strtrim(valueset, rv);
  b_j1 = rv.size(1);
  labelCells = rv.reshape(b_j1);
  matlab::internal::coder::datatypes::cellstr_unique(labelCells, b_rv);
  b_j1 = rv.size(1);
  labelCells = rv.reshape(b_j1);
  categorical_getCategoryNames(labelCells, labelNamesMod_categoryNames);
  if (inData.size(0) == 0) {
    uA.set_size(0);
    cpu_icA.set_size(0);
    gpu_icA.set_size(&cpu_icA);
  } else {
    b_j1 = inData.size(0);
    labelCells = inData.reshape(b_j1);
    matlab::internal::coder::datatypes::cellstr_sort(labelCells, c, idx);
    d.set_size(c.size(0) - 1);
    for (i = 0; i <= c.size(0) - 2; i++) {
      d[i] = !b_strcmp(c[i].f1, c[i + 1].f1);
    }
    r.set_size(d.size(0) + 1);
    r[0] = true;
    b_j1 = d.size(0);
    for (c_i = 0; c_i < b_j1; c_i++) {
      r[c_i + 1] = d[c_i];
    }
    d.set_size(r.size(0));
    for (c_i = 0; c_i < r.size(0); c_i++) {
      d[c_i] = r[c_i];
    }
    j2 = d.size(0);
    b_j1 = d[0];
    for (j = 0; j <= j2 - 2; j++) {
      b_j1 += d[j + 1];
    }
    uA.set_size(static_cast<int>(std::fmin(
        static_cast<double>(b_j1), static_cast<double>(inData.size(0)))));
    eml_find(d, dIdx);
    a__2.set_size(dIdx.size(0));
    j = dIdx.size(0);
    for (i = 0; i < j; i++) {
      if (i + 1 != dIdx.size(0)) {
        if (static_cast<double>(dIdx[i + 1]) - 1.0 < dIdx[i]) {
          d_y.set_size(1, 0);
        } else {
          c_i = dIdx[i];
          b_d = static_cast<double>(dIdx[i + 1]) - 1.0;
          d_y.set_size(
              1, static_cast<int>((static_cast<double>(dIdx[i + 1]) - 1.0) -
                                  static_cast<double>(dIdx[i])) +
                     1);
          for (b_j1 = 0;
               b_j1 <= static_cast<int>(b_d - static_cast<double>(c_i));
               b_j1++) {
            d_y[b_j1] = c_i + b_j1;
          }
        }
        j2 = d_y.size(1);
        b_j1 = idx[d_y[0] - 1];
        for (b_i = 0; b_i <= j2 - 2; b_i++) {
          c_i = d_y[b_i + 1];
          if (b_j1 > idx[c_i - 1]) {
            b_j1 = idx[c_i - 1];
          }
        }
        a__2[i] = b_j1;
      } else {
        if (idx.size(0) < dIdx[i]) {
          c_y.set_size(1, 0);
        } else {
          c_i = dIdx[i];
          c_y.set_size(1, (idx.size(0) - dIdx[i]) + 1);
          for (b_j1 = 0; b_j1 <= idx.size(0) - c_i; b_j1++) {
            c_y[b_j1] = c_i + b_j1;
          }
        }
        j2 = c_y.size(1);
        b_j1 = idx[c_y[0] - 1];
        for (b_i = 0; b_i <= j2 - 2; b_i++) {
          c_i = c_y[b_i + 1];
          if (b_j1 > idx[c_i - 1]) {
            b_j1 = idx[c_i - 1];
          }
        }
        a__2[i] = b_j1;
      }
    }
    cpu_icA.set_size(d.size(0));
    gpu_icA.set_size(&cpu_icA);
    for (c_i = 0; c_i < d.size(0); c_i++) {
      cpu_icA[c_i] = d[c_i];
      icA_outdatedOnGpu = true;
    }
    if (cpu_icA.size(0) != 1) {
      if (icA_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_icA, cpu_icA);
      }
      mwCallThrustScan1D(&(gpu_icA.data())[0], false, cpu_icA.size(0));
      icA_outdatedOnCpu = true;
    }
    icA.set_size(cpu_icA.size(0));
    for (c_i = 0; c_i < cpu_icA.size(0); c_i++) {
      if (icA_outdatedOnCpu) {
        arrayCopyGpuToCpu(cpu_icA, gpu_icA);
      }
      icA_outdatedOnCpu = false;
      icA[c_i] = cpu_icA[c_i];
    }
    for (c_i = 0; c_i < icA.size(0); c_i++) {
      if (icA_outdatedOnCpu) {
        arrayCopyGpuToCpu(cpu_icA, gpu_icA);
      }
      cpu_icA[idx[c_i] - 1] = icA[c_i];
      icA_outdatedOnCpu = false;
    }
    j = dIdx.size(0);
    for (i = 0; i < j; i++) {
      uA[i].f1.set_size(1, inData[a__2[i] - 1].f1.size(1));
      b_j1 = inData[a__2[i] - 1].f1.size(1);
      for (c_i = 0; c_i < b_j1; c_i++) {
        uA[i].f1[c_i] = inData[a__2[i] - 1].f1[c_i];
      }
    }
  }
  b_j1 = rv.size(1);
  labelCells = rv.reshape(b_j1);
  matlab::internal::coder::datatypes::cellstr_unique(labelCells, uB, ib);
  locbOfUA.set_size(uA.size(0));
  for (c_i = 0; c_i < uA.size(0); c_i++) {
    locbOfUA[c_i] = 0.0;
  }
  if (uB.size(0) > 0) {
    bool exitg1;
    j = 0;
    i = 0;
    exitg1 = false;
    while ((!exitg1) && (i <= uA.size(0) - 1)) {
      b_j1 = uA[i].f1.size(1);
      j2 = uB[j].f1.size(1);
      if (b_j1 <= j2) {
        j2 = b_j1;
      }
      if (j2 == 0) {
        icA_outdatedOnGpu = (uA[i].f1.size(1) < uB[j].f1.size(1));
      } else {
        b_i = 0;
        while ((b_i + 1 <= j2) && (uA[i].f1[b_i] == uB[j].f1[b_i])) {
          b_i++;
        }
        if (b_i + 1 == j2 + 1) {
          icA_outdatedOnGpu = (uA[i].f1.size(1) < uB[j].f1.size(1));
        } else {
          icA_outdatedOnGpu = (uA[i].f1[b_i] < uB[j].f1[b_i]);
        }
      }
      if (!icA_outdatedOnGpu) {
        if (b_strcmp(uA[i].f1, uB[j].f1)) {
          locbOfUA[i] = ib[j];
          j++;
        } else {
          bool exitg2;
          exitg2 = false;
          while ((!exitg2) && (j + 1 <= uB.size(0))) {
            b_j1 = uA[i].f1.size(1);
            j2 = uB[j].f1.size(1);
            if (b_j1 <= j2) {
              j2 = b_j1;
            }
            if (j2 == 0) {
              icA_outdatedOnGpu = (uA[i].f1.size(1) > uB[j].f1.size(1));
            } else {
              b_i = 0;
              while ((b_i + 1 <= j2) && (uA[i].f1[b_i] == uB[j].f1[b_i])) {
                b_i++;
              }
              if (b_i + 1 == j2 + 1) {
                icA_outdatedOnGpu = (uA[i].f1.size(1) > uB[j].f1.size(1));
              } else {
                icA_outdatedOnGpu = (uA[i].f1[b_i] > uB[j].f1[b_i]);
              }
            }
            if (icA_outdatedOnGpu) {
              j++;
            } else {
              exitg2 = true;
            }
          }
          if ((j + 1 <= uB.size(0)) && b_strcmp(uA[i].f1, uB[j].f1)) {
            locbOfUA[i] = ib[j];
            j++;
          }
        }
      }
      if (j + 1 > uB.size(0)) {
        exitg1 = true;
      } else {
        i++;
      }
    }
  }
  b_j1 = cpu_icA.size(0);
  labelNamesMod_codes.set_size(1, cpu_icA.size(0));
  for (c_i = 0; c_i < b_j1; c_i++) {
    unsigned int u;
    if (icA_outdatedOnCpu) {
      arrayCopyGpuToCpu(cpu_icA, gpu_icA);
    }
    icA_outdatedOnCpu = false;
    b_d = std::round(locbOfUA[static_cast<int>(cpu_icA[c_i]) - 1]);
    if (b_d < 4.294967296E+9) {
      if (b_d >= 0.0) {
        u = static_cast<unsigned int>(b_d);
      } else {
        u = 0U;
      }
    } else if (b_d >= 4.294967296E+9) {
      u = MAX_uint32_T;
    } else {
      u = 0U;
    }
    labelNamesMod_codes[c_i] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<bool, 3U> a
//                coder::gpu_array<bool, 3U> anchorMask
// Return Type  : void
//
} // namespace internal
} // namespace coder
static __global__ __launch_bounds__(512, 1) void iGenerateDetections_kernel30(
    const coder::gpu_array<bool, 3U> a, coder::gpu_array<bool, 3U> anchorMask)
{
  unsigned long long threadId;
  int b_k;
  int c_k;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  c_k = static_cast<int>(threadId % 2ULL);
  threadId = (threadId - static_cast<unsigned long long>(c_k)) / 2ULL;
  b_k = static_cast<int>(threadId % 216ULL);
  threadId = (threadId - static_cast<unsigned long long>(b_k)) / 216ULL;
  k = static_cast<int>(threadId);
  if ((k < 248) && (b_k < 216) && (c_k < 2)) {
    anchorMask[(b_k + 216 * k) + 53568 * c_k] =
        a[(c_k + a.size(0) * b_k) + a.size(0) * 216 * k];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int result_dim0
//                const coder::gpu_array<float, 2U> intensity
//                const int result_dim1
//                const int result
//                const int sizes_idx_1
//                coder::gpu_array<float, 2U> dataIn
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void pointPillarsObjectDetector_detect_kernel1(
        const int result_dim0, const coder::gpu_array<float, 2U> intensity,
        const int result_dim1, const int result, const int sizes_idx_1,
        coder::gpu_array<float, 2U> dataIn)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<unsigned long long>(result) + 1ULL) *
                static_cast<unsigned long long>(sizes_idx_1) -
            1ULL;
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int i1;
    int i3;
    i3 = static_cast<int>(idx %
                          (static_cast<unsigned long long>(result) + 1ULL));
    i1 = static_cast<int>((idx - static_cast<unsigned long long>(i3)) /
                          (static_cast<unsigned long long>(result) + 1ULL));
    dataIn[i3 + dataIn.size(0) * (i1 + result_dim1)] =
        intensity[i3 + result_dim0 * i1];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float pillarIndices[24000]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void pointPillarsObjectDetector_detect_kernel2(
        float pillarIndices[24000])
{
  unsigned long long threadId;
  int i1;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i1 = static_cast<int>(threadId);
  if (i1 < 24000) {
    pillarIndices[i1] = 0.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float dlPillarFeatures_Data[10800000]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void pointPillarsObjectDetector_detect_kernel3(
        float dlPillarFeatures_Data[10800000])
{
  unsigned long long threadId;
  int i1;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i1 = static_cast<int>(threadId);
  if (i1 < 10800000) {
    dlPillarFeatures_Data[i1] = 0.0F;
  }
}

//
// Arguments    : float u0
//                float u1
// Return Type  : float
//
static float rt_atan2f_snf(float u0, float u1)
{
  float b_y;
  if (std::isnan(u0) || std::isnan(u1)) {
    b_y = rtNaNF;
  } else if (std::isinf(u0) && std::isinf(u1)) {
    int b;
    int c;
    if (u1 > 0.0F) {
      b = 1;
    } else {
      b = -1;
    }
    if (u0 > 0.0F) {
      c = 1;
    } else {
      c = -1;
    }
    b_y = std::atan2(static_cast<float>(c), static_cast<float>(b));
  } else if (u1 == 0.0F) {
    if (u0 > 0.0F) {
      b_y = RT_PIF / 2.0F;
    } else if (u0 < 0.0F) {
      b_y = -(RT_PIF / 2.0F);
    } else {
      b_y = 0.0F;
    }
  } else {
    b_y = std::atan2(u0, u1);
  }
  return b_y;
}

//
// Arguments    : detector0_0 *this_Network
//                const pointCloud *ptCloud
//                array<double, 2U> &bboxes
//                array<float, 1U> &scores
//                array<unsigned int, 2U> &varargout_1_codes
//                array<cell_wrap_6, 1U> &varargout_1_categoryNames
// Return Type  : void
//
namespace coder {
namespace internal {
void pointPillarsObjectDetector_detect(
    detector0_0 *this_Network, const pointCloud *ptCloud,
    array<double, 2U> &bboxes, array<float, 1U> &scores,
    array<unsigned int, 2U> &varargout_1_codes,
    array<cell_wrap_6, 1U> &varargout_1_categoryNames)
{
  static float cpu_dlPillarFeatures_Data[10800000];
  static float cpu_predictions_f1_Data[321408];
  static float cpu_predictions_f2_Data[321408];
  static float cpu_predictions_f3_Data[107136];
  static float cpu_predictions_f4_Data[107136];
  static float cpu_predictions_f5_Data[107136];
  static float cpu_predictions_f6_Data[107136];
  array<float, 2U> cpu_dataIn;
  array<float, 2U> cpu_intensity;
  array<float, 2U> cpu_normals;
  array<float, 2U> cpu_rangeData;
  array<unsigned char, 2U> cpu_color;
  array<bool, 1U> cpu_r;
  gpu_array<float, 2U> gpu_dataIn;
  gpu_array<float, 2U> gpu_intensity;
  gpu_array<float, 2U> gpu_normals;
  gpu_array<float, 2U> gpu_rangeData;
  gpu_array<unsigned char, 2U> gpu_color;
  gpu_array<bool, 1U> gpu_r;
  dim3 block;
  dim3 grid;
  pointCloud ptCloudUnorg;
  float(*gpu_dlPillarFeatures_Data)[10800000];
  float(*gpu_predictions_f1_Data)[321408];
  float(*gpu_predictions_f2_Data)[321408];
  float(*gpu_predictions_f3_Data)[107136];
  float(*gpu_predictions_f4_Data)[107136];
  float(*gpu_predictions_f5_Data)[107136];
  float(*gpu_predictions_f6_Data)[107136];
  float cpu_pillarIndices[24000];
  float(*gpu_pillarIndices)[24000];
  int loop_ub;
  int result;
  int sizes_idx_1;
  signed char input_sizes_idx_1;
  bool color_outdatedOnCpu;
  bool color_outdatedOnGpu;
  bool dataIn_outdatedOnCpu;
  bool dataIn_outdatedOnGpu;
  bool empty_non_axis_sizes;
  bool intensity_outdatedOnCpu;
  bool intensity_outdatedOnGpu;
  bool normals_outdatedOnCpu;
  bool normals_outdatedOnGpu;
  bool r_outdatedOnCpu;
  bool r_outdatedOnGpu;
  bool rangeData_outdatedOnGpu;
  hipMalloc(&gpu_predictions_f6_Data, 428544ULL);
  hipMalloc(&gpu_predictions_f5_Data, 428544ULL);
  hipMalloc(&gpu_predictions_f4_Data, 428544ULL);
  hipMalloc(&gpu_predictions_f3_Data, 428544ULL);
  hipMalloc(&gpu_predictions_f2_Data, 1285632ULL);
  hipMalloc(&gpu_predictions_f1_Data, 1285632ULL);
  hipMalloc(&gpu_dlPillarFeatures_Data, 43200000ULL);
  hipMalloc(&gpu_pillarIndices, 96000ULL);
  dataIn_outdatedOnCpu = false;
  dataIn_outdatedOnGpu = false;
  empty_non_axis_sizes = false;
  rangeData_outdatedOnGpu = false;
  intensity_outdatedOnCpu = false;
  intensity_outdatedOnGpu = false;
  normals_outdatedOnCpu = false;
  normals_outdatedOnGpu = false;
  color_outdatedOnCpu = false;
  color_outdatedOnGpu = false;
  r_outdatedOnCpu = false;
  r_outdatedOnGpu = false;
  ptCloudUnorg.matlabCodegenIsDeleted = true;
  vision::internal::codegen::gpu::PointCloudImpl_extractValidPoints(
      ptCloud->Location, cpu_r, &r_outdatedOnCpu, gpu_r, &r_outdatedOnGpu);
  vision::internal::codegen::gpu::PointCloudImpl_subsetImpl(
      ptCloud->Location, ptCloud->Color, ptCloud->Normal, ptCloud->Intensity,
      ptCloud->RangeData, cpu_r, &r_outdatedOnCpu, gpu_r, &r_outdatedOnGpu,
      ptCloudUnorg.Location, cpu_color, &color_outdatedOnCpu, gpu_color,
      &color_outdatedOnGpu, cpu_normals, &normals_outdatedOnCpu, gpu_normals,
      &normals_outdatedOnGpu, cpu_intensity, &intensity_outdatedOnCpu,
      gpu_intensity, &intensity_outdatedOnGpu, cpu_rangeData,
      &empty_non_axis_sizes, gpu_rangeData, &rangeData_outdatedOnGpu);
  if (color_outdatedOnCpu) {
    arrayCopyGpuToCpu(cpu_color, gpu_color);
  }
  pointCloudBase_set_Color(&ptCloudUnorg, cpu_color);
  if (ptCloudUnorg.Location.size(0) != 0) {
    result = ptCloudUnorg.Location.size(0);
  } else if ((cpu_intensity.size(0) != 0) && (cpu_intensity.size(1) != 0)) {
    result = cpu_intensity.size(0);
  } else {
    result = ptCloudUnorg.Location.size(0);
    if (cpu_intensity.size(0) > ptCloudUnorg.Location.size(0)) {
      result = cpu_intensity.size(0);
    }
  }
  empty_non_axis_sizes = (result == 0);
  if (empty_non_axis_sizes || (ptCloudUnorg.Location.size(0) != 0)) {
    input_sizes_idx_1 = 3;
  } else {
    input_sizes_idx_1 = 0;
  }
  if (empty_non_axis_sizes ||
      ((cpu_intensity.size(0) != 0) && (cpu_intensity.size(1) != 0))) {
    sizes_idx_1 = cpu_intensity.size(1);
  } else {
    sizes_idx_1 = 0;
  }
  cpu_dataIn.set_size(result, input_sizes_idx_1 + sizes_idx_1);
  gpu_dataIn.set_size(&cpu_dataIn);
  loop_ub = input_sizes_idx_1;
  for (int i{0}; i < loop_ub; i++) {
    for (int i2{0}; i2 < result; i2++) {
      cpu_dataIn[i2 + cpu_dataIn.size(0) * i] =
          ptCloudUnorg.Location[i2 + result * i];
      dataIn_outdatedOnGpu = true;
    }
  }
  loop_ub = result - 1;
  empty_non_axis_sizes = mwGetLaunchParameters1D(
      static_cast<double>((loop_ub + 1LL) * sizes_idx_1), &grid, &block, 1024U,
      65535U);
  if (empty_non_axis_sizes) {
    if (intensity_outdatedOnGpu) {
      arrayCopyCpuToGpu(gpu_intensity, cpu_intensity);
    }
    if (dataIn_outdatedOnGpu) {
      arrayCopyCpuToGpu(gpu_dataIn, cpu_dataIn);
    }
    pointPillarsObjectDetector_detect_kernel1<<<grid, block>>>(
        result, gpu_intensity, static_cast<int>(input_sizes_idx_1), result - 1,
        sizes_idx_1, gpu_dataIn);
    dataIn_outdatedOnCpu = true;
  }
  pointPillarsObjectDetector_detect_kernel2<<<dim3(47U, 1U, 1U),
                                              dim3(512U, 1U, 1U)>>>(
      *gpu_pillarIndices);
  pointPillarsObjectDetector_detect_kernel3<<<dim3(21094U, 1U, 1U),
                                              dim3(512U, 1U, 1U)>>>(
      *gpu_dlPillarFeatures_Data);
  hipMemcpy(cpu_dlPillarFeatures_Data, *gpu_dlPillarFeatures_Data, 43200000ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu_pillarIndices, *gpu_pillarIndices, 96000ULL,
             hipMemcpyDeviceToHost);
  if (dataIn_outdatedOnCpu) {
    arrayCopyGpuToCpu(cpu_dataIn, gpu_dataIn);
  }
  createPillarsImplSingle(&cpu_dlPillarFeatures_Data[0], &cpu_pillarIndices[0],
                          &cpu_dataIn[0], cpu_dataIn.size(0), 100, 12000, 0.16F,
                          0.16F, 0.0F, 69.12F, -39.68F, 39.68F, -5.0F, 5.0F);
  hipMemcpy(*gpu_pillarIndices, cpu_pillarIndices, 96000ULL,
             hipMemcpyHostToDevice);
  hipMemcpy(*gpu_dlPillarFeatures_Data, cpu_dlPillarFeatures_Data, 43200000ULL,
             hipMemcpyHostToDevice);
  dlnetwork_predict(this_Network, *gpu_pillarIndices,
                    *gpu_dlPillarFeatures_Data, *gpu_predictions_f1_Data,
                    *gpu_predictions_f2_Data, *gpu_predictions_f3_Data,
                    *gpu_predictions_f4_Data, *gpu_predictions_f5_Data,
                    *gpu_predictions_f6_Data);
  hipMemcpy(cpu_predictions_f1_Data, *gpu_predictions_f1_Data, 1285632ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu_predictions_f2_Data, *gpu_predictions_f2_Data, 1285632ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu_predictions_f3_Data, *gpu_predictions_f3_Data, 428544ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu_predictions_f4_Data, *gpu_predictions_f4_Data, 428544ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu_predictions_f5_Data, *gpu_predictions_f5_Data, 428544ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu_predictions_f6_Data, *gpu_predictions_f6_Data, 428544ULL,
             hipMemcpyDeviceToHost);
  pointPillarsObjectDetector_iPostprocessSingleDetection(
      cpu_predictions_f1_Data, cpu_predictions_f2_Data, cpu_predictions_f3_Data,
      cpu_predictions_f4_Data, cpu_predictions_f5_Data, cpu_predictions_f6_Data,
      cpu_pillarIndices, bboxes, scores, varargout_1_codes,
      varargout_1_categoryNames);
  hipFree(*gpu_pillarIndices);
  hipFree(*gpu_dlPillarFeatures_Data);
  hipFree(*gpu_predictions_f1_Data);
  hipFree(*gpu_predictions_f2_Data);
  hipFree(*gpu_predictions_f3_Data);
  hipFree(*gpu_predictions_f4_Data);
  hipFree(*gpu_predictions_f5_Data);
  hipFree(*gpu_predictions_f6_Data);
}

} // namespace internal
} // namespace coder

//
// File trailer for pointPillarsObjectDetector.cu
//
// [EOF]
//
