#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: deployedPointPillarModel_rtwutil.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "deployedPointPillarModel_rtwutil.h"
#include "rt_nonfinite.h"
#include <cfloat>
#include <cmath>

// Function Definitions
//
// Arguments    : long long start
//                long long end
//                long long stride
// Return Type  : long long
//
long long computeEndIdx(long long start, long long end, long long stride)
{
  long long newEnd;
  newEnd = -1LL;
  if ((stride > 0LL) && (start <= end)) {
    newEnd = (end - start) / stride;
  } else if ((stride < 0LL) && (end <= start)) {
    newEnd = (start - end) / -stride;
  }
  return newEnd;
}

//
// Arguments    : float u0
//                float u1
// Return Type  : float
//
float rt_remf_snf(float u0, float u1)
{
  float b_y;
  if (std::isnan(u0) || std::isnan(u1) || std::isinf(u0)) {
    b_y = rtNaNF;
  } else if (std::isinf(u1)) {
    b_y = u0;
  } else if ((u1 != 0.0F) && (u1 != std::trunc(u1))) {
    float q;
    q = std::abs(u0 / u1);
    if (!(std::abs(q - std::floor(q + 0.5F)) > FLT_EPSILON * q)) {
      b_y = 0.0F * u0;
    } else {
      b_y = std::fmod(u0, u1);
    }
  } else {
    b_y = std::fmod(u0, u1);
  }
  return b_y;
}

//
// File trailer for deployedPointPillarModel_rtwutil.cu
//
// [EOF]
//
