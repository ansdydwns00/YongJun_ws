//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: getCategoryNames.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "getCategoryNames.h"
#include "deployedPointPillarModel_internal_types.h"
#include "rt_nonfinite.h"
#include "coder_array.h"

// Function Definitions
//
// Arguments    : const array<cell_wrap_6, 1U> &valueSet
//                array<cell_wrap_6, 1U> &outCategoryNames
// Return Type  : void
//
namespace coder {
void categorical_getCategoryNames(const array<cell_wrap_6, 1U> &valueSet,
                                  array<cell_wrap_6, 1U> &outCategoryNames)
{
  if (valueSet.size(0) != 0) {
    int i2;
    i2 = valueSet.size(0);
    outCategoryNames.set_size(valueSet.size(0));
    for (int i{0}; i < i2; i++) {
      int loop_ub;
      loop_ub = valueSet[i].f1.size(1);
      outCategoryNames[i].f1.set_size(1, valueSet[i].f1.size(1));
      for (int b_i{0}; b_i < loop_ub; b_i++) {
        outCategoryNames[i].f1[b_i] = valueSet[i].f1[b_i];
      }
    }
  } else {
    outCategoryNames.set_size(0);
  }
}

} // namespace coder

//
// File trailer for getCategoryNames.cu
//
// [EOF]
//
