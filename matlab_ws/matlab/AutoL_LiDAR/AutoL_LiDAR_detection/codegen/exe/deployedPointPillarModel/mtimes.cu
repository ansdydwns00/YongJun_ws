//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: mtimes.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "mtimes.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "coder_array.h"
#include "coder_gpu_array.h"

// Function Definitions
//
// Arguments    : const float A[2]
//                array<float, 2U> &cpu_B
//                bool *B_outdatedOnCpu
//                gpu_array<float, 2U> &gpu_B
//                bool *B_outdatedOnGpu
//                array<float, 2U> &cpu_C
//                bool *C_outdatedOnCpu
//                gpu_array<float, 2U> &gpu_C
//                bool *C_outdatedOnGpu
// Return Type  : void
//
namespace coder {
namespace internal {
namespace blas {
void mtimes(const float A[2], array<float, 2U> &cpu_B, bool *B_outdatedOnCpu,
            gpu_array<float, 2U> &gpu_B, bool *B_outdatedOnGpu,
            array<float, 2U> &cpu_C, bool *C_outdatedOnCpu,
            gpu_array<float, 2U> &gpu_C, bool *C_outdatedOnGpu)
{
  if (cpu_B.size(1) == 0) {
    cpu_C.set_size(1, 0);
    if (!*C_outdatedOnGpu) {
      gpu_C.set_size(&cpu_C);
    }
  } else {
    float alpha1;
    float beta1;
    cpu_C.set_size(1, cpu_B.size(1));
    if (!*C_outdatedOnGpu) {
      gpu_C.set_size(&cpu_C);
    }
    alpha1 = 1.0F;
    beta1 = 0.0F;
    if (*B_outdatedOnGpu) {
      arrayCopyCpuToGpu(gpu_B, cpu_B);
    }
    if (*C_outdatedOnGpu) {
      arrayCopyCpuToGpu(gpu_C, cpu_C);
    }
    hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 1,
                cpu_B.size(1), 2, (float *)&alpha1, (float *)&A[0], 1,
                (float *)&(gpu_B.data())[0], 2, (float *)&beta1,
                (float *)&gpu_C[0], 1);
    *C_outdatedOnGpu = false;
    *C_outdatedOnCpu = true;
    *B_outdatedOnGpu = false;
    *B_outdatedOnCpu = true;
  }
}

} // namespace blas
} // namespace internal
} // namespace coder

//
// File trailer for mtimes.cu
//
// [EOF]
//
