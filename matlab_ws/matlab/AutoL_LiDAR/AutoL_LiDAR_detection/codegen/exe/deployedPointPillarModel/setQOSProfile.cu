//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: setQOSProfile.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "setQOSProfile.h"
#include "rt_nonfinite.h"
#include "mlros2_qos.h"
#include "rmw/qos_profiles.h"
#include "rmw/types.h"
#include <cstddef>

// Function Definitions
//
// Arguments    : rmw_qos_profile_t rmwProfile
//                double qosDepth
//                const char qosReliability[8]
// Return Type  : rmw_qos_profile_t
//
namespace coder {
namespace ros {
namespace ros2 {
namespace internal {
rmw_qos_profile_t setQOSProfile(rmw_qos_profile_t rmwProfile, double qosDepth,
                                const char qosReliability[8])
{
  static const char b_cv[8]{'r', 'e', 'l', 'i', 'a', 'b', 'l', 'e'};
  rmw_qos_reliability_policy_t reliability;
  int k;
  char s[8];
  bool exitg1;
  bool p;
  for (k = 0; k < 8; k++) {
    char c;
    c = qosReliability[k];
    s[k] = c;
    if ((c >= 'A') && (c <= 'Z')) {
      s[k] = static_cast<char>(static_cast<unsigned int>(c) + 32U);
    }
  }
  p = true;
  k = 0;
  exitg1 = false;
  while ((!exitg1) && (k < 8)) {
    if (s[k] != b_cv[k]) {
      p = false;
      exitg1 = true;
    } else {
      k++;
    }
  }
  if (p) {
    reliability = RMW_QOS_POLICY_RELIABILITY_RELIABLE;
  } else {
    reliability = RMW_QOS_POLICY_RELIABILITY_BEST_EFFORT;
  }
  SET_QOS_VALUES(rmwProfile, RMW_QOS_POLICY_HISTORY_KEEP_LAST, (size_t)qosDepth,
                 RMW_QOS_POLICY_DURABILITY_VOLATILE, reliability);
  return rmwProfile;
}

} // namespace internal
} // namespace ros2
} // namespace ros
} // namespace coder

//
// File trailer for setQOSProfile.cu
//
// [EOF]
//
