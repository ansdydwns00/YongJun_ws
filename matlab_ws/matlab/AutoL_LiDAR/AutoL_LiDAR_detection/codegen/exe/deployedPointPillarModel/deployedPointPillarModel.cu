#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: deployedPointPillarModel.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "deployedPointPillarModel.h"
#include "deployedPointPillarModel_data.h"
#include "deployedPointPillarModel_initialize.h"
#include "deployedPointPillarModel_internal_types.h"
#include "deployedPointPillarModel_types.h"
#include "dlnetwork.h"
#include "pointCloudBase.h"
#include "pointPillarsObjectDetector.h"
#include "ros2publisher.h"
#include "ros2rate.h"
#include "ros2subscriber.h"
#include "rosReadField.h"
#include "rosReadXYZ.h"
#include "rt_nonfinite.h"
#include "vision_msgs_Detection3DArrayStruct.h"
#include "MWCudnnCustomLayerBase.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWTensorBase.hpp"
#include "coder_array.h"
#include "mlros2_node.h"
#include "mlros2_pub.h"
#include <cmath>
#include <cstdio>

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void MWScatterLayer_detector0_01::cleanupLayer()
{
}

//
// Arguments    : void
// Return Type  : void
//
void MWResize2DLayer_detector0_01::cleanupLayer()
{
}

//
// Arguments    : void
// Return Type  : void
//
void MWResize2DLayer_detector0_02::cleanupLayer()
{
}

//
// Arguments    : MWCudnnTarget::MWTargetNetworkImpl *b_targetImpl
//                MWTensorBase *b
//                const char *c
//                int d
// Return Type  : void
//
void MWResize2DLayer_detector0_02::createCustomLayer(
    MWCudnnTarget::MWTargetNetworkImpl *b_targetImpl, MWTensorBase *b,
    const char *c, int d)
{
  setupTensors(1, 1, b, c, d);
  setupLayer(b_targetImpl);
}

//
// Arguments    : MWCudnnTarget::MWTargetNetworkImpl *b_targetImpl
//                MWTensorBase *b
//                const char *c
//                int d
// Return Type  : void
//
void MWResize2DLayer_detector0_01::createCustomLayer(
    MWCudnnTarget::MWTargetNetworkImpl *b_targetImpl, MWTensorBase *b,
    const char *c, int d)
{
  setupTensors(1, 1, b, c, d);
  setupLayer(b_targetImpl);
}

//
// Arguments    : MWCudnnTarget::MWTargetNetworkImpl *b_targetImpl
//                MWTensorBase *b
//                MWTensorBase *c
//                const char *d
//                int e
// Return Type  : void
//
void MWScatterLayer_detector0_01::createCustomLayer(
    MWCudnnTarget::MWTargetNetworkImpl *b_targetImpl, MWTensorBase *b,
    MWTensorBase *c, const char *d, int e)
{
  setupTensors(2, 1, b, c, d, e);
  setupLayer(b_targetImpl);
}

//
// Arguments    : void
// Return Type  : void
//
void MWScatterLayer_detector0_01::predict()
{
  reorderInputData(getInputTensor(0), 0);
  reorderInputData(getInputTensor(1), 1);
  coder::internal::dlnetwork_layerPredictWithColMajority(
      m_ntwkImpl->getPermuteBuffer(0), m_ntwkImpl->getPermuteBuffer(1),
      m_ntwkImpl->getPermuteBuffer(2));
  reorderOutputData(getOutputTensor(0), 2);
}

//
// Arguments    : void
// Return Type  : void
//
void MWResize2DLayer_detector0_01::predict()
{
  reorderInputData(getInputTensor(0), 0);
  coder::internal::dlnetwork_layerPredictWithColMajority(
      m_ntwkImpl->getPermuteBuffer(0), m_ntwkImpl->getPermuteBuffer(1));
  reorderOutputData(getOutputTensor(0), 1);
}

//
// Arguments    : void
// Return Type  : void
//
void MWResize2DLayer_detector0_02::predict()
{
  reorderInputData(getInputTensor(0), 0);
  coder::internal::b_dlnetwork_layerPredictWithColMajority(
      m_ntwkImpl->getPermuteBuffer(0), m_ntwkImpl->getPermuteBuffer(1));
  reorderOutputData(getOutputTensor(0), 1);
}

//
// Arguments    : void
// Return Type  : void
//
void MWScatterLayer_detector0_01::propagateSize()
{
  int batchSize;
  MWTensorBase::DIMSLABEL b_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL c_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL d_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL layout[5]{MWTensorBase::S, MWTensorBase::N,
                                    MWTensorBase::C, MWTensorBase::W,
                                    MWTensorBase::H};
  batchSize = getInputTensor(0)->getBatchSize();
  resizeOutputTensor(432, 496, 64, batchSize, 1, 0);
  setupInputDescriptors(getInputTensor(0), b_layout, layout);
  setupInputDescriptors(getInputTensor(1), c_layout, layout);
  setupOutputDescriptors(getOutputTensor(0), layout, d_layout);
}

//
// Arguments    : void
// Return Type  : void
//
void MWResize2DLayer_detector0_01::propagateSize()
{
  int batchSize;
  MWTensorBase::DIMSLABEL b_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL c_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL layout[5]{MWTensorBase::S, MWTensorBase::N,
                                    MWTensorBase::C, MWTensorBase::W,
                                    MWTensorBase::H};
  batchSize = getInputTensor(0)->getBatchSize();
  resizeOutputTensor(216, 248, 128, batchSize, 1, 0);
  setupInputDescriptors(getInputTensor(0), b_layout, layout);
  setupOutputDescriptors(getOutputTensor(0), layout, c_layout);
}

//
// Arguments    : void
// Return Type  : void
//
void MWResize2DLayer_detector0_02::propagateSize()
{
  int batchSize;
  MWTensorBase::DIMSLABEL b_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL c_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL layout[5]{MWTensorBase::S, MWTensorBase::N,
                                    MWTensorBase::C, MWTensorBase::W,
                                    MWTensorBase::H};
  batchSize = getInputTensor(0)->getBatchSize();
  resizeOutputTensor(216, 248, 128, batchSize, 1, 0);
  setupInputDescriptors(getInputTensor(0), b_layout, layout);
  setupOutputDescriptors(getOutputTensor(0), layout, c_layout);
}

//
// load the pre-trained deep learning network from the "pointPillarDetector.mat"
// file
//
// Arguments    : void
// Return Type  : void
//
void deployedPointPillarModel()
{
  static const char cv2[4]{'d', 'r', 'o', 'p'};
  static const char cv3[4]{'s', 'l', 'i', 'p'};
  detector0_0 lobj_6;
  coder::array<cell_wrap_6, 1U> labels_categoryNames;
  coder::array<sensor_msgs_PointFieldStruct_T, 1U> receivedpcdata_fields;
  coder::array<double, 2U> bboxes;
  coder::array<float, 2U> intensity;
  coder::array<float, 1U> a__1;
  coder::array<unsigned int, 2U> labels_codes;
  coder::array<unsigned char, 2U> uv;
  coder::array<unsigned char, 1U> receivedpcdata_data;
  coder::pointCloud ptCloud;
  coder::ros2node nd;
  coder::ros2publisher objDetectPub;
  coder::ros2rate r;
  coder::ros2subscriber pcsub;
  vision_msgs_Detection3DArrayStruct_T objMsg;
  unsigned int receivedpcdata_height;
  unsigned int receivedpcdata_point_step;
  unsigned int receivedpcdata_width;
  char receivedpcdata_MessageType[23];
  if (!isInitialized_deployedPointPillarModel) {
    deployedPointPillarModel_initialize();
  }
  ptCloud.matlabCodegenIsDeleted = true;
  lobj_6.matlabCodegenIsDeleted = true;
  coder::internal::dlnetwork_setup(&lobj_6);
  lobj_6.matlabCodegenIsDeleted = false;
  nd.NodeHandle = MATLAB::getGlobalNodeHandle();
  UNUSED_PARAM(nd.NodeHandle);
  //  create subscribers and publishers
  coder::ros2subscriber_ros2subscriber(&pcsub, &nd);
  coder::ros2publisher_ros2publisher(&objDetectPub, &nd);
  //  create ros message object
  vision_msgs_Detection3DArrayStruct(&objMsg);
  coder::ros2rate_ros2rate(&r);
  while (1) {
    double currentTime;
    int u0;
    std::printf("Waiting for input\n");
    std::fflush(stdout);
    coder::ros2subscriber_receive(
        &pcsub, receivedpcdata_MessageType, &receivedpcdata_height,
        &receivedpcdata_width, receivedpcdata_fields,
        &receivedpcdata_point_step, receivedpcdata_data);
    std::printf("Point Cloud received\n");
    std::fflush(stdout);
    //  Extract XYZ coordinates from ROS point cloud message structure
    coder::rosReadXYZ(receivedpcdata_height, receivedpcdata_width,
                      receivedpcdata_fields, receivedpcdata_point_step,
                      receivedpcdata_data, ptCloud.Location);
    //  Read point cloud data from ROS message structure based on field name
    coder::rosReadField(receivedpcdata_height, receivedpcdata_width,
                        receivedpcdata_fields, receivedpcdata_point_step,
                        receivedpcdata_data, intensity);
    //  Create a 3-D point cloud object
    uv.set(nullptr, 0, 0);
    coder::pointCloudBase_set_Color(&ptCloud, uv);
    ptCloud.Normal.set_size(0, 0);
    ptCloud.Intensity.set_size(intensity.size(0), intensity.size(1));
    for (u0 = 0; u0 < intensity.size(0) * intensity.size(1); u0++) {
      ptCloud.Intensity[u0] = intensity[u0];
    }
    ptCloud.RangeData.set_size(0, 0);
    ptCloud.PointCloudArrayData.set_size(1, 1);
    ptCloud.matlabCodegenIsDeleted = false;
    coder::internal::pointPillarsObjectDetector_detect(
        &lobj_6, &ptCloud, bboxes, a__1, labels_codes, labels_categoryNames);
    //  processing the output of the dectector, before publishing it on the
    //  rostopic publishing the coordinates and lables of detected objects
    std::printf("Publishing the coordinates and labels of detected objects \n");
    std::fflush(stdout);
    u0 = bboxes.size(0);
    if (u0 < 9) {
      u0 = 9;
    }
    if (bboxes.size(0) == 0) {
      u0 = 0;
    }
    for (int i{0}; i < u0; i++) {
      objMsg.detections[i].bbox.center.position.x = bboxes[i];
      objMsg.detections[i].bbox.center.position.y = bboxes[i + bboxes.size(0)];
      objMsg.detections[i].bbox.center.position.z =
          bboxes[i + bboxes.size(0) * 2];
      objMsg.detections[i].bbox.size.x = bboxes[i + bboxes.size(0) * 3];
      objMsg.detections[i].bbox.size.y = bboxes[i + bboxes.size(0) * 4];
      objMsg.detections[i].bbox.size.z = bboxes[i + bboxes.size(0) * 5];
      //  objMsg.detections(i).results.hypothesis.class_id = labels{i,1};
    }
    MATLABROS2Publisher_publish(objDetectPub.PublisherHelper, &objMsg);
    std::printf("Details published \n\n");
    std::fflush(stdout);
    currentTime = coder::ros2rate_getCurrentTime();
    if (currentTime < r.LastWakeTime) {
      r.LastWakeTime = 0.0;
      r.StartTime = coder::ros2rate_getCurrentTime();
      r.NextExecutionIndex = 0.0;
      r.ExecutionStartTime = 0.0;
      r.LastWakeTime = currentTime;
    } else {
      double sleepTime;
      r.NextExecutionIndex++;
      sleepTime =
          (r.NextExecutionIndex * r.DesiredPeriod + r.ExecutionStartTime) -
          currentTime;
      if (sleepTime < 0.0) {
        int exitg1;
        bool result;
        result = false;
        u0 = 0;
        do {
          exitg1 = 0;
          if (u0 + 1 < 5) {
            if (cv2[u0] != r.OverrunAction[u0]) {
              exitg1 = 1;
            } else {
              u0++;
            }
          } else {
            result = true;
            exitg1 = 1;
          }
        } while (exitg1 == 0);
        if (result) {
          u0 = 0;
        } else {
          u0 = 0;
          do {
            exitg1 = 0;
            if (u0 + 1 < 5) {
              if (cv3[u0] != r.OverrunAction[u0]) {
                exitg1 = 1;
              } else {
                u0++;
              }
            } else {
              result = true;
              exitg1 = 1;
            }
          } while (exitg1 == 0);
          if (result) {
            u0 = 1;
          } else {
            u0 = -1;
          }
        }
        switch (u0) {
        case 0:
          sleepTime = (currentTime - r.ExecutionStartTime) / r.DesiredPeriod;
          sleepTime = std::ceil(sleepTime);
          r.NextExecutionIndex = sleepTime;
          sleepTime =
              (r.NextExecutionIndex * r.DesiredPeriod + r.ExecutionStartTime) -
              currentTime;
          break;
        case 1:
          r.NextExecutionIndex = 0.0;
          r.ExecutionStartTime = currentTime;
          sleepTime = 0.0;
          break;
        }
      }
      coder::ros2rate_internalSleep(&r, sleepTime);
      currentTime = coder::ros2rate_getCurrentTime() - r.StartTime;
      if (currentTime > r.LastWakeTime) {
        r.LastWakeTime = currentTime;
      } else {
        r.LastWakeTime = 0.0;
        r.StartTime = coder::ros2rate_getCurrentTime();
        r.NextExecutionIndex = 0.0;
        r.ExecutionStartTime = 0.0;
        r.LastWakeTime = currentTime;
      }
      coder::ros2rate_getCurrentTime();
      r.PreviousPeriod = coder::ros2rate_getCurrentTime();
    }
    ptCloud.matlabCodegenIsDeleted = true;
  }
}

//
// File trailer for deployedPointPillarModel.cu
//
// [EOF]
//
