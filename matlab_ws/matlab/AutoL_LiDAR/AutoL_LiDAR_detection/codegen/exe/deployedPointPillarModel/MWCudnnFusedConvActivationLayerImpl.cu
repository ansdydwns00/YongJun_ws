#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnFusedConvActivationLayerImpl.hpp"
#include <cassert>
#include <cstdio>
#include <vector>
 namespace MWCudnnTarget { 
MWFusedConvActivationLayerImpl::MWFusedConvActivationLayerImpl( MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl, int filt_H, int filt_W, int numGrps, int 
numChnls, int numFilts, int FLuSVNoPhAFKtLUchSvv, int 
FeVcBgtQmTLtmnNcJGMY, int CLOUhPjbgggWoXHTtmjC, int 
CGbFsczkgkhjcHoCKzBx, int ClEhcJFlvGCgiavziIag, int 
CpMjJjtGOeWOzwxpAAQP, int AdmgfUbRAfzFeYHxSnQr, int 
AuqaQHxmPQSyYRemQvyX, int edQOkUJIZbwzEeIcCLzG, const char* 
xHViLEwTujGGrPZZgmbF, const char* JwxFdqOKggeawILBfGgg, double , 
MWActivationFunctionType::ACTIVATION_FCN_ENUM GLpnVFeGjOSrhNqnkdCu) : 
MWCNNLayerImpl(layer, ntwk_impl) , vIWQzNvYZSuxmOTVDFhU(NULL) , IpFhwalnAlrMvcuyQpQD(NULL) , 
TfsmDFpPPOscKZifVzSQ(NULL) , WprSrhAStKGxyXeoxETy(NULL) , 
FrpxvsDMwwgbpqHXWxmN(NULL) , HUdjvMUbhwNBNiIGaMZg(NULL) , 
AwZQzUhuWVLGrWgLHRuM(filt_H) , AzTsxYcYjIEJsGQbeYHm(filt_W) , 
BdqURaHPmdnfzvtUvocl(numGrps) , BRSPqxNffoBYKqpSVHne(numChnls) , 
BUOdotSvmFyUWQKMUdra(numFilts) , EvebzoroiuKkIxwjkGnD(FLuSVNoPhAFKtLUchSvv) , 
FOcStuqCptsGIZXskVpC(FeVcBgtQmTLtmnNcJGMY) , 
CDJtexcMbXMWAmnNZsNf(CLOUhPjbgggWoXHTtmjC) , 
CCKWXUFWgrbBMjwfpOBN(CGbFsczkgkhjcHoCKzBx) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP) , 
AVeZfqOFypgpiqfRYlKc(AdmgfUbRAfzFeYHxSnQr) , 
AjhVZuQXURJimwbnYqDF(AuqaQHxmPQSyYRemQvyX) , 
fSKMHAqIghbYYgyIpNDw(edQOkUJIZbwzEeIcCLzG) , 
HtQBsWTCGEkpylRklilw((CDJtexcMbXMWAmnNZsNf != CCKWXUFWgrbBMjwfpOBN) 
|| (CTCbzQMDaLxINPbODdng != CZNYmBcNFSZWvaCklqeM)) { if 
(GLpnVFeGjOSrhNqnkdCu != 
MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU) { assert(false); }
#if (CUDNN_MAJOR < 6)
 throw std::runtime_error("Fused ConvReLU Layer only supported for cuDNN 6 or greater");
#else
 cQBKlCKXxecGPJrXBXdk = ntwk_impl; 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NGqpeiLeVweDRsOKEtuw)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&PtkeOkuClHzhOfpmBevf)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JgLfgHrHMEMmMYTettJF)); 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&muwRQxtWMMXAPxSuMYBw)); 
MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int 
NMMfJylfQjiIUAKhXCJb = CDJtexcMbXMWAmnNZsNf; int 
NNhshzQGJHLSGjDiVerE = CTCbzQMDaLxINPbODdng; if 
(HtQBsWTCGEkpylRklilw) { NMMfJylfQjiIUAKhXCJb = 0; 
NNhshzQGJHLSGjDiVerE = 0; TfsmDFpPPOscKZifVzSQ = new MWTensor<float>(-1, 
-1, -1, -1, -1, NULL, getLayer(), ipTensor_conv->getDataFormat(), 0); if 
(!TfsmDFpPPOscKZifVzSQ) { MWCNNLayerImpl::throwAllocationError(__LINE__, 
__FILE__); } CUDNN_CALL(hipdnnCreateTensorDescriptor(&XYbzSmRQGatVJtGmDZSo)); } 
else { TfsmDFpPPOscKZifVzSQ = ipTensor_conv; } assert(TfsmDFpPPOscKZifVzSQ != 
NULL); bERCRkGjpaKXMNComoYl = CDJtexcMbXMWAmnNZsNf; bOrQjJTNlssnrexxbHdi = 
CTCbzQMDaLxINPbODdng; CUDNN_CALL(hipdnnSetConvolution2dDescriptor( 
NGqpeiLeVweDRsOKEtuw, NMMfJylfQjiIUAKhXCJb, 
NNhshzQGJHLSGjDiVerE, EvebzoroiuKkIxwjkGnD, FOcStuqCptsGIZXskVpC, 
AVeZfqOFypgpiqfRYlKc, AjhVZuQXURJimwbnYqDF, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
#if (FP16_ENABLED == 1 && (CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2)))
 CUDNN_CALL( hipdnnSetConvolutionMathType(NGqpeiLeVweDRsOKEtuw, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 if (BdqURaHPmdnfzvtUvocl > 1) { 
CUDNN_CALL(hipdnnSetConvolutionGroupCount(NGqpeiLeVweDRsOKEtuw, 
BdqURaHPmdnfzvtUvocl)); } 
CUDNN_CALL(hipdnnSetActivationDescriptor(muwRQxtWMMXAPxSuMYBw, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0)); int 
eWYFXrUazhqiEIscccda = BRSPqxNffoBYKqpSVHne * BdqURaHPmdnfzvtUvocl; 
int eqmVWbEcwBRGnVNDUtrG = BUOdotSvmFyUWQKMUdra * 
BdqURaHPmdnfzvtUvocl; 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(PtkeOkuClHzhOfpmBevf, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, eqmVWbEcwBRGnVNDUtrG, eWYFXrUazhqiEIscccda / 
BdqURaHPmdnfzvtUvocl, AwZQzUhuWVLGrWgLHRuM, AzTsxYcYjIEJsGQbeYHm)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JgLfgHrHMEMmMYTettJF, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, eqmVWbEcwBRGnVNDUtrG, 1, 1)); int weightSize = 
BRSPqxNffoBYKqpSVHne * eqmVWbEcwBRGnVNDUtrG * AwZQzUhuWVLGrWgLHRuM 
* AzTsxYcYjIEJsGQbeYHm; CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, 
sizeof(float) * weightSize)); CUDA_CALL(hipMalloc((void**)&IpFhwalnAlrMvcuyQpQD, 
sizeof(float) * eqmVWbEcwBRGnVNDUtrG)); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(JwxFdqOKggeawILBfGgg); createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
#endif
 } MWFusedConvActivationLayerImpl::~MWFusedConvActivationLayerImpl() { } void 
MWFusedConvActivationLayerImpl::propagateSize() {
#if (CUDNN_MAJOR >= 6)
 MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int inputH; int 
inputW; if (HtQBsWTCGEkpylRklilw) { inputH = 
ipTensor_conv->getHeight() + CDJtexcMbXMWAmnNZsNf + CCKWXUFWgrbBMjwfpOBN; 
inputW = ipTensor_conv->getWidth() + CTCbzQMDaLxINPbODdng + 
CZNYmBcNFSZWvaCklqeM; } else { inputH = ipTensor_conv->getHeight(); inputW = 
ipTensor_conv->getWidth(); } TfsmDFpPPOscKZifVzSQ->setHeight(inputH); 
TfsmDFpPPOscKZifVzSQ->setWidth(inputW); 
TfsmDFpPPOscKZifVzSQ->setChannels(ipTensor_conv->getChannels()); 
TfsmDFpPPOscKZifVzSQ->setBatchSize(ipTensor_conv->getBatchSize()); 
TfsmDFpPPOscKZifVzSQ->setSequenceLength(ipTensor_conv->getSequenceLength()); 
assert(TfsmDFpPPOscKZifVzSQ->getSequenceLength() == 1); if 
(HtQBsWTCGEkpylRklilw) { CUDNN_CALL(hipdnnSetTensor4dDescriptor( 
XYbzSmRQGatVJtGmDZSo, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 
TfsmDFpPPOscKZifVzSQ->getBatchSize(), TfsmDFpPPOscKZifVzSQ->getChannels(), 
TfsmDFpPPOscKZifVzSQ->getHeight(), TfsmDFpPPOscKZifVzSQ->getWidth())); } else { 
XYbzSmRQGatVJtGmDZSo = MWCNNLayerImpl::getCuDNNDescriptor(TfsmDFpPPOscKZifVzSQ); } 
assert(BRSPqxNffoBYKqpSVHne == TfsmDFpPPOscKZifVzSQ->getChannels() / 
BdqURaHPmdnfzvtUvocl); MWTensorBase* opTensor = getLayer()->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));
#if (CUDNN_MAJOR < 7)
 { CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, 
HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NDHPlSVpLroiIBRnjwyO)); }
#else
 { const int maxAlgoCount(3); int returnedAlgoCount(-1); 
hipdnnConvolutionFwdAlgoPerf_t perf_results[maxAlgoCount]; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, maxAlgoCount, 
&returnedAlgoCount, perf_results)); NDHPlSVpLroiIBRnjwyO = perf_results[0].algo; }
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); size_t sxuOMwKXOKfuExclRaSe = 0; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, NDHPlSVpLroiIBRnjwyO, 
&sxuOMwKXOKfuExclRaSe)); if (sxuOMwKXOKfuExclRaSe > 
*cQBKlCKXxecGPJrXBXdk->getProposedWorkSpaceSize()) { 
cQBKlCKXxecGPJrXBXdk->setProposedWorkSpaceSize(sxuOMwKXOKfuExclRaSe); }
#endif
 } void MWFusedConvActivationLayerImpl::allocate() { MWTensorBase* 
ipTensor_conv = getLayer()->getInputTensor(0); if 
(HtQBsWTCGEkpylRklilw) { float* newInput; int inputH = 
ipTensor_conv->getHeight() + CDJtexcMbXMWAmnNZsNf + CCKWXUFWgrbBMjwfpOBN; int 
inputW = ipTensor_conv->getWidth() + CTCbzQMDaLxINPbODdng + 
CZNYmBcNFSZWvaCklqeM; int paddedSize = ipTensor_conv->getBatchSize() * 
ipTensor_conv->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float) * paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float) * paddedSize)); 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->setData(newInput); } 
WprSrhAStKGxyXeoxETy = 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(0))->getData(); 
setalpha2Ptr(getZeroPtr()); int numInputs = getLayer()->getNumInputs(); if 
(numInputs == 2) { setalpha2Ptr(getOnePtr()); WprSrhAStKGxyXeoxETy = 
static_cast<MWTensor<float>*>(getLayer()->getInputTensor(1))->getData(); } if 
(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData() == 
WprSrhAStKGxyXeoxETy) { int xInputTensorSize = 
getLayer()->getInputTensor(0)->getNumElements(); CUDA_CALL( 
hipMalloc((void**)&FrpxvsDMwwgbpqHXWxmN, sizeof(float) * 
xInputTensorSize)); } } void MWFusedConvActivationLayerImpl::deallocate() { if 
(TfsmDFpPPOscKZifVzSQ != getLayer()->getInputTensor(0)) { 
assert(HtQBsWTCGEkpylRklilw); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData()); 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->setData((float*)NULL); } if 
(FrpxvsDMwwgbpqHXWxmN) { CUDA_FREE_CALL(FrpxvsDMwwgbpqHXWxmN); 
FrpxvsDMwwgbpqHXWxmN = NULL; } } void 
MWFusedConvActivationLayerImpl::predict() { MWCNNLayer* fusedConvReluLayer = 
getLayer(); MWTensorBase* ipTensorBase = fusedConvReluLayer->getInputTensor(); 
MWTensorBase* opTensorBase = fusedConvReluLayer->getOutputTensor(); 
MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); if 
(TfsmDFpPPOscKZifVzSQ != fusedConvReluLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
0, sizeof(float) * TfsmDFpPPOscKZifVzSQ->getNumElements())); 
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), TfsmDFpPPOscKZifVzSQ->getHeight(), 
TfsmDFpPPOscKZifVzSQ->getWidth(), bERCRkGjpaKXMNComoYl, bOrQjJTNlssnrexxbHdi, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
ipTensor->getNumElements()); } hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR >= 6)
 assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData() || 
(getLayer()->getNumInputs() == 2)); float* pzUAoBDvaKAtdsmkQuct; if 
(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData() == 
WprSrhAStKGxyXeoxETy) { CUDA_CALL(hipMemcpy(FrpxvsDMwwgbpqHXWxmN, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), sizeof(float) * 
opTensorBase->getNumElements(), hipMemcpyDeviceToDevice)); 
pzUAoBDvaKAtdsmkQuct = FrpxvsDMwwgbpqHXWxmN; } else { 
pzUAoBDvaKAtdsmkQuct = 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(); } 
CUDNN_CALL(cudnnConvolutionBiasActivationForward( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), getOnePtr(), XYbzSmRQGatVJtGmDZSo, 
pzUAoBDvaKAtdsmkQuct, PtkeOkuClHzhOfpmBevf, vIWQzNvYZSuxmOTVDFhU, 
NGqpeiLeVweDRsOKEtuw, NDHPlSVpLroiIBRnjwyO, cQBKlCKXxecGPJrXBXdk->getWorkSpace(), 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize(), getalpha2Ptr(),  *desc,  
WprSrhAStKGxyXeoxETy,  JgLfgHrHMEMmMYTettJF, IpFhwalnAlrMvcuyQpQD, muwRQxtWMMXAPxSuMYBw, 
*desc, opTensor->getData()));
#endif
 } void MWFusedConvActivationLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NGqpeiLeVweDRsOKEtuw)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(PtkeOkuClHzhOfpmBevf)); 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(muwRQxtWMMXAPxSuMYBw)); if 
(vIWQzNvYZSuxmOTVDFhU) { CUDA_FREE_CALL(vIWQzNvYZSuxmOTVDFhU); vIWQzNvYZSuxmOTVDFhU = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JgLfgHrHMEMmMYTettJF)); if 
(IpFhwalnAlrMvcuyQpQD) { CUDA_FREE_CALL(IpFhwalnAlrMvcuyQpQD); IpFhwalnAlrMvcuyQpQD = NULL; } if 
(TfsmDFpPPOscKZifVzSQ != getLayer()->getInputTensor(0)) { 
assert(HtQBsWTCGEkpylRklilw); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(XYbzSmRQGatVJtGmDZSo)); } } void 
MWFusedConvActivationLayerImpl::loadWeights(const char* PmFfARVzoHVAYkfpuvqK) { 
FILE* QMgBqCuvjnbWHWiVPEwn = MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK); 
assert(QMgBqCuvjnbWHWiVPEwn); int cRtIUoZRPICuQEOZOSzT = BRSPqxNffoBYKqpSVHne * 
BdqURaHPmdnfzvtUvocl * BUOdotSvmFyUWQKMUdra * AwZQzUhuWVLGrWgLHRuM * 
AzTsxYcYjIEJsGQbeYHm;  float* KHjdvykTFbUxdfZTFbqy = MALLOC_CALL(sizeof(float) * 
cRtIUoZRPICuQEOZOSzT); MWCNNLayer::call_fread(KHjdvykTFbUxdfZTFbqy, sizeof(float), 
cRtIUoZRPICuQEOZOSzT, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, KHjdvykTFbUxdfZTFbqy, sizeof(float) * 
cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", PmFfARVzoHVAYkfpuvqK, cRtIUoZRPICuQEOZOSzT, KHjdvykTFbUxdfZTFbqy[0]);
#endif
 free(KHjdvykTFbUxdfZTFbqy); fclose(QMgBqCuvjnbWHWiVPEwn); return; } void 
MWFusedConvActivationLayerImpl::loadBias(const char* PmFfARVzoHVAYkfpuvqK) { 
FILE* QMgBqCuvjnbWHWiVPEwn = MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK); 
assert(QMgBqCuvjnbWHWiVPEwn); int cRtIUoZRPICuQEOZOSzT = BdqURaHPmdnfzvtUvocl * 
BUOdotSvmFyUWQKMUdra;  float* KHjdvykTFbUxdfZTFbqy = MALLOC_CALL(sizeof(float) * 
cRtIUoZRPICuQEOZOSzT); MWCNNLayer::call_fread(KHjdvykTFbUxdfZTFbqy, sizeof(float), 
cRtIUoZRPICuQEOZOSzT, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); 
CUDA_CALL(hipMemcpy(IpFhwalnAlrMvcuyQpQD, KHjdvykTFbUxdfZTFbqy, sizeof(float) * 
cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); free(KHjdvykTFbUxdfZTFbqy); 
fclose(QMgBqCuvjnbWHWiVPEwn); return; } void 
MWFusedConvActivationLayerImpl::setLearnables(std::vector<float*> learnables) { 
assert(learnables.size() == 2); int cRtIUoZRPICuQEOZOSzT = BRSPqxNffoBYKqpSVHne * 
BUOdotSvmFyUWQKMUdra * BdqURaHPmdnfzvtUvocl * AwZQzUhuWVLGrWgLHRuM * 
AzTsxYcYjIEJsGQbeYHm;  float* wqggPBXZvtlxnxwngvAq = learnables[0]; 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, wqggPBXZvtlxnxwngvAq, sizeof(float) * 
cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); cRtIUoZRPICuQEOZOSzT = 
BUOdotSvmFyUWQKMUdra * BdqURaHPmdnfzvtUvocl;  float* JLxVZWxVuFDRoIFIqrjf = 
learnables[1]; CUDA_CALL(hipMemcpy(IpFhwalnAlrMvcuyQpQD, JLxVZWxVuFDRoIFIqrjf, 
sizeof(float) * cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); } void 
MWFusedConvActivationLayerImpl::postSetup() { if 
(cQBKlCKXxecGPJrXBXdk->getAutoTune()) { getConvAlgoTuned(); } else { 
getConvAlgoWorkSpaceLimit(); } } void 
MWFusedConvActivationLayerImpl::getConvAlgoTuned() { MWTensorBase* opTensorBase 
= getLayer()->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); 
assert(desc); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
PtkeOkuClHzhOfpmBevf, vIWQzNvYZSuxmOTVDFhU, NGqpeiLeVweDRsOKEtuw, *desc, 
opTensor->getData(), 3, &returnedAlgoCount, &perf_results[0], 
cQBKlCKXxecGPJrXBXdk->getWorkSpace(), 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize())); NDHPlSVpLroiIBRnjwyO = 
perf_results[0].algo; if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvActivationLayerImpl::getConvAlgoWorkSpaceLimit() { 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR < 8)
 CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, 
HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize(), &NDHPlSVpLroiIBRnjwyO));
#else
 int maxAlgoCount(-1); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
&maxAlgoCount)); int returnedAlgoCount(-1); 
std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(maxAlgoCount); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, maxAlgoCount, 
&returnedAlgoCount, &perf_results[0])); hipdnnConvolutionFwdAlgoPerf_t 
nextFastest; bool algoFound(false); for (int i = 0; i < returnedAlgoCount; ++i) 
{ nextFastest = perf_results[i]; if (nextFastest.memory <= 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize()) { NDHPlSVpLroiIBRnjwyO = 
nextFastest.algo; algoFound = true; break; } } if (!algoFound) { assert(false); }
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvActivationLayerImpl::fixConvAlgo() { int inputH = 
TfsmDFpPPOscKZifVzSQ->getHeight(); int inputW = TfsmDFpPPOscKZifVzSQ->getWidth(); 
if (NDHPlSVpLroiIBRnjwyO == HIPDNN_CONVOLUTION_FWD_ALGO_FFT && (inputH > 64 || 
inputW > 64)) { NDHPlSVpLroiIBRnjwyO = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM; 
} } } 