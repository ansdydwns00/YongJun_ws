#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: minOrMax.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "minOrMax.h"
#include "rt_nonfinite.h"
#include "coder_array.h"
#include <cmath>

// Function Definitions
//
// Arguments    : const array<float, 1U> &b_x
// Return Type  : float
//
namespace coder {
namespace internal {
float maximum(const array<float, 1U> &b_x)
{
  float ex;
  int vlen;
  vlen = b_x.size(0);
  ex = b_x[0];
  for (int i{0}; i <= vlen - 2; i++) {
    bool p;
    if (std::isnan(b_x[i + 1])) {
      p = false;
    } else if (std::isnan(ex)) {
      p = true;
    } else {
      p = (ex < b_x[i + 1]);
    }
    if (p) {
      ex = b_x[i + 1];
    }
  }
  return ex;
}

//
// Arguments    : const array<float, 1U> &b_x
// Return Type  : float
//
float minimum(const array<float, 1U> &b_x)
{
  float ex;
  int vlen;
  vlen = b_x.size(0);
  ex = b_x[0];
  for (int i{0}; i <= vlen - 2; i++) {
    bool p;
    if (std::isnan(b_x[i + 1])) {
      p = false;
    } else if (std::isnan(ex)) {
      p = true;
    } else {
      p = (ex > b_x[i + 1]);
    }
    if (p) {
      ex = b_x[i + 1];
    }
  }
  return ex;
}

} // namespace internal
} // namespace coder

//
// File trailer for minOrMax.cu
//
// [EOF]
//
