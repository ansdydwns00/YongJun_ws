#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnCustomLayerBase.hpp"
#include <map>
 MWCudnnCustomLayerBase::MWCudnnCustomLayerBase() : m_ntwkImpl(NULL) { 
m_isCustomLayer = true; } void 
MWCudnnCustomLayerBase::setupLayer(MWCudnnTarget::MWTargetNetworkImpl* 
targetImpl) { m_ntwkImpl = targetImpl; } void 
MWCudnnCustomLayerBase::createInOutDescriptors(MWTensorBase* aTensor) { 
std::map<MWTensorBase*, hipdnnTensorDescriptor_t*>::iterator it = 
m_inOutDescriptor.find(aTensor); if (it == m_inOutDescriptor.end()) { 
m_inOutDescriptor[aTensor] = new hipdnnTensorDescriptor_t; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(m_inOutDescriptor[aTensor])); } } void 
MWCudnnCustomLayerBase::createInOutTransformDescriptors(MWTensorBase* aTensor) 
{ std::map<MWTensorBase*, hipdnnTensorDescriptor_t*>::iterator it = 
m_inOutTransformDescriptor.find(aTensor); if (it == 
m_inOutTransformDescriptor.end()) { m_inOutTransformDescriptor[aTensor] = new 
hipdnnTensorDescriptor_t; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(m_inOutTransformDescriptor[aTensor])); } 
} void MWCudnnCustomLayerBase::setupInputDescriptors(MWTensorBase* aTensor, 
MWTensorBase::DIMSLABEL srcLayout[], MWTensorBase::DIMSLABEL 
customImplLayout[]) { createInOutDescriptors(aTensor); const int size = 5; int 
inDims[size]; aTensor->getDimsByLayout(srcLayout, size, inDims); int 
strides[size]; MWTensorBase::getStrides(inDims, size, strides); 
CUDNN_CALL(hipdnnSetTensorNdDescriptor(*m_inOutDescriptor[aTensor], 
HIPDNN_DATA_FLOAT, size, inDims, strides)); 
createInOutTransformDescriptors(aTensor); int outDims[size]; 
aTensor->getDimsByLayout(customImplLayout, size, outDims); 
MWTensorBase::getTransformStrides( srcLayout, customImplLayout, outDims, size, 
strides);  
CUDNN_CALL(hipdnnSetTensorNdDescriptor(*m_inOutTransformDescriptor[aTensor], 
HIPDNN_DATA_FLOAT, size, inDims, strides));  } void 
MWCudnnCustomLayerBase::setupOutputDescriptors(MWTensorBase* aTensor, 
MWTensorBase::DIMSLABEL customImplLayout[], MWTensorBase::DIMSLABEL 
destLayout[]) { createInOutDescriptors(aTensor); const int size = 5; int 
inDims[size]; aTensor->getDimsByLayout(customImplLayout, size, inDims); int 
strides[size]; MWTensorBase::getStrides(inDims, size, strides); 
CUDNN_CALL(hipdnnSetTensorNdDescriptor(*m_inOutDescriptor[aTensor], 
HIPDNN_DATA_FLOAT, size, inDims, strides)); 
createInOutTransformDescriptors(aTensor); int outDims[size]; 
aTensor->getDimsByLayout(destLayout, size, outDims); 
MWTensorBase::getTransformStrides(customImplLayout, destLayout, outDims, size, 
strides);  
CUDNN_CALL(hipdnnSetTensorNdDescriptor(*m_inOutTransformDescriptor[aTensor], 
HIPDNN_DATA_FLOAT, size, inDims, strides)); } void 
MWCudnnCustomLayerBase::reorderInputData(MWTensorBase* aTensor, int bufIndex) { 
float oneV = 1.0; float zeroV = 0.0; CUDNN_CALL(cudnnTransformTensor( 
*m_ntwkImpl->getCudnnHandle(), &oneV, *m_inOutDescriptor[aTensor], 
static_cast<MWTensor<float>*>(aTensor)->getData(), &zeroV, 
*m_inOutTransformDescriptor[aTensor], m_ntwkImpl->getPermuteBuffer(bufIndex))); 
} void MWCudnnCustomLayerBase::reorderOutputData(MWTensorBase* aTensor, int 
bufIndex) { float oneV = 1.0; float zeroV = 0.0; 
CUDNN_CALL(cudnnTransformTensor( *m_ntwkImpl->getCudnnHandle(), &oneV, 
*m_inOutDescriptor[aTensor], m_ntwkImpl->getPermuteBuffer(bufIndex), &zeroV, 
*m_inOutTransformDescriptor[aTensor], 
static_cast<MWTensor<float>*>(aTensor)->getData())); } void 
MWCudnnCustomLayerBase::allocate() { for (size_t iOut = 0; iOut < 
getNumOutputs(); iOut++) { MWTensorBase* opTensorBase = getOutputTensor(iOut); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); 
opTensor->setData(m_ntwkImpl->getBufferPtr(opTensor->getopBufIndex())); } } 
void MWCudnnCustomLayerBase::deallocate() { for (size_t iOut = 0; iOut < 
getNumOutputs(); iOut++) { 
static_cast<MWTensor<float>*>(getOutputTensor(iOut))->setData((float*)NULL); } 
} void MWCudnnCustomLayerBase::cleanup() { for (std::map<MWTensorBase*, 
hipdnnTensorDescriptor_t*>::iterator it = m_inOutDescriptor.begin(); it != 
m_inOutDescriptor.end(); ++it) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*it->second)); delete it->second; 
it->second = 0; } for (std::map<MWTensorBase*, 
hipdnnTensorDescriptor_t*>::iterator it = m_inOutTransformDescriptor.begin(); it 
!= m_inOutTransformDescriptor.end(); ++it) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*it->second)); delete it->second; 
it->second = 0; } this->cleanupLayer(); this->MWCNNLayer::cleanup(); } 
MWCudnnCustomLayerBase::~MWCudnnCustomLayerBase() { }