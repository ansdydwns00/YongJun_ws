//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: colon.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "colon.h"
#include "rt_nonfinite.h"
#include "coder_array.h"

// Function Definitions
//
// Arguments    : int a
//                int b
//                array<int, 2U> &b_y
// Return Type  : void
//
namespace coder {
void eml_integer_colon_dispatcher(int a, int b, array<int, 2U> &b_y)
{
  int n;
  if (b < a) {
    n = 0;
  } else {
    if ((a < 0) && (b >= 0)) {
      n = b - a;
    } else {
      n = b - a;
    }
    n++;
  }
  b_y.set_size(1, n);
  if (n > 0) {
    int yk;
    b_y[0] = a;
    yk = a;
    for (int k{0}; k <= n - 2; k++) {
      yk++;
      b_y[k + 1] = yk;
    }
  }
}

} // namespace coder

//
// File trailer for colon.cu
//
// [EOF]
//
