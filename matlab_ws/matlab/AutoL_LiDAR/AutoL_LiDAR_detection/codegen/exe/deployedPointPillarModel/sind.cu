#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: sind.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "sind.h"
#include "deployedPointPillarModel_rtwutil.h"
#include "rt_nonfinite.h"
#include "coder_array.h"
#include <cmath>

// Function Definitions
//
// Arguments    : array<float, 1U> &b_x
// Return Type  : void
//
namespace coder {
void b_sind(array<float, 1U> &b_x)
{
  int nx;
  nx = b_x.size(0);
  for (int k{0}; k < nx; k++) {
    float c_x;
    c_x = b_x[k];
    if (std::isinf(c_x) || std::isnan(c_x)) {
      b_x[k] = rtNaNF;
    } else {
      float absx;
      signed char n;
      c_x = rt_remf_snf(c_x, 360.0F);
      absx = std::abs(c_x);
      if (absx > 180.0F) {
        if (c_x > 0.0F) {
          c_x -= 360.0F;
        } else {
          c_x += 360.0F;
        }
        absx = std::abs(c_x);
      }
      if (absx <= 45.0F) {
        c_x *= 0.0174532924F;
        n = 0;
      } else if (absx <= 135.0F) {
        if (c_x > 0.0F) {
          c_x = 0.0174532924F * (c_x - 90.0F);
          n = 1;
        } else {
          c_x = 0.0174532924F * (c_x + 90.0F);
          n = -1;
        }
      } else if (c_x > 0.0F) {
        c_x = 0.0174532924F * (c_x - 180.0F);
        n = 2;
      } else {
        c_x = 0.0174532924F * (c_x + 180.0F);
        n = -2;
      }
      if (n == 0) {
        b_x[k] = std::sin(c_x);
      } else if (n == 1) {
        b_x[k] = std::cos(c_x);
      } else if (n == -1) {
        b_x[k] = -std::cos(c_x);
      } else {
        b_x[k] = -std::sin(c_x);
      }
    }
  }
}

} // namespace coder

//
// File trailer for sind.cu
//
// [EOF]
//
