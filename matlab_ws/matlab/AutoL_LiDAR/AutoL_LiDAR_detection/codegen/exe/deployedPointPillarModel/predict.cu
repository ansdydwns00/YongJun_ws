#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: predict.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "predict.h"
#include "deployedPointPillarModel_internal_types.h"
#include "dlnetwork.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"

// Function Declarations
static __global__ void
dlnetwork_predict_kernel17(const float varargin_1_Data[24000],
                           float inputDataT_f1[24000]);

static __global__ void
dlnetwork_predict_kernel18(const float varargin_2_Data[10800000],
                           float inputDataT_f2[10800000]);

static __global__ void dlnetwork_predict_kernel19(
    const float outputData_f2[321408], const float outputData_f1[321408],
    float varargout_2_Data[321408], float varargout_1_Data[321408]);

static __global__ void dlnetwork_predict_kernel20(
    const float outputData_f6[107136], const float outputData_f5[107136],
    const float outputData_f4[107136], const float outputData_f3[107136],
    float varargout_6_Data[107136], float varargout_5_Data[107136],
    float varargout_4_Data[107136], float varargout_3_Data[107136]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float varargin_1_Data[24000]
//                float inputDataT_f1[24000]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel17(
    const float varargin_1_Data[24000], float inputDataT_f1[24000])
{
  unsigned long long threadId;
  int i;
  int i1;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i1 = static_cast<int>(threadId % 2ULL);
  i = static_cast<int>((threadId - static_cast<unsigned long long>(i1)) / 2ULL);
  if ((i < 12000) && (i1 < 2)) {
    inputDataT_f1[i1 + (i << 1)] = varargin_1_Data[i + 12000 * i1];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float varargin_2_Data[10800000]
//                float inputDataT_f2[10800000]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel18(
    const float varargin_2_Data[10800000], float inputDataT_f2[10800000])
{
  unsigned long long threadId;
  int i;
  int i1;
  int p;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId % 100ULL);
  threadId = (threadId - static_cast<unsigned long long>(i)) / 100ULL;
  i1 = static_cast<int>(threadId % 12000ULL);
  threadId = (threadId - static_cast<unsigned long long>(i1)) / 12000ULL;
  p = static_cast<int>(threadId);
  if ((p < 9) && (i1 < 12000) && (i < 100)) {
    inputDataT_f2[(i + 100 * i1) + 1200000 * p] =
        varargin_2_Data[(i1 + 12000 * i) + 1200000 * p];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float outputData_f2[321408]
//                const float outputData_f1[321408]
//                float varargout_2_Data[321408]
//                float varargout_1_Data[321408]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel19(
    const float outputData_f2[321408], const float outputData_f1[321408],
    float varargout_2_Data[321408], float varargout_1_Data[321408])
{
  unsigned long long threadId;
  int i;
  int i1;
  int p;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId % 216ULL);
  threadId = (threadId - static_cast<unsigned long long>(i)) / 216ULL;
  i1 = static_cast<int>(threadId % 248ULL);
  threadId = (threadId - static_cast<unsigned long long>(i1)) / 248ULL;
  p = static_cast<int>(threadId);
  if ((p < 6) && (i1 < 248) && (i < 216)) {
    varargout_1_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f1[(i1 + 248 * i) + 53568 * p];
    varargout_2_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f2[(i1 + 248 * i) + 53568 * p];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float outputData_f6[107136]
//                const float outputData_f5[107136]
//                const float outputData_f4[107136]
//                const float outputData_f3[107136]
//                float varargout_6_Data[107136]
//                float varargout_5_Data[107136]
//                float varargout_4_Data[107136]
//                float varargout_3_Data[107136]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel20(
    const float outputData_f6[107136], const float outputData_f5[107136],
    const float outputData_f4[107136], const float outputData_f3[107136],
    float varargout_6_Data[107136], float varargout_5_Data[107136],
    float varargout_4_Data[107136], float varargout_3_Data[107136])
{
  unsigned long long threadId;
  int i;
  int i1;
  int p;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId % 216ULL);
  threadId = (threadId - static_cast<unsigned long long>(i)) / 216ULL;
  i1 = static_cast<int>(threadId % 248ULL);
  threadId = (threadId - static_cast<unsigned long long>(i1)) / 248ULL;
  p = static_cast<int>(threadId);
  if ((p < 2) && (i1 < 248) && (i < 216)) {
    varargout_3_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f3[(i1 + 248 * i) + 53568 * p];
    varargout_4_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f4[(i1 + 248 * i) + 53568 * p];
    varargout_5_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f5[(i1 + 248 * i) + 53568 * p];
    varargout_6_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f6[(i1 + 248 * i) + 53568 * p];
  }
}

//
// Arguments    : detector0_0 *obj
//                const float varargin_1_Data[24000]
//                const float varargin_2_Data[10800000]
//                float varargout_1_Data[321408]
//                float varargout_2_Data[321408]
//                float varargout_3_Data[107136]
//                float varargout_4_Data[107136]
//                float varargout_5_Data[107136]
//                float varargout_6_Data[107136]
// Return Type  : void
//
namespace coder {
namespace internal {
void dlnetwork_predict(detector0_0 *obj, const float varargin_1_Data[24000],
                       const float varargin_2_Data[10800000],
                       float varargout_1_Data[321408],
                       float varargout_2_Data[321408],
                       float varargout_3_Data[107136],
                       float varargout_4_Data[107136],
                       float varargout_5_Data[107136],
                       float varargout_6_Data[107136])
{
  float(*gpu_inputDataT_f2)[10800000];
  float(*gpu_outputData_f1)[321408];
  float(*gpu_outputData_f2)[321408];
  float(*gpu_outputData_f3)[107136];
  float(*gpu_outputData_f4)[107136];
  float(*gpu_outputData_f5)[107136];
  float(*gpu_outputData_f6)[107136];
  float(*gpu_inputDataT_f1)[24000];
  hipMalloc(&gpu_outputData_f6, 428544ULL);
  hipMalloc(&gpu_outputData_f5, 428544ULL);
  hipMalloc(&gpu_outputData_f4, 428544ULL);
  hipMalloc(&gpu_outputData_f3, 428544ULL);
  hipMalloc(&gpu_outputData_f2, 1285632ULL);
  hipMalloc(&gpu_outputData_f1, 1285632ULL);
  hipMalloc(&gpu_inputDataT_f2, 43200000ULL);
  hipMalloc(&gpu_inputDataT_f1, 96000ULL);
  dlnetwork_predict_kernel17<<<dim3(47U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      varargin_1_Data, *gpu_inputDataT_f1);
  dlnetwork_predict_kernel18<<<dim3(21094U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      varargin_2_Data, *gpu_inputDataT_f2);
  hipMemcpy(obj->getInputDataPointer(0), *gpu_inputDataT_f1,
             obj->getLayerOutputSize(3, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(obj->getInputDataPointer(1), *gpu_inputDataT_f2,
             obj->getLayerOutputSize(0, 0), hipMemcpyDeviceToDevice);
  obj->activations(56);
  hipMemcpy(*gpu_outputData_f1, obj->getLayerOutput(56, 0),
             obj->getLayerOutputSize(56, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*gpu_outputData_f2, obj->getLayerOutput(53, 0),
             obj->getLayerOutputSize(53, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*gpu_outputData_f3, obj->getLayerOutput(55, 0),
             obj->getLayerOutputSize(55, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*gpu_outputData_f4, obj->getLayerOutput(49, 0),
             obj->getLayerOutputSize(49, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*gpu_outputData_f5, obj->getLayerOutput(52, 0),
             obj->getLayerOutputSize(52, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*gpu_outputData_f6, obj->getLayerOutput(50, 0),
             obj->getLayerOutputSize(50, 0), hipMemcpyDeviceToDevice);
  dlnetwork_predict_kernel19<<<dim3(628U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_outputData_f2, *gpu_outputData_f1, varargout_2_Data,
      varargout_1_Data);
  dlnetwork_predict_kernel20<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_outputData_f6, *gpu_outputData_f5, *gpu_outputData_f4,
      *gpu_outputData_f3, varargout_6_Data, varargout_5_Data, varargout_4_Data,
      varargout_3_Data);
  hipFree(*gpu_inputDataT_f1);
  hipFree(*gpu_inputDataT_f2);
  hipFree(*gpu_outputData_f1);
  hipFree(*gpu_outputData_f2);
  hipFree(*gpu_outputData_f3);
  hipFree(*gpu_outputData_f4);
  hipFree(*gpu_outputData_f5);
  hipFree(*gpu_outputData_f6);
}

} // namespace internal
} // namespace coder

//
// File trailer for predict.cu
//
// [EOF]
//
