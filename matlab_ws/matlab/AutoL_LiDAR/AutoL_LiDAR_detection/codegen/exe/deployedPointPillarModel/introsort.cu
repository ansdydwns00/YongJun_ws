//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: introsort.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "introsort.h"
#include "cellstr_sort.h"
#include "deployedPointPillarModel_internal_types.h"
#include "heapsort.h"
#include "insertionsort.h"
#include "rt_nonfinite.h"
#include "coder_array.h"

// Type Definitions
struct struct_T {
  int xstart;
  int xend;
  int depth;
};

// Function Definitions
//
// Arguments    : array<int, 1U> &b_x
//                int b_xend
//                const array<cell_wrap_6, 1U> &cmp_workspace_c
// Return Type  : void
//
namespace coder {
namespace internal {
void introsort(array<int, 1U> &b_x, int b_xend,
               const array<cell_wrap_6, 1U> &cmp_workspace_c)
{
  array<struct_T, 1U> st_d;
  struct_T frame;
  if (b_xend > 1) {
    if (b_xend <= 32) {
      insertionsort(b_x, 1, b_xend, cmp_workspace_c);
    } else {
      int MAXDEPTH;
      int pivot;
      int pmax;
      int pmin;
      int pow2p;
      bool exitg1;
      pmax = 31;
      pmin = 0;
      exitg1 = false;
      while ((!exitg1) && (pmax - pmin > 1)) {
        pivot = (pmin + pmax) >> 1;
        pow2p = 1 << pivot;
        if (pow2p == b_xend) {
          pmax = pivot;
          exitg1 = true;
        } else if (pow2p > b_xend) {
          pmax = pivot;
        } else {
          pmin = pivot;
        }
      }
      MAXDEPTH = (pmax - 1) << 1;
      frame.xstart = 1;
      frame.xend = b_xend;
      frame.depth = 0;
      pmax = MAXDEPTH << 1;
      st_d.set_size(pmax);
      for (pmin = 0; pmin < pmax; pmin++) {
        st_d[pmin] = frame;
      }
      st_d[0] = frame;
      pow2p = 1;
      while (pow2p > 0) {
        frame = st_d[pow2p - 1];
        pmax = st_d[pow2p - 1].xstart - 1;
        b_xend = st_d[pow2p - 1].xend;
        pow2p--;
        if ((frame.xend - frame.xstart) + 1 <= 32) {
          insertionsort(b_x, frame.xstart, frame.xend, cmp_workspace_c);
        } else if (frame.depth == MAXDEPTH) {
          b_heapsort(b_x, frame.xstart, frame.xend, cmp_workspace_c);
        } else {
          int t;
          pmin = (frame.xstart + (frame.xend - frame.xstart) / 2) - 1;
          if (matlab::internal::coder::datatypes::cellstr_sort_anonFcn1(
                  cmp_workspace_c, b_x[pmin], b_x[pmax])) {
            t = b_x[frame.xstart - 1];
            b_x[frame.xstart - 1] = b_x[pmin];
            b_x[pmin] = t;
          }
          if (matlab::internal::coder::datatypes::cellstr_sort_anonFcn1(
                  cmp_workspace_c, b_x[b_xend - 1], b_x[pmax])) {
            t = b_x[frame.xstart - 1];
            b_x[frame.xstart - 1] = b_x[frame.xend - 1];
            b_x[frame.xend - 1] = t;
          }
          if (matlab::internal::coder::datatypes::cellstr_sort_anonFcn1(
                  cmp_workspace_c, b_x[b_xend - 1], b_x[pmin])) {
            t = b_x[pmin];
            b_x[pmin] = b_x[frame.xend - 1];
            b_x[frame.xend - 1] = t;
          }
          pivot = b_x[pmin];
          b_x[pmin] = b_x[frame.xend - 2];
          b_x[frame.xend - 2] = pivot;
          pmax = frame.xstart - 1;
          pmin = frame.xend - 2;
          int exitg2;
          do {
            exitg2 = 0;
            pmax++;
            while (matlab::internal::coder::datatypes::cellstr_sort_anonFcn1(
                cmp_workspace_c, b_x[pmax], pivot)) {
              pmax++;
            }
            pmin--;
            while (matlab::internal::coder::datatypes::cellstr_sort_anonFcn1(
                cmp_workspace_c, pivot, b_x[pmin])) {
              pmin--;
            }
            if (pmax + 1 >= pmin + 1) {
              exitg2 = 1;
            } else {
              t = b_x[pmax];
              b_x[pmax] = b_x[pmin];
              b_x[pmin] = t;
            }
          } while (exitg2 == 0);
          b_x[frame.xend - 2] = b_x[pmax];
          b_x[pmax] = pivot;
          if (pmax + 2 < frame.xend) {
            st_d[pow2p].xstart = pmax + 2;
            st_d[pow2p].xend = frame.xend;
            st_d[pow2p].depth = frame.depth + 1;
            pow2p++;
          }
          if (frame.xstart < pmax + 1) {
            st_d[pow2p].xstart = frame.xstart;
            st_d[pow2p].xend = pmax + 1;
            st_d[pow2p].depth = frame.depth + 1;
            pow2p++;
          }
        }
      }
    }
  }
}

} // namespace internal
} // namespace coder

//
// File trailer for introsort.cu
//
// [EOF]
//
