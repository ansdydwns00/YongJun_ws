//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: div.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "div.h"
#include "rt_nonfinite.h"
#include "coder_array.h"

// Function Definitions
//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<float, 2U> &in3
//                const coder::array<float, 2U> &in4
//                const coder::array<float, 2U> &in5
// Return Type  : void
//
void binary_expand_op_11(coder::array<float, 2U> &in1,
                         const coder::array<float, 2U> &in2,
                         const coder::array<float, 2U> &in3,
                         const coder::array<float, 2U> &in4,
                         const coder::array<float, 2U> &in5)
{
  int b;
  int i;
  int stride_0_1;
  int stride_1_1;
  int stride_2_1;
  int stride_3_1;
  int stride_4_1;
  int stride_5_1;
  if (in5.size(1) == 1) {
    b = in3.size(1);
    i = in4.size(1);
  } else {
    b = in5.size(1);
    i = in5.size(1);
  }
  if (b == 1) {
    if (i == 1) {
      if (in3.size(1) == 1) {
        b = in2.size(1);
      } else {
        b = in3.size(1);
      }
    } else if (in5.size(1) == 1) {
      b = in4.size(1);
    } else {
      b = in5.size(1);
    }
  } else if (in5.size(1) == 1) {
    b = in3.size(1);
  } else {
    b = in5.size(1);
  }
  in1.set_size(1, b);
  stride_0_1 = (in2.size(1) != 1);
  stride_1_1 = (in3.size(1) != 1);
  stride_2_1 = (in4.size(1) != 1);
  stride_3_1 = (in5.size(1) != 1);
  stride_4_1 = (in3.size(1) != 1);
  stride_5_1 = (in5.size(1) != 1);
  if (in5.size(1) == 1) {
    b = in3.size(1);
    i = in4.size(1);
  } else {
    b = in5.size(1);
    i = in5.size(1);
  }
  if (b == 1) {
    if (i == 1) {
      if (in3.size(1) == 1) {
        b = in2.size(1);
      } else {
        b = in3.size(1);
      }
    } else if (in5.size(1) == 1) {
      b = in4.size(1);
    } else {
      b = in5.size(1);
    }
  } else if (in5.size(1) == 1) {
    b = in3.size(1);
  } else {
    b = in5.size(1);
  }
  for (i = 0; i < b; i++) {
    in1[i] = (in2[i * stride_0_1] * in3[i * stride_1_1] -
              in4[i * stride_2_1] * in5[i * stride_3_1]) /
             (in3[i * stride_4_1] - in5[i * stride_5_1]);
  }
}

//
// File trailer for div.cu
//
// [EOF]
//
