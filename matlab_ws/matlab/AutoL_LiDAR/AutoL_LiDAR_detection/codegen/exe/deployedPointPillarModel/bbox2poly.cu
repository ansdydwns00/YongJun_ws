//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: bbox2poly.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "bbox2poly.h"
#include "rt_nonfinite.h"
#include "coder_array.h"

// Function Definitions
//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<double, 1U> &in3
//                const coder::array<float, 2U> &in4
//                const coder::array<float, 2U> &in5
// Return Type  : void
//
void binary_expand_op(coder::array<float, 2U> &in1,
                      const coder::array<float, 2U> &in2,
                      const coder::array<double, 1U> &in3,
                      const coder::array<float, 2U> &in4,
                      const coder::array<float, 2U> &in5)
{
  int stride_0_1;
  int stride_1_1;
  int stride_2_1;
  int unnamed_idx_1;
  unnamed_idx_1 = in1.size(1);
  stride_0_1 = (in3.size(0) != 1);
  stride_1_1 = (in4.size(0) != 1);
  stride_2_1 = (in5.size(0) != 1);
  for (int i{0}; i < unnamed_idx_1; i++) {
    in1[4 * i + 3] = (in2[i * stride_0_1 + in2.size(0)] -
                      in4[i * stride_1_1 + in4.size(0)]) +
                     in5[i * stride_2_1 + in5.size(0)];
  }
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<double, 1U> &in3
//                const coder::array<float, 2U> &in4
//                const coder::array<float, 2U> &in5
// Return Type  : void
//
void binary_expand_op_1(coder::array<float, 2U> &in1,
                        const coder::array<float, 2U> &in2,
                        const coder::array<double, 1U> &in3,
                        const coder::array<float, 2U> &in4,
                        const coder::array<float, 2U> &in5)
{
  int stride_0_1;
  int stride_1_1;
  int stride_2_1;
  int unnamed_idx_1;
  unnamed_idx_1 = in1.size(1);
  stride_0_1 = (in3.size(0) != 1);
  stride_1_1 = (in4.size(0) != 1);
  stride_2_1 = (in5.size(0) != 1);
  for (int i{0}; i < unnamed_idx_1; i++) {
    in1[4 * i + 3] =
        (in2[i * stride_0_1] - in4[i * stride_1_1]) + in5[i * stride_2_1];
  }
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<double, 1U> &in3
//                const coder::array<float, 2U> &in4
//                const coder::array<float, 2U> &in5
// Return Type  : void
//
void binary_expand_op_2(coder::array<float, 2U> &in1,
                        const coder::array<float, 2U> &in2,
                        const coder::array<double, 1U> &in3,
                        const coder::array<float, 2U> &in4,
                        const coder::array<float, 2U> &in5)
{
  int stride_0_1;
  int stride_1_1;
  int stride_2_1;
  int unnamed_idx_1;
  unnamed_idx_1 = in1.size(1);
  stride_0_1 = (in3.size(0) != 1);
  stride_1_1 = (in4.size(0) != 1);
  stride_2_1 = (in5.size(0) != 1);
  for (int i{0}; i < unnamed_idx_1; i++) {
    in1[4 * i + 2] = (in2[i * stride_0_1 + in2.size(0)] +
                      in4[i * stride_1_1 + in4.size(0)]) +
                     in5[i * stride_2_1 + in5.size(0)];
  }
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<double, 1U> &in3
//                const coder::array<float, 2U> &in4
//                const coder::array<float, 2U> &in5
// Return Type  : void
//
void binary_expand_op_3(coder::array<float, 2U> &in1,
                        const coder::array<float, 2U> &in2,
                        const coder::array<double, 1U> &in3,
                        const coder::array<float, 2U> &in4,
                        const coder::array<float, 2U> &in5)
{
  int stride_0_1;
  int stride_1_1;
  int stride_2_1;
  int unnamed_idx_1;
  unnamed_idx_1 = in1.size(1);
  stride_0_1 = (in3.size(0) != 1);
  stride_1_1 = (in4.size(0) != 1);
  stride_2_1 = (in5.size(0) != 1);
  for (int i{0}; i < unnamed_idx_1; i++) {
    in1[4 * i + 2] =
        (in2[i * stride_0_1] + in4[i * stride_1_1]) + in5[i * stride_2_1];
  }
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<double, 1U> &in3
//                const coder::array<float, 2U> &in4
//                const coder::array<float, 2U> &in5
// Return Type  : void
//
void binary_expand_op_4(coder::array<float, 2U> &in1,
                        const coder::array<float, 2U> &in2,
                        const coder::array<double, 1U> &in3,
                        const coder::array<float, 2U> &in4,
                        const coder::array<float, 2U> &in5)
{
  int stride_0_1;
  int stride_1_1;
  int stride_2_1;
  int unnamed_idx_1;
  unnamed_idx_1 = in1.size(1);
  stride_0_1 = (in3.size(0) != 1);
  stride_1_1 = (in4.size(0) != 1);
  stride_2_1 = (in5.size(0) != 1);
  for (int i{0}; i < unnamed_idx_1; i++) {
    in1[4 * i + 1] = (in2[i * stride_0_1 + in2.size(0)] +
                      in4[i * stride_1_1 + in4.size(0)]) -
                     in5[i * stride_2_1 + in5.size(0)];
  }
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<double, 1U> &in3
//                const coder::array<float, 2U> &in4
//                const coder::array<float, 2U> &in5
// Return Type  : void
//
void binary_expand_op_5(coder::array<float, 2U> &in1,
                        const coder::array<float, 2U> &in2,
                        const coder::array<double, 1U> &in3,
                        const coder::array<float, 2U> &in4,
                        const coder::array<float, 2U> &in5)
{
  int stride_0_1;
  int stride_1_1;
  int stride_2_1;
  int unnamed_idx_1;
  unnamed_idx_1 = in1.size(1);
  stride_0_1 = (in3.size(0) != 1);
  stride_1_1 = (in4.size(0) != 1);
  stride_2_1 = (in5.size(0) != 1);
  for (int i{0}; i < unnamed_idx_1; i++) {
    in1[4 * i + 1] =
        (in2[i * stride_0_1] + in4[i * stride_1_1]) - in5[i * stride_2_1];
  }
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<double, 1U> &in2
//                const coder::array<float, 2U> &in3
//                const coder::array<float, 2U> &in4
//                const coder::array<float, 2U> &in5
// Return Type  : void
//
void binary_expand_op_6(coder::array<float, 2U> &in1,
                        const coder::array<double, 1U> &in2,
                        const coder::array<float, 2U> &in3,
                        const coder::array<float, 2U> &in4,
                        const coder::array<float, 2U> &in5)
{
  int stride_0_1;
  int stride_1_1;
  int stride_2_1;
  int unnamed_idx_1;
  unnamed_idx_1 = in2.size(0);
  stride_0_1 = (in2.size(0) != 1);
  stride_1_1 = (in4.size(0) != 1);
  stride_2_1 = (in5.size(0) != 1);
  for (int i{0}; i < unnamed_idx_1; i++) {
    in1[4 * i] = (in3[i * stride_0_1 + in3.size(0)] -
                  in4[i * stride_1_1 + in4.size(0)]) -
                 in5[i * stride_2_1 + in5.size(0)];
  }
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<double, 1U> &in2
//                const coder::array<float, 2U> &in3
//                const coder::array<float, 2U> &in4
//                const coder::array<float, 2U> &in5
// Return Type  : void
//
void binary_expand_op_7(coder::array<float, 2U> &in1,
                        const coder::array<double, 1U> &in2,
                        const coder::array<float, 2U> &in3,
                        const coder::array<float, 2U> &in4,
                        const coder::array<float, 2U> &in5)
{
  int stride_0_1;
  int stride_1_1;
  int stride_2_1;
  int unnamed_idx_1;
  unnamed_idx_1 = in2.size(0);
  stride_0_1 = (in2.size(0) != 1);
  stride_1_1 = (in4.size(0) != 1);
  stride_2_1 = (in5.size(0) != 1);
  for (int i{0}; i < unnamed_idx_1; i++) {
    in1[4 * i] =
        (in3[i * stride_0_1] - in4[i * stride_1_1]) - in5[i * stride_2_1];
  }
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<double, 1U> &in3
//                const coder::array<float, 1U> &in4
//                const coder::array<float, 1U> &in5
// Return Type  : void
//
void binary_expand_op_8(coder::array<float, 2U> &in1,
                        const coder::array<float, 2U> &in2,
                        const coder::array<double, 1U> &in3,
                        const coder::array<float, 1U> &in4,
                        const coder::array<float, 1U> &in5)
{
  coder::array<float, 2U> b_in2;
  coder::array<float, 2U> b_in4;
  int b;
  int loop_ub;
  int stride_1_0;
  b_in2.set_size(in3.size(0), 2);
  loop_ub = in3.size(0);
  for (int i{0}; i < loop_ub; i++) {
    float f;
    f = in2[i + in2.size(0) * 3];
    b_in2[i] = f / 2.0F;
    b_in2[i + b_in2.size(0)] = f / 2.0F;
  }
  b_in4.set_size(in4.size(0), 2);
  loop_ub = in4.size(0);
  for (int i{0}; i < loop_ub; i++) {
    b_in4[i] = -in4[i];
  }
  loop_ub = in5.size(0);
  for (int i{0}; i < loop_ub; i++) {
    b_in4[i + b_in4.size(0)] = in5[i];
  }
  if (b_in4.size(0) == 1) {
    b = b_in2.size(0);
  } else {
    b = b_in4.size(0);
  }
  in1.set_size(b, 2);
  loop_ub = (b_in2.size(0) != 1);
  stride_1_0 = (b_in4.size(0) != 1);
  for (int i{0}; i < 2; i++) {
    if (b_in4.size(0) == 1) {
      b = b_in2.size(0);
    } else {
      b = b_in4.size(0);
    }
    for (int i1{0}; i1 < b; i1++) {
      in1[i1 + in1.size(0) * i] = b_in2[i1 * loop_ub + b_in2.size(0) * i] *
                                  b_in4[i1 * stride_1_0 + b_in4.size(0) * i];
    }
  }
}

//
// Arguments    : coder::array<float, 2U> &in1
//                const coder::array<float, 2U> &in2
//                const coder::array<double, 1U> &in3
//                const coder::array<float, 1U> &in4
//                const coder::array<float, 1U> &in5
// Return Type  : void
//
void binary_expand_op_9(coder::array<float, 2U> &in1,
                        const coder::array<float, 2U> &in2,
                        const coder::array<double, 1U> &in3,
                        const coder::array<float, 1U> &in4,
                        const coder::array<float, 1U> &in5)
{
  coder::array<float, 2U> b_in2;
  coder::array<float, 2U> b_in4;
  int b;
  int loop_ub;
  int stride_1_0;
  b_in2.set_size(in3.size(0), 2);
  loop_ub = in3.size(0);
  for (int i{0}; i < loop_ub; i++) {
    float f;
    f = in2[i + in2.size(0) * 2];
    b_in2[i] = f / 2.0F;
    b_in2[i + b_in2.size(0)] = f / 2.0F;
  }
  b_in4.set_size(in4.size(0), 2);
  loop_ub = in4.size(0);
  for (int i{0}; i < loop_ub; i++) {
    b_in4[i] = in4[i];
  }
  loop_ub = in5.size(0);
  for (int i{0}; i < loop_ub; i++) {
    b_in4[i + b_in4.size(0)] = in5[i];
  }
  if (b_in4.size(0) == 1) {
    b = b_in2.size(0);
  } else {
    b = b_in4.size(0);
  }
  in1.set_size(b, 2);
  loop_ub = (b_in2.size(0) != 1);
  stride_1_0 = (b_in4.size(0) != 1);
  for (int i{0}; i < 2; i++) {
    if (b_in4.size(0) == 1) {
      b = b_in2.size(0);
    } else {
      b = b_in4.size(0);
    }
    for (int i1{0}; i1 < b; i1++) {
      in1[i1 + in1.size(0) * i] = b_in2[i1 * loop_ub + b_in2.size(0) * i] *
                                  b_in4[i1 * stride_1_0 + b_in4.size(0) * i];
    }
  }
}

//
// File trailer for bbox2poly.cu
//
// [EOF]
//
