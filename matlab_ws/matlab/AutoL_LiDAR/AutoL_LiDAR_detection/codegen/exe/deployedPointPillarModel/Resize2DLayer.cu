#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: Resize2DLayer.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "Resize2DLayer.h"
#include "deployedPointPillarModel_data.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"

// Variable Definitions
static bool gpuConstsCopied_interpAlongSpatialDim;

static bool b_gpuConstsCopied_interpAlongSpatialDim;

// Function Declarations
static __global__ void interp1CustomImpl_kernel24(const double b_x[124],
                                                  const float v[3428352],
                                                  const double xq[248],
                                                  float out[6856704]);

static __global__ void interp1CustomImpl_kernel29(const double b_x[62],
                                                  const float v[1714176],
                                                  const double xq[248],
                                                  float out[6856704]);

static __global__ void
interpAlongSpatialDim_kernel22(const float inTmp[1714176], const double xq[216],
                               float out[3428352]);

static __global__ void interpAlongSpatialDim_kernel27(const float inTmp[428544],
                                                      const double xq[216],
                                                      float out[1714176]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double b_x[124]
//                const float v[3428352]
//                const double xq[248]
//                float out[6856704]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void interp1CustomImpl_kernel24(
    const double b_x[124], const float v[3428352], const double xq[248],
    float out[6856704])
{
  unsigned long long threadId;
  int i;
  int j;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int>(threadId % 27648ULL);
  i = static_cast<int>((threadId - static_cast<unsigned long long>(j)) /
                       27648ULL);
  if ((i < 248) && (j < 27648)) {
    double d;
    double idx;
    double nextIdx;
    d = xq[i];
    idx = floor(d);
    nextIdx = ceil(d);
    if (idx != nextIdx) {
      float slope;
      slope = (v[(static_cast<int>(idx) + 124 * j) - 1] -
               v[(static_cast<int>(nextIdx) + 124 * j) - 1]) /
              static_cast<float>(b_x[static_cast<int>(idx) - 1] -
                                 b_x[static_cast<int>(nextIdx) - 1]);
      out[i + 248 * j] =
          slope * static_cast<float>(d) +
          (v[(static_cast<int>(idx) + 124 * j) - 1] -
           slope * static_cast<float>(b_x[static_cast<int>(idx) - 1]));
    } else {
      out[i + 248 * j] = v[(static_cast<int>(idx) + 124 * j) - 1];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double b_x[62]
//                const float v[1714176]
//                const double xq[248]
//                float out[6856704]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void interp1CustomImpl_kernel29(
    const double b_x[62], const float v[1714176], const double xq[248],
    float out[6856704])
{
  unsigned long long threadId;
  int i;
  int j;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int>(threadId % 27648ULL);
  i = static_cast<int>((threadId - static_cast<unsigned long long>(j)) /
                       27648ULL);
  if ((i < 248) && (j < 27648)) {
    double d;
    double idx;
    double nextIdx;
    d = xq[i];
    idx = floor(d);
    nextIdx = ceil(d);
    if (idx != nextIdx) {
      float slope;
      slope = (v[(static_cast<int>(idx) + 62 * j) - 1] -
               v[(static_cast<int>(nextIdx) + 62 * j) - 1]) /
              static_cast<float>(b_x[static_cast<int>(idx) - 1] -
                                 b_x[static_cast<int>(nextIdx) - 1]);
      out[i + 248 * j] =
          slope * static_cast<float>(d) +
          (v[(static_cast<int>(idx) + 62 * j) - 1] -
           slope * static_cast<float>(b_x[static_cast<int>(idx) - 1]));
    } else {
      out[i + 248 * j] = v[(static_cast<int>(idx) + 62 * j) - 1];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float inTmp[1714176]
//                const double xq[216]
//                float out[3428352]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void interpAlongSpatialDim_kernel22(
    const float inTmp[1714176], const double xq[216], float out[3428352])
{
  unsigned long long threadId;
  int i;
  int j;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int>(threadId % 15872ULL);
  i = static_cast<int>((threadId - static_cast<unsigned long long>(j)) /
                       15872ULL);
  if ((i < 216) && (j < 15872)) {
    double d;
    int idx;
    int nextIdx;
    d = xq[i];
    idx = static_cast<int>(floor(d));
    nextIdx = static_cast<int>(ceil(d));
    if (idx != nextIdx) {
      float slope;
      slope = (inTmp[(idx + 108 * j) - 1] - inTmp[(nextIdx + 108 * j) - 1]) /
              static_cast<float>(idx - nextIdx);
      out[i + 216 * j] = slope * static_cast<float>(d) +
                         (inTmp[(idx + 108 * j) - 1] -
                          slope * ((static_cast<float>(idx) - 1.0F) + 1.0F));
    } else {
      out[i + 216 * j] = inTmp[(idx + 108 * j) - 1];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float inTmp[428544]
//                const double xq[216]
//                float out[1714176]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void interpAlongSpatialDim_kernel27(
    const float inTmp[428544], const double xq[216], float out[1714176])
{
  unsigned long long threadId;
  int i;
  int j;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int>(threadId % 7936ULL);
  i = static_cast<int>((threadId - static_cast<unsigned long long>(j)) /
                       7936ULL);
  if ((i < 216) && (j < 7936)) {
    double d;
    int idx;
    int nextIdx;
    d = xq[i];
    idx = static_cast<int>(floor(d));
    nextIdx = static_cast<int>(ceil(d));
    if (idx != nextIdx) {
      float slope;
      slope = (inTmp[(idx + 54 * j) - 1] - inTmp[(nextIdx + 54 * j) - 1]) /
              static_cast<float>(idx - nextIdx);
      out[i + 216 * j] = slope * static_cast<float>(d) +
                         (inTmp[(idx + 54 * j) - 1] -
                          slope * ((static_cast<float>(idx) - 1.0F) + 1.0F));
    } else {
      out[i + 216 * j] = inTmp[(idx + 54 * j) - 1];
    }
  }
}

//
// Arguments    : const double b_x[62]
//                const float v[1714176]
//                const double xq[248]
//                float out[6856704]
// Return Type  : void
//
namespace coder {
namespace nnet {
namespace internal {
namespace cnn {
namespace coder {
void b_interp1CustomImpl(const double b_x[62], const float v[1714176],
                         const double xq[248], float out[6856704])
{
  interp1CustomImpl_kernel29<<<dim3(13392U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      b_x, v, xq, out);
}

//
// Arguments    : const float inTmp[428544]
//                float out[1714176]
// Return Type  : void
//
void b_interpAlongSpatialDim(const float inTmp[428544], float out[1714176])
{
  static const double cpu_xq[216]{
      1.0,    1.0,    1.125,  1.375,  1.625,  1.875,  2.125,  2.375,  2.625,
      2.875,  3.125,  3.375,  3.625,  3.875,  4.125,  4.375,  4.625,  4.875,
      5.125,  5.375,  5.625,  5.875,  6.125,  6.375,  6.625,  6.875,  7.125,
      7.375,  7.625,  7.875,  8.125,  8.375,  8.625,  8.875,  9.125,  9.375,
      9.625,  9.875,  10.125, 10.375, 10.625, 10.875, 11.125, 11.375, 11.625,
      11.875, 12.125, 12.375, 12.625, 12.875, 13.125, 13.375, 13.625, 13.875,
      14.125, 14.375, 14.625, 14.875, 15.125, 15.375, 15.625, 15.875, 16.125,
      16.375, 16.625, 16.875, 17.125, 17.375, 17.625, 17.875, 18.125, 18.375,
      18.625, 18.875, 19.125, 19.375, 19.625, 19.875, 20.125, 20.375, 20.625,
      20.875, 21.125, 21.375, 21.625, 21.875, 22.125, 22.375, 22.625, 22.875,
      23.125, 23.375, 23.625, 23.875, 24.125, 24.375, 24.625, 24.875, 25.125,
      25.375, 25.625, 25.875, 26.125, 26.375, 26.625, 26.875, 27.125, 27.375,
      27.625, 27.875, 28.125, 28.375, 28.625, 28.875, 29.125, 29.375, 29.625,
      29.875, 30.125, 30.375, 30.625, 30.875, 31.125, 31.375, 31.625, 31.875,
      32.125, 32.375, 32.625, 32.875, 33.125, 33.375, 33.625, 33.875, 34.125,
      34.375, 34.625, 34.875, 35.125, 35.375, 35.625, 35.875, 36.125, 36.375,
      36.625, 36.875, 37.125, 37.375, 37.625, 37.875, 38.125, 38.375, 38.625,
      38.875, 39.125, 39.375, 39.625, 39.875, 40.125, 40.375, 40.625, 40.875,
      41.125, 41.375, 41.625, 41.875, 42.125, 42.375, 42.625, 42.875, 43.125,
      43.375, 43.625, 43.875, 44.125, 44.375, 44.625, 44.875, 45.125, 45.375,
      45.625, 45.875, 46.125, 46.375, 46.625, 46.875, 47.125, 47.375, 47.625,
      47.875, 48.125, 48.375, 48.625, 48.875, 49.125, 49.375, 49.625, 49.875,
      50.125, 50.375, 50.625, 50.875, 51.125, 51.375, 51.625, 51.875, 52.125,
      52.375, 52.625, 52.875, 53.125, 53.375, 53.625, 53.875, 54.0,   54.0};
  if (!b_gpuConstsCopied_interpAlongSpatialDim) {
    b_gpuConstsCopied_interpAlongSpatialDim = true;
    hipMemcpy(*b_global_gpu_xq, cpu_xq, sizeof(double[216]),
               hipMemcpyHostToDevice);
  }
  interpAlongSpatialDim_kernel27<<<dim3(3348U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      inTmp, *b_global_gpu_xq, out);
}

//
// Arguments    : const double b_x[124]
//                const float v[3428352]
//                const double xq[248]
//                float out[6856704]
// Return Type  : void
//
void interp1CustomImpl(const double b_x[124], const float v[3428352],
                       const double xq[248], float out[6856704])
{
  interp1CustomImpl_kernel24<<<dim3(13392U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      b_x, v, xq, out);
}

//
// Arguments    : const float inTmp[1714176]
//                float out[3428352]
// Return Type  : void
//
void interpAlongSpatialDim(const float inTmp[1714176], float out[3428352])
{
  static const double cpu_xq[216]{
      1.0,    1.25,   1.75,   2.25,   2.75,   3.25,   3.75,   4.25,   4.75,
      5.25,   5.75,   6.25,   6.75,   7.25,   7.75,   8.25,   8.75,   9.25,
      9.75,   10.25,  10.75,  11.25,  11.75,  12.25,  12.75,  13.25,  13.75,
      14.25,  14.75,  15.25,  15.75,  16.25,  16.75,  17.25,  17.75,  18.25,
      18.75,  19.25,  19.75,  20.25,  20.75,  21.25,  21.75,  22.25,  22.75,
      23.25,  23.75,  24.25,  24.75,  25.25,  25.75,  26.25,  26.75,  27.25,
      27.75,  28.25,  28.75,  29.25,  29.75,  30.25,  30.75,  31.25,  31.75,
      32.25,  32.75,  33.25,  33.75,  34.25,  34.75,  35.25,  35.75,  36.25,
      36.75,  37.25,  37.75,  38.25,  38.75,  39.25,  39.75,  40.25,  40.75,
      41.25,  41.75,  42.25,  42.75,  43.25,  43.75,  44.25,  44.75,  45.25,
      45.75,  46.25,  46.75,  47.25,  47.75,  48.25,  48.75,  49.25,  49.75,
      50.25,  50.75,  51.25,  51.75,  52.25,  52.75,  53.25,  53.75,  54.25,
      54.75,  55.25,  55.75,  56.25,  56.75,  57.25,  57.75,  58.25,  58.75,
      59.25,  59.75,  60.25,  60.75,  61.25,  61.75,  62.25,  62.75,  63.25,
      63.75,  64.25,  64.75,  65.25,  65.75,  66.25,  66.75,  67.25,  67.75,
      68.25,  68.75,  69.25,  69.75,  70.25,  70.75,  71.25,  71.75,  72.25,
      72.75,  73.25,  73.75,  74.25,  74.75,  75.25,  75.75,  76.25,  76.75,
      77.25,  77.75,  78.25,  78.75,  79.25,  79.75,  80.25,  80.75,  81.25,
      81.75,  82.25,  82.75,  83.25,  83.75,  84.25,  84.75,  85.25,  85.75,
      86.25,  86.75,  87.25,  87.75,  88.25,  88.75,  89.25,  89.75,  90.25,
      90.75,  91.25,  91.75,  92.25,  92.75,  93.25,  93.75,  94.25,  94.75,
      95.25,  95.75,  96.25,  96.75,  97.25,  97.75,  98.25,  98.75,  99.25,
      99.75,  100.25, 100.75, 101.25, 101.75, 102.25, 102.75, 103.25, 103.75,
      104.25, 104.75, 105.25, 105.75, 106.25, 106.75, 107.25, 107.75, 108.0};
  if (!gpuConstsCopied_interpAlongSpatialDim) {
    gpuConstsCopied_interpAlongSpatialDim = true;
    hipMemcpy(*global_gpu_xq, cpu_xq, sizeof(double[216]),
               hipMemcpyHostToDevice);
  }
  interpAlongSpatialDim_kernel22<<<dim3(6696U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      inTmp, *global_gpu_xq, out);
}

} // namespace coder
} // namespace cnn
} // namespace internal
} // namespace nnet
} // namespace coder

//
// File trailer for Resize2DLayer.cu
//
// [EOF]
//
