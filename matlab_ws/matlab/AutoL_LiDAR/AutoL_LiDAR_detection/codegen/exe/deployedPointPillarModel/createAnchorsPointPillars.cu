//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: createAnchorsPointPillars.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "createAnchorsPointPillars.h"
#include "cat.h"
#include "deployedPointPillarModel_internal_types.h"
#include "repelem.h"
#include "rt_nonfinite.h"
#include "coder_array.h"

// Function Definitions
//
// Arguments    : const cell_wrap_5 anchorBoxes[1]
//                array<double, 2U> &anchors3D
//                array<double, 2U> &anchorsBEV
// Return Type  : void
//
namespace coder {
namespace lidar {
namespace internal {
namespace cnn {
void createAnchorsPointPillars(const cell_wrap_5 anchorBoxes[1],
                               array<double, 2U> &anchors3D,
                               array<double, 2U> &anchorsBEV)
{
  static const double dv[248]{-39.52,
                              -39.2,
                              -38.88,
                              -38.56,
                              -38.24,
                              -37.92,
                              -37.6,
                              -37.28,
                              -36.96,
                              -36.64,
                              -36.32,
                              -36.0,
                              -35.68,
                              -35.36,
                              -35.04,
                              -34.72,
                              -34.4,
                              -34.08,
                              -33.76,
                              -33.44,
                              -33.12,
                              -32.8,
                              -32.480000000000004,
                              -32.160000000000004,
                              -31.84,
                              -31.52,
                              -31.2,
                              -30.88,
                              -30.560000000000002,
                              -30.240000000000002,
                              -29.92,
                              -29.6,
                              -29.28,
                              -28.96,
                              -28.64,
                              -28.32,
                              -28.0,
                              -27.68,
                              -27.36,
                              -27.04,
                              -26.72,
                              -26.400000000000002,
                              -26.080000000000002,
                              -25.76,
                              -25.44,
                              -25.12,
                              -24.8,
                              -24.48,
                              -24.16,
                              -23.84,
                              -23.52,
                              -23.2,
                              -22.88,
                              -22.56,
                              -22.240000000000002,
                              -21.92,
                              -21.6,
                              -21.28,
                              -20.96,
                              -20.64,
                              -20.32,
                              -20.0,
                              -19.68,
                              -19.36,
                              -19.04,
                              -18.72,
                              -18.400000000000002,
                              -18.080000000000002,
                              -17.76,
                              -17.44,
                              -17.12,
                              -16.8,
                              -16.48,
                              -16.16,
                              -15.84,
                              -15.52,
                              -15.200000000000001,
                              -14.88,
                              -14.56,
                              -14.24,
                              -13.92,
                              -13.6,
                              -13.280000000000001,
                              -12.96,
                              -12.64,
                              -12.32,
                              -12.0,
                              -11.68,
                              -11.36,
                              -11.040000000000001,
                              -10.72,
                              -10.4,
                              -10.08,
                              -9.76,
                              -9.44,
                              -9.120000000000001,
                              -8.8,
                              -8.48,
                              -8.16,
                              -7.84,
                              -7.5200000000000005,
                              -7.2,
                              -6.88,
                              -6.5600000000000005,
                              -6.24,
                              -5.92,
                              -5.6000000000000005,
                              -5.28,
                              -4.96,
                              -4.64,
                              -4.32,
                              -4.0,
                              -3.68,
                              -3.36,
                              -3.04,
                              -2.72,
                              -2.4,
                              -2.08,
                              -1.76,
                              -1.44,
                              -1.12,
                              -0.8,
                              -0.48,
                              -0.16,
                              0.16,
                              0.48,
                              0.8,
                              1.12,
                              1.44,
                              1.76,
                              2.08,
                              2.4,
                              2.72,
                              3.04,
                              3.36,
                              3.68,
                              4.0,
                              4.32,
                              4.64,
                              4.96,
                              5.28,
                              5.6000000000000005,
                              5.92,
                              6.24,
                              6.5600000000000005,
                              6.88,
                              7.2,
                              7.5200000000000005,
                              7.84,
                              8.16,
                              8.48,
                              8.8,
                              9.120000000000001,
                              9.44,
                              9.76,
                              10.08,
                              10.4,
                              10.72,
                              11.040000000000001,
                              11.36,
                              11.68,
                              12.0,
                              12.32,
                              12.64,
                              12.96,
                              13.280000000000001,
                              13.6,
                              13.92,
                              14.24,
                              14.56,
                              14.88,
                              15.200000000000001,
                              15.52,
                              15.84,
                              16.16,
                              16.48,
                              16.8,
                              17.12,
                              17.44,
                              17.76,
                              18.080000000000002,
                              18.400000000000002,
                              18.72,
                              19.04,
                              19.36,
                              19.68,
                              20.0,
                              20.32,
                              20.64,
                              20.96,
                              21.28,
                              21.6,
                              21.92,
                              22.240000000000002,
                              22.56,
                              22.88,
                              23.2,
                              23.52,
                              23.84,
                              24.16,
                              24.48,
                              24.8,
                              25.12,
                              25.44,
                              25.76,
                              26.080000000000002,
                              26.400000000000002,
                              26.72,
                              27.04,
                              27.36,
                              27.68,
                              28.0,
                              28.32,
                              28.64,
                              28.96,
                              29.28,
                              29.6,
                              29.92,
                              30.240000000000002,
                              30.560000000000002,
                              30.88,
                              31.2,
                              31.52,
                              31.84,
                              32.160000000000004,
                              32.480000000000004,
                              32.8,
                              33.12,
                              33.44,
                              33.76,
                              34.08,
                              34.4,
                              34.72,
                              35.04,
                              35.36,
                              35.68,
                              36.0,
                              36.32,
                              36.64,
                              36.96,
                              37.28,
                              37.6,
                              37.92,
                              38.24,
                              38.56,
                              38.88,
                              39.2,
                              39.52};
  static double tmp[107136];
  static double xx[53568];
  static double yy[53568];
  array<double, 2U> anchors3dMod;
  array<double, 2U> anchorsDim;
  array<double, 2U> b_anchors3dMod;
  array<double, 2U> b_tmp;
  array<double, 2U> c_anchors3dMod;
  array<double, 2U> dv1;
  int iacol;
  int nrows;
  for (nrows = 0; nrows < 248; nrows++) {
    for (iacol = 0; iacol < 216; iacol++) {
      yy[iacol + 216 * nrows] = dv[nrows];
      xx[iacol + 216 * nrows] =
          0.32000000000000006 * static_cast<double>(iacol) + 0.16;
    }
  }
  for (nrows = 0; nrows < 53568; nrows++) {
    tmp[nrows] = xx[nrows];
    tmp[nrows + 53568] = yy[nrows];
  }
  repelem(tmp, 2.0, b_tmp);
  anchors3D.set_size(107136, 7);
  for (nrows = 0; nrows < 749952; nrows++) {
    anchors3D[nrows] = 0.0;
  }
  for (nrows = 0; nrows < 2; nrows++) {
    for (iacol = 0; iacol < 107136; iacol++) {
      anchors3D[iacol + anchors3D.size(0) * nrows] =
          b_tmp[iacol + b_tmp.size(0) * nrows];
    }
  }
  dv1.set_size(2, 5);
  for (nrows = 0; nrows < 5; nrows++) {
    dv1[dv1.size(0) * nrows] = anchorBoxes[0].f1[nrows << 1];
    dv1[dv1.size(0) * nrows + 1] = anchorBoxes[0].f1[(nrows << 1) + 1];
  }
  anchorsDim.set_size(dv1.size(0) * 53568, 5);
  nrows = dv1.size(0);
  for (int jcol{0}; jcol < 5; jcol++) {
    int ibmat;
    iacol = jcol * nrows;
    ibmat = jcol * (nrows * 53568);
    for (int itilerow{0}; itilerow < 53568; itilerow++) {
      int ibcol;
      ibcol = ibmat + itilerow * nrows;
      for (int k{0}; k < nrows; k++) {
        anchorsDim[ibcol + k] = dv1[iacol + k];
      }
    }
  }
  for (nrows = 0; nrows < 107136; nrows++) {
    anchors3D[nrows + anchors3D.size(0) * 2] =
        anchorsDim[nrows + anchorsDim.size(0) * 3];
  }
  for (nrows = 0; nrows < 3; nrows++) {
    for (iacol = 0; iacol < 107136; iacol++) {
      anchors3D[iacol + anchors3D.size(0) * (nrows + 3)] =
          anchorsDim[iacol + anchorsDim.size(0) * nrows];
    }
  }
  for (nrows = 0; nrows < 107136; nrows++) {
    anchors3D[nrows + anchors3D.size(0) * 6] =
        anchorsDim[nrows + anchorsDim.size(0) * 4];
  }
  anchors3dMod.set_size(107136, 7);
  for (nrows = 0; nrows < 749952; nrows++) {
    anchors3dMod[nrows] = anchors3D[nrows];
  }
  b_anchors3dMod.set_size(107136, 2);
  c_anchors3dMod.set_size(107136, 2);
  for (nrows = 0; nrows < 2; nrows++) {
    for (iacol = 0; iacol < 53568; iacol++) {
      anchors3dMod[((iacol << 1) + anchors3dMod.size(0) * (nrows + 3)) + 1] =
          anchors3D[((iacol << 1) + anchors3D.size(0) * (4 - nrows)) + 1];
    }
    for (iacol = 0; iacol < 107136; iacol++) {
      double d;
      double d1;
      d = anchors3dMod[iacol + anchors3dMod.size(0) * nrows];
      d1 = anchors3dMod[iacol + anchors3dMod.size(0) * (nrows + 3)];
      b_anchors3dMod[iacol + b_anchors3dMod.size(0) * nrows] = d - d1 / 2.0;
      c_anchors3dMod[iacol + c_anchors3dMod.size(0) * nrows] = d + d1 / 2.0;
    }
  }
  cat(b_anchors3dMod, c_anchors3dMod, anchorsBEV);
}

} // namespace cnn
} // namespace internal
} // namespace lidar
} // namespace coder

//
// File trailer for createAnchorsPointPillars.cu
//
// [EOF]
//
