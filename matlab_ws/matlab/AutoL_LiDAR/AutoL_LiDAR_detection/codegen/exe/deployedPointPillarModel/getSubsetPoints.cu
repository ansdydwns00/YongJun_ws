#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: getSubsetPoints.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "getSubsetPoints.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include "coder_array.h"
#include "coder_gpu_array.h"

// Function Declarations
static __global__ void
getSubsetPoints_anonFcn2_kernel15(const coder::gpu_array<double, 1U> outIndex,
                                  double *outSize);

static __global__ void getSubsetPoints_anonFcn2_kernel16(
    const double numElements, const coder::gpu_array<float, 2U> property,
    const double outSize, const coder::gpu_array<double, 1U> outIndex,
    const coder::gpu_array<bool, 1U> indices, const int b_indices,
    coder::gpu_array<float, 2U> varargout_1);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const coder::gpu_array<double, 1U> outIndex
//                double *outSize
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void getSubsetPoints_anonFcn2_kernel15(
        const coder::gpu_array<double, 1U> outIndex, double *outSize)
{
  unsigned long long threadId;
  int i;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 2) {
    *outSize = outIndex[outIndex.size(0) - 1];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double numElements
//                const coder::gpu_array<float, 2U> property
//                const double outSize
//                const coder::gpu_array<double, 1U> outIndex
//                const coder::gpu_array<bool, 1U> indices
//                const int b_indices
//                coder::gpu_array<float, 2U> varargout_1
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void getSubsetPoints_anonFcn2_kernel16(
        const double numElements, const coder::gpu_array<float, 2U> property,
        const double outSize, const coder::gpu_array<double, 1U> outIndex,
        const coder::gpu_array<bool, 1U> indices, const int b_indices,
        coder::gpu_array<float, 2U> varargout_1)
{
  unsigned long long loopEnd;
  unsigned long long threadId;
  unsigned long long threadStride;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<unsigned long long>(b_indices) + 1ULL) * 3ULL - 1ULL;
  for (unsigned long long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int col;
    int itr;
    itr = static_cast<int>(idx %
                           (static_cast<unsigned long long>(b_indices) + 1ULL));
    col = static_cast<int>((idx - static_cast<unsigned long long>(itr)) /
                           (static_cast<unsigned long long>(b_indices) + 1ULL));
    if (indices[itr]) {
      varargout_1[static_cast<int>(outIndex[itr] +
                                   static_cast<double>(col) * outSize) -
                  1] =
          property[static_cast<int>((static_cast<double>(itr) + 1.0) +
                                    static_cast<double>(col) * numElements) -
                   1];
    }
  }
}

//
// Arguments    : array<float, 2U> &cpu_property
//                gpu_array<float, 2U> &gpu_property
//                bool *property_outdatedOnGpu
//                array<bool, 1U> &cpu_indices
//                bool *indices_outdatedOnCpu
//                gpu_array<bool, 1U> &gpu_indices
//                bool *indices_outdatedOnGpu
//                array<float, 2U> &cpu_varargout_1
//                bool *varargout_1_outdatedOnCpu
//                gpu_array<float, 2U> &gpu_varargout_1
//                bool *varargout_1_outdatedOnGpu
// Return Type  : void
//
namespace coder {
namespace vision {
namespace internal {
namespace codegen {
namespace gpu {
void getSubsetPoints_anonFcn2(
    array<float, 2U> &cpu_property, gpu_array<float, 2U> &gpu_property,
    bool *property_outdatedOnGpu, array<bool, 1U> &cpu_indices,
    bool *indices_outdatedOnCpu, gpu_array<bool, 1U> &gpu_indices,
    bool *indices_outdatedOnGpu, array<float, 2U> &cpu_varargout_1,
    bool *varargout_1_outdatedOnCpu, gpu_array<float, 2U> &gpu_varargout_1,
    bool *varargout_1_outdatedOnGpu)
{
  array<double, 1U> cpu_outIndex;
  gpu_array<double, 1U> gpu_outIndex;
  dim3 block;
  dim3 grid;
  double cpu_outSize;
  double *gpu_outSize;
  bool outIndex_outdatedOnGpu;
  hipMalloc(&gpu_outSize, 8ULL);
  outIndex_outdatedOnGpu = false;
  if ((cpu_property.size(0) == 0) || (cpu_property.size(1) == 0) ||
      (cpu_indices.size(0) == 0)) {
    cpu_varargout_1.set_size(0, 3);
    if (!*varargout_1_outdatedOnGpu) {
      gpu_varargout_1.set_size(&cpu_varargout_1);
    }
  } else {
    double numElements;
    int i;
    numElements =
        static_cast<double>(cpu_property.size(0) * cpu_property.size(1)) / 3.0;
    cpu_outIndex.set_size(cpu_indices.size(0));
    gpu_outIndex.set_size(&cpu_outIndex);
    for (i = 0; i < cpu_indices.size(0); i++) {
      if (*indices_outdatedOnCpu) {
        arrayCopyGpuToCpu(cpu_indices, gpu_indices);
      }
      *indices_outdatedOnCpu = false;
      cpu_outIndex[i] = cpu_indices[i];
      outIndex_outdatedOnGpu = true;
    }
    if (cpu_outIndex.size(0) != 1) {
      if (outIndex_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_outIndex, cpu_outIndex);
      }
      mwCallThrustScan1D(&(gpu_outIndex.data())[0], false,
                         cpu_outIndex.size(0));
      outIndex_outdatedOnGpu = false;
    }
    if (outIndex_outdatedOnGpu) {
      arrayCopyCpuToGpu(gpu_outIndex, cpu_outIndex);
    }
    getSubsetPoints_anonFcn2_kernel15<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        gpu_outIndex, gpu_outSize);
    hipMemcpy(&cpu_outSize, gpu_outSize, 8ULL, hipMemcpyDeviceToHost);
    cpu_varargout_1.set_size(static_cast<int>(cpu_outSize), 3);
    if (!*varargout_1_outdatedOnGpu) {
      gpu_varargout_1.set_size(&cpu_varargout_1);
    }
    i = cpu_indices.size(0) - 1;
    outIndex_outdatedOnGpu = mwGetLaunchParameters1D(
        static_cast<double>((i + 1LL) * 3LL), &grid, &block, 1024U, 65535U);
    if (outIndex_outdatedOnGpu) {
      if (*property_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_property, cpu_property);
      }
      *property_outdatedOnGpu = false;
      if (*indices_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_indices, cpu_indices);
      }
      *indices_outdatedOnGpu = false;
      if (*varargout_1_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_varargout_1, cpu_varargout_1);
      }
      getSubsetPoints_anonFcn2_kernel16<<<grid, block>>>(
          numElements, gpu_property, cpu_outSize, gpu_outIndex, gpu_indices, i,
          gpu_varargout_1);
      *varargout_1_outdatedOnGpu = false;
      *varargout_1_outdatedOnCpu = true;
    }
  }
  hipFree(gpu_outSize);
}

} // namespace gpu
} // namespace codegen
} // namespace internal
} // namespace vision
} // namespace coder

//
// File trailer for getSubsetPoints.cu
//
// [EOF]
//
