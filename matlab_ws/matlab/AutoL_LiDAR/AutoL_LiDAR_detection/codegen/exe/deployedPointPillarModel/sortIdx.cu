//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: sortIdx.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "sortIdx.h"
#include "deployedPointPillarModel_rtwutil.h"
#include "rt_nonfinite.h"
#include "coder_array.h"

// Function Declarations
namespace coder {
namespace internal {
static void merge(array<int, 2U> &idx, array<float, 2U> &b_x, int b_offset,
                  int np, int nq, array<int, 1U> &iwork,
                  array<float, 1U> &xwork);

}
} // namespace coder

// Function Definitions
//
// Arguments    : array<int, 2U> &idx
//                array<float, 2U> &b_x
//                int b_offset
//                int np
//                int nq
//                array<int, 1U> &iwork
//                array<float, 1U> &xwork
// Return Type  : void
//
namespace coder {
namespace internal {
static void merge(array<int, 2U> &idx, array<float, 2U> &b_x, int b_offset,
                  int np, int nq, array<int, 1U> &iwork,
                  array<float, 1U> &xwork)
{
  if ((np != 0) && (nq != 0)) {
    int iout;
    int n;
    int p;
    int qend;
    n = np + nq;
    for (qend = 0; qend < n; qend++) {
      iwork[qend] = idx[b_offset + qend];
      xwork[qend] = b_x[b_offset + qend];
    }
    p = 1;
    n = np;
    qend = np + nq;
    iout = b_offset - 1;
    int exitg1;
    do {
      exitg1 = 0;
      iout++;
      if (xwork[p - 1] <= xwork[n]) {
        idx[iout] = iwork[p - 1];
        b_x[iout] = xwork[p - 1];
        if (p < np) {
          p++;
        } else {
          exitg1 = 1;
        }
      } else {
        idx[iout] = iwork[n];
        b_x[iout] = xwork[n];
        if (n + 1 < qend) {
          n++;
        } else {
          long long b;
          n = (iout - p) + 1;
          b = computeEndIdx(static_cast<long long>(p),
                            static_cast<long long>(np), 1LL);
          for (long long j{0LL}; j <= b; j++) {
            idx[n + static_cast<int>(p + j)] =
                iwork[static_cast<int>(p + j) - 1];
            b_x[n + static_cast<int>(p + j)] =
                xwork[static_cast<int>(p + j) - 1];
          }
          exitg1 = 1;
        }
      }
    } while (exitg1 == 0);
  }
}

//
// Arguments    : array<int, 2U> &idx
//                array<float, 2U> &b_x
//                int b_offset
//                int n
//                int preSortLevel
//                array<int, 1U> &iwork
//                array<float, 1U> &xwork
// Return Type  : void
//
void merge_block(array<int, 2U> &idx, array<float, 2U> &b_x, int b_offset,
                 int n, int preSortLevel, array<int, 1U> &iwork,
                 array<float, 1U> &xwork)
{
  int bLen;
  int nPairs;
  nPairs = n >> preSortLevel;
  bLen = 1 << preSortLevel;
  while (nPairs > 1) {
    int nTail;
    int tailOffset;
    if ((nPairs & 1) != 0) {
      nPairs--;
      tailOffset = bLen * nPairs;
      nTail = n - tailOffset;
      if (nTail > bLen) {
        merge(idx, b_x, b_offset + tailOffset, bLen, nTail - bLen, iwork,
              xwork);
      }
    }
    tailOffset = bLen << 1;
    nPairs >>= 1;
    for (nTail = 0; nTail < nPairs; nTail++) {
      merge(idx, b_x, b_offset + nTail * tailOffset, bLen, bLen, iwork, xwork);
    }
    bLen = tailOffset;
  }
  if (n > bLen) {
    merge(idx, b_x, b_offset, bLen, n - bLen, iwork, xwork);
  }
}

} // namespace internal
} // namespace coder

//
// File trailer for sortIdx.cu
//
// [EOF]
//
