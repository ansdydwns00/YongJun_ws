#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: inpolygon.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "inpolygon.h"
#include "colon.h"
#include "find.h"
#include "minOrMax.h"
#include "rt_nonfinite.h"
#include "coder_array.h"
#include <cmath>

// Function Declarations
static void binary_expand_op_13(coder::array<bool, 1U> &in1,
                                const coder::array<bool, 1U> &in2, int in3);

static void binary_expand_op_14(coder::array<bool, 1U> &in1,
                                const coder::array<bool, 1U> &in2,
                                const coder::array<float, 1U> &in3);

static void binary_expand_op_15(coder::array<bool, 1U> &in1,
                                const coder::array<float, 1U> &in2, float in3,
                                float in4, const coder::array<float, 1U> &in5,
                                float in6, float in7);

// Function Definitions
//
// Arguments    : coder::array<bool, 1U> &in1
//                const coder::array<bool, 1U> &in2
//                int in3
// Return Type  : void
//
static void binary_expand_op_13(coder::array<bool, 1U> &in1,
                                const coder::array<bool, 1U> &in2, int in3)
{
  coder::array<bool, 1U> r;
  int b;
  int stride_0_0;
  int stride_1_0;
  r.set_size(in3 + 2);
  r[0] = true;
  for (int i{0}; i <= in3; i++) {
    r[i + 1] = in2[i];
  }
  if (r.size(0) == 1) {
    b = in2.size(0);
  } else {
    b = r.size(0);
  }
  in1.set_size(b);
  stride_0_0 = (in2.size(0) != 1);
  stride_1_0 = (r.size(0) != 1);
  if (r.size(0) == 1) {
    b = in2.size(0);
  } else {
    b = r.size(0);
  }
  for (int i{0}; i < b; i++) {
    in1[i] = (in2[i * stride_0_0] && r[i * stride_1_0]);
  }
}

//
// Arguments    : coder::array<bool, 1U> &in1
//                const coder::array<bool, 1U> &in2
//                const coder::array<float, 1U> &in3
// Return Type  : void
//
static void binary_expand_op_14(coder::array<bool, 1U> &in1,
                                const coder::array<bool, 1U> &in2,
                                const coder::array<float, 1U> &in3)
{
  int b;
  int stride_0_0;
  int stride_1_0;
  if (in3.size(0) == 1) {
    b = in2.size(0);
  } else {
    b = in3.size(0);
  }
  in1.set_size(b);
  stride_0_0 = (in2.size(0) != 1);
  stride_1_0 = (in3.size(0) != 1);
  if (in3.size(0) == 1) {
    b = in2.size(0);
  } else {
    b = in3.size(0);
  }
  for (int i{0}; i < b; i++) {
    in1[i] = (in2[i * stride_0_0] || std::isnan(in3[i * stride_1_0]));
  }
}

//
// Arguments    : coder::array<bool, 1U> &in1
//                const coder::array<float, 1U> &in2
//                float in3
//                float in4
//                const coder::array<float, 1U> &in5
//                float in6
//                float in7
// Return Type  : void
//
static void binary_expand_op_15(coder::array<bool, 1U> &in1,
                                const coder::array<float, 1U> &in2, float in3,
                                float in4, const coder::array<float, 1U> &in5,
                                float in6, float in7)
{
  int b;
  int stride_0_0;
  int stride_1_0;
  int stride_2_0;
  if (in5.size(0) == 1) {
    b = in2.size(0);
  } else {
    b = in5.size(0);
  }
  in1.set_size(b);
  stride_0_0 = (in2.size(0) != 1);
  stride_1_0 = (in5.size(0) != 1);
  stride_2_0 = (in5.size(0) != 1);
  if (in5.size(0) == 1) {
    b = in2.size(0);
  } else {
    b = in5.size(0);
  }
  for (int i{0}; i < b; i++) {
    in1[i] = ((in2[i * stride_0_0] >= in3) && (in2[i * stride_0_0] <= in4) &&
              (in5[i * stride_1_0] >= in6) && (in5[i * stride_2_0] <= in7));
  }
}

//
// Arguments    : const array<float, 1U> &points_x
//                const array<float, 1U> &points_y
//                const array<float, 1U> &polygon_x
//                const array<float, 1U> &polygon_y
//                array<bool, 2U> &in
// Return Type  : void
//
namespace coder {
namespace images {
namespace internal {
namespace coder {
void inpolygon(const array<float, 1U> &points_x,
               const array<float, 1U> &points_y,
               const array<float, 1U> &polygon_x,
               const array<float, 1U> &polygon_y, array<bool, 2U> &in)
{
  array<double, 2U> signCrossProd;
  array<float, 2U> dotProd;
  array<float, 2U> translatedPolygon_x;
  array<float, 2U> translatedPolygon_y;
  array<float, 1U> closedPolygon_x;
  array<float, 1U> closedPolygon_y;
  array<float, 1U> newPolygon_x;
  array<float, 1U> newPolygon_y;
  array<float, 1U> scaledEps;
  array<int, 2U> iv3;
  array<int, 2U> iv4;
  array<int, 1U> NaNLocations;
  array<int, 1U> inBoundsPointsIdx;
  array<bool, 1U> b_in;
  array<bool, 1U> b_x;
  array<bool, 1U> mask;
  array<bool, 1U> r;
  array<bool, 1U> r1;
  array<bool, 1U> redundantNaNPositions;
  array<bool, 1U> xNaNPositions;
  float avx;
  float avy;
  float maxval;
  float minval;
  unsigned int inputSize_idx_0;
  int ix;
  bool addTrailingNaN;
  bool exitg1;
  inputSize_idx_0 = static_cast<unsigned int>(points_x.size(0));
  avx = ::coder::internal::minimum(polygon_x);
  avy = ::coder::internal::maximum(polygon_x);
  minval = ::coder::internal::minimum(polygon_y);
  maxval = ::coder::internal::maximum(polygon_y);
  if (points_x.size(0) == 1) {
    ix = points_y.size(0);
  } else {
    ix = points_x.size(0);
  }
  if ((points_x.size(0) == points_y.size(0)) && (ix == points_y.size(0))) {
    mask.set_size(points_x.size(0));
    for (int i{0}; i < points_x.size(0); i++) {
      mask[i] = ((points_x[i] >= avx) && (points_x[i] <= avy) &&
                 (points_y[i] >= minval) && (points_y[i] <= maxval));
    }
  } else {
    binary_expand_op_15(mask, points_x, avx, avy, points_y, minval, maxval);
  }
  addTrailingNaN = false;
  ix = 1;
  exitg1 = false;
  while ((!exitg1) && (ix <= mask.size(0))) {
    if (mask[ix - 1]) {
      addTrailingNaN = true;
      exitg1 = true;
    } else {
      ix++;
    }
  }
  if (!addTrailingNaN) {
    in.set_size(points_x.size(0), 1);
    for (int i{0}; i < static_cast<int>(inputSize_idx_0); i++) {
      in[i] = false;
    }
  } else {
    double newNumberOfVertices;
    int b_i;
    int idxOffset;
    int k;
    int p;
    xNaNPositions.set_size(polygon_x.size(0));
    for (int i{0}; i < polygon_x.size(0); i++) {
      xNaNPositions[i] = std::isnan(polygon_x[i]);
    }
    if (xNaNPositions.size(0) == polygon_y.size(0)) {
      b_x.set_size(xNaNPositions.size(0));
      for (int i{0}; i < xNaNPositions.size(0); i++) {
        b_x[i] = (xNaNPositions[i] || std::isnan(polygon_y[i]));
      }
    } else {
      binary_expand_op_14(b_x, xNaNPositions, polygon_y);
    }
    addTrailingNaN = false;
    ix = 1;
    exitg1 = false;
    while ((!exitg1) && (ix <= b_x.size(0))) {
      if (b_x[ix - 1]) {
        addTrailingNaN = true;
        exitg1 = true;
      } else {
        ix++;
      }
    }
    if (!addTrailingNaN) {
      if (polygon_x.size(0) < 3) {
        closedPolygon_x.set_size(polygon_x.size(0));
        for (int i{0}; i < polygon_x.size(0); i++) {
          closedPolygon_x[i] = polygon_x[i];
        }
        closedPolygon_y.set_size(polygon_y.size(0));
        for (int i{0}; i < polygon_y.size(0); i++) {
          closedPolygon_y[i] = polygon_y[i];
        }
      } else if ((polygon_x[0] != polygon_x[polygon_x.size(0) - 1]) ||
                 (polygon_y[0] != polygon_y[polygon_y.size(0) - 1])) {
        closedPolygon_x.set_size(polygon_x.size(0) + 1);
        ix = polygon_x.size(0);
        for (int i{0}; i < ix; i++) {
          closedPolygon_x[i] = polygon_x[i];
        }
        closedPolygon_x[polygon_x.size(0)] = polygon_x[0];
        closedPolygon_y.set_size(polygon_y.size(0) + 1);
        ix = polygon_y.size(0);
        for (int i{0}; i < ix; i++) {
          closedPolygon_y[i] = polygon_y[i];
        }
        closedPolygon_y[polygon_y.size(0)] = polygon_y[0];
      } else {
        closedPolygon_x.set_size(polygon_x.size(0));
        for (int i{0}; i < polygon_x.size(0); i++) {
          closedPolygon_x[i] = polygon_x[i];
        }
        closedPolygon_y.set_size(polygon_y.size(0));
        for (int i{0}; i < polygon_y.size(0); i++) {
          closedPolygon_y[i] = polygon_y[i];
        }
      }
    } else {
      int startIdx;
      if (xNaNPositions.size(0) - 1 < 1) {
        b_i = 1;
      } else {
        b_i = xNaNPositions.size(0);
      }
      if (xNaNPositions.size(0) == b_i) {
        r.set_size(b_i);
        r[0] = true;
        for (int i{0}; i <= b_i - 2; i++) {
          r[i + 1] = xNaNPositions[i];
        }
        redundantNaNPositions.set_size(xNaNPositions.size(0));
        for (int i{0}; i < xNaNPositions.size(0); i++) {
          redundantNaNPositions[i] = (xNaNPositions[i] && r[i]);
        }
      } else {
        binary_expand_op_13(redundantNaNPositions, xNaNPositions, b_i - 2);
      }
      addTrailingNaN = !std::isnan(polygon_x[polygon_x.size(0) - 1]);
      ix = 0;
      b_i = redundantNaNPositions.size(0);
      for (k = 0; k < b_i; k++) {
        if (redundantNaNPositions[k]) {
          ix++;
        }
      }
      newNumberOfVertices = static_cast<double>(polygon_x.size(0) - ix) +
                            static_cast<double>(addTrailingNaN);
      newPolygon_x.set_size(static_cast<int>(newNumberOfVertices));
      newPolygon_y.set_size(static_cast<int>(newNumberOfVertices));
      p = 0;
      ix = polygon_x.size(0);
      for (k = 0; k < ix; k++) {
        if (!redundantNaNPositions[k]) {
          newPolygon_x[p] = polygon_x[k];
          newPolygon_y[p] = polygon_y[k];
          p++;
        }
      }
      if (addTrailingNaN) {
        newPolygon_x[p] = rtNaNF;
        newPolygon_y[p] = rtNaNF;
      }
      r1.set_size(newPolygon_x.size(0));
      for (int i{0}; i < newPolygon_x.size(0); i++) {
        r1[i] = std::isnan(newPolygon_x[i]);
      }
      eml_find(r1, NaNLocations);
      startIdx = 0;
      newNumberOfVertices = 0.0;
      ix = NaNLocations.size(0);
      for (k = 0; k < ix; k++) {
        if ((newPolygon_x[startIdx] == newPolygon_x[NaNLocations[k] - 2]) &&
            (newPolygon_y[startIdx] == newPolygon_y[NaNLocations[k] - 2])) {
          addTrailingNaN = true;
        } else {
          addTrailingNaN = false;
        }
        newNumberOfVertices += static_cast<double>(!addTrailingNaN);
        startIdx = NaNLocations[k];
      }
      if (newNumberOfVertices > 0.0) {
        newNumberOfVertices += static_cast<double>(newPolygon_x.size(0));
        closedPolygon_x.set_size(static_cast<int>(newNumberOfVertices));
        closedPolygon_y.set_size(static_cast<int>(newNumberOfVertices));
        startIdx = 0;
        idxOffset = -1;
        p = NaNLocations.size(0);
        for (k = 0; k < p; k++) {
          if (startIdx + 1 > NaNLocations[k] - 1) {
            ix = 0;
            b_i = 1;
          } else {
            ix = startIdx;
            b_i = NaNLocations[k];
          }
          eml_integer_colon_dispatcher(startIdx + 1, NaNLocations[k] - 1, iv3);
          for (int i{0}; i <= (b_i - ix) - 2; i++) {
            closedPolygon_x[iv3[i] + idxOffset] = newPolygon_x[ix + i];
          }
          if (startIdx + 1 > NaNLocations[k] - 1) {
            ix = 0;
            b_i = 1;
          } else {
            ix = startIdx;
            b_i = NaNLocations[k];
          }
          eml_integer_colon_dispatcher(startIdx + 1, NaNLocations[k] - 1, iv4);
          for (int i{0}; i <= (b_i - ix) - 2; i++) {
            closedPolygon_y[iv4[i] + idxOffset] = newPolygon_y[ix + i];
          }
          if ((!(newPolygon_x[startIdx] ==
                 newPolygon_x[NaNLocations[k] - 2])) ||
              (!(newPolygon_y[startIdx] ==
                 newPolygon_y[NaNLocations[k] - 2]))) {
            idxOffset++;
            closedPolygon_x[(idxOffset + NaNLocations[k]) - 1] =
                newPolygon_x[startIdx];
            closedPolygon_y[(idxOffset + NaNLocations[k]) - 1] =
                newPolygon_y[startIdx];
          }
          closedPolygon_x[idxOffset + NaNLocations[k]] = rtNaNF;
          closedPolygon_y[idxOffset + NaNLocations[k]] = rtNaNF;
          startIdx = NaNLocations[k];
        }
      } else {
        if (newPolygon_x.size(0) - 1 < 1) {
          ix = 0;
        } else {
          ix = newPolygon_x.size(0) - 1;
        }
        closedPolygon_x.set_size(ix);
        for (int i{0}; i < ix; i++) {
          closedPolygon_x[i] = newPolygon_x[i];
        }
        if (newPolygon_y.size(0) - 1 < 1) {
          ix = 0;
        } else {
          ix = newPolygon_y.size(0) - 1;
        }
        closedPolygon_y.set_size(ix);
        for (int i{0}; i < ix; i++) {
          closedPolygon_y[i] = newPolygon_y[i];
        }
      }
    }
    eml_find(mask, inBoundsPointsIdx);
    translatedPolygon_x.set_size(closedPolygon_x.size(0),
                                 inBoundsPointsIdx.size(0));
    translatedPolygon_y.set_size(closedPolygon_x.size(0),
                                 inBoundsPointsIdx.size(0));
    ix = inBoundsPointsIdx.size(0);
    for (p = 0; p < ix; p++) {
      b_i = closedPolygon_x.size(0);
      for (idxOffset = 0; idxOffset < b_i; idxOffset++) {
        translatedPolygon_x[idxOffset + translatedPolygon_x.size(0) * p] =
            closedPolygon_x[idxOffset] - points_x[inBoundsPointsIdx[p] - 1];
        translatedPolygon_y[idxOffset + translatedPolygon_y.size(0) * p] =
            closedPolygon_y[idxOffset] - points_y[inBoundsPointsIdx[p] - 1];
      }
    }
    scaledEps.set_size(closedPolygon_x.size(0) - 1);
    ix = closedPolygon_x.size(0);
    for (k = 0; k <= ix - 2; k++) {
      avx = std::abs(0.5F * (closedPolygon_x[k] + closedPolygon_x[k + 1]));
      avy = std::abs(0.5F * (closedPolygon_y[k] + closedPolygon_y[k + 1]));
      scaledEps[k] =
          std::fmax(std::fmax(avx, avy), avx * avy) * 2.22044605E-16F * 3.0F;
    }
    signCrossProd.set_size(translatedPolygon_x.size(0) - 1,
                           translatedPolygon_x.size(1));
    dotProd.set_size(translatedPolygon_x.size(0) - 1,
                     translatedPolygon_x.size(1));
    ix = translatedPolygon_x.size(1);
    b_in.set_size(translatedPolygon_x.size(1));
    for (p = 0; p < ix; p++) {
      double sumDiffQuad;
      b_i = translatedPolygon_x.size(0);
      sumDiffQuad = 0.0;
      for (idxOffset = 0; idxOffset <= b_i - 2; idxOffset++) {
        double d;
        double quadNum2;
        float b_y;
        float crossProd;
        bool isPositive_y;
        avx = translatedPolygon_x[idxOffset + translatedPolygon_x.size(0) * p];
        avy =
            translatedPolygon_x[(idxOffset + translatedPolygon_x.size(0) * p) +
                                1];
        minval =
            translatedPolygon_y[idxOffset + translatedPolygon_y.size(0) * p];
        maxval =
            translatedPolygon_y[(idxOffset + translatedPolygon_y.size(0) * p) +
                                1];
        dotProd[idxOffset + dotProd.size(0) * p] = avx * avy + minval * maxval;
        crossProd = maxval * avx - avy * minval;
        if (std::isnan(crossProd)) {
          b_y = rtNaNF;
        } else if (crossProd < 0.0F) {
          b_y = -1.0F;
        } else {
          b_y = (crossProd > 0.0F);
        }
        d = b_y;
        signCrossProd[idxOffset + signCrossProd.size(0) * p] = b_y;
        if (std::abs(crossProd) < scaledEps[idxOffset]) {
          d = 0.0;
          signCrossProd[idxOffset + signCrossProd.size(0) * p] = 0.0;
        }
        addTrailingNaN = (avx > 0.0F);
        isPositive_y = (minval > 0.0F);
        newNumberOfVertices =
            (static_cast<double>((!addTrailingNaN) && isPositive_y) +
             2.0 * static_cast<double>((!addTrailingNaN) && (!isPositive_y))) +
            3.0 * static_cast<double>(addTrailingNaN && (!isPositive_y));
        if (std::isnan(avx) || std::isnan(minval)) {
          newNumberOfVertices = rtNaN;
        }
        addTrailingNaN = (avy > 0.0F);
        isPositive_y = (maxval > 0.0F);
        quadNum2 =
            (static_cast<double>((!addTrailingNaN) && isPositive_y) +
             2.0 * static_cast<double>((!addTrailingNaN) && (!isPositive_y))) +
            3.0 * static_cast<double>(addTrailingNaN && (!isPositive_y));
        if (std::isnan(avy) || std::isnan(maxval)) {
          quadNum2 = rtNaN;
        }
        newNumberOfVertices = quadNum2 - newNumberOfVertices;
        if (std::abs(newNumberOfVertices) == 3.0) {
          newNumberOfVertices = -newNumberOfVertices / 3.0;
        } else if (std::abs(newNumberOfVertices) == 2.0) {
          newNumberOfVertices = 2.0 * d;
        } else if (std::isnan(newNumberOfVertices)) {
          newNumberOfVertices = 0.0;
        }
        sumDiffQuad += newNumberOfVertices;
      }
      b_in[p] = (sumDiffQuad != 0.0);
    }
    ix = signCrossProd.size(1);
    for (p = 0; p < ix; p++) {
      k = 0;
      exitg1 = false;
      while ((!exitg1) && (k <= signCrossProd.size(0) - 1)) {
        if ((signCrossProd[k + signCrossProd.size(0) * p] == 0.0) &&
            (dotProd[k + dotProd.size(0) * p] <= 0.0F)) {
          b_in[p] = true;
          exitg1 = true;
        } else {
          k++;
        }
      }
    }
    ix = b_in.size(0) - 1;
    for (b_i = 0; b_i <= ix; b_i++) {
      if (!b_in[b_i]) {
        mask[inBoundsPointsIdx[b_i] - 1] = false;
      }
    }
    in.set_size(points_x.size(0), 1);
    for (int i{0}; i < static_cast<int>(inputSize_idx_0); i++) {
      in[i] = mask[i];
    }
  }
}

} // namespace coder
} // namespace internal
} // namespace images
} // namespace coder

//
// File trailer for inpolygon.cu
//
// [EOF]
//
