//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: deployedPointPillarModel_initialize.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "deployedPointPillarModel_initialize.h"
#include "deployedPointPillarModel_data.h"
#include "pause.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void deployedPointPillarModel_initialize()
{
  cpause_init();
  cublasEnsureInitialization(HIPBLAS_POINTER_MODE_HOST);
  hipMalloc(&global_gpu_dv, sizeof(double[248]));
  hipMalloc(&global_gpu_xq, sizeof(double[216]));
  hipMalloc(&b_global_gpu_dv, sizeof(double[248]));
  hipMalloc(&b_global_gpu_xq, sizeof(double[216]));
  hipGetLastError();
  isInitialized_deployedPointPillarModel = true;
}

//
// File trailer for deployedPointPillarModel_initialize.cu
//
// [EOF]
//
