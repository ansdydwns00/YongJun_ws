#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: permute.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "permute.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"

// Function Declarations
static __global__ void permute_kernel23(const float a[3428352],
                                        float b[3428352]);

static __global__ void permute_kernel28(const float a[1714176],
                                        float b[1714176]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float a[3428352]
//                float b[3428352]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void permute_kernel23(const float a[3428352],
                                                    float b[3428352])
{
  unsigned long long threadId;
  int b_k;
  int c_k;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  c_k = static_cast<int>(threadId % 216ULL);
  threadId = (threadId - static_cast<unsigned long long>(c_k)) / 216ULL;
  b_k = static_cast<int>(threadId % 124ULL);
  threadId = (threadId - static_cast<unsigned long long>(b_k)) / 124ULL;
  k = static_cast<int>(threadId);
  if ((k < 128) && (b_k < 124) && (c_k < 216)) {
    b[(b_k + 124 * c_k) + 26784 * k] = a[(c_k + 216 * b_k) + 26784 * k];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float a[1714176]
//                float b[1714176]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void permute_kernel28(const float a[1714176],
                                                    float b[1714176])
{
  unsigned long long threadId;
  int b_k;
  int c_k;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  c_k = static_cast<int>(threadId % 216ULL);
  threadId = (threadId - static_cast<unsigned long long>(c_k)) / 216ULL;
  b_k = static_cast<int>(threadId % 62ULL);
  threadId = (threadId - static_cast<unsigned long long>(b_k)) / 62ULL;
  k = static_cast<int>(threadId);
  if ((k < 128) && (b_k < 62) && (c_k < 216)) {
    b[(b_k + 62 * c_k) + 13392 * k] = a[(c_k + 216 * b_k) + 13392 * k];
  }
}

//
// Arguments    : const float a[1714176]
//                float b[1714176]
// Return Type  : void
//
namespace coder {
void b_permute(const float a[1714176], float b[1714176])
{
  permute_kernel28<<<dim3(3348U, 1U, 1U), dim3(512U, 1U, 1U)>>>(a, b);
}

//
// Arguments    : const float a[3428352]
//                float b[3428352]
// Return Type  : void
//
void permute(const float a[3428352], float b[3428352])
{
  permute_kernel23<<<dim3(6696U, 1U, 1U), dim3(512U, 1U, 1U)>>>(a, b);
}

} // namespace coder

//
// File trailer for permute.cu
//
// [EOF]
//
