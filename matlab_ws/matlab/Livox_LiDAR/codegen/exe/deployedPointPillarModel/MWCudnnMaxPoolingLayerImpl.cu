#include "hip/hip_runtime.h"
#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnMaxPoolingLayerImpl.hpp"
#include <cmath>
 namespace MWCudnnTarget { void __global__ MWSetDyForBackPropImpl(float* 
OwortPcLToImGdYFtbSF, const int eqOmMKQRpqBqRQCnJmxt); void __global__ 
doMWMaxPoolingLayerImpl(float* URgvgDXnZskIYGdtimcU, float* 
UKtMXCCqdjeyaVHabkxg, const int BNrGqqHwfmYKIqbDbnjx); 
MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DCdZnqpcBnvXVgEsLBnz, int 
DqxLTLaJwwgQqmrtCDuu, int FLuSVNoPhAFKtLUchSvv, int FeVcBgtQmTLtmnNcJGMY, 
int CLOUhPjbgggWoXHTtmjC, int CGbFsczkgkhjcHoCKzBx, int 
ClEhcJFlvGCgiavziIag, int CpMjJjtGOeWOzwxpAAQP, bool 
FwLnexHgxHRquTKmNpoa, int eqUIJyhXTwRqtPfXapcx) : MWCNNLayerImpl(layer, 
ntwk_impl) , BHuHNDGoRwGRouCxeMbw(FwLnexHgxHRquTKmNpoa) , 
URgvgDXnZskIYGdtimcU(0) , OwortPcLToImGdYFtbSF(0) , 
CufLFODQDXTAPyRqYodN(DCdZnqpcBnvXVgEsLBnz) , 
DSsxcjIrUgZCKZovyNQf(DqxLTLaJwwgQqmrtCDuu) , 
CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz) , 
DRzwhbNPpftRRIXXfHzd(DqxLTLaJwwgQqmrtCDuu) , 
CDJtexcMbXMWAmnNZsNf(CLOUhPjbgggWoXHTtmjC) , 
CCKWXUFWgrbBMjwfpOBN(CGbFsczkgkhjcHoCKzBx) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP) , 
EvebzoroiuKkIxwjkGnD(FLuSVNoPhAFKtLUchSvv) , 
FOcStuqCptsGIZXskVpC(FeVcBgtQmTLtmnNcJGMY) , 
puSFZkRJmyuFPfQRswDK(std::pow(2.0f, 
static_cast<float>(layer->getScalingExponent()))) , 
vFNECEAeLZsYsUxvlgqL(std::string{"INT8x4"}.compare(layer->getAccelMode()) 
== 0) , etjQLJVQCaeAXRWYtqOl(eqUIJyhXTwRqtPfXapcx) { 
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&lHtftnmGBvlSSoGOXVui)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&DGzdAcREJHGXjyRzNjJV)); } 
MWMaxPoolingLayerImpl::~MWMaxPoolingLayerImpl() { } void 
MWMaxPoolingLayerImpl::propagateSize() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); if ((CufLFODQDXTAPyRqYodN == -1) && 
(DSsxcjIrUgZCKZovyNQf == -1)) { CqtPRJvHlGJFssiPzsOm = 
ipTensorBase->getHeight(); DRzwhbNPpftRRIXXfHzd = ipTensorBase->getWidth(); } int 
lWJYwWaFPmWNQDPrlqER = CDJtexcMbXMWAmnNZsNf; int 
lXJKIOEATumoVKStGbVy = CTCbzQMDaLxINPbODdng; 
CUDNN_CALL(hipdnnSetPooling2dDescriptor( lHtftnmGBvlSSoGOXVui, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, CqtPRJvHlGJFssiPzsOm, DRzwhbNPpftRRIXXfHzd, 
lWJYwWaFPmWNQDPrlqER, lXJKIOEATumoVKStGbVy, EvebzoroiuKkIxwjkGnD, 
FOcStuqCptsGIZXskVpC)); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); if 
(ipTensorBase->isInt8()) { if (vFNECEAeLZsYsUxvlgqL) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(DGzdAcREJHGXjyRzNjJV, 
HIPDNN_TENSOR_NCHW_VECT_C,  HIPDNN_DATA_INT8x4,  ipTensorBase->getBatchSize(),  
ipTensorBase->getChannels(),  ipTensorBase->getHeight(),  
ipTensorBase->getWidth())  ); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8x4, 
HIPDNN_TENSOR_NCHW_VECT_C); } else { CUDNN_CALL(hipdnnSetTensor4dDescriptor( 
DGzdAcREJHGXjyRzNjJV, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_INT8, 
ipTensorBase->getBatchSize(), ipTensorBase->getChannels(), 
ipTensorBase->getHeight(), ipTensorBase->getWidth())); 
MWCNNLayerImpl::setDescriptorForINT8(*desc, static_cast<MWTensor<signed 
char>*>(opTensorBase), HIPDNN_DATA_INT8, HIPDNN_TENSOR_NCHW); } } else { 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensorBase)); } } 
void MWMaxPoolingLayerImpl::allocate() { MWCNNLayer* maxpoolLayer = getLayer(); 
MWTensorBase* ipTensor = maxpoolLayer->getInputTensor(0); MWTensorBase* 
opTensor = maxpoolLayer->getOutputTensor(0); if (BHuHNDGoRwGRouCxeMbw) { const 
int dJcdBfQQLhIAYHPxwQeg = ipTensor->getNumElements(); CUDA_CALL( 
hipMalloc((void**)&URgvgDXnZskIYGdtimcU, sizeof(float) * 
dJcdBfQQLhIAYHPxwQeg)); const int eqOmMKQRpqBqRQCnJmxt = 
opTensor->getNumElements(); CUDA_CALL(hipMalloc((void**)&OwortPcLToImGdYFtbSF, 
sizeof(float) * eqOmMKQRpqBqRQCnJmxt)); int shEncNmxJsMuJKwbrwok = 
(eqOmMKQRpqBqRQCnJmxt < 1024) ? eqOmMKQRpqBqRQCnJmxt : 1024; int 
KCudOrFMfgCzUPMcdePX = (eqOmMKQRpqBqRQCnJmxt + shEncNmxJsMuJKwbrwok - 
1) / shEncNmxJsMuJKwbrwok; 
MWSetDyForBackPropImpl<<<KCudOrFMfgCzUPMcdePX, 
shEncNmxJsMuJKwbrwok>>>( OwortPcLToImGdYFtbSF, eqOmMKQRpqBqRQCnJmxt); } } void 
MWMaxPoolingLayerImpl::deallocate() { if (URgvgDXnZskIYGdtimcU) { 
CUDA_FREE_CALL(URgvgDXnZskIYGdtimcU); URgvgDXnZskIYGdtimcU = 
NULL; } if (OwortPcLToImGdYFtbSF) { CUDA_FREE_CALL(OwortPcLToImGdYFtbSF); OwortPcLToImGdYFtbSF = 
NULL; } } void MWMaxPoolingLayerImpl::predict() { MWCNNLayer* maxpoolLayer = 
getLayer(); MWTensorBase* ipTensorBase = maxpoolLayer->getInputTensor(0); 
MWTensorBase* opTensorBase = maxpoolLayer->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t XYbzSmRQGatVJtGmDZSo; if (opTensorBase->isInt8()) { 
XYbzSmRQGatVJtGmDZSo = DGzdAcREJHGXjyRzNjJV; MWTensor<signed char>* ipTensor = 
static_cast<MWTensor<signed char>*>(ipTensorBase); MWTensor<signed char>* 
opTensor = static_cast<MWTensor<signed char>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
lHtftnmGBvlSSoGOXVui, &puSFZkRJmyuFPfQRswDK, XYbzSmRQGatVJtGmDZSo, 
ipTensor->getData(), getZeroPtr(), *desc, opTensor->getData())); } else { 
XYbzSmRQGatVJtGmDZSo = MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
lHtftnmGBvlSSoGOXVui, getOnePtr(), XYbzSmRQGatVJtGmDZSo, ipTensor->getData(), 
getZeroPtr(), *desc, opTensor->getData())); if (BHuHNDGoRwGRouCxeMbw) { 
CUDNN_CALL(hipdnnPoolingBackward( *cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
lHtftnmGBvlSSoGOXVui, getOnePtr(), *desc, opTensor->getData(), *desc, 
OwortPcLToImGdYFtbSF, XYbzSmRQGatVJtGmDZSo, ipTensor->getData(), getZeroPtr(), 
XYbzSmRQGatVJtGmDZSo, URgvgDXnZskIYGdtimcU)); int dJcdBfQQLhIAYHPxwQeg = 
ipTensor->getNumElements(); int shEncNmxJsMuJKwbrwok = 
(dJcdBfQQLhIAYHPxwQeg < 1024) ? dJcdBfQQLhIAYHPxwQeg : 1024; int 
KCudOrFMfgCzUPMcdePX = (dJcdBfQQLhIAYHPxwQeg + shEncNmxJsMuJKwbrwok - 
1) / shEncNmxJsMuJKwbrwok; 
doMWMaxPoolingLayerImpl<<<KCudOrFMfgCzUPMcdePX, 
shEncNmxJsMuJKwbrwok>>>( URgvgDXnZskIYGdtimcU, 
static_cast<MWTensor<float>*>(maxpoolLayer->getOutputTensor(1))->getData(), 
dJcdBfQQLhIAYHPxwQeg); } } return; } void MWMaxPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(lHtftnmGBvlSSoGOXVui)); MWTensorBase* 
opTensorBase = getLayer()->getOutputTensor(0); if (opTensorBase->isInt8()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(DGzdAcREJHGXjyRzNjJV)); } } float* 
MWMaxPoolingLayerImpl::getIndexData() { return 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(1))->getData(); } 
void __global__ __launch_bounds__(1024) MWSetDyForBackPropImpl(float* 
OwortPcLToImGdYFtbSF, const int eqOmMKQRpqBqRQCnJmxt) { for (int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < eqOmMKQRpqBqRQCnJmxt; i += blockDim.x * 
gridDim.x) { OwortPcLToImGdYFtbSF[i] = i + 1; } } void __global__ 
__launch_bounds__(1024) doMWMaxPoolingLayerImpl(float* 
URgvgDXnZskIYGdtimcU, float* UKtMXCCqdjeyaVHabkxg, const int 
BNrGqqHwfmYKIqbDbnjx) { for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
BNrGqqHwfmYKIqbDbnjx; i += blockDim.x * gridDim.x) { if 
(static_cast<int>(URgvgDXnZskIYGdtimcU[i]) != 0) { 
UKtMXCCqdjeyaVHabkxg[static_cast<int>(URgvgDXnZskIYGdtimcU[i]) - 1] = 
i; } } } } 