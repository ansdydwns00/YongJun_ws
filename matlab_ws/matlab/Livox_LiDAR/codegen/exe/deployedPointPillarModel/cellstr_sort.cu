#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: cellstr_sort.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "cellstr_sort.h"
#include "deployedPointPillarModel_internal_types.h"
#include "introsort.h"
#include "rt_nonfinite.h"
#include "coder_array.h"
#include <cmath>

// Function Definitions
//
// Arguments    : const array<cell_wrap_6, 1U> &c
//                array<cell_wrap_6, 1U> &sorted
//                array<int, 1U> &idx
// Return Type  : void
//
namespace coder {
namespace matlab {
namespace internal {
namespace coder {
namespace datatypes {
void cellstr_sort(const array<cell_wrap_6, 1U> &c,
                  array<cell_wrap_6, 1U> &sorted, array<int, 1U> &idx)
{
  array<int, 2U> b_y;
  int n;
  int yk;
  n = c.size(0);
  b_y.set_size(1, c.size(0));
  if (c.size(0) > 0) {
    b_y[0] = 1;
    yk = 1;
    for (int k{0}; k <= n - 2; k++) {
      yk++;
      b_y[k + 1] = yk;
    }
  }
  idx.set_size(b_y.size(1));
  for (int k{0}; k < b_y.size(1); k++) {
    idx[k] = b_y[k];
  }
  ::coder::internal::introsort(idx, c.size(0), c);
  sorted.set_size(c.size(0));
  n = c.size(0);
  for (int i{0}; i < n; i++) {
    sorted[i].f1.set_size(1, c[idx[i] - 1].f1.size(1));
    yk = c[idx[i] - 1].f1.size(1);
    for (int k{0}; k < yk; k++) {
      sorted[i].f1[k] = c[idx[i] - 1].f1[k];
    }
  }
}

//
// Arguments    : const array<cell_wrap_6, 1U> &c
//                int i
//                int j
// Return Type  : bool
//
bool cellstr_sort_anonFcn1(const array<cell_wrap_6, 1U> &c, int i, int j)
{
  int k;
  int n;
  bool varargout_1;
  n = static_cast<int>(std::fmin(static_cast<double>(c[i - 1].f1.size(1)),
                                 static_cast<double>(c[j - 1].f1.size(1))));
  varargout_1 = (c[i - 1].f1.size(1) < c[j - 1].f1.size(1));
  k = 0;
  int exitg1;
  do {
    exitg1 = 0;
    if (k <= n - 1) {
      if (c[i - 1].f1[k] != c[j - 1].f1[k]) {
        varargout_1 = (c[i - 1].f1[k] < c[j - 1].f1[k]);
        exitg1 = 1;
      } else {
        k++;
      }
    } else {
      if (c[i - 1].f1.size(1) == c[j - 1].f1.size(1)) {
        varargout_1 = (i < j);
      }
      exitg1 = 1;
    }
  } while (exitg1 == 0);
  return varargout_1;
}

} // namespace datatypes
} // namespace coder
} // namespace internal
} // namespace matlab
} // namespace coder

//
// File trailer for cellstr_sort.cu
//
// [EOF]
//
