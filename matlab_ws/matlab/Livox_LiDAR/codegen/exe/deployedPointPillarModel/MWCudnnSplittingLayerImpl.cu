#include "hip/hip_runtime.h"
#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnSplittingLayerImpl.hpp"
#include <cmath>
#include <cassert>
 namespace MWCudnnTarget { 
MWSplittingLayerImpl::MWSplittingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int numOutputs, int* channelSizes) : 
MWCNNLayerImpl(layer, ntwk_impl) , etjQLJVQCaeAXRWYtqOl(numOutputs) , 
MdSWZSOAjugbWppryHbR(channelSizes) { } 
MWSplittingLayerImpl::~MWSplittingLayerImpl() { } void 
MWSplittingLayerImpl::propagateSize() { } void __global__ 
__launch_bounds__(1024) SplittingLayerImpl(float* inputBuffer, float* 
outputBuffer, int MW_startingChannel, int MW_channelWidth, double 
YNmJhGSUszJKxsodxiuV, double YNDVziqpDddiXQKYZZhX, double 
YMNbgnUYZspjMLjwcIOS, double hljcfGWsvZXJZNrImpJB, const long 
int BNrGqqHwfmYKIqbDbnjx) { int idx = blockDim.x * blockIdx.x + threadIdx.x; if (idx < 
BNrGqqHwfmYKIqbDbnjx) { double cAUupmktEnGPfLHyWfFm = floor(idx / 
YMNbgnUYZspjMLjwcIOS); double EfvWctmlsWAPsxXgdKWf = idx - 
(YMNbgnUYZspjMLjwcIOS * cAUupmktEnGPfLHyWfFm); double LgxABSJPBXdCozJkFqTg = 
floor(EfvWctmlsWAPsxXgdKWf / YNDVziqpDddiXQKYZZhX); if 
(static_cast<int>(LgxABSJPBXdCozJkFqTg) >= MW_startingChannel && 
static_cast<int>(LgxABSJPBXdCozJkFqTg) < MW_startingChannel + MW_channelWidth) { double 
LklYEpYUjaLTgcFFAaJX = LgxABSJPBXdCozJkFqTg - (MW_startingChannel); double 
EpwuhXsRcwdqXSjBpUeO = EfvWctmlsWAPsxXgdKWf - 
(YNDVziqpDddiXQKYZZhX * LgxABSJPBXdCozJkFqTg); long int opIdx = cAUupmktEnGPfLHyWfFm 
* hljcfGWsvZXJZNrImpJB + LklYEpYUjaLTgcFFAaJX * 
YNDVziqpDddiXQKYZZhX + EpwuhXsRcwdqXSjBpUeO; outputBuffer[opIdx] 
= inputBuffer[idx]; } } } void MWSplittingLayerImpl::predict() { MWTensorBase* 
ipTensorBase = getLayer()->getInputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); if (getLayer()->getInPlaceIndex(0) 
!= -1) { assert(ipTensor->getBatchSize() == 1); 
assert(ipTensor->getSequenceLength() == 1); } else { int YNmJhGSUszJKxsodxiuV = 
ipTensor->getWidth(); long int YNDVziqpDddiXQKYZZhX = 
ipTensor->getHeight() * ipTensor->getWidth(); long int 
YMNbgnUYZspjMLjwcIOS = YNDVziqpDddiXQKYZZhX * 
(ipTensor->getChannels()); long int YGiQICncmsGZkNUyiQyg = 
YMNbgnUYZspjMLjwcIOS * ipTensor->getBatchSize(); long int 
rlQsibXJSWJVnUVpdNeL = ((YGiQICncmsGZkNUyiQyg + 31) / 32) * 32; long int 
shEncNmxJsMuJKwbrwok = (rlQsibXJSWJVnUVpdNeL < 1024) ? rlQsibXJSWJVnUVpdNeL : 1024; 
long int KCudOrFMfgCzUPMcdePX = (YGiQICncmsGZkNUyiQyg + 
shEncNmxJsMuJKwbrwok - 1) / shEncNmxJsMuJKwbrwok; int 
MW_startingChannel = 0; for (int i = 0; i < this->etjQLJVQCaeAXRWYtqOl; i++) { 
long int hljcfGWsvZXJZNrImpJB = YNDVziqpDddiXQKYZZhX * 
(getLayer()->getOutputTensor(i)->getChannels()); 
SplittingLayerImpl<<<KCudOrFMfgCzUPMcdePX, shEncNmxJsMuJKwbrwok>>>( 
ipTensor->getData(), 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(i))->getData(), 
MW_startingChannel, getLayer()->getOutputTensor(i)->getChannels(), 
YNmJhGSUszJKxsodxiuV, YNDVziqpDddiXQKYZZhX, 
YMNbgnUYZspjMLjwcIOS, hljcfGWsvZXJZNrImpJB, 
YGiQICncmsGZkNUyiQyg); MW_startingChannel += 
getLayer()->getOutputTensor(i)->getChannels(); } } return; } void 
MWSplittingLayerImpl::cleanup() { } float* 
MWSplittingLayerImpl::offsetOutputPointer(float* initialPointer, int outIdx) { 
if (getLayer()->getInPlaceIndex(0) != -1) { 
assert(getLayer()->getInputTensor(0)->getBatchSize() == 1); 
assert(getLayer()->getInputTensor(0)->getSequenceLength() == 1); long int 
accumulatedC = 0; long int HW = (getLayer()->getOutputTensor(0)->getHeight()) * 
(getLayer()->getOutputTensor(0)->getWidth()); assert(outIdx < 
this->etjQLJVQCaeAXRWYtqOl); for (int i = 0; i < outIdx; i++) { accumulatedC 
+= getLayer()->getOutputTensor(i)->getChannels(); } return initialPointer + (HW 
* accumulatedC); } else { return initialPointer; } } signed char* 
MWSplittingLayerImpl::offsetOutputPointer(signed char* , int ) { assert(false); 
return NULL; } } 