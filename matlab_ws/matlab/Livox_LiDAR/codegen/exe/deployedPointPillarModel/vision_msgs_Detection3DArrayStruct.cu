//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: vision_msgs_Detection3DArrayStruct.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "vision_msgs_Detection3DArrayStruct.h"
#include "deployedPointPillarModel_types.h"
#include "rt_nonfinite.h"
#include "std_msgs_HeaderStruct.h"
#include "vision_msgs_Detection3DStruct.h"
#include "coder_array.h"

// Function Definitions
//
// Message struct definition for vision_msgs/Detection3DArray
//
// Arguments    : vision_msgs_Detection3DArrayStruct_T *msg
// Return Type  : void
//
void vision_msgs_Detection3DArrayStruct(
    vision_msgs_Detection3DArrayStruct_T *msg)
{
  static const char b_cv[28]{'v', 'i', 's', 'i', 'o', 'n', '_', 'm', 's', 'g',
                             's', '/', 'D', 'e', 't', 'e', 'c', 't', 'i', 'o',
                             'n', '3', 'D', 'A', 'r', 'r', 'a', 'y'};
  vision_msgs_Detection3DStruct_T expl_temp;
  for (int i{0}; i < 28; i++) {
    msg->MessageType[i] = b_cv[i];
  }
  std_msgs_HeaderStruct(&msg->header);
  vision_msgs_Detection3DStruct(&expl_temp);
  msg->detections.set_size(1);
  for (int i{0}; i < 23; i++) {
    msg->detections[0].MessageType[i] = expl_temp.MessageType[i];
  }
  for (int i{0}; i < 15; i++) {
    msg->detections[0].header.MessageType[i] = expl_temp.header.MessageType[i];
  }
  for (int i{0}; i < 23; i++) {
    msg->detections[0].header.stamp.MessageType[i] =
        expl_temp.header.stamp.MessageType[i];
  }
  msg->detections[0].header.stamp.sec = expl_temp.header.stamp.sec;
  msg->detections[0].header.stamp.nanosec = expl_temp.header.stamp.nanosec;
  msg->detections[0].header.frame_id.set_size(
      1, expl_temp.header.frame_id.size(1));
  for (int i{0}; i < expl_temp.header.frame_id.size(1); i++) {
    msg->detections[0].header.frame_id[i] = expl_temp.header.frame_id[i];
  }
  msg->detections[0].results.set_size(expl_temp.results.size(0));
  for (int i{0}; i < expl_temp.results.size(0); i++) {
    msg->detections[0].results[i] = expl_temp.results[i];
  }
  for (int i{0}; i < 25; i++) {
    msg->detections[0].bbox.MessageType[i] = expl_temp.bbox.MessageType[i];
  }
  for (int i{0}; i < 18; i++) {
    msg->detections[0].bbox.center.MessageType[i] =
        expl_temp.bbox.center.MessageType[i];
  }
  for (int i{0}; i < 19; i++) {
    msg->detections[0].bbox.center.position.MessageType[i] =
        expl_temp.bbox.center.position.MessageType[i];
  }
  msg->detections[0].bbox.center.position.x = expl_temp.bbox.center.position.x;
  msg->detections[0].bbox.center.position.y = expl_temp.bbox.center.position.y;
  msg->detections[0].bbox.center.position.z = expl_temp.bbox.center.position.z;
  for (int i{0}; i < 24; i++) {
    msg->detections[0].bbox.center.orientation.MessageType[i] =
        expl_temp.bbox.center.orientation.MessageType[i];
  }
  msg->detections[0].bbox.center.orientation.x =
      expl_temp.bbox.center.orientation.x;
  msg->detections[0].bbox.center.orientation.y =
      expl_temp.bbox.center.orientation.y;
  msg->detections[0].bbox.center.orientation.z =
      expl_temp.bbox.center.orientation.z;
  msg->detections[0].bbox.center.orientation.w =
      expl_temp.bbox.center.orientation.w;
  for (int i{0}; i < 21; i++) {
    msg->detections[0].bbox.size.MessageType[i] =
        expl_temp.bbox.size.MessageType[i];
  }
  msg->detections[0].bbox.size.x = expl_temp.bbox.size.x;
  msg->detections[0].bbox.size.y = expl_temp.bbox.size.y;
  msg->detections[0].bbox.size.z = expl_temp.bbox.size.z;
  msg->detections[0].id.set_size(1, expl_temp.id.size(1));
  for (int i{0}; i < expl_temp.id.size(1); i++) {
    msg->detections[0].id[i] = expl_temp.id[i];
  }
  msg->detections.set_size(0);
  //(msg);
}

//
// File trailer for vision_msgs_Detection3DArrayStruct.cu
//
// [EOF]
//
