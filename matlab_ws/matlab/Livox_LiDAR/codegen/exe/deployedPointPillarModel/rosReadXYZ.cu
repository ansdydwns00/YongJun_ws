#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: rosReadXYZ.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "rosReadXYZ.h"
#include "PointCloud2Types.h"
#include "SpecialMsgUtil.h"
#include "deployedPointPillarModel_internal_types.h"
#include "deployedPointPillarModel_types.h"
#include "rt_nonfinite.h"
#include "coder_array.h"
#include <cmath>

// Type Definitions
struct cell_wrap_28 {
  coder::array<float, 1U> f1;
};

// Function Definitions
//
// Arguments    : unsigned int msg_height
//                unsigned int msg_width
//                const array<sensor_msgs_PointFieldStruct_T, 1U> &msg_fields
//                unsigned int msg_point_step
//                const array<unsigned char, 1U> &msg_data
//                array<float, 2U> &xyz
// Return Type  : void
//
namespace coder {
void rosReadXYZ(unsigned int msg_height, unsigned int msg_width,
                const array<sensor_msgs_PointFieldStruct_T, 1U> &msg_fields,
                unsigned int msg_point_step,
                const array<unsigned char, 1U> &msg_data, array<float, 2U> &xyz)
{
  array<cell_wrap_6, 2U> allFieldNames;
  array<cell_wrap_6, 2U> b_allFieldNames;
  array<cell_wrap_6, 2U> c_allFieldNames;
  array<double, 2U> b_byteIdx;
  array<double, 2U> b_y;
  array<double, 2U> byteIdx;
  array<double, 2U> c;
  array<double, 2U> c_byteIdx;
  array<unsigned int, 2U> pointIndices;
  array<int, 2U> r;
  array<unsigned int, 1U> a;
  array<char, 2U> a__4;
  array<bool, 2U> pointIdxIsValid;
  cell_wrap_28 reshapes[3];
  double numBytes;
  double zOff;
  unsigned long long u;
  int acoef;
  int bcoef;
  int end;
  unsigned int numPointsActual;
  int varargin_3;
  int xIdx;
  int yIdx;
  int zIdx;
  bool b_bool;
  if (msg_fields.size(0) == 0) {
    allFieldNames.set_size(0, 0);
  } else {
    bcoef = msg_fields.size(0);
    allFieldNames.set_size(1, msg_fields.size(0));
    for (acoef = 0; acoef < bcoef; acoef++) {
      allFieldNames[acoef].f1.set_size(1, msg_fields[acoef].name.size(1));
      for (varargin_3 = 0; varargin_3 < msg_fields[acoef].name.size(1);
           varargin_3++) {
        allFieldNames[acoef].f1[varargin_3] =
            msg_fields[acoef].name[varargin_3];
      }
    }
  }
  xIdx = -1;
  if ((allFieldNames.size(0) == 0) || (allFieldNames.size(1) == 0)) {
    bcoef = 0;
  } else {
    bcoef = allFieldNames.size(1);
  }
  for (acoef = 0; acoef < bcoef; acoef++) {
    b_bool = false;
    if ((allFieldNames[acoef].f1.size(1) == 1) &&
        (allFieldNames[acoef].f1[0] == 'x')) {
      b_bool = true;
    }
    if (b_bool) {
      xIdx = acoef;
    }
  }
  if (msg_fields.size(0) == 0) {
    b_allFieldNames.set_size(0, 0);
  } else {
    bcoef = msg_fields.size(0);
    b_allFieldNames.set_size(1, msg_fields.size(0));
    for (acoef = 0; acoef < bcoef; acoef++) {
      b_allFieldNames[acoef].f1.set_size(1, msg_fields[acoef].name.size(1));
      for (varargin_3 = 0; varargin_3 < msg_fields[acoef].name.size(1);
           varargin_3++) {
        b_allFieldNames[acoef].f1[varargin_3] =
            msg_fields[acoef].name[varargin_3];
      }
    }
  }
  yIdx = -1;
  if ((b_allFieldNames.size(0) == 0) || (b_allFieldNames.size(1) == 0)) {
    bcoef = 0;
  } else {
    bcoef = b_allFieldNames.size(1);
  }
  for (acoef = 0; acoef < bcoef; acoef++) {
    b_bool = false;
    if ((b_allFieldNames[acoef].f1.size(1) == 1) &&
        (b_allFieldNames[acoef].f1[0] == 'y')) {
      b_bool = true;
    }
    if (b_bool) {
      yIdx = acoef;
    }
  }
  if (msg_fields.size(0) == 0) {
    c_allFieldNames.set_size(0, 0);
  } else {
    bcoef = msg_fields.size(0);
    c_allFieldNames.set_size(1, msg_fields.size(0));
    for (acoef = 0; acoef < bcoef; acoef++) {
      c_allFieldNames[acoef].f1.set_size(1, msg_fields[acoef].name.size(1));
      for (varargin_3 = 0; varargin_3 < msg_fields[acoef].name.size(1);
           varargin_3++) {
        c_allFieldNames[acoef].f1[varargin_3] =
            msg_fields[acoef].name[varargin_3];
      }
    }
  }
  zIdx = -1;
  if ((c_allFieldNames.size(0) == 0) || (c_allFieldNames.size(1) == 0)) {
    bcoef = 0;
  } else {
    bcoef = c_allFieldNames.size(1);
  }
  for (acoef = 0; acoef < bcoef; acoef++) {
    b_bool = false;
    if ((c_allFieldNames[acoef].f1.size(1) == 1) &&
        (c_allFieldNames[acoef].f1[0] == 'z')) {
      b_bool = true;
    }
    if (b_bool) {
      zIdx = acoef;
    }
  }
  u = static_cast<unsigned long long>(msg_width) * msg_height;
  if (u > 4294967295ULL) {
    u = 4294967295ULL;
  }
  if (static_cast<unsigned int>(u) < 1U) {
    bcoef = 0;
  } else {
    bcoef = static_cast<int>(u);
  }
  pointIndices.set_size(1, bcoef);
  for (int k{0}; k < bcoef; k++) {
    pointIndices[k] = static_cast<unsigned int>(k + 1);
  }
  u = static_cast<unsigned long long>(msg_height) * msg_width;
  if (u > 4294967295ULL) {
    u = 4294967295ULL;
  }
  numBytes = std::trunc(static_cast<double>(msg_data.size(0)) /
                        static_cast<double>(msg_point_step));
  if (numBytes < 4.294967296E+9) {
    numPointsActual = static_cast<unsigned int>(numBytes);
  } else if (numBytes >= 4.294967296E+9) {
    numPointsActual = MAX_uint32_T;
  } else {
    numPointsActual = 0U;
  }
  if (std::isnan(numBytes) || (!(static_cast<unsigned int>(u) > numBytes))) {
    numPointsActual = static_cast<unsigned int>(u);
  }
  ros::msg::sensor_msgs::internal::PointCloud2Types_rosToMATLABType(
      msg_fields[xIdx].datatype, a__4, &numBytes);
  byteIdx.set_size(pointIndices.size(1), static_cast<int>(numBytes));
  for (varargin_3 = 0;
       varargin_3 < pointIndices.size(1) * static_cast<int>(numBytes);
       varargin_3++) {
    byteIdx[varargin_3] = 0.0;
  }
  pointIdxIsValid.set_size(1, pointIndices.size(1));
  for (varargin_3 = 0; varargin_3 < pointIndices.size(1); varargin_3++) {
    pointIdxIsValid[varargin_3] =
        ((static_cast<int>(pointIndices[varargin_3]) > 0) &&
         (pointIndices[varargin_3] <= numPointsActual));
  }
  if (numBytes < 1.0) {
    b_y.set_size(1, 0);
  } else {
    b_y.set_size(1, static_cast<int>(numBytes - 1.0) + 1);
    for (varargin_3 = 0; varargin_3 <= static_cast<int>(numBytes - 1.0);
         varargin_3++) {
      b_y[varargin_3] = static_cast<double>(varargin_3) + 1.0;
    }
  }
  end = pointIdxIsValid.size(1) - 1;
  bcoef = 0;
  for (acoef = 0; acoef <= end; acoef++) {
    if (pointIdxIsValid[acoef]) {
      bcoef++;
    }
  }
  a.set_size(bcoef);
  bcoef = 0;
  for (acoef = 0; acoef <= end; acoef++) {
    if (pointIdxIsValid[acoef]) {
      unsigned int qY;
      numPointsActual = msg_fields[xIdx].offset;
      u = static_cast<unsigned long long>(msg_point_step) *
          (pointIndices[acoef] - 1U);
      if (u > 4294967295ULL) {
        u = 4294967295ULL;
      }
      qY = numPointsActual + static_cast<unsigned int>(u);
      if (qY < numPointsActual) {
        qY = MAX_uint32_T;
      }
      a[bcoef] = qY;
      bcoef++;
    }
  }
  c.set_size(a.size(0), b_y.size(1));
  if ((a.size(0) != 0) && (b_y.size(1) != 0)) {
    bcoef = (b_y.size(1) != 1);
    end = b_y.size(1) - 1;
    acoef = (a.size(0) != 1);
    for (int k{0}; k <= end; k++) {
      int i7;
      varargin_3 = bcoef * k;
      i7 = c.size(0) - 1;
      for (int b_k{0}; b_k <= i7; b_k++) {
        c[b_k + c.size(0) * k] =
            static_cast<double>(a[acoef * b_k]) + b_y[varargin_3];
      }
    }
  }
  end = pointIdxIsValid.size(1) - 1;
  bcoef = 0;
  for (acoef = 0; acoef <= end; acoef++) {
    if (pointIdxIsValid[acoef]) {
      bcoef++;
    }
  }
  r.set_size(1, bcoef);
  bcoef = 0;
  for (acoef = 0; acoef <= end; acoef++) {
    if (pointIdxIsValid[acoef]) {
      r[bcoef] = acoef;
      bcoef++;
    }
  }
  bcoef = r.size(1);
  end = static_cast<int>(numBytes);
  for (varargin_3 = 0; varargin_3 < end; varargin_3++) {
    for (acoef = 0; acoef < bcoef; acoef++) {
      byteIdx[r[acoef] + byteIdx.size(0) * varargin_3] =
          c[acoef + bcoef * varargin_3];
    }
  }
  numBytes = static_cast<double>(msg_fields[yIdx].offset) -
             static_cast<double>(msg_fields[xIdx].offset);
  zOff = static_cast<double>(msg_fields[zIdx].offset) -
         static_cast<double>(msg_fields[xIdx].offset);
  ros::internal::ros2::SpecialMsgUtil_readFieldFromData(
      msg_data, byteIdx, pointIdxIsValid, reshapes[0].f1);
  b_byteIdx.set_size(byteIdx.size(0), byteIdx.size(1));
  for (varargin_3 = 0; varargin_3 < byteIdx.size(0) * byteIdx.size(1);
       varargin_3++) {
    b_byteIdx[varargin_3] = byteIdx[varargin_3] + numBytes;
  }
  ros::internal::ros2::SpecialMsgUtil_readFieldFromData(
      msg_data, b_byteIdx, pointIdxIsValid, reshapes[1].f1);
  c_byteIdx.set_size(byteIdx.size(0), byteIdx.size(1));
  for (varargin_3 = 0; varargin_3 < byteIdx.size(0) * byteIdx.size(1);
       varargin_3++) {
    c_byteIdx[varargin_3] = byteIdx[varargin_3] + zOff;
  }
  ros::internal::ros2::SpecialMsgUtil_readFieldFromData(
      msg_data, c_byteIdx, pointIdxIsValid, reshapes[2].f1);
  xyz.set_size(reshapes[0].f1.size(0), 3);
  bcoef = reshapes[0].f1.size(0);
  for (varargin_3 = 0; varargin_3 < bcoef; varargin_3++) {
    xyz[varargin_3] = reshapes[0].f1[varargin_3];
  }
  bcoef = reshapes[1].f1.size(0);
  for (varargin_3 = 0; varargin_3 < bcoef; varargin_3++) {
    xyz[varargin_3 + xyz.size(0)] = reshapes[1].f1[varargin_3];
  }
  bcoef = reshapes[2].f1.size(0);
  for (varargin_3 = 0; varargin_3 < bcoef; varargin_3++) {
    xyz[varargin_3 + xyz.size(0) * 2] = reshapes[2].f1[varargin_3];
  }
}

} // namespace coder

//
// File trailer for rosReadXYZ.cu
//
// [EOF]
//
