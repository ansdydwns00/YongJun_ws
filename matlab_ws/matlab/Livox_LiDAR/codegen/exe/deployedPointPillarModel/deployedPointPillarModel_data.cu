//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: deployedPointPillarModel_data.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "deployedPointPillarModel_data.h"
#include "rt_nonfinite.h"

// Variable Definitions
unsigned char pauseState;

const bool bv[128]{
    false, false, false, false, false, false, false, false, false, true,  true,
    true,  true,  true,  false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, true,  true,  true,  true,  true,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false};

double (*global_gpu_dv)[248];

double (*global_gpu_xq)[216];

double (*b_global_gpu_dv)[248];

double (*b_global_gpu_xq)[216];

bool isInitialized_deployedPointPillarModel{false};

//
// File trailer for deployedPointPillarModel_data.cu
//
// [EOF]
//
