#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: arrayfun.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "arrayfun.h"
#include "deployedPointPillarModel_internal_types.h"
#include "rt_nonfinite.h"
#include "coder_array.h"
#include <cmath>

// Function Definitions
//
// Arguments    : const float fun_workspace_predLoc[321408]
//                const array<cell_wrap_60, 2U> &fun_workspace_newAnchors
//                const array<double, 1U> &varargin_1
//                const array<double, 1U> &varargin_2
//                const array<double, 1U> &varargin_3
//                const array<double, 1U> &varargin_4
//                array<float, 1U> &varargout_1
// Return Type  : void
//
namespace coder {
void applyArrayFunction(const float fun_workspace_predLoc[321408],
                        const array<cell_wrap_60, 2U> &fun_workspace_newAnchors,
                        const array<double, 1U> &varargin_1,
                        const array<double, 1U> &varargin_2,
                        const array<double, 1U> &varargin_3,
                        const array<double, 1U> &varargin_4,
                        array<float, 1U> &varargout_1)
{
  int n;
  varargout_1.set_size(varargin_1.size(0));
  n = varargin_1.size(0);
  for (int k{0}; k < n; k++) {
    varargout_1[k] =
        fun_workspace_predLoc[((static_cast<int>(varargin_1[k]) +
                                216 * (static_cast<int>(varargin_2[k]) - 1)) +
                               53568 * (static_cast<int>(varargin_3[k]) - 1)) -
                              1] *
            static_cast<float>(std::sqrt(
                fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) - 1]
                        .f1[0] *
                    fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) -
                                             1]
                        .f1[0] +
                fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) - 1]
                        .f1[1] *
                    fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) -
                                             1]
                        .f1[1])) +
        static_cast<float>(varargin_4[k]);
  }
}

//
// Arguments    : const float fun_workspace_predLoc[321408]
//                const array<cell_wrap_60, 2U> &fun_workspace_newAnchors
//                const array<double, 1U> &varargin_1
//                const array<double, 1U> &varargin_2
//                const array<double, 1U> &varargin_3
//                array<float, 1U> &varargout_1
// Return Type  : void
//
void arrayfun(const float fun_workspace_predLoc[321408],
              const array<cell_wrap_60, 2U> &fun_workspace_newAnchors,
              const array<double, 1U> &varargin_1,
              const array<double, 1U> &varargin_2,
              const array<double, 1U> &varargin_3,
              array<float, 1U> &varargout_1)
{
  int n;
  varargout_1.set_size(varargin_1.size(0));
  n = varargin_1.size(0);
  for (int k{0}; k < n; k++) {
    varargout_1[k] =
        fun_workspace_predLoc[((static_cast<int>(varargin_1[k]) +
                                216 * (static_cast<int>(varargin_2[k]) - 1)) +
                               53568 * (static_cast<int>(varargin_3[k]) - 1)) +
                              214271] *
            static_cast<float>(
                fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) - 1]
                    .f1[2]) +
        static_cast<float>(
            fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) - 1]
                .f1[3]);
  }
}

//
// Arguments    : const float fun_workspace_predAngle[107136]
//                const array<cell_wrap_60, 2U> &fun_workspace_newAnchors
//                const array<double, 1U> &varargin_1
//                const array<double, 1U> &varargin_2
//                const array<double, 1U> &varargin_3
//                const array<float, 1U> &varargin_4
//                array<float, 1U> &varargout_1
// Return Type  : void
//
void arrayfun(const float fun_workspace_predAngle[107136],
              const array<cell_wrap_60, 2U> &fun_workspace_newAnchors,
              const array<double, 1U> &varargin_1,
              const array<double, 1U> &varargin_2,
              const array<double, 1U> &varargin_3,
              const array<float, 1U> &varargin_4, array<float, 1U> &varargout_1)
{
  int n;
  varargout_1.set_size(varargin_1.size(0));
  n = varargin_1.size(0);
  for (int k{0}; k < n; k++) {
    varargout_1[k] =
        varargin_4[k] *
            std::asin(fun_workspace_predAngle
                          [((static_cast<int>(varargin_1[k]) +
                             216 * (static_cast<int>(varargin_2[k]) - 1)) +
                            53568 * (static_cast<int>(varargin_3[k]) - 1)) -
                           1]) +
        static_cast<float>(
            fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) - 1]
                .f1[4]);
  }
}

//
// Arguments    : const float fun_workspace_predLoc[321408]
//                const array<cell_wrap_60, 2U> &fun_workspace_newAnchors
//                const array<double, 1U> &varargin_1
//                const array<double, 1U> &varargin_2
//                const array<double, 1U> &varargin_3
//                const array<double, 1U> &varargin_4
//                array<float, 1U> &varargout_1
// Return Type  : void
//
void b_applyArrayFunction(
    const float fun_workspace_predLoc[321408],
    const array<cell_wrap_60, 2U> &fun_workspace_newAnchors,
    const array<double, 1U> &varargin_1, const array<double, 1U> &varargin_2,
    const array<double, 1U> &varargin_3, const array<double, 1U> &varargin_4,
    array<float, 1U> &varargout_1)
{
  int n;
  varargout_1.set_size(varargin_1.size(0));
  n = varargin_1.size(0);
  for (int k{0}; k < n; k++) {
    varargout_1[k] =
        fun_workspace_predLoc[((static_cast<int>(varargin_1[k]) +
                                216 * (static_cast<int>(varargin_2[k]) - 1)) +
                               53568 * (static_cast<int>(varargin_3[k]) - 1)) +
                              107135] *
            static_cast<float>(std::sqrt(
                fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) - 1]
                        .f1[0] *
                    fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) -
                                             1]
                        .f1[0] +
                fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) - 1]
                        .f1[1] *
                    fun_workspace_newAnchors[static_cast<int>(varargin_3[k]) -
                                             1]
                        .f1[1])) +
        static_cast<float>(varargin_4[k]);
  }
}

} // namespace coder

//
// File trailer for arrayfun.cu
//
// [EOF]
//
