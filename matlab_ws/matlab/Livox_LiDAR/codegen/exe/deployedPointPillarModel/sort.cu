#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: sort.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "sort.h"
#include "deployedPointPillarModel_rtwutil.h"
#include "rt_nonfinite.h"
#include "sortIdx.h"
#include "coder_array.h"
#include <cmath>

// Function Definitions
//
// Arguments    : array<float, 2U> &b_x
//                array<int, 2U> &idx
// Return Type  : void
//
namespace coder {
namespace internal {
void sort(array<float, 2U> &b_x, array<int, 2U> &idx)
{
  array<float, 1U> xwork;
  array<int, 1U> iwork;
  float b_xwork[256];
  float x4[4];
  int b_iwork[256];
  int idx4[4];
  int nPairs;
  signed char perm[4];
  idx.set_size(1, b_x.size(1));
  for (nPairs = 0; nPairs < b_x.size(1); nPairs++) {
    idx[nPairs] = 0;
  }
  if (b_x.size(1) != 0) {
    int bLen2;
    int i1;
    int i2;
    int i3;
    int i4;
    int ib;
    int n;
    int nNaNs;
    int nNonNaN;
    n = b_x.size(1) - 1;
    x4[0] = 0.0F;
    idx4[0] = 0;
    x4[1] = 0.0F;
    idx4[1] = 0;
    x4[2] = 0.0F;
    idx4[2] = 0;
    x4[3] = 0.0F;
    idx4[3] = 0;
    iwork.set_size(b_x.size(1));
    xwork.set_size(b_x.size(1));
    for (nPairs = 0; nPairs < b_x.size(1); nPairs++) {
      iwork[nPairs] = 0;
      xwork[nPairs] = 0.0F;
    }
    nNaNs = 0;
    ib = 0;
    for (int k{0}; k <= n; k++) {
      if (std::isnan(b_x[k])) {
        idx[n - nNaNs] = k + 1;
        xwork[n - nNaNs] = b_x[k];
        nNaNs++;
      } else {
        ib++;
        idx4[ib - 1] = k + 1;
        x4[ib - 1] = b_x[k];
        if (ib == 4) {
          ib = k - nNaNs;
          if (x4[0] <= x4[1]) {
            i1 = 1;
            i2 = 2;
          } else {
            i1 = 2;
            i2 = 1;
          }
          if (x4[2] <= x4[3]) {
            i3 = 3;
            i4 = 4;
          } else {
            i3 = 4;
            i4 = 3;
          }
          if (x4[i1 - 1] <= x4[i3 - 1]) {
            if (x4[i2 - 1] <= x4[i3 - 1]) {
              nPairs = i1;
              bLen2 = i2;
              i1 = i3;
              i2 = i4;
            } else if (x4[i2 - 1] <= x4[i4 - 1]) {
              nPairs = i1;
              bLen2 = i3;
              i1 = i2;
              i2 = i4;
            } else {
              nPairs = i1;
              bLen2 = i3;
              i1 = i4;
            }
          } else if (x4[i1 - 1] <= x4[i4 - 1]) {
            if (x4[i2 - 1] <= x4[i4 - 1]) {
              nPairs = i3;
              bLen2 = i1;
              i1 = i2;
              i2 = i4;
            } else {
              nPairs = i3;
              bLen2 = i1;
              i1 = i4;
            }
          } else {
            nPairs = i3;
            bLen2 = i4;
          }
          idx[ib - 3] = idx4[nPairs - 1];
          idx[ib - 2] = idx4[bLen2 - 1];
          idx[ib - 1] = idx4[i1 - 1];
          idx[ib] = idx4[i2 - 1];
          b_x[ib - 3] = x4[nPairs - 1];
          b_x[ib - 2] = x4[bLen2 - 1];
          b_x[ib - 1] = x4[i1 - 1];
          b_x[ib] = x4[i2 - 1];
          ib = 0;
        }
      }
    }
    i2 = b_x.size(1) - nNaNs;
    if (ib > 0) {
      perm[1] = 0;
      perm[2] = 0;
      perm[3] = 0;
      if (ib == 1) {
        perm[0] = 1;
      } else if (ib == 2) {
        if (x4[0] <= x4[1]) {
          perm[0] = 1;
          perm[1] = 2;
        } else {
          perm[0] = 2;
          perm[1] = 1;
        }
      } else if (x4[0] <= x4[1]) {
        if (x4[1] <= x4[2]) {
          perm[0] = 1;
          perm[1] = 2;
          perm[2] = 3;
        } else if (x4[0] <= x4[2]) {
          perm[0] = 1;
          perm[1] = 3;
          perm[2] = 2;
        } else {
          perm[0] = 3;
          perm[1] = 1;
          perm[2] = 2;
        }
      } else if (x4[0] <= x4[2]) {
        perm[0] = 2;
        perm[1] = 1;
        perm[2] = 3;
      } else if (x4[1] <= x4[2]) {
        perm[0] = 2;
        perm[1] = 3;
        perm[2] = 1;
      } else {
        perm[0] = 3;
        perm[1] = 2;
        perm[2] = 1;
      }
      for (int k{0}; k < ib; k++) {
        nPairs = perm[k];
        idx[(i2 - ib) + k] = idx4[nPairs - 1];
        b_x[(i2 - ib) + k] = x4[nPairs - 1];
      }
    }
    i1 = nNaNs >> 1;
    for (int k{0}; k < i1; k++) {
      ib = idx[i2 + k];
      idx[i2 + k] = idx[n - k];
      idx[n - k] = ib;
      b_x[i2 + k] = xwork[n - k];
      b_x[n - k] = xwork[i2 + k];
    }
    if ((nNaNs & 1) != 0) {
      b_x[i2 + i1] = xwork[i2 + i1];
    }
    nNonNaN = b_x.size(1) - nNaNs;
    ib = 2;
    if (nNonNaN > 1) {
      if (b_x.size(1) >= 256) {
        n = nNonNaN >> 8;
        if (n > 0) {
          for (nNaNs = 0; nNaNs < n; nNaNs++) {
            i3 = nNaNs << 8;
            for (int b{0}; b < 6; b++) {
              i4 = 1 << (b + 2);
              bLen2 = i4 << 1;
              nPairs = 256 >> (b + 3);
              for (int k{0}; k < nPairs; k++) {
                ib = i3 + k * bLen2;
                for (i1 = 0; i1 < bLen2; i1++) {
                  b_iwork[i1] = idx[ib + i1];
                  b_xwork[i1] = b_x[ib + i1];
                }
                i2 = 1;
                i1 = i4;
                ib--;
                int exitg1;
                do {
                  exitg1 = 0;
                  ib++;
                  if (b_xwork[i2 - 1] <= b_xwork[i1]) {
                    idx[ib] = b_iwork[i2 - 1];
                    b_x[ib] = b_xwork[i2 - 1];
                    if (i2 < i4) {
                      i2++;
                    } else {
                      exitg1 = 1;
                    }
                  } else {
                    idx[ib] = b_iwork[i1];
                    b_x[ib] = b_xwork[i1];
                    if (i1 + 1 < bLen2) {
                      i1++;
                    } else {
                      long long c;
                      ib = (ib - i2) + 1;
                      c = computeEndIdx(static_cast<long long>(i2),
                                        static_cast<long long>(i4), 1LL);
                      for (long long j{0LL}; j <= c; j++) {
                        idx[ib + static_cast<int>(i2 + j)] =
                            b_iwork[static_cast<int>(i2 + j) - 1];
                        b_x[ib + static_cast<int>(i2 + j)] =
                            b_xwork[static_cast<int>(i2 + j) - 1];
                      }
                      exitg1 = 1;
                    }
                  }
                } while (exitg1 == 0);
              }
            }
          }
          ib = n << 8;
          i1 = nNonNaN - ib;
          if (i1 > 0) {
            merge_block(idx, b_x, ib, i1, 2, iwork, xwork);
          }
          ib = 8;
        }
      }
      merge_block(idx, b_x, 0, nNonNaN, ib, iwork, xwork);
    }
  }
}

} // namespace internal
} // namespace coder

//
// File trailer for sort.cu
//
// [EOF]
//
