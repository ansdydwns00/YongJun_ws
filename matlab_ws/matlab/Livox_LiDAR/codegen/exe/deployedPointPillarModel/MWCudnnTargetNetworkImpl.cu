#ifdef PRECOMPILE_LAYERFILES
#include "layer/MWTargetNetworkImplBase.hpp"
#include "layer/MWTargetTypes.hpp"
#else
#include "MWTargetNetworkImplBase.hpp"
#include "MWTargetTypes.hpp"
#endif
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#ifdef PRECOMPILE_LAYERFILES
#include "MWCudnnLayerImplFactory_precompile.hpp"
#else
#include "MWCudnnLayerImplFactory.hpp"
#endif
#include <cassert>
#include <cmath>
#include <algorithm>
 namespace MWCudnnTarget { MWTargetNetworkImpl::MWTargetNetworkImpl() : 
MWTargetNetworkImplBase(MWTargetType::CUDNN_TARGET, new 
MWCudnnLayerImplFactory) , xcusoQxPPodcHwVviCWI(0) , 
NXruhrCCiguRjAgSNDuz(NULL) , NbunkIVaMPVYgAQHXXYd(NULL) , 
MW_autoTune(true) , lkGLRakytrdNuJCcpYWt(0) , 
GnxRkpzrPZimKtYYHSuG(0) { } void MWTargetNetworkImpl::allocate(int 
BufSize, int numBufsToAlloc) { numBufs = numBufsToAlloc; 
memBuffer.reserve(numBufs); for (int i = 0; i < numBufs; i++) { float* memPtr = 
NULL; CUDA_CALL(hipMalloc((void**)&memPtr, sizeof(float) * BufSize)); 
memBuffer.push_back(memPtr); } } void 
MWTargetNetworkImpl::allocatePermuteBuffers(int bufSize, int numBufsToAlloc) { 
for (int i = 0; i < numBufsToAlloc; i++) { float* memPtr = NULL; 
CUDA_CALL(hipMalloc((void**)&memPtr, sizeof(float) * bufSize)); 
kqftrrQBBOgGsrDSkIUk.push_back(memPtr); } } void 
MWTargetNetworkImpl::preSetup() { NXruhrCCiguRjAgSNDuz = new 
hipblasHandle_t(); if (!NXruhrCCiguRjAgSNDuz) { 
MWCNNLayerImpl::throwAllocationError(__LINE__, __FILE__); } 
CUBLAS_CALL(hipblasCreate(NXruhrCCiguRjAgSNDuz)); NbunkIVaMPVYgAQHXXYd 
= new hipdnnHandle_t(); if (!NbunkIVaMPVYgAQHXXYd) { 
MWCNNLayerImpl::throwAllocationError(__LINE__, __FILE__); } 
CUDNN_CALL(hipdnnCreate(NbunkIVaMPVYgAQHXXYd)); } void 
MWTargetNetworkImpl::postSetup(MWCNNLayer* layers[], int numLayers) { if 
(*getProposedWorkSpaceSize() > *getAllocatedWorkSpaceSize()) { if 
(xcusoQxPPodcHwVviCWI) { destroyWorkSpace(xcusoQxPPodcHwVviCWI); } 
createWorkSpace(xcusoQxPPodcHwVviCWI); while ((!xcusoQxPPodcHwVviCWI) && 
(*getProposedWorkSpaceSize() > 0)) { setProposedWorkSpaceSize( 
MWTargetNetworkImpl::getNextProposedWorkSpaceSize(*getProposedWorkSpaceSize())); 
createWorkSpace(xcusoQxPPodcHwVviCWI); } } for (int i = 0; i < numLayers; i++) 
{ layers[i]->postSetup(); } } size_t 
MWTargetNetworkImpl::getNextProposedWorkSpaceSize(size_t failedWorkSpaceSize) { 
assert(failedWorkSpaceSize > 0); return failedWorkSpaceSize / 2; } void 
MWTargetNetworkImpl::createWorkSpace(float*& xkUNToJIgvoLoUQuzKRF) { 
hipError_t rMMjgjGRAiLVlTlRSByU = hipMalloc((void**)&xkUNToJIgvoLoUQuzKRF, 
*getProposedWorkSpaceSize()); if (rMMjgjGRAiLVlTlRSByU != hipSuccess) { 
xkUNToJIgvoLoUQuzKRF = NULL; setAllocatedWorkSpaceSize(0); rMMjgjGRAiLVlTlRSByU 
= hipGetLastError();  } else { 
setAllocatedWorkSpaceSize(*getProposedWorkSpaceSize()); } } void 
MWTargetNetworkImpl::destroyWorkSpace(float*& xkUNToJIgvoLoUQuzKRF) { 
CUDA_FREE_CALL(xkUNToJIgvoLoUQuzKRF); xkUNToJIgvoLoUQuzKRF = NULL; 
setAllocatedWorkSpaceSize(0); } void 
MWTargetNetworkImpl::setProposedWorkSpaceSize(size_t wss) { 
lkGLRakytrdNuJCcpYWt = wss; } size_t* 
MWTargetNetworkImpl::getProposedWorkSpaceSize() { return 
&lkGLRakytrdNuJCcpYWt; } void 
MWTargetNetworkImpl::setAllocatedWorkSpaceSize(size_t wss) { 
GnxRkpzrPZimKtYYHSuG = wss; } size_t* 
MWTargetNetworkImpl::getAllocatedWorkSpaceSize() { return 
&GnxRkpzrPZimKtYYHSuG; } float* 
MWTargetNetworkImpl::getWorkSpace() { return xcusoQxPPodcHwVviCWI; } float* 
MWTargetNetworkImpl::getPermuteBuffer(int bufIndex) { return 
kqftrrQBBOgGsrDSkIUk[bufIndex]; } hipblasHandle_t* 
MWTargetNetworkImpl::getCublasHandle() { return NXruhrCCiguRjAgSNDuz; } 
hipdnnHandle_t* MWTargetNetworkImpl::getCudnnHandle() { return 
NbunkIVaMPVYgAQHXXYd; } void MWTargetNetworkImpl::setAutoTune(bool 
autotune) { MW_autoTune = autotune; } bool MWTargetNetworkImpl::getAutoTune() 
const { return MW_autoTune; } void MWTargetNetworkImpl::deallocate() { for (int 
i = 0; i < static_cast<int>(memBuffer.size()); i++) { float* memPtr = 
memBuffer[i]; if (memPtr) { CUDA_FREE_CALL(memPtr); } } memBuffer.clear(); for 
(int i = 0; i < static_cast<int>(kqftrrQBBOgGsrDSkIUk.size()); i++) { 
float* memPtr = kqftrrQBBOgGsrDSkIUk[i]; if (memPtr) { 
CUDA_FREE_CALL(memPtr); } } kqftrrQBBOgGsrDSkIUk.clear(); } void 
MWTargetNetworkImpl::cleanup() { if (xcusoQxPPodcHwVviCWI) { 
destroyWorkSpace(xcusoQxPPodcHwVviCWI); } if (NXruhrCCiguRjAgSNDuz) { if 
(*NXruhrCCiguRjAgSNDuz) { hipError_t hipError_t = hipPeekAtLastError(); 
if (hipError_t != hipErrorDeinitialized) { 
CUBLAS_CALL(hipblasDestroy(*NXruhrCCiguRjAgSNDuz)); } 
*NXruhrCCiguRjAgSNDuz = NULL; } delete NXruhrCCiguRjAgSNDuz; 
NXruhrCCiguRjAgSNDuz = NULL; } if (NbunkIVaMPVYgAQHXXYd) { if 
(*NbunkIVaMPVYgAQHXXYd) {
#if (CUDNN_MAJOR < 8)
 CUDNN_CALL(hipdnnDestroy(*NbunkIVaMPVYgAQHXXYd));
#else
 hipError_t hipError_t = hipPeekAtLastError(); if (hipError_t != 
hipErrorDeinitialized) { 
CUDNN_CALL(hipdnnDestroy(*NbunkIVaMPVYgAQHXXYd)); }
#endif
 *NbunkIVaMPVYgAQHXXYd = NULL; } delete NbunkIVaMPVYgAQHXXYd; 
NbunkIVaMPVYgAQHXXYd = NULL; } } float* 
MWTargetNetworkImpl::getBufferPtr(int bufferIndex) { 
assert(static_cast<size_t>(bufferIndex) < memBuffer.size()); return 
memBuffer[bufferIndex]; } } 