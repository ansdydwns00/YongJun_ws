//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: geometry_msgs_QuaternionStruct.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "geometry_msgs_QuaternionStruct.h"
#include "deployedPointPillarModel_types.h"
#include "rt_nonfinite.h"

// Function Definitions
//
// Message struct definition for geometry_msgs/Quaternion
//
// Arguments    : void
// Return Type  : geometry_msgs_QuaternionStruct_T
//
geometry_msgs_QuaternionStruct_T geometry_msgs_QuaternionStruct()
{
  static const geometry_msgs_QuaternionStruct_T b_msg{
      {'g', 'e', 'o', 'm', 'e', 't', 'r', 'y', '_', 'm', 's', 'g', 's',
       '/', 'Q', 'u', 'a', 't', 'e', 'r', 'n', 'i', 'o', 'n'}, // MessageType
      0.0,                                                     // x
      0.0,                                                     // y
      0.0,                                                     // z
      0.0                                                      // w
  };
  geometry_msgs_QuaternionStruct_T msg;
  msg = b_msg;
  //(&b_msg);
  return msg;
}

//
// File trailer for geometry_msgs_QuaternionStruct.cu
//
// [EOF]
//
