//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: ros2publisher.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "ros2publisher.h"
#include "deployedPointPillarModel_internal_types.h"
#include "deployedPointPillarModel_types.h"
#include "rt_nonfinite.h"
#include "setQOSProfile.h"
#include "vision_msgs_Detection3DArrayStruct.h"
#include "mlros2_node.h"
#include "mlros2_pub.h"
#include "rmw/qos_profiles.h"

// Function Definitions
//
// Arguments    : ros2publisher *obj
//                const ros2node *node
// Return Type  : ros2publisher *
//
namespace coder {
ros2publisher *ros2publisher_ros2publisher(ros2publisher *obj,
                                           const ros2node *node)
{
  static const char resolvedTopic[17]{'/', 'o', 'b', 'j', 'D', 'e',
                                      't', 'e', 'c', 't', 'i', 'o',
                                      'n', 'I', 'n', 'f', 'o'};
  static const char b_cv[8]{'k', 'e', 'e', 'p', 'l', 'a', 's', 't'};
  static const char cv1[8]{'r', 'e', 'l', 'i', 'a', 'b', 'l', 'e'};
  static const char cv2[8]{'v', 'o', 'l', 'a', 't', 'i', 'l', 'e'};
  ros2publisher *b_obj;
  vision_msgs_Detection3DArrayStruct_T r;
  b_obj = obj;
  for (int i{0}; i < 17; i++) {
    b_obj->TopicName[i] = resolvedTopic[i];
  }
  b_obj->Depth = 1.0;
  for (int i{0}; i < 8; i++) {
    b_obj->History[i] = b_cv[i];
    b_obj->Reliability[i] = cv1[i];
    b_obj->Durability[i] = cv2[i];
  }
  rmw_qos_profile_t qos_profile;
  qos_profile = ros::ros2::internal::setQOSProfile(
      rmw_qos_profile_default, b_obj->Depth, b_obj->Reliability);
  vision_msgs_Detection3DArrayStruct(&r);
  b_obj->PublisherHelper = std::unique_ptr<
      MATLABROS2Publisher<vision_msgs::msg::Detection3DArray,
                          vision_msgs_Detection3DArrayStruct_T>>(
      new MATLABROS2Publisher<vision_msgs::msg::Detection3DArray,
                              vision_msgs_Detection3DArrayStruct_T>{}); //();
  MATLABROS2Publisher_createPublisher(b_obj->PublisherHelper, node->NodeHandle,
                                      &b_obj->TopicName[0], 17.0, qos_profile);
  return b_obj;
}

} // namespace coder

//
// File trailer for ros2publisher.cu
//
// [EOF]
//
