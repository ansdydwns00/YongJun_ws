#include "hip/hip_runtime.h"
#ifdef PRECOMPILE_LAYERFILES
#include "layer/MWCNNLayer.hpp"
#include "layer/MWTensorBase.hpp"
#include "layer/MWTensor.hpp"
#else
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#endif
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCudnnConcatenationLayerImpl.hpp"
#include <cstdarg>
#include <cassert>
 namespace MWCudnnTarget { class MWTargetNetworkImpl; 
MWConcatenationLayerImpl::MWConcatenationLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int dimension) : MWCNNLayerImpl(layer, 
ntwk_impl) , OPnUFarBGxQhYRrGpeBQ(dimension) { } 
MWConcatenationLayerImpl::~MWConcatenationLayerImpl() { } void 
MWConcatenationLayerImpl::propagateSize() { } void __global__ 
__launch_bounds__(1024) concatHeightImpl(float* in, float* out, double W,  
double WH,  double WHC,  double WHCN,  double WHCNS,  double W_totalH,  double 
W_totalH_C,  double W_totalH_CN,  double accumulatedH) { long int i = 
blockDim.x * blockIdx.x + threadIdx.x; if (i < WHCNS) { double sequenceIdx = 
floor(i / WHCN); double i_sequence = i - (sequenceIdx * WHCN); double batchIdx 
= floor(i_sequence / WHC); double i_batch = i_sequence - (batchIdx * WHC); 
double channelIdx = floor(i_batch / WH); double i_channel = i_batch - 
(channelIdx * WH); double heightIdx = floor(i_channel / W); double widthIdx = 
i_channel - (heightIdx * W); long int outIdx = sequenceIdx * W_totalH_CN + 
batchIdx * W_totalH_C + channelIdx * W_totalH + (heightIdx + accumulatedH) * W 
+ widthIdx; out[outIdx] = in[i]; } } void __global__ __launch_bounds__(1024) 
concatWidthImpl(float* in, float* out, double W,  double WH,  double WHC,  
double WHCN,  double WHCNS,  double totalW,  double totalW_H,  double 
totalW_HC,  double totalW_HCN,  double accumulatedW) { long int i = blockDim.x 
* blockIdx.x + threadIdx.x; if (i < WHCNS) { double sequenceIdx = floor(i / 
WHCN); double i_sequence = i - (sequenceIdx * WHCN); double batchIdx = 
floor(i_sequence / WHC); double i_batch = i_sequence - (batchIdx * WHC); double 
channelIdx = floor(i_batch / WH); double i_channel = i_batch - (channelIdx * 
WH); double heightIdx = floor(i_channel / W); double widthIdx = i_channel - 
(heightIdx * W); long int outIdx = sequenceIdx * totalW_HCN + batchIdx * 
totalW_HC + channelIdx * totalW_H + heightIdx * totalW + (widthIdx + 
accumulatedW); out[outIdx] = in[i]; } } void __global__ __launch_bounds__(1024) 
concatChannelImpl(float* in, float* out, double WH,  double WHC,  double WHCN,  
double WHCNS,  double WH_totalC,  double WH_totalC_N,  double accumulatedC) { 
long int i = blockDim.x * blockIdx.x + threadIdx.x; if (i < WHCNS) { double 
sequenceIdx = floor(i / WHCN); double i_sequence = i - (sequenceIdx * WHCN); 
double batchIdx = floor(i_sequence / WHC); double i_batch = i_sequence - 
(batchIdx * WHC); double channelIdx = floor(i_batch / WH); double i_channel = 
i_batch - (channelIdx * WH); long int outIdx = sequenceIdx * WH_totalC_N + 
batchIdx * WH_totalC + (channelIdx + accumulatedC) * WH + i_channel; 
out[outIdx] = in[i]; } } void MWConcatenationLayerImpl::predict() { MWCNNLayer* 
concatLayer = getLayer(); MWTensorBase* opTensorBase = 
concatLayer->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int accumulatedH = 0; int 
accumulatedW = 0; int accumulatedC = 0; long int accumulatedElements = 0; int 
KCudOrFMfgCzUPMcdePX = 0; int shEncNmxJsMuJKwbrwok = 0; for (int k = 
0; k < static_cast<int>(concatLayer->getNumInputs()); k++) { MWTensorBase* 
ipTensorBase = concatLayer->getInputTensor(k); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); switch (OPnUFarBGxQhYRrGpeBQ) { 
case 1:  { if ((ipTensor->getBatchSize() == 1 && ipTensor->getSequenceLength() 
== 1) && ipTensor->getChannels() == 1) { hipMemcpy(opTensor->getData() + 
accumulatedElements, ipTensor->getData(), ipTensor->getNumElements() * 
sizeof(float), hipMemcpyDeviceToDevice); accumulatedElements += (long 
int)ipTensor->getNumElements(); } else { double W = ipTensor->getWidth(); 
double WH = W * ipTensor->getHeight(); double WHC = WH * 
ipTensor->getChannels(); double WHCN = WHC * ipTensor->getBatchSize(); double 
WHCNS = WHCN * ipTensor->getSequenceLength(); double W_totalH = W * 
opTensor->getHeight(); double W_totalH_C = W_totalH * ipTensor->getChannels(); 
double W_totalH_CN = W_totalH_C * ipTensor->getBatchSize(); 
prepareKernelInputs(KCudOrFMfgCzUPMcdePX, shEncNmxJsMuJKwbrwok, 
ipTensor->getNumElements()); concatHeightImpl<<<KCudOrFMfgCzUPMcdePX, 
shEncNmxJsMuJKwbrwok>>>( ipTensor->getData(), opTensor->getData(), W, WH, 
WHC, WHCN, WHCNS, W_totalH, W_totalH_C, W_totalH_CN, accumulatedH); 
accumulatedH += ipTensor->getHeight(); } } break; case 2:  { if 
(((ipTensor->getBatchSize() == 1 && ipTensor->getSequenceLength() == 1) && 
ipTensor->getChannels() == 1) && ipTensor->getHeight() == 1) { 
hipMemcpy(opTensor->getData() + accumulatedElements, ipTensor->getData(), 
ipTensor->getNumElements() * sizeof(float), hipMemcpyDeviceToDevice); 
accumulatedElements += (long int)ipTensor->getNumElements(); } else { double W 
= ipTensor->getWidth(); double WH = W * ipTensor->getHeight(); double WHC = WH 
* ipTensor->getChannels(); double WHCN = WHC * ipTensor->getBatchSize(); double 
WHCNS = WHCN * ipTensor->getSequenceLength(); double totalW = 
opTensor->getWidth(); double totalW_H = totalW * ipTensor->getHeight(); double 
totalW_HC = totalW_H * ipTensor->getChannels(); double totalW_HCN = totalW_HC * 
ipTensor->getBatchSize(); prepareKernelInputs(KCudOrFMfgCzUPMcdePX, 
shEncNmxJsMuJKwbrwok, ipTensor->getNumElements()); 
concatWidthImpl<<<KCudOrFMfgCzUPMcdePX, shEncNmxJsMuJKwbrwok>>>( 
ipTensor->getData(), opTensor->getData(), W, WH, WHC, WHCN, WHCNS, totalW, 
totalW_H, totalW_HC, totalW_HCN, accumulatedW); accumulatedW += 
ipTensor->getWidth(); } } break; case 3:  { if (ipTensor->getBatchSize() == 1 
&& ipTensor->getSequenceLength() == 1) { hipMemcpy(opTensor->getData() + 
accumulatedElements, ipTensor->getData(), ipTensor->getNumElements() * 
sizeof(float), hipMemcpyDeviceToDevice); accumulatedElements += (long 
int)ipTensor->getNumElements(); } else { double WH = ipTensor->getWidth() * 
ipTensor->getHeight(); double WHC = WH * ipTensor->getChannels(); double WHCN = 
WHC * ipTensor->getBatchSize(); double WHCNS = WHCN * 
ipTensor->getSequenceLength(); double WH_totalC = WH * opTensor->getChannels(); 
double WH_totalC_N = WH_totalC * ipTensor->getBatchSize(); 
prepareKernelInputs(KCudOrFMfgCzUPMcdePX, shEncNmxJsMuJKwbrwok, 
ipTensor->getNumElements()); concatChannelImpl<<<KCudOrFMfgCzUPMcdePX, 
shEncNmxJsMuJKwbrwok>>>( ipTensor->getData(), opTensor->getData(), WH, 
WHC, WHCN, WHCNS, WH_totalC, WH_totalC_N, accumulatedC); accumulatedC += 
ipTensor->getChannels(); } } break; default: assert((OPnUFarBGxQhYRrGpeBQ == 1 
|| OPnUFarBGxQhYRrGpeBQ == 2) || OPnUFarBGxQhYRrGpeBQ == 3); } } } void 
MWConcatenationLayerImpl::prepareKernelInputs(int& KCudOrFMfgCzUPMcdePX, 
int& shEncNmxJsMuJKwbrwok, int dJcdBfQQLhIAYHPxwQeg) { const int 
threadsPerWarp = 32; shEncNmxJsMuJKwbrwok = ((dJcdBfQQLhIAYHPxwQeg + 
threadsPerWarp - 1)/threadsPerWarp) * threadsPerWarp; 
shEncNmxJsMuJKwbrwok = (shEncNmxJsMuJKwbrwok < 1024) ? 
shEncNmxJsMuJKwbrwok : 1024; KCudOrFMfgCzUPMcdePX = 
(dJcdBfQQLhIAYHPxwQeg + shEncNmxJsMuJKwbrwok - 1) / 
shEncNmxJsMuJKwbrwok; } void MWConcatenationLayerImpl::cleanup() { } } 