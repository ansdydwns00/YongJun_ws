#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnConvLayerImpl.hpp"
#include <cassert>
#include <cstdio>
#include <vector>
 namespace MWCudnnTarget { MWConvLayerImpl::MWConvLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int filt_H, int filt_W, int numGrps, int 
numChnls, int numFilts, int FLuSVNoPhAFKtLUchSvv, int 
FeVcBgtQmTLtmnNcJGMY, int CLOUhPjbgggWoXHTtmjC, int 
CGbFsczkgkhjcHoCKzBx, int ClEhcJFlvGCgiavziIag, int 
CpMjJjtGOeWOzwxpAAQP, int AdmgfUbRAfzFeYHxSnQr, int 
AuqaQHxmPQSyYRemQvyX, const char* xHViLEwTujGGrPZZgmbF, const 
char* JwxFdqOKggeawILBfGgg) : MWCNNLayerImpl(layer, ntwk_impl) , 
vIWQzNvYZSuxmOTVDFhU(NULL) , IpFhwalnAlrMvcuyQpQD(NULL) , SugesRlPIbOVzRgNWRnl(NULL) , 
AwZQzUhuWVLGrWgLHRuM(filt_H) , AzTsxYcYjIEJsGQbeYHm(filt_W) , 
BdqURaHPmdnfzvtUvocl(numGrps) , BRSPqxNffoBYKqpSVHne(numChnls) , 
BUOdotSvmFyUWQKMUdra(numFilts) , 
AVeZfqOFypgpiqfRYlKc(AdmgfUbRAfzFeYHxSnQr) , 
AjhVZuQXURJimwbnYqDF(AuqaQHxmPQSyYRemQvyX) , 
CDJtexcMbXMWAmnNZsNf(CLOUhPjbgggWoXHTtmjC) , 
CCKWXUFWgrbBMjwfpOBN(CGbFsczkgkhjcHoCKzBx) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP) , 
EvebzoroiuKkIxwjkGnD(FLuSVNoPhAFKtLUchSvv) , 
FOcStuqCptsGIZXskVpC(FeVcBgtQmTLtmnNcJGMY) , 
HtQBsWTCGEkpylRklilw((CDJtexcMbXMWAmnNZsNf != CCKWXUFWgrbBMjwfpOBN) 
|| (CTCbzQMDaLxINPbODdng != CZNYmBcNFSZWvaCklqeM)) { cQBKlCKXxecGPJrXBXdk = 
ntwk_impl; CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NGqpeiLeVweDRsOKEtuw)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&PtkeOkuClHzhOfpmBevf)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JgLfgHrHMEMmMYTettJF)); MWCNNLayer* 
convLayer = getLayer(); MWTensorBase* ipTensor = convLayer->getInputTensor(0); 
if (HtQBsWTCGEkpylRklilw) { SugesRlPIbOVzRgNWRnl = new MWTensor<float>(-1, 
-1, -1, -1, -1, NULL, getLayer(), ipTensor->getDataFormat(), 0); if 
(!SugesRlPIbOVzRgNWRnl) { MWCNNLayerImpl::throwAllocationError(__LINE__, __FILE__); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&XYbzSmRQGatVJtGmDZSo)); } else { 
SugesRlPIbOVzRgNWRnl = ipTensor; } assert(SugesRlPIbOVzRgNWRnl != NULL); int 
NMMfJylfQjiIUAKhXCJb; int NNhshzQGJHLSGjDiVerE; if 
(HtQBsWTCGEkpylRklilw) { NMMfJylfQjiIUAKhXCJb = 0; 
NNhshzQGJHLSGjDiVerE = 0; } else { NMMfJylfQjiIUAKhXCJb = 
CDJtexcMbXMWAmnNZsNf; NNhshzQGJHLSGjDiVerE = CTCbzQMDaLxINPbODdng; } 
bERCRkGjpaKXMNComoYl = CDJtexcMbXMWAmnNZsNf; bOrQjJTNlssnrexxbHdi = CTCbzQMDaLxINPbODdng;
#if (CUDNN_MAJOR <= 5)
 { if ((AVeZfqOFypgpiqfRYlKc != 1) && (AjhVZuQXURJimwbnYqDF != 1)) 
{ printf("Dilated Convolution only supported for cuDNN 6 or greater "); throw 
std::runtime_error("Unsupported Dilation Factor"); } 
CUDNN_CALL(hipdnnSetConvolution2dDescriptor( NGqpeiLeVweDRsOKEtuw, 
NMMfJylfQjiIUAKhXCJb, NNhshzQGJHLSGjDiVerE, EvebzoroiuKkIxwjkGnD, 
FOcStuqCptsGIZXskVpC, 1, 1, HIPDNN_CROSS_CORRELATION));  }
#else
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor( NGqpeiLeVweDRsOKEtuw, 
NMMfJylfQjiIUAKhXCJb, NNhshzQGJHLSGjDiVerE, EvebzoroiuKkIxwjkGnD, 
FOcStuqCptsGIZXskVpC, AVeZfqOFypgpiqfRYlKc, AjhVZuQXURJimwbnYqDF, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT)); }
#endif
#if (FP16_ENABLED == 1 && (CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2)))
 CUDNN_CALL( hipdnnSetConvolutionMathType(NGqpeiLeVweDRsOKEtuw, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 if (BdqURaHPmdnfzvtUvocl > 1) { 
CUDNN_CALL(hipdnnSetConvolutionGroupCount(NGqpeiLeVweDRsOKEtuw, 
BdqURaHPmdnfzvtUvocl)); } int eWYFXrUazhqiEIscccda = 
BRSPqxNffoBYKqpSVHne * BdqURaHPmdnfzvtUvocl; int eqmVWbEcwBRGnVNDUtrG 
= BUOdotSvmFyUWQKMUdra * BdqURaHPmdnfzvtUvocl; 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(PtkeOkuClHzhOfpmBevf, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, eqmVWbEcwBRGnVNDUtrG, eWYFXrUazhqiEIscccda / 
BdqURaHPmdnfzvtUvocl, AwZQzUhuWVLGrWgLHRuM, AzTsxYcYjIEJsGQbeYHm)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JgLfgHrHMEMmMYTettJF, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, eqmVWbEcwBRGnVNDUtrG, 1, 1)); int weightSize = 
BRSPqxNffoBYKqpSVHne * eqmVWbEcwBRGnVNDUtrG * AwZQzUhuWVLGrWgLHRuM 
* AzTsxYcYjIEJsGQbeYHm; CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, 
sizeof(float) * weightSize)); CUDA_CALL(hipMalloc((void**)&IpFhwalnAlrMvcuyQpQD, 
sizeof(float) * eqmVWbEcwBRGnVNDUtrG)); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(JwxFdqOKggeawILBfGgg); 
createAndAddDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); } 
void MWConvLayerImpl::propagateSize() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); int inputH; int inputW; if 
(HtQBsWTCGEkpylRklilw) { inputH = ipTensor->getHeight() + 
CDJtexcMbXMWAmnNZsNf + CCKWXUFWgrbBMjwfpOBN; inputW = ipTensor->getWidth() + 
CTCbzQMDaLxINPbODdng + CZNYmBcNFSZWvaCklqeM; } else { inputH = 
ipTensor->getHeight(); inputW = ipTensor->getWidth(); } 
SugesRlPIbOVzRgNWRnl->setHeight(inputH); SugesRlPIbOVzRgNWRnl->setWidth(inputW); 
SugesRlPIbOVzRgNWRnl->setChannels(ipTensor->getChannels()); 
SugesRlPIbOVzRgNWRnl->setBatchSize(ipTensor->getBatchSize()); 
SugesRlPIbOVzRgNWRnl->setSequenceLength(ipTensor->getSequenceLength()); 
assert(SugesRlPIbOVzRgNWRnl->getSequenceLength() == 1); if 
(HtQBsWTCGEkpylRklilw) { CUDNN_CALL(hipdnnSetTensor4dDescriptor( 
XYbzSmRQGatVJtGmDZSo, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 
SugesRlPIbOVzRgNWRnl->getBatchSize(), SugesRlPIbOVzRgNWRnl->getChannels(), 
SugesRlPIbOVzRgNWRnl->getHeight(), SugesRlPIbOVzRgNWRnl->getWidth())); } else { 
XYbzSmRQGatVJtGmDZSo = MWCNNLayerImpl::getCuDNNDescriptor(SugesRlPIbOVzRgNWRnl); } 
MWTensorBase* opTensor = getLayer()->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));
#if (CUDNN_MAJOR < 7)
 { CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, 
HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NDHPlSVpLroiIBRnjwyO)); }
#else
 { const int maxAlgoCount(3); int returnedAlgoCount(-1); 
hipdnnConvolutionFwdAlgoPerf_t perf_results[maxAlgoCount]; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, maxAlgoCount, 
&returnedAlgoCount, perf_results)); NDHPlSVpLroiIBRnjwyO = perf_results[0].algo; }
#endif
 size_t sxuOMwKXOKfuExclRaSe = 0; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, NDHPlSVpLroiIBRnjwyO, 
&sxuOMwKXOKfuExclRaSe)); if (sxuOMwKXOKfuExclRaSe > 
*cQBKlCKXxecGPJrXBXdk->getProposedWorkSpaceSize()) { 
cQBKlCKXxecGPJrXBXdk->setProposedWorkSpaceSize(sxuOMwKXOKfuExclRaSe); } } void 
MWConvLayerImpl::allocate() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); if (HtQBsWTCGEkpylRklilw) { float* 
newInput; int inputH = ipTensor->getHeight() + CDJtexcMbXMWAmnNZsNf + 
CCKWXUFWgrbBMjwfpOBN; int inputW = ipTensor->getWidth() + 
CTCbzQMDaLxINPbODdng + CZNYmBcNFSZWvaCklqeM; int paddedSize = 
ipTensor->getBatchSize() * ipTensor->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float) * paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float) * paddedSize)); 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->setData(newInput); } } void 
MWConvLayerImpl::deallocate() { if (SugesRlPIbOVzRgNWRnl != 
getLayer()->getInputTensor(0)) { assert(HtQBsWTCGEkpylRklilw); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData()); 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->setData((float*)NULL); } } void 
MWConvLayerImpl::predict() { MWCNNLayer* convLayer = getLayer(); MWTensorBase* 
ipTensorBase = convLayer->getInputTensor(); MWTensorBase* opTensorBase = 
convLayer->getOutputTensor(); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); if (SugesRlPIbOVzRgNWRnl != 
convLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), 
0, sizeof(float) * SugesRlPIbOVzRgNWRnl->getNumElements())); MWCNNLayerImpl::padInput( 
ipTensor->getData(), ipTensor->getHeight(), ipTensor->getWidth(), 
ipTensor->getChannels(), SugesRlPIbOVzRgNWRnl->getHeight(), SugesRlPIbOVzRgNWRnl->getWidth(), 
bERCRkGjpaKXMNComoYl, bOrQjJTNlssnrexxbHdi, 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), 
ipTensor->getNumElements()); } assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData()); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); CUDNN_CALL(hipdnnConvolutionForward( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), getOnePtr(), XYbzSmRQGatVJtGmDZSo, 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), PtkeOkuClHzhOfpmBevf, 
vIWQzNvYZSuxmOTVDFhU, NGqpeiLeVweDRsOKEtuw, NDHPlSVpLroiIBRnjwyO, 
cQBKlCKXxecGPJrXBXdk->getWorkSpace(), 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize(), getZeroPtr(), *desc, 
opTensor->getData())); 
CUDNN_CALL(hipdnnAddTensor(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), getOnePtr(), 
JgLfgHrHMEMmMYTettJF, IpFhwalnAlrMvcuyQpQD, getOnePtr(), *desc, opTensor->getData())); } 
void MWConvLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NGqpeiLeVweDRsOKEtuw)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(PtkeOkuClHzhOfpmBevf)); if 
(vIWQzNvYZSuxmOTVDFhU) { CUDA_FREE_CALL(vIWQzNvYZSuxmOTVDFhU); vIWQzNvYZSuxmOTVDFhU = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JgLfgHrHMEMmMYTettJF)); if 
(IpFhwalnAlrMvcuyQpQD) { CUDA_FREE_CALL(IpFhwalnAlrMvcuyQpQD); IpFhwalnAlrMvcuyQpQD = NULL; } if 
(SugesRlPIbOVzRgNWRnl != getLayer()->getInputTensor(0)) { 
assert(HtQBsWTCGEkpylRklilw); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(XYbzSmRQGatVJtGmDZSo)); } } void 
MWConvLayerImpl::loadWeights(const char* PmFfARVzoHVAYkfpuvqK) { FILE* 
QMgBqCuvjnbWHWiVPEwn = MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK); 
assert(QMgBqCuvjnbWHWiVPEwn); int cRtIUoZRPICuQEOZOSzT = BRSPqxNffoBYKqpSVHne * 
BUOdotSvmFyUWQKMUdra * BdqURaHPmdnfzvtUvocl * AwZQzUhuWVLGrWgLHRuM * 
AzTsxYcYjIEJsGQbeYHm;  float* KHjdvykTFbUxdfZTFbqy = MALLOC_CALL(sizeof(float) * 
cRtIUoZRPICuQEOZOSzT); MWCNNLayer::call_fread(KHjdvykTFbUxdfZTFbqy, sizeof(float), 
cRtIUoZRPICuQEOZOSzT, QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, KHjdvykTFbUxdfZTFbqy, sizeof(float) * 
cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); fclose(QMgBqCuvjnbWHWiVPEwn); 
free(KHjdvykTFbUxdfZTFbqy); } void MWConvLayerImpl::loadBias(const char* 
PmFfARVzoHVAYkfpuvqK) { FILE* QMgBqCuvjnbWHWiVPEwn = 
MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK); assert(QMgBqCuvjnbWHWiVPEwn); int 
cRtIUoZRPICuQEOZOSzT = BUOdotSvmFyUWQKMUdra * BdqURaHPmdnfzvtUvocl;  float* 
KHjdvykTFbUxdfZTFbqy = MALLOC_CALL(sizeof(float) * cRtIUoZRPICuQEOZOSzT); 
MWCNNLayer::call_fread(KHjdvykTFbUxdfZTFbqy, sizeof(float), cRtIUoZRPICuQEOZOSzT, 
QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); CUDA_CALL(hipMemcpy(IpFhwalnAlrMvcuyQpQD, 
KHjdvykTFbUxdfZTFbqy, sizeof(float) * cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); 
free(KHjdvykTFbUxdfZTFbqy); fclose(QMgBqCuvjnbWHWiVPEwn); } void 
MWConvLayerImpl::setLearnables(std::vector<float*> learnables) { 
assert(learnables.size() == 2); int cRtIUoZRPICuQEOZOSzT = BRSPqxNffoBYKqpSVHne * 
BUOdotSvmFyUWQKMUdra * BdqURaHPmdnfzvtUvocl * AwZQzUhuWVLGrWgLHRuM * 
AzTsxYcYjIEJsGQbeYHm;  float* wqggPBXZvtlxnxwngvAq = learnables[0]; 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, wqggPBXZvtlxnxwngvAq, sizeof(float) * 
cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); cRtIUoZRPICuQEOZOSzT = 
BUOdotSvmFyUWQKMUdra * BdqURaHPmdnfzvtUvocl;  float* JLxVZWxVuFDRoIFIqrjf = 
learnables[1]; CUDA_CALL(hipMemcpy(IpFhwalnAlrMvcuyQpQD, JLxVZWxVuFDRoIFIqrjf, 
sizeof(float) * cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); } void 
MWConvLayerImpl::postSetup() { if (cQBKlCKXxecGPJrXBXdk->getAutoTune()) { 
getConvAlgoTuned(); } else { getConvAlgoWorkSpaceLimit(); } } void 
MWConvLayerImpl::getConvAlgoTuned() { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); 
assert(desc); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), PtkeOkuClHzhOfpmBevf, 
vIWQzNvYZSuxmOTVDFhU, NGqpeiLeVweDRsOKEtuw, *desc, opTensor->getData(), 3, 
&returnedAlgoCount, &perf_results[0], cQBKlCKXxecGPJrXBXdk->getWorkSpace(), 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize())); NDHPlSVpLroiIBRnjwyO = 
perf_results[0].algo; } void MWConvLayerImpl::getConvAlgoWorkSpaceLimit() { 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR < 8)
 CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, 
HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize(), &NDHPlSVpLroiIBRnjwyO));
#else
 int maxAlgoCount(-1); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
&maxAlgoCount)); int returnedAlgoCount(-1); 
std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(maxAlgoCount); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NGqpeiLeVweDRsOKEtuw, *desc, maxAlgoCount, 
&returnedAlgoCount, &perf_results[0])); hipdnnConvolutionFwdAlgoPerf_t 
nextFastest; bool algoFound(false); for (int i = 0; i < returnedAlgoCount; ++i) 
{ nextFastest = perf_results[i]; if (nextFastest.memory <= 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize()) { NDHPlSVpLroiIBRnjwyO = 
nextFastest.algo; algoFound = true; break; } } if (!algoFound) { assert(false); }
#endif
 } } 