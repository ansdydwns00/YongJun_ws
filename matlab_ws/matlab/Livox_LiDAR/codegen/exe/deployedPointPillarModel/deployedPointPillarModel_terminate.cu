//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: deployedPointPillarModel_terminate.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "deployedPointPillarModel_terminate.h"
#include "deployedPointPillarModel_data.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void deployedPointPillarModel_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorName(errCode),
            hipGetErrorString(errCode));
    exit(errCode);
  }
  cublasEnsureDestruction();
  hipFree(*global_gpu_dv);
  hipFree(*global_gpu_xq);
  hipFree(*b_global_gpu_dv);
  hipFree(*b_global_gpu_xq);
  isInitialized_deployedPointPillarModel = false;
}

//
// File trailer for deployedPointPillarModel_terminate.cu
//
// [EOF]
//
