#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: cellstr_unique.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "cellstr_unique.h"
#include "cellstr_sort.h"
#include "deployedPointPillarModel_internal_types.h"
#include "find.h"
#include "rt_nonfinite.h"
#include "strcmp.h"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include "coder_array.h"
#include "coder_gpu_array.h"
#include <cmath>

// Function Definitions
//
// Arguments    : const array<cell_wrap_6, 1U> &a
//                array<cell_wrap_6, 1U> &u
// Return Type  : void
//
namespace coder {
namespace matlab {
namespace internal {
namespace coder {
namespace datatypes {
void cellstr_unique(const array<cell_wrap_6, 1U> &a, array<cell_wrap_6, 1U> &u)
{
  array<cell_wrap_6, 1U> c;
  array<double, 1U> cpu_ic;
  array<int, 2U> b_y;
  array<int, 2U> c_y;
  array<int, 1U> dIdx;
  array<int, 1U> ia;
  array<int, 1U> idx;
  array<bool, 1U> d;
  array<bool, 1U> r;
  gpu_array<double, 1U> gpu_ic;
  bool ic_outdatedOnGpu;
  ic_outdatedOnGpu = false;
  if (a.size(0) == 0) {
    u.set_size(0);
  } else {
    int b_i;
    int k;
    int loop_ub;
    int vlen;
    cellstr_sort(a, c, idx);
    d.set_size(c.size(0) - 1);
    for (int i{0}; i <= c.size(0) - 2; i++) {
      d[i] = !::coder::internal::b_strcmp(c[i].f1, c[i + 1].f1);
    }
    r.set_size(d.size(0) + 1);
    r[0] = true;
    loop_ub = d.size(0);
    for (b_i = 0; b_i < loop_ub; b_i++) {
      r[b_i + 1] = d[b_i];
    }
    d.set_size(r.size(0));
    for (b_i = 0; b_i < r.size(0); b_i++) {
      d[b_i] = r[b_i];
    }
    vlen = d.size(0);
    loop_ub = d[0];
    for (k = 0; k <= vlen - 2; k++) {
      loop_ub += d[k + 1];
    }
    u.set_size(static_cast<int>(std::fmin(static_cast<double>(loop_ub),
                                          static_cast<double>(a.size(0)))));
    eml_find(d, dIdx);
    ia.set_size(dIdx.size(0));
    k = dIdx.size(0);
    for (int i{0}; i < k; i++) {
      if (i + 1 != dIdx.size(0)) {
        if (static_cast<double>(dIdx[i + 1]) - 1.0 < dIdx[i]) {
          c_y.set_size(1, 0);
        } else {
          double b_d;
          b_i = dIdx[i];
          b_d = static_cast<double>(dIdx[i + 1]) - 1.0;
          c_y.set_size(
              1, static_cast<int>((static_cast<double>(dIdx[i + 1]) - 1.0) -
                                  static_cast<double>(dIdx[i])) +
                     1);
          for (loop_ub = 0;
               loop_ub <= static_cast<int>(b_d - static_cast<double>(b_i));
               loop_ub++) {
            c_y[loop_ub] = b_i + loop_ub;
          }
        }
        vlen = c_y.size(1);
        loop_ub = idx[c_y[0] - 1];
        for (int c_i{0}; c_i <= vlen - 2; c_i++) {
          b_i = c_y[c_i + 1];
          if (loop_ub > idx[b_i - 1]) {
            loop_ub = idx[b_i - 1];
          }
        }
        ia[i] = loop_ub;
      } else {
        if (idx.size(0) < dIdx[i]) {
          b_y.set_size(1, 0);
        } else {
          b_i = dIdx[i];
          b_y.set_size(1, (idx.size(0) - dIdx[i]) + 1);
          for (loop_ub = 0; loop_ub <= idx.size(0) - b_i; loop_ub++) {
            b_y[loop_ub] = b_i + loop_ub;
          }
        }
        vlen = b_y.size(1);
        loop_ub = idx[b_y[0] - 1];
        for (int c_i{0}; c_i <= vlen - 2; c_i++) {
          b_i = b_y[c_i + 1];
          if (loop_ub > idx[b_i - 1]) {
            loop_ub = idx[b_i - 1];
          }
        }
        ia[i] = loop_ub;
      }
    }
    cpu_ic.set_size(d.size(0));
    gpu_ic.set_size(&cpu_ic);
    for (b_i = 0; b_i < d.size(0); b_i++) {
      cpu_ic[b_i] = d[b_i];
      ic_outdatedOnGpu = true;
    }
    if (cpu_ic.size(0) != 1) {
      if (ic_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_ic, cpu_ic);
      }
      mwCallThrustScan1D(&(gpu_ic.data())[0], false, cpu_ic.size(0));
    }
    k = dIdx.size(0);
    for (int i{0}; i < k; i++) {
      u[i].f1.set_size(1, a[ia[i] - 1].f1.size(1));
      loop_ub = a[ia[i] - 1].f1.size(1);
      for (b_i = 0; b_i < loop_ub; b_i++) {
        u[i].f1[b_i] = a[ia[i] - 1].f1[b_i];
      }
    }
  }
}

//
// Arguments    : const array<cell_wrap_6, 1U> &a
//                array<cell_wrap_6, 1U> &u
//                array<double, 1U> &ia
// Return Type  : void
//
void cellstr_unique(const array<cell_wrap_6, 1U> &a, array<cell_wrap_6, 1U> &u,
                    array<double, 1U> &ia)
{
  array<cell_wrap_6, 1U> c;
  array<double, 1U> cpu_ic;
  array<int, 2U> b_y;
  array<int, 2U> c_y;
  array<int, 1U> dIdx;
  array<int, 1U> idx;
  array<bool, 1U> d;
  array<bool, 1U> r;
  gpu_array<double, 1U> gpu_ic;
  bool ic_outdatedOnGpu;
  ic_outdatedOnGpu = false;
  if (a.size(0) == 0) {
    u.set_size(0);
    ia.set_size(0);
  } else {
    int b_i;
    int k;
    int loop_ub;
    int vlen;
    cellstr_sort(a, c, idx);
    d.set_size(c.size(0) - 1);
    for (int i{0}; i <= c.size(0) - 2; i++) {
      d[i] = !::coder::internal::b_strcmp(c[i].f1, c[i + 1].f1);
    }
    r.set_size(d.size(0) + 1);
    r[0] = true;
    loop_ub = d.size(0);
    for (b_i = 0; b_i < loop_ub; b_i++) {
      r[b_i + 1] = d[b_i];
    }
    d.set_size(r.size(0));
    for (b_i = 0; b_i < r.size(0); b_i++) {
      d[b_i] = r[b_i];
    }
    vlen = d.size(0);
    loop_ub = d[0];
    for (k = 0; k <= vlen - 2; k++) {
      loop_ub += d[k + 1];
    }
    u.set_size(static_cast<int>(std::fmin(static_cast<double>(loop_ub),
                                          static_cast<double>(a.size(0)))));
    eml_find(d, dIdx);
    ia.set_size(dIdx.size(0));
    k = dIdx.size(0);
    for (int i{0}; i < k; i++) {
      if (i + 1 != dIdx.size(0)) {
        if (static_cast<double>(dIdx[i + 1]) - 1.0 < dIdx[i]) {
          c_y.set_size(1, 0);
        } else {
          double b_d;
          b_i = dIdx[i];
          b_d = static_cast<double>(dIdx[i + 1]) - 1.0;
          c_y.set_size(
              1, static_cast<int>((static_cast<double>(dIdx[i + 1]) - 1.0) -
                                  static_cast<double>(dIdx[i])) +
                     1);
          for (loop_ub = 0;
               loop_ub <= static_cast<int>(b_d - static_cast<double>(b_i));
               loop_ub++) {
            c_y[loop_ub] = b_i + loop_ub;
          }
        }
        vlen = c_y.size(1);
        loop_ub = idx[c_y[0] - 1];
        for (int c_i{0}; c_i <= vlen - 2; c_i++) {
          b_i = c_y[c_i + 1];
          if (loop_ub > idx[b_i - 1]) {
            loop_ub = idx[b_i - 1];
          }
        }
        ia[i] = loop_ub;
      } else {
        if (idx.size(0) < dIdx[i]) {
          b_y.set_size(1, 0);
        } else {
          b_i = dIdx[i];
          b_y.set_size(1, (idx.size(0) - dIdx[i]) + 1);
          for (loop_ub = 0; loop_ub <= idx.size(0) - b_i; loop_ub++) {
            b_y[loop_ub] = b_i + loop_ub;
          }
        }
        vlen = b_y.size(1);
        loop_ub = idx[b_y[0] - 1];
        for (int c_i{0}; c_i <= vlen - 2; c_i++) {
          b_i = b_y[c_i + 1];
          if (loop_ub > idx[b_i - 1]) {
            loop_ub = idx[b_i - 1];
          }
        }
        ia[i] = loop_ub;
      }
    }
    cpu_ic.set_size(d.size(0));
    gpu_ic.set_size(&cpu_ic);
    for (b_i = 0; b_i < d.size(0); b_i++) {
      cpu_ic[b_i] = d[b_i];
      ic_outdatedOnGpu = true;
    }
    if (cpu_ic.size(0) != 1) {
      if (ic_outdatedOnGpu) {
        arrayCopyCpuToGpu(gpu_ic, cpu_ic);
      }
      mwCallThrustScan1D(&(gpu_ic.data())[0], false, cpu_ic.size(0));
    }
    k = dIdx.size(0);
    for (int i{0}; i < k; i++) {
      u[i].f1.set_size(1, a[static_cast<int>(ia[i]) - 1].f1.size(1));
      loop_ub = a[static_cast<int>(ia[i]) - 1].f1.size(1);
      for (b_i = 0; b_i < loop_ub; b_i++) {
        u[i].f1[b_i] = a[static_cast<int>(ia[i]) - 1].f1[b_i];
      }
    }
  }
}

} // namespace datatypes
} // namespace coder
} // namespace internal
} // namespace matlab
} // namespace coder

//
// File trailer for cellstr_unique.cu
//
// [EOF]
//
