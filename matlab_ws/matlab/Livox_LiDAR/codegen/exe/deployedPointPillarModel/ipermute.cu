#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: ipermute.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "ipermute.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"

// Function Declarations
static __global__ void ipermute_kernel25(const float b[6856704],
                                         float a[6856704]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float b[6856704]
//                float a[6856704]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void ipermute_kernel25(const float b[6856704],
                                                     float a[6856704])
{
  unsigned long long threadId;
  int b_k;
  int c_k;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  c_k = static_cast<int>(threadId % 248ULL);
  threadId = (threadId - static_cast<unsigned long long>(c_k)) / 248ULL;
  b_k = static_cast<int>(threadId % 216ULL);
  threadId = (threadId - static_cast<unsigned long long>(b_k)) / 216ULL;
  k = static_cast<int>(threadId);
  if ((k < 128) && (b_k < 216) && (c_k < 248)) {
    a[(b_k + 216 * c_k) + 53568 * k] = b[(c_k + 248 * b_k) + 53568 * k];
  }
}

//
// Arguments    : const float b[6856704]
//                float a[6856704]
// Return Type  : void
//
namespace coder {
void ipermute(const float b[6856704], float a[6856704])
{
  ipermute_kernel25<<<dim3(13392U, 1U, 1U), dim3(512U, 1U, 1U)>>>(b, a);
}

} // namespace coder

//
// File trailer for ipermute.cu
//
// [EOF]
//
