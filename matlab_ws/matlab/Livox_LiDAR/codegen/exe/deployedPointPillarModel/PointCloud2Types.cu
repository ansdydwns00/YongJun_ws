//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: PointCloud2Types.cu
//
// GPU Coder version                    : 23.2
// CUDA/C/C++ source code generated on  : 04-Jun-2024 16:24:05
//

// Include Files
#include "PointCloud2Types.h"
#include "rt_nonfinite.h"
#include "coder_array.h"

// Function Definitions
//
// Arguments    : unsigned char type
//                array<char, 2U> &mlType
//                double *numBytes
// Return Type  : void
//
namespace coder {
namespace ros {
namespace msg {
namespace sensor_msgs {
namespace internal {
void PointCloud2Types_rosToMATLABType(unsigned char type,
                                      array<char, 2U> &mlType, double *numBytes)
{
  static const char cv3[6]{'u', 'i', 'n', 't', '1', '6'};
  static const char cv5[6]{'u', 'i', 'n', 't', '3', '2'};
  static const char cv6[6]{'s', 'i', 'n', 'g', 'l', 'e'};
  static const char cv7[6]{'d', 'o', 'u', 'b', 'l', 'e'};
  static const char cv1[5]{'u', 'i', 'n', 't', '8'};
  static const char cv2[5]{'i', 'n', 't', '1', '6'};
  static const char cv4[5]{'i', 'n', 't', '3', '2'};
  int b_numBytes;
  switch (type) {
  case 1U:
    mlType.set_size(1, 4);
    mlType[0] = 'i';
    mlType[1] = 'n';
    mlType[2] = 't';
    mlType[3] = '8';
    b_numBytes = 1;
    break;
  case 2U:
    mlType.set_size(1, 5);
    for (b_numBytes = 0; b_numBytes < 5; b_numBytes++) {
      mlType[b_numBytes] = cv1[b_numBytes];
    }
    b_numBytes = 1;
    break;
  case 3U:
    mlType.set_size(1, 5);
    for (b_numBytes = 0; b_numBytes < 5; b_numBytes++) {
      mlType[b_numBytes] = cv2[b_numBytes];
    }
    b_numBytes = 2;
    break;
  case 4U:
    mlType.set_size(1, 6);
    for (b_numBytes = 0; b_numBytes < 6; b_numBytes++) {
      mlType[b_numBytes] = cv3[b_numBytes];
    }
    b_numBytes = 2;
    break;
  case 5U:
    mlType.set_size(1, 5);
    for (b_numBytes = 0; b_numBytes < 5; b_numBytes++) {
      mlType[b_numBytes] = cv4[b_numBytes];
    }
    b_numBytes = 4;
    break;
  case 6U:
    mlType.set_size(1, 6);
    for (b_numBytes = 0; b_numBytes < 6; b_numBytes++) {
      mlType[b_numBytes] = cv5[b_numBytes];
    }
    b_numBytes = 4;
    break;
  case 7U:
    mlType.set_size(1, 6);
    for (b_numBytes = 0; b_numBytes < 6; b_numBytes++) {
      mlType[b_numBytes] = cv6[b_numBytes];
    }
    b_numBytes = 4;
    break;
  case 8U:
    mlType.set_size(1, 6);
    for (b_numBytes = 0; b_numBytes < 6; b_numBytes++) {
      mlType[b_numBytes] = cv7[b_numBytes];
    }
    b_numBytes = 8;
    break;
  }
  *numBytes = b_numBytes;
}

} // namespace internal
} // namespace sensor_msgs
} // namespace msg
} // namespace ros
} // namespace coder

//
// File trailer for PointCloud2Types.cu
//
// [EOF]
//
